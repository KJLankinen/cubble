#include "hip/hip_runtime.h"
// -*- C++ -*-

#include "CudaKernelWrapper.h"
#include "Macros.h"
#include "CudaContainer.h"

#include <iostream>
#include <hiprand.h>

cubble::CudaKernelWrapper::CudaKernelWrapper(std::shared_ptr<BubbleManager> bm)
{
    bubbleManager = bm;
}

cubble::CudaKernelWrapper::~CudaKernelWrapper()
{}

void cubble::CudaKernelWrapper::generateBubblesOnGPU(size_t n,
						     int rngSeed,
						     double avgRad,
						     double stdDevRad,
						     dvec lbb,
						     dvec tfr)
{
    size_t blockSize = n > 1024 ? 1024 : n;
    size_t numBlocks = n > 1024 ? (n % 1024 != 0 ? n / 1024 + 1 : n / 1024) : 1;
    
    CudaContainer<float> x(n);
    CudaContainer<float> y(n);
#if (NUM_DIM == 3)
    CudaContainer<float> z(n);
#endif
    
    CudaContainer<float> r(n);
    CudaContainer<Bubble> b(n);

    hiprandGenerator_t generator;
    CURAND_CALL(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_MTGP32));
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(generator, rngSeed));
    
    CURAND_CALL(hiprandGenerateUniform(generator, x.data(), n));
    CURAND_CALL(hiprandGenerateUniform(generator, y.data(), n));
#if (NUM_DIM == 3)
    CURAND_CALL(hiprandGenerateUniform(generator, z.data(), n));
#endif
    CURAND_CALL(hiprandGenerateNormal(generator, r.data(), n, avgRad, stdDevRad));

    assignDataToBubbles<<<numBlocks, blockSize>>>(x.data(),
						  y.data(),
#if (NUM_DIM == 3)
						  z.data(),
#endif
						  r.data(),
						  b.data(),
						  lbb,
						  tfr);
    
    b.copyDeviceDataToVec(bubbleManager->bubbles);
}

// ----------------------
// Kernel implementations
// ----------------------

__global__
void cubble::assignDataToBubbles(float *x,
				 float *y,
#if (NUM_DIM == 3)
				 float *z,
#endif
				 float *r,
				 Bubble *b,
				 dvec lbb,
				 dvec tfr)
{
    size_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    b[tid].pos.x = lbb.x + (double)x[tid] * tfr.x;
    b[tid].pos.y = lbb.y + (double)y[tid] * tfr.y;
#if (NUM_DIM == 3)
    b[tid].pos.z = lbb.z + (double)z[tid] * tfr.z;
#endif
    b[tid].radius = (double)r[tid];
}
