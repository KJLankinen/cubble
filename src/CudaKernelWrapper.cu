#include "hip/hip_runtime.h"
// -*- C++ -*-

#include "CudaKernelWrapper.h"
#include "Macros.h"
#include "CudaContainer.h"
#include "Fileio.h"

#include <iostream>
#include <hiprand.h>

cubble::CudaKernelWrapper::CudaKernelWrapper(std::shared_ptr<BubbleManager> bm)
{
    bubbleManager = bm;
}

cubble::CudaKernelWrapper::~CudaKernelWrapper()
{}

void cubble::CudaKernelWrapper::generateBubblesOnGPU(size_t n,
						     size_t numBlocksPerDim,
						     int rngSeed,
						     double avgRad,
						     double stdDevRad,
						     dvec lbb,
						     dvec tfr)
{
    int totalNumBlocks = numBlocksPerDim * numBlocksPerDim;
#if (NUM_DIM == 3)
    totalNumBlocks *= numBlocksPerDim;
    int numThreadsPerDim = (int)std::ceil(std::cbrt(n / (float)totalNumBlocks));
    dim3 blockSize = dim3(numThreadsPerDim * numThreadsPerDim * numThreadsPerDim, 1, 1);
    dim3 gridSize = dim3(numBlocksPerDim, numBlocksPerDim, numBlocksPerDim);
    int recommendedNumCells = (int)(std::ceil(std::cbrt(n / 1024.0f) * 0.125f)) * 8;
#else
    int numThreadsPerDim = (int)std::ceil(std::sqrt(n / (float)totalNumBlocks));
    dim3 blockSize = dim3(numThreadsPerDim * numThreadsPerDim, 1, 1);
    dim3 gridSize = dim3(numBlocksPerDim, numBlocksPerDim, 1);
    int recommendedNumCells = (int)(std::ceil(std::sqrt(n / 1024.0f) * 0.125f)) * 8;
#endif
    int numCells = gridSize.x * gridSize.y * gridSize.z;
    int numThreads = blockSize.x * blockSize.y * blockSize.z;
    if (numThreads > 1024)
    {
	std::cerr << "Too many bubbles to simulate w.r.t. the number of blocks."
		  << "\nAmount of threads per block " << numThreads
		  << " exceeds the maximum, which is 1024."
		  << "\nIncrease the number of blocks (cells) per dimension."
		  << "\nRecommended number of cells with given number of bubbles: "
		  << recommendedNumCells
		  << std::endl;
	
	std::exit(EXIT_FAILURE);
    }

    std::cout << "Grid size: (" << gridSize.x
	      << ", " << gridSize.y
	      << ", " << gridSize.z
	      << "), block size: (" << blockSize.x
	      << ", " << blockSize.y
	      << ", " << blockSize.z
	      << ")\nRecommended number of cells with given number of bubbles: "
	      << recommendedNumCells
	      << std::endl;
    
    CudaContainer<float> x(n);
    CudaContainer<float> y(n);
#if (NUM_DIM == 3)
    CudaContainer<float> z(n);
#endif
    CudaContainer<float> r(n);
    CudaContainer<Bubble> b(n);
    CudaContainer<int> bubbleIndices(n);
    CudaContainer<int> bubblesPerCell(numCells);
    CudaContainer<int> offsets(numCells);

    hiprandGenerator_t generator;
    CURAND_CALL(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_MTGP32));
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(generator, rngSeed));
    CURAND_CALL(hiprandGenerateUniform(generator, x.getDevicePtr(), n));
    CURAND_CALL(hiprandGenerateUniform(generator, y.getDevicePtr(), n));
    CURAND_CALL(hiprandGenerateNormal(generator, r.getDevicePtr(), n, avgRad, stdDevRad));
#if (NUM_DIM == 3)
    CURAND_CALL(hiprandGenerateUniform(generator, z.getDevicePtr(), n));
#endif

    // Assign generated data to bubbles
    assignDataToBubbles<<<gridSize, blockSize>>>(x.getDevicePtr(),
						 y.getDevicePtr(),
#if (NUM_DIM == 3)
						 z.getDevicePtr(),
#endif
						 r.getDevicePtr(),
						 b.getDevicePtr(),
						 bubblesPerCell.getDevicePtr(),
						 lbb,
						 tfr,
						 n);
    bubblesPerCell.toHost();
    bubbleManager->cellEnds.resize(numCells);
    bubbleManager->cellBegins.resize(numCells);

    // Bubbles are stored in a 1D array, even if the domain is 2D or 3D.
    // Here we calculate the offsets (begin and end indices) to the memory
    // location where the bubbles of the current cell start.
    int offset = 0;
    for (size_t i = 0; i < numCells; ++i)
    {
	offsets[i] = offset;
	bubbleManager->cellBegins[i] = offset;
	offset += bubblesPerCell[i];
	bubbleManager->cellEnds[i] = offset;
    }
    offsets.toDevice();

    // Reset bubblesPerCell to 0, and reuse it.
    bubblesPerCell.fillHostWith(0);
    bubblesPerCell.toDevice();

    assignBubblesToCells<<<gridSize, blockSize>>>(b.getDevicePtr(),
						  bubbleIndices.getDevicePtr(),
						  offsets.getDevicePtr(),
						  bubblesPerCell.getDevicePtr(),
						  lbb,
						  tfr,
						  n);
    bubbleIndices.toHost();
    b.toHost();

    // Create a new host vector and add the bubbles in to it
    // s.t. the bubbles of any one cell are continuously stored.
    // Note that the memory locality could probably be improved
    // for nearby cells by saving the cells as a Hilbert curve,
    // instead of left to right, bottom to top, back to front.
    std::vector<Bubble> bubbles;
    bubbles.resize(n);
    for (size_t i = 0; i < n; ++i)
	bubbles[i] = b[bubbleIndices[i]];

    bubbleManager->setBubbles(bubbles);
}

__forceinline__ __device__
int getTid()
{
    int threadsPerBlock = blockDim.x * blockDim.y * blockDim.z;
    int blocksBefore = blockIdx.z * (gridDim.y * gridDim.x)
	+ blockIdx.y * gridDim.x
	+ blockIdx.x;
    int threadsBefore = blockDim.y * blockDim.x * threadIdx.z + blockDim.x * threadIdx.y;
    int tid = blocksBefore * threadsPerBlock + threadsBefore + threadIdx.x;

    return tid;
}

__global__
void cubble::assignDataToBubbles(float *x,
				 float *y,
#if (NUM_DIM == 3)
				 float *z,
#endif
				 float *r,
				 Bubble *b,
				 int *bubblesPerCell,
				 dvec lbb,
				 dvec tfr,
				 int numBubbles)
{
    int tid = getTid();
    
    if (tid < numBubbles)
    {
	b[tid].pos.x = lbb.x + (double)x[tid] * tfr.x;
	b[tid].pos.y = lbb.y + (double)y[tid] * tfr.y;
	b[tid].radius = (double)r[tid];
	
	int bbi = ((int)(y[tid] * gridDim.y) * gridDim.x) + (int)(x[tid] * gridDim.x);
#if (NUM_DIM == 3)
	b[tid].pos.z = lbb.z + (double)z[tid] * tfr.z;
	bbi += ((int)(z[tid] * gridDim.z) * gridDim.y * gridDim.x);
#endif
        int indexToCellContainer = atomicAdd(&bubblesPerCell[bbi], 1);
    }
}

__global__
void cubble::assignBubblesToCells(Bubble *b,
				  int *bubbleIndices,
				  int *offsets,
				  int *currentIndices,
				  dvec lbb,
				  dvec tfr,
				  int numBubbles)
{
    int tid = getTid();
    dvec interval = tfr - lbb;
    dvec normPos = (b[tid].pos - lbb) / interval;

    if (tid < numBubbles)
    {
	int bbi = ((int)(normPos.y * gridDim.y) * gridDim.x) + (int)(normPos.x * gridDim.x);
#if (NUM_DIM == 3)
	bbi += (int)(normPos.z * gridDim.z) * gridDim.y * gridDim.x;
#endif
	int offset = atomicAdd(&currentIndices[bbi], 1) + offsets[bbi];
	bubbleIndices[offset] = tid;
    }
}
