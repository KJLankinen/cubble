#include "hip/hip_runtime.h"
// -*- C++ -*-

#include "CudaKernelWrapper.h"
#include "Macros.h"
#include "CudaContainer.h"
#include "Cell.h"

#include <iostream>
#include <hiprand.h>

cubble::CudaKernelWrapper::CudaKernelWrapper(std::shared_ptr<BubbleManager> bm,
					     std::shared_ptr<Env> e)
{
    bubbleManager = bm;
    env = e;

    printRelevantInfoOfCurrentDevice();
}

cubble::CudaKernelWrapper::~CudaKernelWrapper()
{}

void cubble::CudaKernelWrapper::generateBubbles(std::vector<Bubble> &outBubbles)
{
    std::cout << "Generating bubbles..." << std::endl;
    
    // Get necessary parameters
    int n = env->getNumBubbles();
    int rngSeed = env->getRngSeed();
    double avgRad = env->getAvgRad();
    double stdDevRad = env->getStdDevRad();
    dvec lbb = env->getLbb();
    dvec tfr = env->getTfr();	

    int numThreads = 1024;
    int numBlocks = (int)std::ceil(n / (float)numThreads);
    
    CudaContainer<float> x(n);
    CudaContainer<float> y(n);
    CudaContainer<float> r(n);
    
    CudaContainer<Bubble> b(n);

    // Generate random positions & radii
    hiprandGenerator_t generator;
    CURAND_CALL(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_MTGP32));
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(generator, rngSeed));
    CURAND_CALL(hiprandGenerateUniform(generator, x.getDevPtr(), n));
    CURAND_CALL(hiprandGenerateUniform(generator, y.getDevPtr(), n));
    CURAND_CALL(hiprandGenerateNormal(generator, r.getDevPtr(), n, avgRad, stdDevRad));
    
#if (NUM_DIM == 3)
    CudaContainer<float> z(n);
    CURAND_CALL(hiprandGenerateUniform(generator, z.getDevPtr(), n));
#endif
    
    // Assign generated data to bubbles
    assignDataToBubbles<<<numBlocks, numThreads>>>(x.getDevPtr(),
						 y.getDevPtr(),
#if (NUM_DIM == 3)
						 z.getDevPtr(),
#endif
						 r.getDevPtr(),
						 b.getDevPtr(),
						 lbb,
						 tfr,
						 n);
    CUDA_CALL(hipPeekAtLastError());
    CUDA_CALL(hipDeviceSynchronize());

    b.copyDeviceDataToVec(outBubbles);
}

void cubble::CudaKernelWrapper::assignBubblesToCells(const std::vector<Bubble> &b)
{
    std::cout << "Assigning bubbles to cells..." << std::endl;
    
    int numBubblesPerCell = env->getNumBubblesPerCell();
    dvec tfr = env->getTfr();
    dvec lbb = env->getLbb();
    dim3 gridSize = getGridSize(b.size());
    int numCells = gridSize.x * gridSize.y * gridSize.z;

    std::cout << "Grid size: (" << gridSize.x
	      << ", " << gridSize.y
	      << ", " << gridSize.z
	      << "), numCells: " << numCells
	      << std::endl;
    
    CudaContainer<Bubble> bubbles(b);
    CudaContainer<Cell> cells(numCells);
    CudaContainer<int> indices(bubbles.size());

    bubbles.toDevice();
    cells.toDevice();
    
    calculateOffsets<<<gridSize, numBubblesPerCell>>>(bubbles.getDevPtr(),
						      cells.getDevPtr(),
						      lbb,
						      tfr,
						      bubbles.size());

    cells.toHost();

    int cumulativeSum = 0;
    for (size_t i = 0; i < cells.size(); ++i)
    {
	int numBubbles = cells[i].offset;
        cells[i].offset = cumulativeSum;
	cumulativeSum += numBubbles;
    }
    cells.toDevice();
    
    cubble::assignBubblesToCells<<<gridSize,numBubblesPerCell>>>(bubbles.getDevPtr(),
								 indices.getDevPtr(),
								 cells.getDevPtr(),
								 bubbles.size());
    
    bubbleManager->setBubblesFromDevice(bubbles);
    bubbleManager->setIndicesFromDevice(indices);
    bubbleManager->setCellsFromDevice(cells);
}

void cubble::CudaKernelWrapper::removeIntersectingBubbles(const std::vector<Bubble> &b)
{
    std::cout << "Removing intersecing bubbles..." << std::endl;
    
    CudaContainer<Bubble> bubbles(b);
    CudaContainer<int> indices(bubbles.size());
    CudaContainer<int> intersections(bubbles.size() + 1); // <-- First value gives count
    CudaContainer<Cell> cells(bubbleManager->getCellsSize());
    bubbleManager->getCells(cells);
    bubbleManager->getIndices(indices);
    intersections.fillHostWith(0);

    int numThreads = 1024;
    int numDomains = (CUBBLE_NUM_NEIGHBORS + 1) * 4;
    dim3 gridSize = getGridSize(bubbles.size());
    int originalGridZ = gridSize.z;
    gridSize.z *= numDomains;

    assertGridSizeBelowLimit(gridSize);

    bubbles.toDevice();
    indices.toDevice();
    intersections.toDevice();
    cells.toDevice();

    int sharedMemSize = 0;
    for (size_t i = 0; i < cells.size(); ++i)
    {
	int temp = cells[i].size;
	sharedMemSize = sharedMemSize < temp ? temp : sharedMemSize;
    }
    sharedMemSize = (int)(sharedMemSize * 0.5f) + 1;
    sharedMemSize *= sizeof(Bubble);

    assertMemBelowLimit(sharedMemSize);
    
    findIntersections<<<numThreads, gridSize, sharedMemSize>>>(bubbles.getDevPtr(),
							       indices.getDevPtr(),
							       cells.getDevPtr(),
							       intersections.getDevPtr(),
							       bubbles.size(),
							       numDomains);
}

dim3 cubble::CudaKernelWrapper::getGridSize(int numBubbles)
{
    int numBubblesPerCell = env->getNumBubblesPerCell();
#if NUM_DIM == 3
    int numCellsPerDim = (int)std::ceil(std::cbrt(numBubbles / numBubblesPerCell));
    dim3 gridSize(numCellsPerDim, numCellsPerDim, numCellsPerDim);
#else
    int numCellsPerDim = (int)std::ceil(std::sqrt(numBubbles / numBubblesPerCell));
    dim3 gridSize(numCellsPerDim, numCellsPerDim, 1);
#endif

    return gridSize;
}

__forceinline__ __device__
int cubble::geNeighborCellIndex(ivec cellIdx, ivec dim, int neighborNum)
{
    // Switch statements and ifs that diverge inside one warp/block are
    // detrimental for performance. However, this should never diverge,
    // as all the threads of one block should always be in the same cell
    // going for the same neighbor.
    ivec idxVec = cellIdx;
    switch(neighborNum)
    {
    case 0:
	// self
	break;
    case 1:
	idxVec += ivec(-1, 1, 0);
	break;
    case 2:
	idxVec += ivec(-1, 0, 0);
	break;
    case 3:
	idxVec += ivec(-1, -1, 0);
	break;
    case 4:
	idxVec += ivec(0, -1, 0);
	break;
#if NUM_DIM == 3
    case 5:
	idxVec += ivec(-1, 1, -1);
	break;
    case 6:
	idxVec += ivec(-1, 0, -1);
	break;
    case 7:
	idxVec += ivec(-1, -1, -1);
	break;
    case 8:
	idxVec += ivec(0, 1, -1);
	break;
    case 9:
	idxVec += ivec(0, 0, -1);
	break;
    case 10:
	idxVec += ivec(0, -1, -1);
	break;
    case 11:
	idxVec += ivec(1, 1, -1);
	break;
    case 12:
	idxVec += ivec(1, 0, -1);
	break;
    case 13:
	idxVec += ivec(1, -1, -1);
	break;
#endif
    default:
	printf("Should never end up here!");
	break;
    }

    idxVec += dim;
    idxVec %= dim;

    return idxVec.z * dim.y * dim.x + idxVec.y * dim.x + idxVec.x;
}

__forceinline__ __device__
int cubble::getGlobalTid()
{
    // Simple helper function for calculating a 1D coordinate
    // from 1, 2 or 3 dimensional coordinates.
    int threadsPerBlock = blockDim.x * blockDim.y * blockDim.z;
    int blocksBefore = blockIdx.z * (gridDim.y * gridDim.x)
	+ blockIdx.y * gridDim.x
	+ blockIdx.x;
    int threadsBefore = blockDim.y * blockDim.x * threadIdx.z + blockDim.x * threadIdx.y;
    int tid = blocksBefore * threadsPerBlock + threadsBefore + threadIdx.x;

    return tid;
}

__global__
void cubble::assignDataToBubbles(float *x,
				 float *y,
#if (NUM_DIM == 3)
				 float *z,
#endif
				 float *r,
				 Bubble *b,
				 dvec lbb,
				 dvec tfr,
				 int numBubbles)
{
    int tid = getGlobalTid();
    if (tid < numBubbles)
    {
	dvec pos;
        pos.x = (double)x[tid];
	pos.y = (double)y[tid];
#if (NUM_DIM == 3)
	pos.z = (double)z[tid];
#endif	
	// Scale position
	pos = pos * tfr + lbb;
	
	b[tid].setPos(pos);
	b[tid].setRadius((double)r[tid]);
    }
}

__global__
void cubble::calculateOffsets(Bubble *bubbles,
			      Cell *cells,
			      dvec lbb,
			      dvec tfr,
			      int numBubbles)
{   
    int tid = getGlobalTid();
    dvec invInterval = 1.0 / (tfr - lbb);
    
    if (tid < numBubbles)
    {
	dvec pos = (bubbles[tid].getPos() - lbb) * invInterval;
	ivec indexVec(gridDim.x * pos.x, gridDim.y * pos.y, gridDim.z * pos.z);
	int index = gridDim.x * gridDim.y * indexVec.z
	    + gridDim.x * indexVec.y
	    + indexVec.x;

	fvec color = fvec(indexVec.x / (float)gridDim.x,
			  indexVec.y / (float)gridDim.y,
			  indexVec.z / (float)gridDim.z);
	bubbles[tid].setCellIndex(index);
        bubbles[tid].setColor(color);
	
	atomicAdd(&cells[index].offset, 1);
    }
}


__global__
void cubble::assignBubblesToCells(Bubble *bubbles,
				  int *indices,
				  Cell *cells,
				  int numBubbles)
{
    int tid = getGlobalTid();

    if (tid < numBubbles)
    {
	int index = bubbles[tid].getCellIndex();
	int offset = cells[index].offset + atomicAdd(&cells[index].size, 1);
        indices[offset] = bubbles[tid].getCellIndex();
    }
}

__global__
void cubble::findIntersections(Bubble *bubbles,
			       int *indices,
			       Cell *cells,
			       int *intersectingIndices,
			       int numBubbles,
			       int numDomains)
{
    extern __shared__ Bubble localBubbles[];
    ivec cellIdx(blockIdx.x, blockIdx.y, blockIdx.z / numDomains);
    ivec boxDim(gridDim.x, gridDim.y, gridDim.z / numDomains);
    int domain = blockIdx.z % numDomains;
    int halfNumDomains = numDomains / 2;
    int di = domain / halfNumDomains;
    int dj = domain % halfNumDomains;
    int djMod2 = dj % 2;

    __shared__ Cell neighbor;
    __shared__ Cell self;
    if (threadIdx.x == 0)
    {
	int temp = cellIdx.z * boxDim.x * boxDim.y + cellIdx.y * boxDim.x + cellIdx.x;
	self = cells[temp];
	neighbor = cells[getNeighborCell(cellIdx, boxDim, dj / 2)];
    }
    __syncthreads();

    int xBegin = neighbor.offset;
    xBegin += djMod2 * neighbor.size * 0.5f;
    int xEnd = xBegin + neighbor.size * (0.5f + 0.5f * djMod2);
    int xInterval = xEnd - xBegin;

    int yBegin = self.offset;
    yBegin += di * self.size * 0.5f;
    int yEnd = YBegin + self.size * (0.5f + 0.5f * di);
    int yInterval = yEnd - yBegin;

    if (threadIdx.x < xInterval && xBegin + threadIdx.x < numBubbles)
	localBubbles[threadIdx.x] = bubbles[indices[xBegin + threadIdx.x]];
    else if (threadIdx.x < xInterval + yInterval && yBegin + threadIdx.x < numBubbles)
	localBubbles[threadIdx.x] = bubbles[indices[yBegin + threadIdx.x]];

    __syncthreads();

    int numPairs = xInterval * yInterval;
    int numRounds = numPairs / blockDim.x + 1;

    for (int round = 0; round < numRounds; ++round)
    {
        int tempIdx = round * blockDim.x + threadIdx.x;
	if (tempIdx < numPairs)
	{
	    int x = tempIdx % xInterval;
	    int y = tempIdx / xInterval;

	    if (x > xInterval)
		printf("", );
	    
	    if (y > yInterval)
		printf("", );
	    
	    Bubble *b1 = &localBubbles[x];
	    Bubble *b2 = &localBubbles[xInterval + y];

	    // Skip self-intersections
	    if (b1 == b2)
	    {
		THIS DOES NOT DETECT SAME BUBBLE CORRECTLY, SINCE THERE ARE DUPLICATES.
		continue;
	    }
	    
	    double radii = b1->getRadius() + b2->getRadius();
	    if (radii * radii < (b1->getPos() - b2->getPos).getSquaredLength())
		intersectingIndices[indices[xBegin + x]] = 1;
	}
    }
}
