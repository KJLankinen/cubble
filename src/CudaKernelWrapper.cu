#include "hip/hip_runtime.h"
// -*- C++ -*-

#include "CudaKernelWrapper.h"
#include "Macros.h"
#include "CudaContainer.h"

#include <iostream>
#include <hiprand.h>

cubble::CudaKernelWrapper::CudaKernelWrapper(std::shared_ptr<BubbleManager> bm,
					     std::shared_ptr<Env> e)
{
    bubbleManager = bm;
    env = e;

    printRelevantInfoOfCurrentDevice();
}

cubble::CudaKernelWrapper::~CudaKernelWrapper()
{}

void cubble::CudaKernelWrapper::generateBubbles(std::vector<Bubble> &outBubbles)
{
    // Get necessary parameters
    int n = env->getNumBubbles();
    int rngSeed = env->getRngSeed();
    double avgRad = env->getAvgRad();
    double stdDevRad = env->getStdDevRad();
    dvec lbb = env->getLbb();
    dvec tfr = env->getTfr();	

    int numThreads = 1024;
    int numBlocks = (int)std::ceil(n / (float)numThreads);
    
    CudaContainer<float> x(n);
    CudaContainer<float> y(n);
    CudaContainer<float> r(n);
    
    CudaContainer<Bubble> b(n);

    // Generate random positions & radii
    hiprandGenerator_t generator;
    CURAND_CALL(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_MTGP32));
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(generator, rngSeed));
    CURAND_CALL(hiprandGenerateUniform(generator, x.getDevPtr(), n));
    CURAND_CALL(hiprandGenerateUniform(generator, y.getDevPtr(), n));
    CURAND_CALL(hiprandGenerateNormal(generator, r.getDevPtr(), n, avgRad, stdDevRad));
    
#if (NUM_DIM == 3)
    CudaContainer<float> z(n);
    CURAND_CALL(hiprandGenerateUniform(generator, z.getDevPtr(), n));
#endif
    
    // Assign generated data to bubbles
    assignDataToBubbles<<<numBlocks, numThreads>>>(x.getDevPtr(),
						 y.getDevPtr(),
#if (NUM_DIM == 3)
						 z.getDevPtr(),
#endif
						 r.getDevPtr(),
						 b.getDevPtr(),
						 lbb,
						 tfr,
						 n);
    CUDA_CALL(hipPeekAtLastError());
    CUDA_CALL(hipDeviceSynchronize());

    b.copyDeviceDataToVec(outBubbles);
}

void cubble::CudaKernelWrapper::assignBubblesToCells(const std::vector<Bubble> &b)
{
    int numBubblesPerCell = env->getNumBubblesPerCell();
    dvec tfr = env->getTfr();
    dvec lbb = env->getLbb();

#if NUM_DIM == 3
    int numCellsPerDim = (int)std::ceil(std::cbrt(b.size() / numBubblesPerCell));
    dim3 gridSize(numCellsPerDim, numCellsPerDim, numCellsPerDim);
#else
    int numCellsPerDim = (int)std::ceil(std::sqrt(b.size() / numBubblesPerCell));
    dim3 gridSize(numCellsPerDim, numCellsPerDim, 1);
#endif

    int numCells = gridSize.x * gridSize.y * gridSize.z;

    std::cout << "Grid size: (" << gridSize.x
	      << ", " << gridSize.y
	      << ", " << gridSize.z
	      << "), numCells: " << numCells
	      << ", size of bubbles: " << sizeof(Bubble) * b.size()
	      << std::endl;
    
    CudaContainer<Bubble> bubbles(b);
    CudaContainer<int> offsets(numCells);

    offsets.fillHostWith(0);
    offsets.toDevice();
    bubbles.toDevice();

    calculateOffsets<<<gridSize, numBubblesPerCell>>>(bubbles.getDevPtr(),
						      offsets.getDevPtr(),
						      lbb,
						      tfr,
						      bubbles.size());

    offsets.toHost();

    int cumulativeSum = 0;
    for (size_t i = 0; i < offsets.size(); ++i)
    {
	int numBubbles = offsets[i];
	offsets[i] = cumulativeSum;
	cumulativeSum += numBubbles;
    }
    offsets.toDevice();

    CudaContainer<int> currentIndex(offsets.size());
    CudaContainer<Bubble> reorganizedBubbles(bubbles.size());
    
    currentIndex.fillHostWith(0);
    currentIndex.toDevice();

    reorganizeBubbles<<<gridSize,numBubblesPerCell>>>(bubbles.getDevPtr(),
						      reorganizedBubbles.getDevPtr(),
						      offsets.getDevPtr(),
						      currentIndex.getDevPtr(),
						      bubbles.size());

    bubbleManager->setBubblesFromDevice(reorganizedBubbles);
    bubbleManager->setOffsetsFromDevice(offsets);
}

__forceinline__ __device__
int cubble::getGlobalTid()
{
    // Simple helper function for calculating a 1D coordinate
    // from 1, 2 or 3 dimensional coordinates.
    int threadsPerBlock = blockDim.x * blockDim.y * blockDim.z;
    int blocksBefore = blockIdx.z * (gridDim.y * gridDim.x)
	+ blockIdx.y * gridDim.x
	+ blockIdx.x;
    int threadsBefore = blockDim.y * blockDim.x * threadIdx.z + blockDim.x * threadIdx.y;
    int tid = blocksBefore * threadsPerBlock + threadsBefore + threadIdx.x;

    return tid;
}

__global__
void cubble::assignDataToBubbles(float *x,
				 float *y,
#if (NUM_DIM == 3)
				 float *z,
#endif
				 float *r,
				 Bubble *b,
				 dvec lbb,
				 dvec tfr,
				 int numBubbles)
{
    int tid = getGlobalTid();
    if (tid < numBubbles)
    {
	dvec pos;
        pos.x = (double)x[tid];
	pos.y = (double)y[tid];
#if (NUM_DIM == 3)
	pos.z = (double)z[tid];
#endif	
	// Scale position
	pos = pos * tfr + lbb;
	
	b[tid].setPos(pos);
	b[tid].setRadius((double)r[tid]);
    }
}

__global__
void cubble::calculateOffsets(Bubble *bubbles,
			      int *offsets,
			      dvec lbb,
			      dvec tfr,
			      int numBubbles)
{   
    int tid = getGlobalTid();
    dvec invInterval = 1.0 / (tfr - lbb);
    
    if (tid < numBubbles)
    {
	dvec pos = (bubbles[tid].getPos() - lbb) * invInterval;
	ivec indexVec(gridDim.x * pos.x, gridDim.y * pos.y, gridDim.z * pos.z);
	int index = gridDim.x * gridDim.y * indexVec.z
	    + gridDim.x * indexVec.y
	    + indexVec.x;

	fvec color = fvec(indexVec.x / (float)gridDim.x,
			  indexVec.y / (float)gridDim.y,
			  indexVec.z / (float)gridDim.z);
	bubbles[tid].setCellIndex(index);
        bubbles[tid].setColor(color);
	
	atomicAdd(&offsets[index], 1);
    }
}


__global__
void cubble::reorganizeBubbles(Bubble *bubbles,
			       Bubble *reorganizedBubbles,
			       int *offsets,
			       int *currentIndex,
			       int numBubbles)
{
    int tid = getGlobalTid();

    if (tid < numBubbles)
    {
	int index = bubbles[tid].getCellIndex();
	int offset = offsets[index] + atomicAdd(&currentIndex[index], 1);
	reorganizedBubbles[offset] = bubbles[tid];
    }
}
