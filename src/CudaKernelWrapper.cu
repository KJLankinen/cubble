#include "hip/hip_runtime.h"
// -*- C++ -*-

#include "CudaKernelWrapper.h"
#include "Macros.h"
#include "CudaContainer.h"

#include <iostream>
#include <hiprand.h>

cubble::CudaKernelWrapper::CudaKernelWrapper(std::shared_ptr<BubbleManager> bm,
					     std::shared_ptr<Env> e)
{
    bubbleManager = bm;
    env = e;
}

cubble::CudaKernelWrapper::~CudaKernelWrapper()
{}

void cubble::CudaKernelWrapper::generateBubblesOnGPU()
{
    // Get necessary parameters
    int n = env->getNumBubbles();
    int numBlocksPerDim = env->getNumCellsPerDim();
    int rngSeed = env->getRngSeed();
    double avgRad = env->getAvgRad();
    double stdDevRad = env->getStdDevRad();
    dvec lbb = env->getLbb();
    dvec tfr = env->getTfr();
	
    int totalNumBlocks = numBlocksPerDim * numBlocksPerDim;
#if (NUM_DIM == 3)
    totalNumBlocks *= numBlocksPerDim;
    int numThreadsPerDim = (int)std::ceil(std::cbrt(n / (float)totalNumBlocks));
    dim3 blockSize = dim3(numThreadsPerDim * numThreadsPerDim * numThreadsPerDim, 1, 1);
    dim3 gridSize = dim3(numBlocksPerDim, numBlocksPerDim, numBlocksPerDim);
#else
    int numThreadsPerDim = (int)std::ceil(std::sqrt(n / (float)totalNumBlocks));
    dim3 blockSize = dim3(numThreadsPerDim * numThreadsPerDim, 1, 1);
    dim3 gridSize = dim3(numBlocksPerDim, numBlocksPerDim, 1);
#endif
    int minNumCells = (int)std::ceil(n / 1024.0f);
    int numCells = gridSize.x * gridSize.y * gridSize.z;
    int numThreads = blockSize.x * blockSize.y * blockSize.z;
    if (numThreads > 1024)
    {
	std::cerr << "Too many bubbles to simulate w.r.t. the number of blocks."
		  << "\nAmount of threads per block " << numThreads
		  << " exceeds the maximum, which is 1024."
		  << "\nIncrease the number of blocks (cells) per dimension."
		  << "\nMinimum (total) number of cells with given number of bubbles: "
		  << minNumCells
		  << std::endl;
	
	std::exit(EXIT_FAILURE);
    }

    std::cout << "Grid size: (" << gridSize.x
	      << ", " << gridSize.y
	      << ", " << gridSize.z
	      << "), block size: (" << blockSize.x
	      << ", " << blockSize.y
	      << ", " << blockSize.z
	      << ")\nMinimum (total) number of cells with given number of bubbles: "
	      << minNumCells
	      << std::endl;
    
    CudaContainer<float> x(n);
    CudaContainer<float> y(n);
#if (NUM_DIM == 3)
    CudaContainer<float> z(n);
#endif
    CudaContainer<float> r(n);
    CudaContainer<Bubble> b(n);
    CudaContainer<int> bubbleIndices(n);
    CudaContainer<int> bubblesPerCell(numCells);
    CudaContainer<int> offsets(numCells);

    hiprandGenerator_t generator;
    CURAND_CALL(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_MTGP32));
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(generator, rngSeed));
    CURAND_CALL(hiprandGenerateUniform(generator, x.getDevicePtr(), n));
    CURAND_CALL(hiprandGenerateUniform(generator, y.getDevicePtr(), n));
    CURAND_CALL(hiprandGenerateNormal(generator, r.getDevicePtr(), n, avgRad, stdDevRad));
#if (NUM_DIM == 3)
    CURAND_CALL(hiprandGenerateUniform(generator, z.getDevicePtr(), n));
#endif

    // Assign generated data to bubbles
    assignDataToBubbles<<<gridSize, blockSize>>>(x.getDevicePtr(),
						 y.getDevicePtr(),
#if (NUM_DIM == 3)
						 z.getDevicePtr(),
#endif
						 r.getDevicePtr(),
						 b.getDevicePtr(),
						 bubblesPerCell.getDevicePtr(),
						 lbb,
						 tfr,
						 n);
    bubblesPerCell.toHost();
    bubbleManager->cellEnds.resize(numCells);
    bubbleManager->cellBegins.resize(numCells);

    // Bubbles are stored in a 1D array, even if the domain is 2D or 3D.
    // Here we calculate the offsets (begin and end indices) to the memory
    // location where the bubbles of the current cell start (and end).
    int offset = 0;
    for (size_t i = 0; i < numCells; ++i)
    {
	offsets[i] = offset;
	bubbleManager->cellBegins[i] = offset;
	offset += bubblesPerCell[i];
	bubbleManager->cellEnds[i] = offset;
    }
    offsets.toDevice();

    // Reset bubblesPerCell to 0, and reuse it.
    bubblesPerCell.fillHostWith(0);
    bubblesPerCell.toDevice();

    assignBubblesToCells<<<gridSize, blockSize>>>(b.getDevicePtr(),
						  bubbleIndices.getDevicePtr(),
						  offsets.getDevicePtr(),
						  bubblesPerCell.getDevicePtr(),
						  lbb,
						  tfr,
						  n);
    bubbleIndices.toHost();
    b.toHost();

    // Create a new host vector and add the bubbles in to it
    // s.t. the bubbles of any one cell are continuously stored.
    // Note that the memory locality could probably be improved
    // for nearby cells by saving the cells as a Hilbert curve,
    // instead of left to right, bottom to top, back to front.
    std::vector<Bubble> bubbles;
    bubbles.resize(n);
    for (size_t i = 0; i < n; ++i)
	bubbles[i] = b[bubbleIndices[i]];

    bubbleManager->setBubbles(bubbles);
}

__forceinline__ __device__
int getTid()
{
    // Simple helper function for calculating a 1D coordinate
    // from 1, 2 or 3 dimensional coordinates.
    int threadsPerBlock = blockDim.x * blockDim.y * blockDim.z;
    int blocksBefore = blockIdx.z * (gridDim.y * gridDim.x)
	+ blockIdx.y * gridDim.x
	+ blockIdx.x;
    int threadsBefore = blockDim.y * blockDim.x * threadIdx.z + blockDim.x * threadIdx.y;
    int tid = blocksBefore * threadsPerBlock + threadsBefore + threadIdx.x;

    return tid;
}

__global__
void cubble::assignDataToBubbles(float *x,
				 float *y,
#if (NUM_DIM == 3)
				 float *z,
#endif
				 float *r,
				 Bubble *b,
				 int *bubblesPerCell,
				 dvec lbb,
				 dvec tfr,
				 int numBubbles)
{
    int tid = getTid();
    if (tid < numBubbles)
    {
	dvec pos;
        pos.x = (double)x[tid];
	pos.y = (double)y[tid];
	
	int bbi = ((int)(pos.y * gridDim.y) * gridDim.x) + (int)(pos.x * gridDim.x);
	
#if (NUM_DIM == 3)
	pos.z = (double)z[tid];
	bbi += ((int)(pos.z * gridDim.z) * gridDim.y * gridDim.x);
#endif
	// Scale position
	pos = pos * tfr + lbb;

	b[tid].setPos(pos);
	b[tid].setRadius((double)r[tid]);
	
        atomicAdd(&bubblesPerCell[bbi], 1);
    }
}

__global__
void cubble::assignBubblesToCells(Bubble *b,
				  int *bubbleIndices,
				  int *offsets,
				  int *currentIndices,
				  dvec lbb,
				  dvec tfr,
				  int numBubbles)
{
    int tid = getTid();
    dvec interval = tfr - lbb;
    dvec normPos = (b[tid].getPos() - lbb) / interval;

    if (tid < numBubbles)
    {
	int bbi = ((int)(normPos.y * gridDim.y) * gridDim.x) + (int)(normPos.x * gridDim.x);
#if (NUM_DIM == 3)
	bbi += (int)(normPos.z * gridDim.z) * gridDim.y * gridDim.x;
#endif
	int offset = atomicAdd(&currentIndices[bbi], 1) + offsets[bbi];
	bubbleIndices[offset] = tid;
    }
}
