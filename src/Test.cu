#include "hip/hip_runtime.h"
// -*- C++ -*-

#include "Test.h"

#include <vector>
#include <numeric>
#include <iostream>

#define CUDA_CALL(x)							\
    do									\
    {									\
	hipError_t result = x;						\
	if (result != hipSuccess)					\
	{								\
	    std::cerr << "Error at " << __FILE__ << ":" << __LINE__;	\
	    std::cerr << hipGetErrorName(result) << "\n"		\
		      << hipGetErrorString(result) << std::endl;	\
	}								\
    }									\
    while(0)

__global__
void cubble::testFunction(float *a, float *b)
{
    int tid = threadIdx.x;
    printf("Asd asd asd asd from thread %d", tid);
    b[tid] = a[tid] * a[tid];
}

void cubble::Test::testFunctionWrapper()
{
    const size_t n = 1024;
    std::vector<float> a(n);
    std::vector<float> b;
    b.resize(a.size());
    std::iota(a.begin(), a.end(), 0);
    /*
    for (auto it : a)
	std::cout << it << " ";
    
    std::cout << std::endl;
    */
    float *d_a, *d_b;
    CUDA_CALL(hipMalloc((void**)&d_a, n * sizeof(float)));
    CUDA_CALL(hipMalloc((void**)&d_b, n * sizeof(float)));

    CUDA_CALL(hipMemcpy((void*)d_a, (void*)a.data(), n * sizeof(float), hipMemcpyHostToDevice));

    std::cout << "Before" << std::endl;
    cubble::testFunction<<<1, n>>>(d_a, d_b);
    CUDA_CALL(hipMemcpy((void*)b.data(), (void*)d_b, n * sizeof(float), hipMemcpyDeviceToHost));
    std::cout << "After" << std::endl;

    /*
    for (auto it : b)
	std::cout << it << " ";
    
    std::cout << std::endl;
    */
    CUDA_CALL(hipFree(d_a));
    CUDA_CALL(hipFree(d_b));
}
