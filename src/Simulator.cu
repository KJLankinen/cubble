#include "hip/hip_runtime.h"
#include "CubWrapper.h"
#include "Kernels.cuh"
#include "Util.h"
#include "Vec.h"
#include "cub/hipcub/hipcub.hpp"
#include "nlohmann/json.hpp"
#include <array>
#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <fstream>
#include <iostream>
#include <nvToolsExt.h>
#include <sstream>
#include <string>
#include <vector>

namespace cubble {
// Device double pointer names
enum class DDP {
    X,
    Y,
    Z,
    R,

    XP,
    YP,
    ZP,
    RP,

    DXDT,
    DYDT,
    DZDT,
    DRDT,

    DXDTP,
    DYDTP,
    DZDTP,
    DRDTP,

    DXDTO,
    DYDTO,
    DZDTO,
    DRDTO,

    X0,
    Y0,
    Z0,

    PATH,
    DISTANCE,
    ERROR,
    TEMP_DATA,

    FLOW_VX,
    FLOW_VY,
    FLOW_VZ,

    SAVED_X,
    SAVED_Y,
    SAVED_Z,
    SAVED_R,

    NUM_VALUES
};

// Device int pointer names
enum class DIP {
    TEMP,

    WRAP_COUNT_X,
    WRAP_COUNT_Y,
    WRAP_COUNT_Z,

    INDEX,
    NUM_NEIGHBORS,

    PAIR1,
    PAIR2,

    PAIR1COPY,
    PAIR2COPY,

    NUM_VALUES
};

struct SimulationState {
    dvec lbb = dvec(0.0, 0.0, 0.0);
    dvec tfr = dvec(0.0, 0.0, 0.0);
    dvec interval = dvec(0.0, 0.0, 0.0);
    dvec flowLbb = dvec(0.0, 0.0, 0.0);
    dvec flowTfr = dvec(0.0, 0.0, 0.0);
    dvec flowVel = dvec(0.0, 0.0, 0.0);
    double timeStep = 0.0;
    double averageSurfaceAreaIn = 0.0;
    double avgRad = 0.0;
    double minRad = 0.0;
    double fZeroPerMuZero = 0.0;
    double kParameter = 0.0;
    double kappa = 0.0;
    double wallDragStrength = 0.0;
    double skinRadius = 0.0;

    uint64_t memReqD = 0;
    uint64_t memReqI = 0;
    uint64_t numIntegrationSteps = 0;
    uint64_t numNeighborsSearched = 0;
    uint64_t numStepsInTimeStep = 0;
    uint64_t timeInteger = 0;
    double timeFraction = 0.0;
    double energy1 = 0.0;
    double energy2 = 0.0;
    double maxBubbleRadius = 0.0;
    double timeScalingFactor = 0.0;
    double errorTolerance = 0.0;
    double snapshotFrequency = 0.0;

    int numBubbles = 0;
    int numPairs = 0;

    int minNumBubbles = 0;
    uint32_t numSnapshots = 0;
    uint32_t timesPrinted = 0;
    uint32_t dataStride = 0;
    uint32_t pairStride = 0;
};

struct Params {
    SimulationState state;
    CubWrapper cw;

    hipStream_t velocityStream;
    hipStream_t gasStream;

    hipEvent_t event1;

    KernelSize pairKernelSize = KernelSize(dim3(1024, 1, 1), dim3(128, 1, 1));
    KernelSize defaultKernelSize;

    // Device memory & arrays of pointers to those memory chunks.
    int *deviceIntMemory = nullptr;
    double *deviceDoubleMemory = nullptr;
    int *pinnedInt = nullptr;
    double *pinnedDouble = nullptr;
    std::array<double *, (uint64_t)DDP::NUM_VALUES> ddps;
    std::array<int *, (uint64_t)DIP::NUM_VALUES> dips;

    std::vector<double> previousX;
    std::vector<double> previousY;
    std::vector<double> previousZ;

    int *numToBeDeleted = nullptr;
};

} // namespace cubble

namespace // anonymous
{
using namespace hipcubble;

#if (USE_PROFILING == 1)
void startProfiling(bool start) {
    if (start) {
        CUDA_CALL(hipProfilerStart());
    }
}

void stopProfiling(bool stop, bool &continueIntegration) {
    if (stop) {
        CUDA_CALL(hipDeviceSynchronize());
        CUDA_CALL(hipProfilerStop());
        continueIntegration = false;
    }
}
#endif

void updateCellsAndNeighbors(Params &params) {
    NVTX_RANGE_PUSH_A("Neighbors");
    params.state.numNeighborsSearched++;
    // Boundary wrap
    KERNEL_LAUNCH(wrapKernel, params.pairKernelSize, 0, 0,
                  params.state.numBubbles, params.state.lbb, params.state.tfr,
                  params.ddps[(uint32_t)DDP::X], params.ddps[(uint32_t)DDP::Y],
                  params.ddps[(uint32_t)DDP::Z],
                  params.dips[(uint32_t)DIP::WRAP_COUNT_X],
                  params.dips[(uint32_t)DIP::WRAP_COUNT_Y],
                  params.dips[(uint32_t)DIP::WRAP_COUNT_Z]);

    // Update saved values
    CUDA_CALL(hipMemcpyAsync(
        static_cast<void *>(params.ddps[(uint32_t)DDP::SAVED_X]),
        static_cast<void *>(params.ddps[(uint32_t)DDP::X]),
        sizeof(double) * params.state.dataStride, hipMemcpyDeviceToDevice, 0));
    CUDA_CALL(hipMemcpyAsync(
        static_cast<void *>(params.ddps[(uint32_t)DDP::SAVED_Y]),
        static_cast<void *>(params.ddps[(uint32_t)DDP::Y]),
        sizeof(double) * params.state.dataStride, hipMemcpyDeviceToDevice, 0));
    CUDA_CALL(hipMemcpyAsync(
        static_cast<void *>(params.ddps[(uint32_t)DDP::SAVED_Z]),
        static_cast<void *>(params.ddps[(uint32_t)DDP::Z]),
        sizeof(double) * params.state.dataStride, hipMemcpyDeviceToDevice, 0));
    CUDA_CALL(hipMemcpyAsync(
        static_cast<void *>(params.ddps[(uint32_t)DDP::SAVED_R]),
        static_cast<void *>(params.ddps[(uint32_t)DDP::R]),
        sizeof(double) * params.state.dataStride, hipMemcpyDeviceToDevice, 0));

    // Minimum size of cell is twice the sum of the skin and max bubble radius
    ivec cellDim =
        (params.state.interval /
         (2 * (params.state.maxBubbleRadius + params.state.skinRadius)))
            .floor();
    cellDim.z = cellDim.z > 0 ? cellDim.z : 1;
    dim3 gridSize = dim3(cellDim.x, cellDim.y, cellDim.z);

    // Determine the maximum number of Morton numbers for the simulation box
    const int maxGridSize =
        gridSize.x > gridSize.y
            ? (gridSize.x > gridSize.z ? gridSize.x : gridSize.z)
            : (gridSize.y > gridSize.z ? gridSize.y : gridSize.z);
    int maxNumCells = 1;
    while (maxNumCells < maxGridSize)
        maxNumCells = maxNumCells << 1;

    if (NUM_DIM == 3)
        maxNumCells = maxNumCells * maxNumCells * maxNumCells;
    else
        maxNumCells = maxNumCells * maxNumCells;

    std::cout << "Max num cells: " << maxNumCells << ", grid size: ("
              << gridSize.x << ", " << gridSize.y << ", " << gridSize.z
              << "), avg num bubbles per cell: "
              << params.state.numBubbles /
                     (gridSize.x * gridSize.y * gridSize.z)
              << std::endl;

    int *offsets = params.dips[(uint32_t)DIP::PAIR1];
    int *sizes = params.dips[(uint32_t)DIP::PAIR1] + maxNumCells;
    int *cellIndices =
        params.dips[(uint32_t)DIP::PAIR1COPY] + 0 * params.state.dataStride;
    int *bubbleIndices =
        params.dips[(uint32_t)DIP::PAIR1COPY] + 1 * params.state.dataStride;
    int *sortedCellIndices =
        params.dips[(uint32_t)DIP::PAIR1COPY] + 2 * params.state.dataStride;
    int *sortedBubbleIndices =
        params.dips[(uint32_t)DIP::PAIR1COPY] + 3 * params.state.dataStride;

    const uint64_t resetBytes =
        sizeof(int) * params.state.pairStride *
        ((uint64_t)DIP::NUM_VALUES - (uint64_t)DIP::PAIR1);
    CUDA_CALL(hipMemset(params.dips[(uint32_t)DIP::PAIR1], 0, resetBytes));

    // Reset number of neighbors to zero as they will be calculated again
    CUDA_CALL(hipMemset(params.dips[(uint32_t)DIP::NUM_NEIGHBORS], 0,
                         sizeof(int) * params.state.dataStride));

    KERNEL_LAUNCH(assignBubblesToCells, params.pairKernelSize, 0, 0,
                  params.ddps[(uint32_t)DDP::X], params.ddps[(uint32_t)DDP::Y],
                  params.ddps[(uint32_t)DDP::Z], cellIndices, bubbleIndices,
                  params.state.lbb, params.state.tfr, cellDim,
                  params.state.numBubbles);

    params.cw.sortPairs<int, int>(
        &hipcub::DeviceRadixSort::SortPairs, const_cast<const int *>(cellIndices),
        sortedCellIndices, const_cast<const int *>(bubbleIndices),
        sortedBubbleIndices, params.state.numBubbles);

    params.cw.histogram<int *, int, int, int>(
        &hipcub::DeviceHistogram::HistogramEven, cellIndices, sizes,
        maxNumCells + 1, 0, maxNumCells, params.state.numBubbles);

    params.cw.scan<int *, int *>(&hipcub::DeviceScan::ExclusiveSum, sizes, offsets,
                                 maxNumCells);

    auto copyAndSwap = [](Params &params, int *inds, auto &&arr, uint32_t from,
                          uint32_t to) {
        KERNEL_LAUNCH(copyKernel, params.defaultKernelSize, 0, 0,
                      params.state.numBubbles, ReorganizeType::COPY_FROM_INDEX,
                      inds, inds, arr[from], arr[to]);

        auto *swapper = arr[from];
        arr[from] = arr[to];
        arr[to] = swapper;
    };

    copyAndSwap(params, sortedBubbleIndices, params.ddps, (uint32_t)DDP::X,
                (uint32_t)DDP::TEMP_DATA);
    copyAndSwap(params, sortedBubbleIndices, params.ddps, (uint32_t)DDP::Y,
                (uint32_t)DDP::TEMP_DATA);
    copyAndSwap(params, sortedBubbleIndices, params.ddps, (uint32_t)DDP::Z,
                (uint32_t)DDP::TEMP_DATA);
    copyAndSwap(params, sortedBubbleIndices, params.ddps, (uint32_t)DDP::R,
                (uint32_t)DDP::TEMP_DATA);
    copyAndSwap(params, sortedBubbleIndices, params.ddps, (uint32_t)DDP::DXDT,
                (uint32_t)DDP::TEMP_DATA);
    copyAndSwap(params, sortedBubbleIndices, params.ddps, (uint32_t)DDP::DYDT,
                (uint32_t)DDP::TEMP_DATA);
    copyAndSwap(params, sortedBubbleIndices, params.ddps, (uint32_t)DDP::DZDT,
                (uint32_t)DDP::TEMP_DATA);
    copyAndSwap(params, sortedBubbleIndices, params.ddps, (uint32_t)DDP::DRDT,
                (uint32_t)DDP::TEMP_DATA);
    copyAndSwap(params, sortedBubbleIndices, params.ddps, (uint32_t)DDP::DXDTO,
                (uint32_t)DDP::TEMP_DATA);
    copyAndSwap(params, sortedBubbleIndices, params.ddps, (uint32_t)DDP::DYDTO,
                (uint32_t)DDP::TEMP_DATA);
    copyAndSwap(params, sortedBubbleIndices, params.ddps, (uint32_t)DDP::DZDTO,
                (uint32_t)DDP::TEMP_DATA);
    copyAndSwap(params, sortedBubbleIndices, params.ddps, (uint32_t)DDP::DRDTO,
                (uint32_t)DDP::TEMP_DATA);
    copyAndSwap(params, sortedBubbleIndices, params.ddps, (uint32_t)DDP::X0,
                (uint32_t)DDP::TEMP_DATA);
    copyAndSwap(params, sortedBubbleIndices, params.ddps, (uint32_t)DDP::Y0,
                (uint32_t)DDP::TEMP_DATA);
    copyAndSwap(params, sortedBubbleIndices, params.ddps, (uint32_t)DDP::Z0,
                (uint32_t)DDP::TEMP_DATA);
    copyAndSwap(params, sortedBubbleIndices, params.ddps, (uint32_t)DDP::PATH,
                (uint32_t)DDP::TEMP_DATA);
    copyAndSwap(params, sortedBubbleIndices, params.ddps,
                (uint32_t)DDP::DISTANCE, (uint32_t)DDP::TEMP_DATA);
    copyAndSwap(params, sortedBubbleIndices, params.ddps,
                (uint32_t)DDP::SAVED_X, (uint32_t)DDP::TEMP_DATA);
    copyAndSwap(params, sortedBubbleIndices, params.ddps,
                (uint32_t)DDP::SAVED_Y, (uint32_t)DDP::TEMP_DATA);
    copyAndSwap(params, sortedBubbleIndices, params.ddps,
                (uint32_t)DDP::SAVED_Z, (uint32_t)DDP::TEMP_DATA);
    copyAndSwap(params, sortedBubbleIndices, params.ddps,
                (uint32_t)DDP::SAVED_R, (uint32_t)DDP::TEMP_DATA);
    copyAndSwap(params, sortedBubbleIndices, params.ddps, (uint32_t)DDP::ERROR,
                (uint32_t)DDP::TEMP_DATA);
    copyAndSwap(params, sortedBubbleIndices, params.dips,
                (uint32_t)DIP::WRAP_COUNT_X, (uint32_t)DIP::TEMP);
    copyAndSwap(params, sortedBubbleIndices, params.dips,
                (uint32_t)DIP::WRAP_COUNT_Y, (uint32_t)DIP::TEMP);
    copyAndSwap(params, sortedBubbleIndices, params.dips,
                (uint32_t)DIP::WRAP_COUNT_Z, (uint32_t)DIP::TEMP);
    copyAndSwap(params, sortedBubbleIndices, params.dips, (uint32_t)DIP::INDEX,
                (uint32_t)DIP::TEMP);

    KernelSize kernelSizeNeighbor = KernelSize(gridSize, dim3(128, 1, 1));

    int *dnp = nullptr;
    CUDA_ASSERT(
        hipGetSymbolAddress(reinterpret_cast<void **>(&dnp), dNumPairs));
    CUDA_CALL(hipMemset(dnp, 0, sizeof(int)));

    for (int i = 0; i < CUBBLE_NUM_NEIGHBORS + 1; ++i) {
        hipStream_t stream =
            (i % 2) ? params.velocityStream : params.gasStream;
        KERNEL_LAUNCH(neighborSearch, kernelSizeNeighbor, 0, stream, i,
                      params.state.numBubbles, maxNumCells,
                      (int)params.state.pairStride, params.state.skinRadius,
                      offsets, sizes, params.dips[(uint32_t)DIP::PAIR1COPY],
                      params.dips[(uint32_t)DIP::PAIR2COPY],
                      params.ddps[(uint32_t)DDP::R], params.state.interval,
                      params.ddps[(uint32_t)DDP::X],
                      params.ddps[(uint32_t)DDP::Y],
                      params.ddps[(uint32_t)DDP::Z],
                      params.dips[(uint32_t)DIP::NUM_NEIGHBORS]);
    }

    CUDA_CALL(hipMemcpy(static_cast<void *>(&params.state.numPairs),
                         static_cast<void *>(dnp), sizeof(int),
                         hipMemcpyDeviceToHost));

    params.cw.sortPairs<int, int>(
        &hipcub::DeviceRadixSort::SortPairs,
        const_cast<const int *>(params.dips[(uint32_t)DIP::PAIR1COPY]),
        params.dips[(uint32_t)DIP::PAIR1],
        const_cast<const int *>(params.dips[(uint32_t)DIP::PAIR2COPY]),
        params.dips[(uint32_t)DIP::PAIR2], params.state.numPairs);
    NVTX_RANGE_POP();
}

void deleteSmallBubbles(Params &params, int numToBeDeleted) {
    NVTX_RANGE_PUSH_A("BubbleRemoval");

    KERNEL_LAUNCH(
        swapDataCountPairs, params.pairKernelSize, 0, 0,
        params.state.numBubbles, params.state.minRad,
        params.dips[(uint32_t)DIP::PAIR1], params.dips[(uint32_t)DIP::PAIR2],
        params.dips[(uint32_t)DIP::TEMP], params.ddps[(uint32_t)DDP::R],
        params.ddps[(uint32_t)DDP::X], params.ddps[(uint32_t)DDP::Y],
        params.ddps[(uint32_t)DDP::Z], params.ddps[(uint32_t)DDP::DXDT],
        params.ddps[(uint32_t)DDP::DYDT], params.ddps[(uint32_t)DDP::DZDT],
        params.ddps[(uint32_t)DDP::DRDT], params.ddps[(uint32_t)DDP::DXDTO],
        params.ddps[(uint32_t)DDP::DYDTO], params.ddps[(uint32_t)DDP::DZDTO],
        params.ddps[(uint32_t)DDP::DRDTO], params.ddps[(uint32_t)DDP::X0],
        params.ddps[(uint32_t)DDP::Y0], params.ddps[(uint32_t)DDP::Z0],
        params.ddps[(uint32_t)DDP::PATH], params.ddps[(uint32_t)DDP::DISTANCE],
        params.ddps[(uint32_t)DDP::SAVED_X],
        params.ddps[(uint32_t)DDP::SAVED_Y],
        params.ddps[(uint32_t)DDP::SAVED_Z],
        params.ddps[(uint32_t)DDP::SAVED_R], params.ddps[(uint32_t)DDP::ERROR],
        params.dips[(uint32_t)DIP::WRAP_COUNT_X],
        params.dips[(uint32_t)DIP::WRAP_COUNT_Y],
        params.dips[(uint32_t)DIP::WRAP_COUNT_Z],
        params.dips[(uint32_t)DIP::INDEX],
        params.dips[(uint32_t)DIP::NUM_NEIGHBORS]);

    KERNEL_LAUNCH(
        addVolumeFixPairs, params.pairKernelSize, 0, 0, params.state.numBubbles,
        params.dips[(uint32_t)DIP::PAIR1], params.dips[(uint32_t)DIP::PAIR2],
        params.dips[(uint32_t)DIP::TEMP], params.ddps[(uint32_t)DDP::R]);

    // Update kernel sizes based on number of remaining bubbles
    params.state.numBubbles -= numToBeDeleted;
    params.defaultKernelSize = KernelSize(128, params.state.numBubbles);
    int numBlocks =
        std::min(1024, (int)std::ceil(params.state.numBubbles / 128.0));
    params.pairKernelSize = KernelSize(dim3(numBlocks, 1, 1), dim3(128, 1, 1));

    NVTX_RANGE_POP();
}

void saveSnapshotToFile(Params &params) {
    // Calculate total energy
    KERNEL_LAUNCH(resetKernel, params.defaultKernelSize, 0, 0, 0.0,
                  params.state.numBubbles,
                  params.ddps[(uint32_t)DDP::TEMP_DATA]);

    KERNEL_LAUNCH(potentialEnergyKernel, params.pairKernelSize, 0, 0,
                  params.state.numBubbles, params.dips[(uint32_t)DIP::PAIR1],
                  params.dips[(uint32_t)DIP::PAIR2],
                  params.ddps[(uint32_t)DDP::R],
                  params.ddps[(uint32_t)DDP::TEMP_DATA], params.state.interval,
                  params.ddps[(uint32_t)DDP::X], params.ddps[(uint32_t)DDP::Y],
                  params.ddps[(uint32_t)DDP::Z]);

    std::stringstream ss;
    ss << "snapshot.csv." << params.state.numSnapshots;
    std::ofstream file(ss.str().c_str(), std::ios::out);
    if (file.is_open()) {
        std::vector<double> doubleData;
        doubleData.resize(params.state.dataStride * (uint32_t)DDP::NUM_VALUES);
        for (uint32_t i = 0; i < (uint32_t)DDP::NUM_VALUES; ++i) {
            CUDA_CALL(hipMemcpy(&doubleData[i * params.state.dataStride],
                                 params.ddps[i],
                                 sizeof(double) * params.state.dataStride,
                                 hipMemcpyDeviceToHost));
        }

        std::vector<int> intData;
        intData.resize(params.state.dataStride);
        CUDA_CALL(hipMemcpy(intData.data(), params.dips[(uint32_t)DIP::INDEX],
                             sizeof(intData[0]) * intData.size(),
                             hipMemcpyDeviceToHost));

        if (params.state.numSnapshots == 0) {
            for (uint64_t i = 0; i < (uint64_t)params.state.numBubbles; ++i) {
                params.previousX[intData[i]] =
                    doubleData[i + 0 * params.state.dataStride];
                params.previousY[intData[i]] =
                    doubleData[i + 1 * params.state.dataStride];
                params.previousZ[intData[i]] =
                    doubleData[i + 2 * params.state.dataStride];
            }
        }

        file << "x,y,z,r,vx,vy,vz,vtot,vr,path,distance,energy,displacement,"
                "error,index\n ";
        for (uint64_t i = 0; i < (uint64_t)params.state.numBubbles; ++i) {
            const double x =
                doubleData[i + (uint32_t)DDP::X * params.state.dataStride];
            const double y =
                doubleData[i + (uint32_t)DDP::Y * params.state.dataStride];
            const double z =
                doubleData[i + (uint32_t)DDP::Z * params.state.dataStride];
            const double r =
                doubleData[i + (uint32_t)DDP::R * params.state.dataStride];
            const double vx =
                doubleData[i + (uint32_t)DDP::DXDT * params.state.dataStride];
            const double vy =
                doubleData[i + (uint32_t)DDP::DYDT * params.state.dataStride];
            const double vz =
                doubleData[i + (uint32_t)DDP::DZDT * params.state.dataStride];
            const double vr =
                doubleData[i + (uint32_t)DDP::DRDT * params.state.dataStride];
            const double path =
                doubleData[i + (uint32_t)DDP::PATH * params.state.dataStride];
            const double distance = doubleData[i + (uint32_t)DDP::DISTANCE *
                                                       params.state.dataStride];
            const double error =
                doubleData[i + (uint32_t)DDP::ERROR * params.state.dataStride];
            const double energy = doubleData[i + (uint32_t)DDP::TEMP_DATA *
                                                     params.state.dataStride];
            const double px = params.previousX[intData[i]];
            const double py = params.previousY[intData[i]];
            const double pz = params.previousZ[intData[i]];

            double displX = abs(x - px);
            displX = displX > 0.5 * params.state.interval.x
                         ? displX - params.state.interval.x
                         : displX;
            double displY = abs(y - py);
            displY = displY > 0.5 * params.state.interval.y
                         ? displY - params.state.interval.y
                         : displY;
            double displZ = abs(z - pz);
            displZ = displZ > 0.5 * params.state.interval.z
                         ? displZ - params.state.interval.z
                         : displZ;

            file << x;
            file << ",";
            file << y;
            file << ",";
            file << z;
            file << ",";
            file << r;
            file << ",";
            file << vx;
            file << ",";
            file << vy;
            file << ",";
            file << vz;
            file << ",";
            file << sqrt(vx * vx + vy * vy + vz * vz);
            file << ",";
            file << vr;
            file << ",";
            file << path;
            file << ",";
            file << distance;
            file << ",";
            file << energy;
            file << ",";
            file << sqrt(displX * displX + displY * displY + displZ * displZ);
            file << ",";
            file << error;
            file << ",";
            file << intData[i + 0 * params.state.dataStride];
            file << "\n";

            params.previousX[intData[i]] = x;
            params.previousY[intData[i]] = y;
            params.previousZ[intData[i]] = z;
        }

        ++params.state.numSnapshots;
    }
}

double stabilize(Params &params, int numStepsToRelax) {
    // This function integrates only the positions of the bubbles.
    // Gas exchange is not used. This is used for equilibrating the foam.

    double elapsedTime = 0.0;
    double error = 100000;

    // Energy before stabilization
    KERNEL_LAUNCH(resetKernel, params.defaultKernelSize, 0, 0, 0.0,
                  params.state.numBubbles,
                  params.ddps[(uint32_t)DDP::TEMP_DATA]);

    KERNEL_LAUNCH(potentialEnergyKernel, params.pairKernelSize, 0, 0,
                  params.state.numBubbles, params.dips[(uint32_t)DIP::PAIR1],
                  params.dips[(uint32_t)DIP::PAIR2],
                  params.ddps[(uint32_t)DDP::R],
                  params.ddps[(uint32_t)DDP::TEMP_DATA], params.state.interval,
                  params.ddps[(uint32_t)DDP::X], params.ddps[(uint32_t)DDP::Y],
                  params.ddps[(uint32_t)DDP::Z]);

    params.state.energy1 = params.cw.reduce<double, double *, double *>(
        &hipcub::DeviceReduce::Sum, params.ddps[(uint32_t)DDP::TEMP_DATA],
        params.state.numBubbles);

    for (int i = 0; i < numStepsToRelax; ++i) {
        do {
            KERNEL_LAUNCH(resetKernel, params.defaultKernelSize, 0, 0, 0.0,
                          params.state.numBubbles,
                          params.ddps[(uint32_t)DDP::DXDTP],
                          params.ddps[(uint32_t)DDP::DYDTP],
                          params.ddps[(uint32_t)DDP::DZDTP]);

            KERNEL_LAUNCH(
                predictKernel, params.pairKernelSize, 0, 0,
                params.state.numBubbles, params.state.timeStep, false,
                params.ddps[(uint32_t)DDP::XP], params.ddps[(uint32_t)DDP::X],
                params.ddps[(uint32_t)DDP::DXDT],
                params.ddps[(uint32_t)DDP::DXDTO],
                params.ddps[(uint32_t)DDP::YP], params.ddps[(uint32_t)DDP::Y],
                params.ddps[(uint32_t)DDP::DYDT],
                params.ddps[(uint32_t)DDP::DYDTO],
                params.ddps[(uint32_t)DDP::ZP], params.ddps[(uint32_t)DDP::Z],
                params.ddps[(uint32_t)DDP::DZDT],
                params.ddps[(uint32_t)DDP::DZDTO],
                params.ddps[(uint32_t)DDP::RP], params.ddps[(uint32_t)DDP::R],
                params.ddps[(uint32_t)DDP::DRDT],
                params.ddps[(uint32_t)DDP::DRDTO]);

            KERNEL_LAUNCH(
                velocityPairKernel, params.pairKernelSize, 0, 0,
                params.state.fZeroPerMuZero, params.dips[(uint32_t)DIP::PAIR1],
                params.dips[(uint32_t)DIP::PAIR2],
                params.ddps[(uint32_t)DDP::R], params.state.interval,
                params.ddps[(uint32_t)DDP::XP], params.ddps[(uint32_t)DDP::YP],
                params.ddps[(uint32_t)DDP::ZP],
                params.ddps[(uint32_t)DDP::DXDTP],
                params.ddps[(uint32_t)DDP::DYDTP],
                params.ddps[(uint32_t)DDP::DZDTP]);

#if (PBC_X == 0 || PBC_Y == 0 || PBC_Z == 0)
            KERNEL_LAUNCH(
                velocityWallKernel, params.pairKernelSize, 0,
                params.velocityStream, params.state.numBubbles,
                params.ddps[(uint32_t)DDP::R], params.ddps[(uint32_t)DDP::XP],
                params.ddps[(uint32_t)DDP::YP], params.ddps[(uint32_t)DDP::ZP],
                params.ddps[(uint32_t)DDP::DXDTP],
                params.ddps[(uint32_t)DDP::DYDTP],
                params.ddps[(uint32_t)DDP::DZDTP], params.state.lbb,
                params.state.tfr, params.state.fZeroPerMuZero,
                params.state.wallDragStrength);
#endif
            // Correct
            KERNEL_LAUNCH(
                correctKernel, params.pairKernelSize, 0, 0,
                params.state.numBubbles, params.state.timeStep, false,
                params.state.minRad, params.ddps[(uint32_t)DDP::ERROR],
                params.ddps[(uint32_t)DDP::TEMP_DATA],
                params.dips[(uint32_t)DIP::TEMP],
                params.ddps[(uint32_t)DDP::XP], params.ddps[(uint32_t)DDP::X],
                params.ddps[(uint32_t)DDP::DXDT],
                params.ddps[(uint32_t)DDP::DXDTP],
                params.ddps[(uint32_t)DDP::YP], params.ddps[(uint32_t)DDP::Y],
                params.ddps[(uint32_t)DDP::DYDT],
                params.ddps[(uint32_t)DDP::DYDTP],
                params.ddps[(uint32_t)DDP::ZP], params.ddps[(uint32_t)DDP::Z],
                params.ddps[(uint32_t)DDP::DZDT],
                params.ddps[(uint32_t)DDP::DZDTP],
                params.ddps[(uint32_t)DDP::RP], params.ddps[(uint32_t)DDP::R],
                params.ddps[(uint32_t)DDP::DRDT],
                params.ddps[(uint32_t)DDP::DRDTP],
                params.ddps[(uint32_t)DDP::SAVED_X],
                params.ddps[(uint32_t)DDP::SAVED_Y],
                params.ddps[(uint32_t)DDP::SAVED_Z],
                params.ddps[(uint32_t)DDP::SAVED_R]);

            KERNEL_LAUNCH(endStepKernel, params.pairKernelSize, 0,
                          params.gasStream, params.state.numBubbles,
                          params.ddps[(uint32_t)DDP::TEMP_DATA],
                          params.ddps[(uint32_t)DDP::SAVED_X],
                          params.ddps[(uint32_t)DDP::SAVED_Y],
                          params.ddps[(uint32_t)DDP::SAVED_Z],
                          params.ddps[(uint32_t)DDP::SAVED_R],
                          (int)params.pairKernelSize.grid.x);

            CUDA_CALL(hipMemcpyAsync(
                static_cast<void *>(params.pinnedDouble),
                params.ddps[(uint32_t)DDP::TEMP_DATA], 3 * sizeof(double),
                hipMemcpyDeviceToHost, params.gasStream));

            CUDA_CALL(hipEventRecord(params.event1, params.gasStream));

            // Error
            // Wait for event
            CUDA_CALL(hipEventSynchronize(params.event1));
            error = params.pinnedDouble[0];

            if (error < params.state.errorTolerance &&
                params.state.timeStep < 0.1)
                params.state.timeStep *= 1.9;
            else if (error > params.state.errorTolerance)
                params.state.timeStep *= 0.5;

        } while (error > params.state.errorTolerance);

        // Update the current values with the calculated predictions
        double *swapper = params.ddps[(uint32_t)DDP::DXDTO];
        params.ddps[(uint32_t)DDP::DXDTO] = params.ddps[(uint32_t)DDP::DXDT];
        params.ddps[(uint32_t)DDP::DXDT] = params.ddps[(uint32_t)DDP::DXDTP];
        params.ddps[(uint32_t)DDP::DXDTP] = swapper;

        swapper = params.ddps[(uint32_t)DDP::DYDTO];
        params.ddps[(uint32_t)DDP::DYDTO] = params.ddps[(uint32_t)DDP::DYDT];
        params.ddps[(uint32_t)DDP::DYDT] = params.ddps[(uint32_t)DDP::DYDTP];
        params.ddps[(uint32_t)DDP::DYDTP] = swapper;

        swapper = params.ddps[(uint32_t)DDP::DZDTO];
        params.ddps[(uint32_t)DDP::DZDTO] = params.ddps[(uint32_t)DDP::DZDT];
        params.ddps[(uint32_t)DDP::DZDT] = params.ddps[(uint32_t)DDP::DZDTP];
        params.ddps[(uint32_t)DDP::DZDTP] = swapper;

        swapper = params.ddps[(uint32_t)DDP::X];
        params.ddps[(uint32_t)DDP::X] = params.ddps[(uint32_t)DDP::XP];
        params.ddps[(uint32_t)DDP::XP] = swapper;

        swapper = params.ddps[(uint32_t)DDP::Y];
        params.ddps[(uint32_t)DDP::Y] = params.ddps[(uint32_t)DDP::YP];
        params.ddps[(uint32_t)DDP::YP] = swapper;

        swapper = params.ddps[(uint32_t)DDP::Z];
        params.ddps[(uint32_t)DDP::Z] = params.ddps[(uint32_t)DDP::ZP];
        params.ddps[(uint32_t)DDP::ZP] = swapper;

        elapsedTime += params.state.timeStep;

        if (2 * params.pinnedDouble[2] >= params.state.skinRadius) {
            updateCellsAndNeighbors(params);
        }
    }

    // Energy after stabilization
    KERNEL_LAUNCH(resetKernel, params.defaultKernelSize, 0, 0, 0.0,
                  params.state.numBubbles,
                  params.ddps[(uint32_t)DDP::TEMP_DATA]);

    KERNEL_LAUNCH(potentialEnergyKernel, params.pairKernelSize, 0, 0,
                  params.state.numBubbles, params.dips[(uint32_t)DIP::PAIR1],
                  params.dips[(uint32_t)DIP::PAIR2],
                  params.ddps[(uint32_t)DDP::R],
                  params.ddps[(uint32_t)DDP::TEMP_DATA], params.state.interval,
                  params.ddps[(uint32_t)DDP::X], params.ddps[(uint32_t)DDP::Y],
                  params.ddps[(uint32_t)DDP::Z]);

    params.state.energy2 = params.cw.reduce<double, double *, double *>(
        &hipcub::DeviceReduce::Sum, params.ddps[(uint32_t)DDP::TEMP_DATA],
        params.state.numBubbles);

    return elapsedTime;
}

void velocityCalculation(Params &params) {
    // Velocity
    KERNEL_LAUNCH(
        velocityPairKernel, params.pairKernelSize, 0, params.velocityStream,
        params.state.fZeroPerMuZero, params.dips[(uint32_t)DIP::PAIR1],
        params.dips[(uint32_t)DIP::PAIR2], params.ddps[(uint32_t)DDP::RP],
        params.state.interval, params.ddps[(uint32_t)DDP::XP],
        params.ddps[(uint32_t)DDP::YP], params.ddps[(uint32_t)DDP::ZP],
        params.ddps[(uint32_t)DDP::DXDTP], params.ddps[(uint32_t)DDP::DYDTP],
        params.ddps[(uint32_t)DDP::DZDTP]);

    // Flow velocity
#if (USE_FLOW == 1)
    {
        KERNEL_LAUNCH(neighborVelocityKernel, params.pairKernelSize, 0,
                      params.velocityStream, params.dips[(uint32_t)DIP::PAIR1],
                      params.dips[(uint32_t)DIP::PAIR2],
                      params.ddps[(uint32_t)DDP::FLOW_VX],
                      params.ddps[(uint32_t)DDP::FLOW_VY],
                      params.ddps[(uint32_t)DDP::FLOW_VZ],
                      params.ddps[(uint32_t)DDP::DXDTO],
                      params.ddps[(uint32_t)DDP::DYDTO],
                      params.ddps[(uint32_t)DDP::DZDTO]);

        KERNEL_LAUNCH(
            flowVelocityKernel, params.pairKernelSize, 0, params.velocityStream,
            params.state.numBubbles, params.dips[(uint32_t)DIP::NUM_NEIGHBORS],
            params.ddps[(uint32_t)DDP::DXDTP],
            params.ddps[(uint32_t)DDP::DYDTP],
            params.ddps[(uint32_t)DDP::DZDTP],
            params.ddps[(uint32_t)DDP::FLOW_VX],
            params.ddps[(uint32_t)DDP::FLOW_VY],
            params.ddps[(uint32_t)DDP::FLOW_VZ], params.ddps[(uint32_t)DDP::XP],
            params.ddps[(uint32_t)DDP::YP], params.ddps[(uint32_t)DDP::ZP],
            params.ddps[(uint32_t)DDP::RP], params.state.flowVel,
            params.state.flowTfr, params.state.flowLbb);
    }
#endif

#if (PBC_X == 0 || PBC_Y == 0 || PBC_Z == 0)
    // Wall velocity, should be after flow so that possible drag is applied
    // correctly
    KERNEL_LAUNCH(
        velocityWallKernel, params.pairKernelSize, 0, params.velocityStream,
        params.state.numBubbles, params.ddps[(uint32_t)DDP::RP],
        params.ddps[(uint32_t)DDP::XP], params.ddps[(uint32_t)DDP::YP],
        params.ddps[(uint32_t)DDP::ZP], params.ddps[(uint32_t)DDP::DXDTP],
        params.ddps[(uint32_t)DDP::DYDTP], params.ddps[(uint32_t)DDP::DZDTP],
        params.state.lbb, params.state.tfr, params.state.fZeroPerMuZero,
        params.state.wallDragStrength);
#endif
}

void gasExchangeCalculation(Params &params) {
    // Gas exchange
    KERNEL_LAUNCH(
        gasExchangeKernel, params.pairKernelSize, 0, params.gasStream,
        params.state.numBubbles, params.dips[(uint32_t)DIP::PAIR1],
        params.dips[(uint32_t)DIP::PAIR2], params.state.interval,
        params.ddps[(uint32_t)DDP::RP], params.ddps[(uint32_t)DDP::DRDTP],
        params.ddps[(uint32_t)DDP::TEMP_DATA], params.ddps[(uint32_t)DDP::XP],
        params.ddps[(uint32_t)DDP::YP], params.ddps[(uint32_t)DDP::ZP]);

    KERNEL_LAUNCH(finalRadiusChangeRateKernel, params.pairKernelSize, 0,
                  params.gasStream, params.ddps[(uint32_t)DDP::DRDTP],
                  params.ddps[(uint32_t)DDP::RP],
                  params.ddps[(uint32_t)DDP::TEMP_DATA],
                  params.state.numBubbles, params.state.kappa,
                  params.state.kParameter, params.state.averageSurfaceAreaIn);
}

bool integrate(Params &params) {
    NVTX_RANGE_PUSH_A("Integration function");

    double error = 100000;
    uint32_t numLoopsDone = 0;

    do {
        NVTX_RANGE_PUSH_A("Integration step");

        // Reset
        KERNEL_LAUNCH(resetKernel, params.defaultKernelSize, 0,
                      params.velocityStream, 0.0, params.state.numBubbles,
                      params.ddps[(uint32_t)DDP::DXDTP],
                      params.ddps[(uint32_t)DDP::DYDTP],
                      params.ddps[(uint32_t)DDP::DZDTP],
                      params.ddps[(uint32_t)DDP::DRDTP],
                      params.ddps[(uint32_t)DDP::TEMP_DATA],
                      params.ddps[(uint32_t)DDP::FLOW_VX],
                      params.ddps[(uint32_t)DDP::FLOW_VY],
                      params.ddps[(uint32_t)DDP::FLOW_VZ]);

        // Predict
        KERNEL_LAUNCH(
            predictKernel, params.pairKernelSize, 0, params.gasStream,
            params.state.numBubbles, params.state.timeStep, true,
            params.ddps[(uint32_t)DDP::XP], params.ddps[(uint32_t)DDP::X],
            params.ddps[(uint32_t)DDP::DXDT], params.ddps[(uint32_t)DDP::DXDTO],
            params.ddps[(uint32_t)DDP::YP], params.ddps[(uint32_t)DDP::Y],
            params.ddps[(uint32_t)DDP::DYDT], params.ddps[(uint32_t)DDP::DYDTO],
            params.ddps[(uint32_t)DDP::ZP], params.ddps[(uint32_t)DDP::Z],
            params.ddps[(uint32_t)DDP::DZDT], params.ddps[(uint32_t)DDP::DZDTO],
            params.ddps[(uint32_t)DDP::RP], params.ddps[(uint32_t)DDP::R],
            params.ddps[(uint32_t)DDP::DRDT],
            params.ddps[(uint32_t)DDP::DRDTO]);

        CUDA_CALL(hipEventRecord(params.event1, params.gasStream));
        gasExchangeCalculation(params);
        CUDA_CALL(hipStreamWaitEvent(params.velocityStream, params.event1, 0));
        velocityCalculation(params);

        // Correct
        KERNEL_LAUNCH(
            correctKernel, params.pairKernelSize, 0, 0, params.state.numBubbles,
            params.state.timeStep, true, params.state.minRad,
            params.ddps[(uint32_t)DDP::ERROR],
            params.ddps[(uint32_t)DDP::FLOW_VX],
            params.dips[(uint32_t)DIP::TEMP], params.ddps[(uint32_t)DDP::XP],
            params.ddps[(uint32_t)DDP::X], params.ddps[(uint32_t)DDP::DXDT],
            params.ddps[(uint32_t)DDP::DXDTP], params.ddps[(uint32_t)DDP::YP],
            params.ddps[(uint32_t)DDP::Y], params.ddps[(uint32_t)DDP::DYDT],
            params.ddps[(uint32_t)DDP::DYDTP], params.ddps[(uint32_t)DDP::ZP],
            params.ddps[(uint32_t)DDP::Z], params.ddps[(uint32_t)DDP::DZDT],
            params.ddps[(uint32_t)DDP::DZDTP], params.ddps[(uint32_t)DDP::RP],
            params.ddps[(uint32_t)DDP::R], params.ddps[(uint32_t)DDP::DRDT],
            params.ddps[(uint32_t)DDP::DRDTP],
            params.ddps[(uint32_t)DDP::SAVED_X],
            params.ddps[(uint32_t)DDP::SAVED_Y],
            params.ddps[(uint32_t)DDP::SAVED_Z],
            params.ddps[(uint32_t)DDP::SAVED_R]);

        // Copy numToBeDeleted to pinned memory
        CUDA_CALL(hipMemcpyAsync(static_cast<void *>(params.pinnedInt),
                                  static_cast<void *>(params.numToBeDeleted),
                                  sizeof(int), hipMemcpyDeviceToHost,
                                  params.gasStream));

        KERNEL_LAUNCH(endStepKernel, params.pairKernelSize, 0,
                      params.velocityStream, params.state.numBubbles,
                      params.ddps[(uint32_t)DDP::FLOW_VX],
                      params.ddps[(uint32_t)DDP::SAVED_X],
                      params.ddps[(uint32_t)DDP::SAVED_Y],
                      params.ddps[(uint32_t)DDP::SAVED_Z],
                      params.ddps[(uint32_t)DDP::SAVED_R],
                      (int)params.pairKernelSize.grid.x);

        // Copy maximum error, maximum radius and maximum boundary expansion to
        // pinned memory. See correctKernel and endStepKernel for details.
        CUDA_CALL(hipMemcpyAsync(static_cast<void *>(params.pinnedDouble),
                                  params.ddps[(uint32_t)DDP::FLOW_VX],
                                  3 * sizeof(double), hipMemcpyDeviceToHost,
                                  params.velocityStream));

        CUDA_CALL(hipEventRecord(params.event1, params.velocityStream));

        // Path lenghts & distances
        KERNEL_LAUNCH(
            pathLengthDistanceKernel, params.pairKernelSize, 0,
            params.gasStream, params.state.numBubbles, params.state.interval,
            params.ddps[(uint32_t)DDP::TEMP_DATA],
            params.ddps[(uint32_t)DDP::PATH],
            params.ddps[(uint32_t)DDP::DISTANCE],
            params.ddps[(uint32_t)DDP::XP], params.ddps[(uint32_t)DDP::X],
            params.ddps[(uint32_t)DDP::X0],
            params.dips[(uint32_t)DIP::WRAP_COUNT_X],
            params.ddps[(uint32_t)DDP::YP], params.ddps[(uint32_t)DDP::Y],
            params.ddps[(uint32_t)DDP::Y0],
            params.dips[(uint32_t)DIP::WRAP_COUNT_Y],
            params.ddps[(uint32_t)DDP::ZP], params.ddps[(uint32_t)DDP::Z],
            params.ddps[(uint32_t)DDP::Z0],
            params.dips[(uint32_t)DIP::WRAP_COUNT_Z]);

        // Wait for event
        CUDA_CALL(hipEventSynchronize(params.event1));

        error = params.pinnedDouble[0];
        if (error < params.state.errorTolerance && params.state.timeStep < 0.1)
            params.state.timeStep *= 1.9;
        else if (error > params.state.errorTolerance)
            params.state.timeStep *= 0.5;

        ++numLoopsDone;

        NVTX_RANGE_POP();
    } while (error > params.state.errorTolerance);

    // Update values
    double *swapper = params.ddps[(uint32_t)DDP::DXDTO];
    params.ddps[(uint32_t)DDP::DXDTO] = params.ddps[(uint32_t)DDP::DXDT];
    params.ddps[(uint32_t)DDP::DXDT] = params.ddps[(uint32_t)DDP::DXDTP];
    params.ddps[(uint32_t)DDP::DXDTP] = swapper;

    swapper = params.ddps[(uint32_t)DDP::DYDTO];
    params.ddps[(uint32_t)DDP::DYDTO] = params.ddps[(uint32_t)DDP::DYDT];
    params.ddps[(uint32_t)DDP::DYDT] = params.ddps[(uint32_t)DDP::DYDTP];
    params.ddps[(uint32_t)DDP::DYDTP] = swapper;

    swapper = params.ddps[(uint32_t)DDP::DZDTO];
    params.ddps[(uint32_t)DDP::DZDTO] = params.ddps[(uint32_t)DDP::DZDT];
    params.ddps[(uint32_t)DDP::DZDT] = params.ddps[(uint32_t)DDP::DZDTP];
    params.ddps[(uint32_t)DDP::DZDTP] = swapper;

    swapper = params.ddps[(uint32_t)DDP::DRDTO];
    params.ddps[(uint32_t)DDP::DRDTO] = params.ddps[(uint32_t)DDP::DRDT];
    params.ddps[(uint32_t)DDP::DRDT] = params.ddps[(uint32_t)DDP::DRDTP];
    params.ddps[(uint32_t)DDP::DRDTP] = swapper;

    swapper = params.ddps[(uint32_t)DDP::X];
    params.ddps[(uint32_t)DDP::X] = params.ddps[(uint32_t)DDP::XP];
    params.ddps[(uint32_t)DDP::XP] = swapper;

    swapper = params.ddps[(uint32_t)DDP::Y];
    params.ddps[(uint32_t)DDP::Y] = params.ddps[(uint32_t)DDP::YP];
    params.ddps[(uint32_t)DDP::YP] = swapper;

    swapper = params.ddps[(uint32_t)DDP::Z];
    params.ddps[(uint32_t)DDP::Z] = params.ddps[(uint32_t)DDP::ZP];
    params.ddps[(uint32_t)DDP::ZP] = swapper;

    swapper = params.ddps[(uint32_t)DDP::R];
    params.ddps[(uint32_t)DDP::R] = params.ddps[(uint32_t)DDP::RP];
    params.ddps[(uint32_t)DDP::RP] = swapper;

    swapper = params.ddps[(uint32_t)DDP::PATH];
    params.ddps[(uint32_t)DDP::PATH] = params.ddps[(uint32_t)DDP::TEMP_DATA];
    params.ddps[(uint32_t)DDP::TEMP_DATA] = swapper;

    ++params.state.numIntegrationSteps;

    // As the total simulation time can reach very large numbers as the
    // simulation goes on it's better to keep track of the time as two separate
    // values. One large integer for the integer part and a double that is
    // <= 1.0 to which the potentially very small timeStep gets added. This
    // keeps the precision of the time relatively constant even when the
    // simulation has run a long time.
    params.state.timeFraction += params.state.timeStep;
    params.state.timeInteger += (uint64_t)params.state.timeFraction;
    params.state.timeFraction =
        params.state.timeFraction - (uint64_t)params.state.timeFraction;

    params.state.maxBubbleRadius = params.pinnedDouble[1];

    // Delete, if there are nonzero amount of bubbles with a radius
    // smaller than the minimum radius. See correctKernel for the
    // comparison & calculation.
    if (params.pinnedInt[0] > 0) {
        deleteSmallBubbles(params, params.pinnedInt[0]);
    }

    // If the boundary of the bubble with maximum sum of movement & expansion
    // has moved more than half of the "skin radius", reorder bubbles.
    // See correctKernel, comparePair for details.
    if (params.pinnedDouble[2] >= 0.5 * params.state.skinRadius) {
        updateCellsAndNeighbors(params);
    }

    bool continueSimulation =
        params.state.numBubbles > params.state.minNumBubbles;
    continueSimulation &=
        (NUM_DIM == 3)
            ? params.state.maxBubbleRadius <
                  0.5 * (params.state.tfr - params.state.lbb).getMinComponent()
            : true;

    NVTX_RANGE_POP();

    return continueSimulation;
}

void transformPositions(Params &params, bool normalize) {
    KERNEL_LAUNCH(transformPositionsKernel, params.pairKernelSize, 0, 0,
                  normalize, params.state.numBubbles, params.state.lbb,
                  params.state.tfr, params.ddps[(uint32_t)DDP::X],
                  params.ddps[(uint32_t)DDP::Y], params.ddps[(uint32_t)DDP::Z]);
}

double calculateVolumeOfBubbles(Params &params) {
    KERNEL_LAUNCH(calculateVolumes, params.pairKernelSize, 0, 0,
                  params.ddps[(uint32_t)DDP::R],
                  params.ddps[(uint32_t)DDP::TEMP_DATA],
                  params.state.numBubbles);

    return params.cw.reduce<double, double *, double *>(
        &hipcub::DeviceReduce::Sum, params.ddps[(uint32_t)DDP::TEMP_DATA],
        params.state.numBubbles);
}

void deinit(Params &params) {
    CUDA_CALL(hipDeviceSynchronize());

    CUDA_CALL(hipFree(static_cast<void *>(params.deviceDoubleMemory)));
    CUDA_CALL(hipFree(static_cast<void *>(params.deviceIntMemory)));
    CUDA_CALL(hipHostFree(static_cast<void *>(params.pinnedInt)));
    CUDA_CALL(hipHostFree(static_cast<void *>(params.pinnedDouble)));

    CUDA_CALL(hipEventDestroy(params.event1));

    CUDA_CALL(hipStreamDestroy(params.velocityStream));
    CUDA_CALL(hipStreamDestroy(params.gasStream));
}

double getSimulationBoxVolume(Params &params) {
    dvec temp = params.state.tfr - params.state.lbb;
    return (NUM_DIM == 3) ? temp.x * temp.y * temp.z : temp.x * temp.y;
}

void commonSetup(Params &params) {
    params.defaultKernelSize = KernelSize(128, params.state.numBubbles);

    // Streams
    CUDA_ASSERT(hipStreamCreate(&params.velocityStream));
    CUDA_ASSERT(hipStreamCreate(&params.gasStream));

    printRelevantInfoOfCurrentDevice();

    CUDA_CALL(hipEventCreate(&params.event1));

    CUDA_CALL(hipGetSymbolAddress(
        reinterpret_cast<void **>(&params.numToBeDeleted), dNumToBeDeleted));

    // Set device globals to zero
    double zero = 0.0;
    int zeroI = 0;
    bool falseB = false;
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dTotalArea), reinterpret_cast<void *>(&zero),
                                 sizeof(double)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(
        dTotalOverlapArea), reinterpret_cast<void *>(&zero), sizeof(double)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dTotalOverlapAreaPerRadius),
                                 reinterpret_cast<void *>(&zero),
                                 sizeof(double)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(
        dTotalAreaPerRadius), reinterpret_cast<void *>(&zero), sizeof(double)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dTotalVolume), reinterpret_cast<void *>(&zero),
                                 sizeof(double)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(
        dVolumeMultiplier), reinterpret_cast<void *>(&zero), sizeof(double)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(
        dErrorEncountered), reinterpret_cast<void *>(&falseB), sizeof(bool)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dNumPairs), reinterpret_cast<void *>(&zeroI),
                                 sizeof(int)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dNumPairsNew), reinterpret_cast<void *>(&zeroI),
                                 sizeof(int)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(
        dNumToBeDeleted), reinterpret_cast<void *>(&zeroI), sizeof(int)));

    std::cout << "Reserving device memory to hold data." << std::endl;

    CUDA_CALL(hipHostMalloc(reinterpret_cast<void **>(&params.pinnedDouble),
                             sizeof(double) * 3));
    CUDA_CALL(hipHostMalloc(reinterpret_cast<void **>(&params.pinnedInt),
                             sizeof(int)));

    // Calculate the length of 'rows'. Will be divisible by 32, as that's the
    // warp size.
    params.state.dataStride =
        params.state.numBubbles +
        !!(params.state.numBubbles % 32) * (32 - params.state.numBubbles % 32);

    // Doubles
    params.state.memReqD = sizeof(double) * (uint64_t)params.state.dataStride *
                           (uint64_t)DDP::NUM_VALUES;
    CUDA_ASSERT(
        hipMalloc(reinterpret_cast<void **>(&params.deviceDoubleMemory),
                   params.state.memReqD));

    for (uint32_t i = 0; i < (uint32_t)DDP::NUM_VALUES; ++i)
        params.ddps[i] =
            params.deviceDoubleMemory + i * params.state.dataStride;

    // Integers
    // It seems to roughly hold that in 3 dimensions the total number of
    // neighbors is < (10 x numBubbles) and in 2D < (3.5 x numBubbles)
    // Note that these numbers depend on the "skin radius", i.e.
    // from how far are the neighbors looked for.
    const uint32_t avgNumNeighbors = (NUM_DIM == 3) ? 24 : 4;
    params.state.pairStride = avgNumNeighbors * params.state.dataStride;

    params.state.memReqI =
        sizeof(int) * (uint64_t)params.state.dataStride *
        ((uint64_t)DIP::PAIR1 +
         avgNumNeighbors * ((uint64_t)DIP::NUM_VALUES - (uint64_t)DIP::PAIR1));
    CUDA_ASSERT(hipMalloc(reinterpret_cast<void **>(&params.deviceIntMemory),
                           params.state.memReqI));

    for (uint32_t i = 0; i < (uint32_t)DIP::PAIR2; ++i)
        params.dips[i] = params.deviceIntMemory + i * params.state.dataStride;

    uint32_t j = 0;
    for (uint32_t i = (uint32_t)DIP::PAIR2; i < (uint32_t)DIP::NUM_VALUES; ++i)
        params.dips[i] = params.dips[(uint32_t)DIP::PAIR1] +
                         avgNumNeighbors * ++j * params.state.dataStride;

    params.previousX.resize(params.state.dataStride);
    params.previousY.resize(params.state.dataStride);
    params.previousZ.resize(params.state.dataStride);

    std::cout << "Memory requirement for data:\n\tdouble: "
              << params.state.memReqD
              << " bytes\n\tint: " << params.state.memReqI
              << " bytes\n\ttotal: "
              << params.state.memReqI + params.state.memReqD << " bytes"
              << std::endl;
}

void generateStartingData(Params &params, ivec bubblesPerDim, double stdDevRad,
                          int rngSeed) {
    std::cout << "Starting to generate data for bubbles." << std::endl;
    const double avgRad = params.state.avgRad;

    hiprandGenerator_t generator;
    CURAND_CALL(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_MTGP32));
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(generator, rngSeed));
    if (NUM_DIM == 3)
        CURAND_CALL(hiprandGenerateUniformDouble(
            generator, params.ddps[(uint32_t)DDP::Z], params.state.numBubbles));
    CURAND_CALL(hiprandGenerateUniformDouble(
        generator, params.ddps[(uint32_t)DDP::X], params.state.numBubbles));
    CURAND_CALL(hiprandGenerateUniformDouble(
        generator, params.ddps[(uint32_t)DDP::Y], params.state.numBubbles));
    CURAND_CALL(hiprandGenerateUniformDouble(
        generator, params.ddps[(uint32_t)DDP::RP], params.state.numBubbles));
    CURAND_CALL(
        hiprandGenerateNormalDouble(generator, params.ddps[(uint32_t)DDP::R],
                                   params.state.numBubbles, avgRad, stdDevRad));
    CURAND_CALL(hiprandDestroyGenerator(generator));

    KERNEL_LAUNCH(assignDataToBubbles, params.pairKernelSize, 0, 0,
                  params.ddps[(uint32_t)DDP::X], params.ddps[(uint32_t)DDP::Y],
                  params.ddps[(uint32_t)DDP::Z], params.ddps[(uint32_t)DDP::R],
                  params.ddps[(uint32_t)DDP::RP],
                  params.dips[(uint32_t)DIP::INDEX], bubblesPerDim,
                  params.state.tfr, params.state.lbb, avgRad,
                  params.state.minRad, params.state.numBubbles);

    params.state.averageSurfaceAreaIn =
        params.cw.reduce<double, double *, double *>(
            &hipcub::DeviceReduce::Sum, params.ddps[(uint32_t)DDP::RP],
            params.state.numBubbles, 0);

    params.state.maxBubbleRadius = params.cw.reduce<double, double *, double *>(
        &hipcub::DeviceReduce::Max, params.ddps[(uint32_t)DDP::R],
        params.state.numBubbles, 0);

    std::cout << "Updating neighbor lists." << std::endl;
    updateCellsAndNeighbors(params);

    // Calculate some initial values which are needed
    // for the two-step Adams-Bashforth-Moulton predictor-corrector method
    KERNEL_LAUNCH(
        resetKernel, params.defaultKernelSize, 0, 0, 0.0,
        params.state.numBubbles, params.ddps[(uint32_t)DDP::DXDTO],
        params.ddps[(uint32_t)DDP::DYDTO], params.ddps[(uint32_t)DDP::DZDTO],
        params.ddps[(uint32_t)DDP::DRDTO], params.ddps[(uint32_t)DDP::DISTANCE],
        params.ddps[(uint32_t)DDP::PATH]);

    std::cout << "Calculating some initial values as a part of setup."
              << std::endl;

    KERNEL_LAUNCH(
        velocityPairKernel, params.pairKernelSize, 0, 0,
        params.state.fZeroPerMuZero, params.dips[(uint32_t)DIP::PAIR1],
        params.dips[(uint32_t)DIP::PAIR2], params.ddps[(uint32_t)DDP::R],
        params.state.interval, params.ddps[(uint32_t)DDP::X],
        params.ddps[(uint32_t)DDP::Y], params.ddps[(uint32_t)DDP::Z],
        params.ddps[(uint32_t)DDP::DXDTO], params.ddps[(uint32_t)DDP::DYDTO],
        params.ddps[(uint32_t)DDP::DZDTO]);

    KERNEL_LAUNCH(
        eulerKernel, params.pairKernelSize, 0, 0, params.state.numBubbles,
        params.state.timeStep, params.ddps[(uint32_t)DDP::X],
        params.ddps[(uint32_t)DDP::DXDTO], params.ddps[(uint32_t)DDP::Y],
        params.ddps[(uint32_t)DDP::DYDTO], params.ddps[(uint32_t)DDP::Z],
        params.ddps[(uint32_t)DDP::DZDTO]);

    KERNEL_LAUNCH(resetKernel, params.defaultKernelSize, 0, 0, 0.0,
                  params.state.numBubbles, params.ddps[(uint32_t)DDP::DXDTO],
                  params.ddps[(uint32_t)DDP::DYDTO],
                  params.ddps[(uint32_t)DDP::DZDTO],
                  params.ddps[(uint32_t)DDP::DRDTO]);

    KERNEL_LAUNCH(
        velocityPairKernel, params.pairKernelSize, 0, 0,
        params.state.fZeroPerMuZero, params.dips[(uint32_t)DIP::PAIR1],
        params.dips[(uint32_t)DIP::PAIR2], params.ddps[(uint32_t)DDP::R],
        params.state.interval, params.ddps[(uint32_t)DDP::X],
        params.ddps[(uint32_t)DDP::Y], params.ddps[(uint32_t)DDP::Z],
        params.ddps[(uint32_t)DDP::DXDTO], params.ddps[(uint32_t)DDP::DYDTO],
        params.ddps[(uint32_t)DDP::DZDTO]);
}

void initializeFromJson(const char *inputFileName, Params &params) {
    std::cout << "\n=====\nSetup\n====="
              << "Reading inputs from file \"" << inputFileName << "\""
              << std::endl;

    nlohmann::json inputJson;
    std::fstream file(inputFileName, std::ios::in);
    if (file.is_open()) {
        file >> inputJson;

        const double mu = inputJson["muZero"];
        assert(mu > 0);
        assert(inputJson["wallDragStrength"] >= 0.0 &&
               inputJson["wallDragStrength"] <= 1.0);

        params.state.avgRad = inputJson["avgRad"];
        params.state.minRad = 0.1 * params.state.avgRad;
        params.state.fZeroPerMuZero =
            (float)inputJson["sigmaZero"] * params.state.avgRad / mu;
        params.state.flowLbb = inputJson["flowLbb"];
        params.state.flowTfr = inputJson["flowTfr"];
        params.state.flowVel = inputJson["flowVel"];
        params.state.flowVel *= params.state.fZeroPerMuZero;
        params.state.kParameter = inputJson["kParameter"];
        params.state.kappa = inputJson["kappa"];
        params.state.skinRadius =
            (float)inputJson["skinRadius"] * params.state.avgRad;
        params.state.timeScalingFactor =
            params.state.kParameter /
            (params.state.avgRad * params.state.avgRad);
        params.state.errorTolerance = inputJson["errorTolerance"];
        params.state.wallDragStrength = inputJson["wallDragStrength"];
        params.state.snapshotFrequency = inputJson["snapshotFrequency"];
        params.state.minNumBubbles = inputJson["minNumBubbles"];
    } else
        throw std::runtime_error("Couldn't open input file!");

    // First calculate the size of the box and the starting number of bubbles
    dvec relDim = inputJson["boxRelDim"];
    assert(relDim.x > 0);
    assert(relDim.y > 0);
    assert(relDim.z > 0);

    relDim = relDim / relDim.x;
    const float d = 2 * params.state.avgRad;
    float x = (float)inputJson["numBubblesIn"] * d * d / relDim.y;
    ivec bubblesPerDim = ivec(0, 0, 0);

    if (NUM_DIM == 3) {
        x = x * d / relDim.z;
        x = std::cbrt(x);
        relDim = relDim * x;
        bubblesPerDim = ivec(std::ceil(relDim.x / d), std::ceil(relDim.y / d),
                             std::ceil(relDim.z / d));
        params.state.numBubbles =
            bubblesPerDim.x * bubblesPerDim.y * bubblesPerDim.z;
    } else {
        x = std::sqrt(x);
        relDim = relDim * x;
        bubblesPerDim =
            ivec(std::ceil(relDim.x / d), std::ceil(relDim.y / d), 0);
        params.state.numBubbles = bubblesPerDim.x * bubblesPerDim.y;
    }

    params.state.tfr = d * bubblesPerDim.asType<double>() + params.state.lbb;
    params.state.interval = params.state.tfr - params.state.lbb;
    params.state.timeStep = inputJson["timeStepIn"];

    // Reserve memory etc.
    commonSetup(params);
    generateStartingData(params, bubblesPerDim, inputJson["stdDevRad"],
                         inputJson["rngSeed"]);

    std::cout << "Letting bubbles settle after they've been created and before "
                 "scaling or stabilization."
              << std::endl;

    for (uint32_t i = 0; i < 3; ++i)
        stabilize(params, inputJson["numStepsToRelax"]);

    const double bubbleVolume = calculateVolumeOfBubbles(params);
    std::cout << "Volume ratios: current: "
              << bubbleVolume / getSimulationBoxVolume(params)
              << ", target: " << inputJson["phiTarget"]
              << "\nScaling the simulation box." << std::endl;

    transformPositions(params, true);

    relDim = inputJson["boxRelDim"];
    relDim.z = (NUM_DIM == 2) ? 1 : relDim.z;
    double t = bubbleVolume /
               ((float)inputJson["phiTarget"] * relDim.x * relDim.y * relDim.z);
    t = (NUM_DIM == 3) ? std::cbrt(t) : std::sqrt(t);
    params.state.tfr = dvec(t, t, t) * relDim;
    params.state.interval = params.state.tfr - params.state.lbb;
    params.state.flowTfr =
        params.state.interval * params.state.flowTfr + params.state.lbb;
    params.state.flowLbb =
        params.state.interval * params.state.flowLbb + params.state.lbb;

    transformPositions(params, false);

    for (uint32_t i = 0; i < 3; ++i)
        stabilize(params, inputJson["numStepsToRelax"]);

    std::cout << "Volume ratios: current: "
              << bubbleVolume / getSimulationBoxVolume(params)
              << ", target: " << inputJson["phiTarget"]
              << "\n\n=============\nStabilization\n=============" << std::endl;

    int numSteps = 0;
    const int failsafe = 500;

    std::cout << std::setw(10) << std::left << "#steps" << std::setw(12)
              << std::left << "dE" << std::setw(15) << std::left << "e1"
              << std::setw(15) << std::left << "e2" << std::setw(5) << std::left
              << "#searches" << std::endl;

    while (true) {
        double time = stabilize(params, inputJson["numStepsToRelax"]);
        double deltaEnergy =
            std::abs(params.state.energy2 - params.state.energy1) / time;
        deltaEnergy *= 0.5 * (float)inputJson["sigmaZero"];

        if (deltaEnergy < inputJson["maxDeltaEnergy"]) {
            std::cout << "Final delta energy " << deltaEnergy << " after "
                      << (numSteps + 1) * (int)inputJson["numStepsToRelax"]
                      << " steps."
                      << "\nEnergy before: " << params.state.energy1
                      << ", energy after: " << params.state.energy2
                      << ", time: " << time * params.state.timeScalingFactor
                      << std::endl;
            break;
        } else if (numSteps > failsafe) {
            std::cout << "Over " << failsafe * (int)inputJson["numStepsToRelax"]
                      << " steps taken and required delta energy not reached."
                      << " Check parameters." << std::endl;
            break;
        } else {
            std::cout << std::setw(10) << std::left
                      << (numSteps + 1) * (int)inputJson["numStepsToRelax"]
                      << std::setw(12) << std::left << std::setprecision(5)
                      << std::scientific << deltaEnergy << std::setw(15)
                      << std::left << std::setprecision(5) << std::fixed
                      << params.state.energy1 << std::setw(15) << std::left
                      << std::setprecision(5) << std::fixed
                      << params.state.energy2 << std::setw(5) << std::left
                      << params.state.numNeighborsSearched << std::endl;
            params.state.numNeighborsSearched = 0;
        }

        ++numSteps;
    }

    // Set starting positions
    // Avoiding batched memset, because the pointers might not be in order
    const uint64_t numBytesToCopy = sizeof(double) * params.state.dataStride;
    CUDA_CALL(hipMemcpy(params.ddps[(uint32_t)DDP::X0],
                         params.ddps[(uint32_t)DDP::X], numBytesToCopy,
                         hipMemcpyDeviceToDevice));
    CUDA_CALL(hipMemcpy(params.ddps[(uint32_t)DDP::Y0],
                         params.ddps[(uint32_t)DDP::Y], numBytesToCopy,
                         hipMemcpyDeviceToDevice));
    CUDA_CALL(hipMemcpy(params.ddps[(uint32_t)DDP::Z0],
                         params.ddps[(uint32_t)DDP::Z], numBytesToCopy,
                         hipMemcpyDeviceToDevice));

    // Reset wrap counts to 0
    // Again avoiding batched memset, because the pointers might not be in order
    CUDA_CALL(hipMemset(params.dips[(uint32_t)DIP::WRAP_COUNT_X], 0,
                         params.state.dataStride * sizeof(int)));

    CUDA_CALL(hipMemset(params.dips[(uint32_t)DIP::WRAP_COUNT_Y], 0,
                         params.state.dataStride * sizeof(int)));

    CUDA_CALL(hipMemset(params.dips[(uint32_t)DIP::WRAP_COUNT_Z], 0,
                         params.state.dataStride * sizeof(int)));

    // Reset temp for energy, and errors since integration starts after this
    KERNEL_LAUNCH(resetKernel, params.defaultKernelSize, 0, 0, 0.0,
                  params.state.numBubbles,
                  params.ddps[(uint32_t)DDP::TEMP_DATA],
                  params.ddps[(uint32_t)DDP::ERROR]);

    // Calculate the energy at starting positions
    KERNEL_LAUNCH(potentialEnergyKernel, params.pairKernelSize, 0, 0,
                  params.state.numBubbles, params.dips[(uint32_t)DIP::PAIR1],
                  params.dips[(uint32_t)DIP::PAIR2],
                  params.ddps[(uint32_t)DDP::R],
                  params.ddps[(uint32_t)DDP::TEMP_DATA], params.state.interval,
                  params.ddps[(uint32_t)DDP::X], params.ddps[(uint32_t)DDP::Y],
                  params.ddps[(uint32_t)DDP::Z]);

    params.state.energy1 = params.cw.reduce<double, double *, double *>(
        &hipcub::DeviceReduce::Sum, params.ddps[(uint32_t)DDP::TEMP_DATA],
        params.state.numBubbles);
    params.state.timeInteger = 0;
    params.state.timeFraction = 0.0;
    params.state.timesPrinted = 1;
    params.state.numIntegrationSteps = 0;
}

} // namespace

namespace cubble {

void run(std::string &&inputFileName) {
    Params params;
    initializeFromJson(inputFileName.c_str(), params);
    if (params.state.snapshotFrequency > 0.0)
        saveSnapshotToFile(params);

    std::cout << "\n==========\nIntegration\n==========" << std::endl;

    std::cout << std::setw(10) << std::left << "T" << std::setw(10) << std::left
              << "phi" << std::setw(10) << std::left << "R" << std::setw(10)
              << std::left << "#b" << std::setw(10) << std::left << "#pairs"
              << std::setw(10) << std::left << "#steps" << std::setw(10)
              << std::left << "#searches" << std::setw(10) << std::left
              << "min ts" << std::setw(10) << std::left << "max ts"
              << std::setw(10) << std::left << "avg ts" << std::endl;

    bool continueIntegration = true;
    double minTimestep = 9999999.9;
    double maxTimestep = -1.0;
    double avgTimestep = 0.0;
    bool resetErrors = false;

    // This is the simulation loop, which runs until (at least) one
    // end condition is met
    while (continueIntegration) {
        // Define two lambas for later use
        auto getSum = [](double *p, Params &params) -> double {
            return params.cw.reduce<double, double *, double *>(
                &hipcub::DeviceReduce::Sum, p, params.state.numBubbles);
        };

        auto getAvg = [getSum](double *p, Params &params) -> double {
            return getSum(p, params) / params.state.numBubbles;
        };

        // Perform one integration step
        continueIntegration = integrate(params);

        // When profiling, we don't want to run the entire simulation until the
        // end, but rather just enough simulation steps to get a representative
        // view of the entire simulation
        CUDA_PROFILER_START(params.state.numIntegrationSteps == 2000);
        CUDA_PROFILER_STOP(params.state.numIntegrationSteps == 12000,
                           continueIntegration);

        // Track timestep
        minTimestep = params.state.timeStep < minTimestep
                          ? params.state.timeStep
                          : minTimestep;
        maxTimestep = params.state.timeStep > maxTimestep
                          ? params.state.timeStep
                          : maxTimestep;
        avgTimestep += params.state.timeStep;

        // Here we compare potentially very large integers (> 10e6) to each
        // other and small doubles (<= 1.0) to each other to preserve precision.
        const double nextPrintTime =
            params.state.timesPrinted / params.state.timeScalingFactor;
        const uint64_t nextPrintTimeInteger = (uint64_t)nextPrintTime;
        const double nextPrintTimeFraction =
            nextPrintTime - nextPrintTimeInteger;

        // Print stuff to stdout at the earliest possible moment
        // when simulation time is larger than scaled time
        if (params.state.timeInteger >= nextPrintTimeInteger &&
            params.state.timeFraction >= nextPrintTimeFraction) {
            // Calculate total energy
            KERNEL_LAUNCH(resetKernel, params.defaultKernelSize, 0, 0, 0.0,
                          params.state.numBubbles,
                          params.ddps[(uint32_t)DDP::TEMP_DATA]);

            KERNEL_LAUNCH(
                potentialEnergyKernel, params.pairKernelSize, 0, 0,
                params.state.numBubbles, params.dips[(uint32_t)DIP::PAIR1],
                params.dips[(uint32_t)DIP::PAIR2],
                params.ddps[(uint32_t)DDP::R],
                params.ddps[(uint32_t)DDP::TEMP_DATA], params.state.interval,
                params.ddps[(uint32_t)DDP::X], params.ddps[(uint32_t)DDP::Y],
                params.ddps[(uint32_t)DDP::Z]);

            params.state.energy2 =
                getSum(params.ddps[(uint32_t)DDP::TEMP_DATA], params);
            const double dE = (params.state.energy2 - params.state.energy1) /
                              params.state.energy2;
            const double relRad =
                getAvg(params.ddps[(uint32_t)DDP::R], params) /
                params.state.avgRad;

            // Add values to data stream
            std::ofstream resultFile("results.dat", std::ios_base::app);
            if (resultFile.is_open()) {
                const double vx =
                    getAvg(params.ddps[(uint32_t)DDP::DXDT], params);
                const double vy =
                    getAvg(params.ddps[(uint32_t)DDP::DYDT], params);
                const double vz =
                    getAvg(params.ddps[(uint32_t)DDP::DZDT], params);
                const double vr =
                    getAvg(params.ddps[(uint32_t)DDP::DRDT], params);

                resultFile << params.state.timesPrinted << " " << relRad << " "
                           << params.state.numBubbles << " "
                           << getAvg(params.ddps[(uint32_t)DDP::PATH], params)
                           << " "
                           << getAvg(params.ddps[(uint32_t)DDP::DISTANCE],
                                     params)
                           << " " << params.state.energy2 << " " << dE << " "
                           << vx << " " << vy << " " << vz << " "
                           << sqrt(vx * vx + vy * vy + vz * vz) << " " << vr
                           << "\n";
            } else {
                std::cout << "Couldn't open file stream to append results to!"
                          << std::endl;
            }

            const double phi = calculateVolumeOfBubbles(params) /
                               getSimulationBoxVolume(params);

            // Print some values
            std::cout << std::setw(10) << std::left << params.state.timesPrinted
                      << std::setw(10) << std::left << std::setprecision(6)
                      << std::fixed << phi << std::setw(10) << std::left
                      << std::setprecision(6) << std::fixed << relRad
                      << std::setw(10) << std::left << params.state.numBubbles
                      << std::setw(10) << std::left << params.state.numPairs
                      << std::setw(10) << std::left
                      << params.state.numStepsInTimeStep << std::setw(10)
                      << std::left << params.state.numNeighborsSearched
                      << std::setw(10) << std::left << minTimestep
                      << std::setw(10) << std::left << maxTimestep
                      << std::setw(10) << std::left
                      << avgTimestep / params.state.numStepsInTimeStep
                      << std::endl;

            ++params.state.timesPrinted;
            params.state.numStepsInTimeStep = 0;
            params.state.energy1 = params.state.energy2;
            params.state.numNeighborsSearched = 0;
            minTimestep = 9999999.9;
            maxTimestep = -1.0;
            avgTimestep = 0.0;
            resetErrors = true;
        }

        // Save snapshot
        if (params.state.snapshotFrequency > 0.0) {
            const double nextSnapshotTime = params.state.numSnapshots /
                                            params.state.snapshotFrequency /
                                            params.state.timeScalingFactor;
            const uint64_t nextSnapshotTimeInteger = (uint64_t)nextSnapshotTime;
            const double nextSnapshotTimeFraction =
                nextSnapshotTime - nextSnapshotTimeInteger;

            if (params.state.timeInteger >= nextSnapshotTimeInteger &&
                params.state.timeFraction >= nextSnapshotTimeFraction)
                saveSnapshotToFile(params);
        }

        if (resetErrors) {
            KERNEL_LAUNCH(resetKernel, params.defaultKernelSize, 0, 0, 0.0,
                          params.state.numBubbles,
                          params.ddps[(uint32_t)DDP::ERROR]);
            resetErrors = false;
        }

        ++params.state.numStepsInTimeStep;
    }

    if (params.state.snapshotFrequency > 0.0)
        saveSnapshotToFile(params);

    deinit(params);
}
} // namespace cubble
