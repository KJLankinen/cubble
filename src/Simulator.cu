#include "hip/hip_runtime.h"
// -*- C++ -*-

#include "Simulator.cuh"
#include "Macros.h"
#include "Vec.h"
#include "Util.h"
#include "BubbleKernels.cuh"
#include "UtilityKernels.cuh"
#include "IntegrationKernels.cuh"

#include "cub/hipcub/hipcub.hpp"

#include <iostream>
#include <sstream>
#include <chrono>
#include <algorithm>
#include <vector>
#include <hiprand.h>

#include <hip/hip_runtime_api.h>
#include <nvToolsExt.h>

namespace cubble
{
typedef BubbleProperty BP;
__device__ double deviceMaxError;
Simulator::Simulator(std::shared_ptr<Env> e)
{
    env = e;
    dvec relDim = env->getBoxRelativeDimensions();
    relDim /= relDim.x;
    const float d = 2 * env->getAvgRad();
#if (NUM_DIM == 3)
    const float x = std::cbrt(env->getNumBubbles() * d * d * d / (relDim.y * relDim.z));
    dvec tfr = relDim * x;
    const ivec bubblesPerDim(std::ceil(tfr.x / d), std::ceil(tfr.y / d), std::ceil(tfr.z / d));
    numBubbles = bubblesPerDim.x * bubblesPerDim.y * bubblesPerDim.z;
#else
    const float x = std::sqrt(env->getNumBubbles() * d * d / relDim.y);
    dvec tfr = relDim * x;
    tfr.z = 0;
    const ivec bubblesPerDim(std::ceil(tfr.x / d), std::ceil(tfr.y / d), 0);
    numBubbles = bubblesPerDim.x * bubblesPerDim.y;
#endif
    numBubblesAboveMinRad = numBubbles;
    bubblesPerDimAtStart = bubblesPerDim;
    tfr = d * bubblesPerDim.asType<double>();
    env->setTfr(tfr + env->getLbb());

    cubWrapper = std::make_shared<CubWrapper>(env, numBubbles);
    bubbleData = FixedSizeDeviceArray<double>(numBubbles, (size_t)BP::NUM_VALUES);
    aboveMinRadFlags = FixedSizeDeviceArray<int>(numBubbles, 2);
    bubbleCellIndices = FixedSizeDeviceArray<int>(numBubbles, 4);

    // TODO: Figure out a more sensible value for this.
    const int maxNumPairs = (CUBBLE_NUM_NEIGHBORS + 1) * env->getNumBubblesPerCell() * numBubbles;
    neighborPairIndices = FixedSizeDeviceArray<int>(maxNumPairs, 2);

    const dim3 gridSize = getGridSize();
    size_t numCells = gridSize.x * gridSize.y * gridSize.z;
    cellData = FixedSizeDeviceArray<int>(numCells, (size_t)CellProperty::NUM_VALUES);

    hostData.resize(bubbleData.getSize(), 0);

    printRelevantInfoOfCurrentDevice();

    CUDA_CALL(hipGetSymbolAddress(&dnp, HIP_SYMBOL(deviceNumPairs)));
    CUDA_CALL(hipGetSymbolAddress(&dtfa, HIP_SYMBOL(deviceTotalFreeArea)));
    CUDA_CALL(hipGetSymbolAddress(&dtfapr, HIP_SYMBOL(deviceTotalFreeAreaPerRadius)));
    CUDA_CALL(hipGetSymbolAddress(&dme, HIP_SYMBOL(deviceMaxError)));
    CUDA_CALL(hipGetSymbolAddress(&dtv, HIP_SYMBOL(deviceTotalVolume)));
    CUDA_CALL(hipGetSymbolAddress(&dvm, HIP_SYMBOL(deviceVolumeMultiplier)));
    assert(dnp != nullptr);
    assert(dtfa != nullptr);
    assert(dtfapr != nullptr);
    assert(dme != nullptr);
    assert(dtv != nullptr);
    assert(dvm != nullptr);
}

Simulator::~Simulator() {}

void Simulator::setupSimulation()
{
    generateBubbles();
    if (numBubblesAboveMinRad < numBubbles)
        deleteSmallBubbles();
    updateCellsAndNeighbors();

    // Calculate some initial values which are needed
    // for the two-step Adams-Bashforth-Moulton perdictor-corrector method (ABMpc).

    double *x = bubbleData.getRowPtr((size_t)BP::X);
    double *y = bubbleData.getRowPtr((size_t)BP::Y);
    double *z = bubbleData.getRowPtr((size_t)BP::Z);
    double *r = bubbleData.getRowPtr((size_t)BP::R);

    double *dxdt = bubbleData.getRowPtr((size_t)BP::DXDT);
    double *dydt = bubbleData.getRowPtr((size_t)BP::DYDT);
    double *dzdt = bubbleData.getRowPtr((size_t)BP::DZDT);
    double *drdt = bubbleData.getRowPtr((size_t)BP::DRDT);

    double *dxdtOld = bubbleData.getRowPtr((size_t)BP::DXDT_OLD);
    double *dydtOld = bubbleData.getRowPtr((size_t)BP::DYDT_OLD);
    double *dzdtOld = bubbleData.getRowPtr((size_t)BP::DZDT_OLD);
    double *drdtOld = bubbleData.getRowPtr((size_t)BP::DRDT_OLD);

    double *energies = bubbleData.getRowPtr((size_t)BP::ENERGY);
    double *freeArea = bubbleData.getRowPtr((size_t)BP::FREE_AREA);

    int *firstIndices = neighborPairIndices.getRowPtr(0);
    int *secondIndices = neighborPairIndices.getRowPtr(1);

    const dvec tfr = env->getTfr();
    const dvec lbb = env->getLbb();
    ExecutionPolicy defaultPolicy(128, numBubbles);
    ExecutionPolicy accPolicy(128, hostNumPairs);

    double timeStep = env->getTimeStep();

    hipLaunchByPtr(defaultPolicy, resetKernel,
               0.0, numBubbles,
               dxdtOld, dydtOld, dzdtOld, drdtOld);

    std::cout << "Calculating some initial values as a part of setup."
              << " Num bubbles: " << numBubbles
              << ", host num pairs: " << hostNumPairs
              << std::endl;

    hipLaunchByPtr(accPolicy, calculateVelocityAndGasExchange,
               x, y, z, r, dxdtOld, dydtOld, dzdtOld, drdtOld, energies, freeArea,
               firstIndices, secondIndices, numBubbles, env->getFZeroPerMuZero(), env->getPi(), tfr - lbb, false, false);

    hipLaunchByPtr(defaultPolicy, eulerKernel,
               numBubbles, timeStep,
               x, dxdtOld,
               y, dydtOld,
               z, dzdtOld);

    hipLaunchByPtr(defaultPolicy, boundaryWrapKernel,
               numBubbles,
               x, lbb.x, tfr.x,
               y, lbb.y, tfr.y,
               z, lbb.z, tfr.z);

    hipLaunchByPtr(defaultPolicy, resetKernel,
               0.0, numBubbles,
               dxdtOld, dydtOld, dzdtOld, drdtOld);

    hipLaunchByPtr(accPolicy, calculateVelocityAndGasExchange,
               x, y, z, r, dxdtOld, dydtOld, dzdtOld, drdtOld, energies, freeArea,
               firstIndices, secondIndices, numBubbles, env->getFZeroPerMuZero(), env->getPi(), tfr - lbb, false, false);
}

bool Simulator::integrate(bool useGasExchange, bool calculateEnergy)
{
    const dvec tfr = env->getTfr();
    const dvec lbb = env->getLbb();
    const double minRad = env->getMinRad();
    ExecutionPolicy defaultPolicy(128, numBubbles);
    ExecutionPolicy accPolicy(128, hostNumPairs);

    double timeStep = env->getTimeStep();
    double maxError = 1000000;

    double *x = bubbleData.getRowPtr((size_t)BP::X);
    double *y = bubbleData.getRowPtr((size_t)BP::Y);
    double *z = bubbleData.getRowPtr((size_t)BP::Z);
    double *r = bubbleData.getRowPtr((size_t)BP::R);

    double *xPrd = bubbleData.getRowPtr((size_t)BP::X_PRD);
    double *yPrd = bubbleData.getRowPtr((size_t)BP::Y_PRD);
    double *zPrd = bubbleData.getRowPtr((size_t)BP::Z_PRD);
    double *rPrd = bubbleData.getRowPtr((size_t)BP::R_PRD);

    double *dxdt = bubbleData.getRowPtr((size_t)BP::DXDT);
    double *dydt = bubbleData.getRowPtr((size_t)BP::DYDT);
    double *dzdt = bubbleData.getRowPtr((size_t)BP::DZDT);
    double *drdt = bubbleData.getRowPtr((size_t)BP::DRDT);

    double *dxdtPrd = bubbleData.getRowPtr((size_t)BP::DXDT_PRD);
    double *dydtPrd = bubbleData.getRowPtr((size_t)BP::DYDT_PRD);
    double *dzdtPrd = bubbleData.getRowPtr((size_t)BP::DZDT_PRD);
    double *drdtPrd = bubbleData.getRowPtr((size_t)BP::DRDT_PRD);

    double *dxdtOld = bubbleData.getRowPtr((size_t)BP::DXDT_OLD);
    double *dydtOld = bubbleData.getRowPtr((size_t)BP::DYDT_OLD);
    double *dzdtOld = bubbleData.getRowPtr((size_t)BP::DZDT_OLD);
    double *drdtOld = bubbleData.getRowPtr((size_t)BP::DRDT_OLD);

    double *energies = bubbleData.getRowPtr((size_t)BP::ENERGY);
    double *errors = bubbleData.getRowPtr((size_t)BP::ERROR);
    double *volumes = bubbleData.getRowPtr((size_t)BP::VOLUME);
    double *freeArea = bubbleData.getRowPtr((size_t)BP::FREE_AREA);

    int *firstIndices = neighborPairIndices.getRowPtr(0);
    int *secondIndices = neighborPairIndices.getRowPtr(1);
    int *flag = aboveMinRadFlags.getRowPtr(0);

    size_t numLoopsDone = 0;
    do
    {
        NVTX_RANGE_PUSH_A("Integration step");

        hipLaunchByPtr(defaultPolicy, resetKernel,
                   0.0, numBubbles,
                   dxdtPrd, dydtPrd, dzdtPrd, drdtPrd, freeArea, energies, errors);

        //HACK:  This is REALLY stupid, but doing it temporarily.
        if (useGasExchange)
            hipLaunchByPtr(defaultPolicy, predictKernel,
                       numBubbles, timeStep,
                       xPrd, x, dxdt, dxdtOld,
                       yPrd, y, dydt, dydtOld,
                       zPrd, z, dzdt, dzdtOld,
                       rPrd, r, drdt, drdtOld);
        else
            hipLaunchByPtr(defaultPolicy, predictKernel,
                       numBubbles, timeStep,
                       xPrd, x, dxdt, dxdtOld,
                       yPrd, y, dydt, dydtOld,
                       zPrd, z, dzdt, dzdtOld);

        hipLaunchByPtr(accPolicy, calculateVelocityAndGasExchange,
                   xPrd, yPrd, zPrd, rPrd, dxdtPrd, dydtPrd, dzdtPrd, drdtPrd,
                   energies, freeArea, firstIndices, secondIndices, numBubbles,
                   env->getFZeroPerMuZero(), env->getPi(), env->getTfr() - env->getLbb(), calculateEnergy, useGasExchange);

        if (useGasExchange)
        {
            hipLaunchByPtr(defaultPolicy, calculateFreeAreaPerRadius,
                       rPrd, freeArea, errors, env->getPi(), numBubbles);

            cubWrapper->reduce<double, double *, double *>(&hipcub::DeviceReduce::Sum, errors, static_cast<double *>(dtfapr), numBubbles);
            cubWrapper->reduce<double, double *, double *>(&hipcub::DeviceReduce::Sum, freeArea, static_cast<double *>(dtfa), numBubbles);
            hipLaunchByPtr(defaultPolicy, calculateFinalRadiusChangeRate,
                       drdtPrd, rPrd, freeArea, numBubbles, 1.0 / env->getPi(), env->getKappa(), env->getKParameter());
        }

        //HACK:  This is REALLY stupid, but doing it temporarily.
        if (useGasExchange)
            hipLaunchByPtr(defaultPolicy, correctKernel,
                       numBubbles, timeStep, errors,
                       xPrd, x, dxdt, dxdtPrd,
                       yPrd, y, dydt, dydtPrd,
                       zPrd, z, dzdt, dzdtPrd,
                       rPrd, r, drdt, drdtPrd);
        else
            hipLaunchByPtr(defaultPolicy, correctKernel,
                       numBubbles, timeStep, errors,
                       xPrd, x, dxdt, dxdtPrd,
                       yPrd, y, dydt, dydtPrd,
                       zPrd, z, dzdt, dzdtPrd);

        cubWrapper->reduce<double, double *, double *>(&hipcub::DeviceReduce::Max, errors, static_cast<double *>(dme), numBubbles);

        hipLaunchByPtr(defaultPolicy, boundaryWrapKernel,
                   numBubbles,
                   xPrd, lbb.x, tfr.x,
                   yPrd, lbb.y, tfr.y,
                   zPrd, lbb.z, tfr.z);

        hipLaunchByPtr(defaultPolicy, setFlagIfGreaterThanConstantKernel,
                   numBubbles, flag, rPrd, env->getMinRad());

        cubWrapper->reduce<int, int *, int *>(&hipcub::DeviceReduce::Sum, flag, static_cast<int *>(dnp), numBubbles);
        CUDA_CALL(hipMemcpy(static_cast<void *>(&numBubblesAboveMinRad), dnp, sizeof(int), hipMemcpyDeviceToHost));
        CUDA_CALL(hipMemcpy(static_cast<void *>(&maxError), dme, sizeof(double), hipMemcpyDeviceToHost));

        if (maxError < env->getErrorTolerance() && timeStep < 0.1)
            timeStep *= 1.9;
        else if (maxError > env->getErrorTolerance())
            timeStep *= 0.5;

        ++numLoopsDone;

        if (numLoopsDone > 1000)
        {
            std::cout << "Done " << numLoopsDone << " loops, and error is " << maxError << std::endl;
            throw std::runtime_error("Error.");
        }
        NVTX_RANGE_POP();
    } while (maxError > env->getErrorTolerance());

    updateData();

    ++integrationStep;
    env->setTimeStep(timeStep);
    SimulationTime += timeStep;

    if (calculateEnergy)
    {
        cubWrapper->reduce<double, double *, double *>(&hipcub::DeviceReduce::Sum, energies, static_cast<double *>(dme), numBubbles);
        CUDA_CALL(hipMemcpy(static_cast<void *>(&ElasticEnergy), dme, sizeof(double), hipMemcpyDeviceToHost));
    }

    const bool shouldDelete = numBubblesAboveMinRad < numBubbles;
    if (shouldDelete)
        deleteSmallBubbles();

    if (shouldDelete || integrationStep % 50 == 0)
        updateCellsAndNeighbors();

    return numBubbles > env->getMinNumBubbles();
}

void Simulator::generateBubbles()
{
    std::cout << "Starting to generate data for bubbles." << std::endl;

    const int rngSeed = env->getRngSeed();
    const double avgRad = env->getAvgRad();
    const double stdDevRad = env->getStdDevRad();
    const dvec tfr = env->getTfr();
    const dvec lbb = env->getLbb();

    double *x = bubbleData.getRowPtr((size_t)BP::X);
    double *y = bubbleData.getRowPtr((size_t)BP::Y);
    double *z = bubbleData.getRowPtr((size_t)BP::Z);

    double *xPrd = bubbleData.getRowPtr((size_t)BP::X_PRD);
    double *yPrd = bubbleData.getRowPtr((size_t)BP::Y_PRD);
    double *zPrd = bubbleData.getRowPtr((size_t)BP::Z_PRD);

    double *r = bubbleData.getRowPtr((size_t)BP::R);
    double *w = bubbleData.getRowPtr((size_t)BP::R_PRD);

    hiprandGenerator_t generator;
    CURAND_CALL(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_MTGP32));
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(generator, rngSeed));

    CURAND_CALL(hiprandGenerateUniformDouble(generator, x, numBubbles));
    CURAND_CALL(hiprandGenerateUniformDouble(generator, y, numBubbles));
    CURAND_CALL(hiprandGenerateUniformDouble(generator, z, numBubbles));
    CURAND_CALL(hiprandGenerateUniformDouble(generator, w, numBubbles));
    CURAND_CALL(hiprandGenerateNormalDouble(generator, r, numBubbles, avgRad, stdDevRad));

    CURAND_CALL(hiprandDestroyGenerator(generator));

    ExecutionPolicy defaultPolicy(128, numBubbles);
    assert(bubblesPerDimAtStart.x > 0);
    assert(bubblesPerDimAtStart.y > 0);
#if (NUM_DIM == 3)
    assert(bubblesPerDimAtStart.z > 0);
#endif

    int *flag = aboveMinRadFlags.getRowPtr(0);
    hipLaunchByPtr(defaultPolicy, assignDataToBubbles,
               x, y, z, xPrd, yPrd, zPrd, r, w, flag, bubblesPerDimAtStart, tfr, lbb, avgRad, env->getMinRad(), numBubbles);

    cubWrapper->reduce<int, int *, int *>(&hipcub::DeviceReduce::Sum, flag, static_cast<int *>(dnp), numBubbles);
    CUDA_CALL(hipMemcpy(static_cast<void *>(&numBubblesAboveMinRad), dnp, sizeof(int), hipMemcpyDeviceToHost));
}

void Simulator::updateCellsAndNeighbors()
{
    dim3 gridSize = getGridSize();
    const int numCells = gridSize.x * gridSize.y * gridSize.z;
    const ivec cellDim(gridSize.x, gridSize.y, gridSize.z);

    double *x = bubbleData.getRowPtr((size_t)BP::X);
    double *y = bubbleData.getRowPtr((size_t)BP::Y);
    double *z = bubbleData.getRowPtr((size_t)BP::Z);
    double *r = bubbleData.getRowPtr((size_t)BP::R);
    int *offsets = cellData.getRowPtr((size_t)CellProperty::OFFSET);
    int *sizes = cellData.getRowPtr((size_t)CellProperty::SIZE);

    ExecutionPolicy defaultPolicy(128, numBubbles);
    hipLaunchByPtr(defaultPolicy, assignBubblesToCells,
               x, y, z, bubbleCellIndices.getRowPtr(2), bubbleCellIndices.getRowPtr(3), env->getLbb(), env->getTfr(), cellDim, numBubbles);

    int *cellIndices = bubbleCellIndices.getRowPtr(0);
    int *bubbleIndices = bubbleCellIndices.getRowPtr(1);

    cubWrapper->sortPairs<int, int>(&hipcub::DeviceRadixSort::SortPairs,
                                    const_cast<const int *>(bubbleCellIndices.getRowPtr(2)),
                                    cellIndices,
                                    const_cast<const int *>(bubbleCellIndices.getRowPtr(3)),
                                    bubbleIndices,
                                    numBubbles);

    hipLaunchByPtr(defaultPolicy, findOffsets,
               cellIndices, offsets, numCells, numBubbles);

    hipLaunchByPtr(defaultPolicy, findSizes,
               offsets, sizes, numCells, numBubbles);

    hipLaunchByPtr(defaultPolicy, reorganizeKernel,
               numBubbles, ReorganizeType::COPY_FROM_INDEX, bubbleIndices, bubbleIndices,
               bubbleData.getRowPtr((size_t)BP::X), bubbleData.getRowPtr((size_t)BP::X_PRD),
               bubbleData.getRowPtr((size_t)BP::Y), bubbleData.getRowPtr((size_t)BP::Y_PRD),
               bubbleData.getRowPtr((size_t)BP::Z), bubbleData.getRowPtr((size_t)BP::Z_PRD),
               bubbleData.getRowPtr((size_t)BP::R), bubbleData.getRowPtr((size_t)BP::R_PRD),
               bubbleData.getRowPtr((size_t)BP::DXDT), bubbleData.getRowPtr((size_t)BP::DXDT_PRD),
               bubbleData.getRowPtr((size_t)BP::DYDT), bubbleData.getRowPtr((size_t)BP::DYDT_PRD),
               bubbleData.getRowPtr((size_t)BP::DZDT), bubbleData.getRowPtr((size_t)BP::DZDT_PRD),
               bubbleData.getRowPtr((size_t)BP::DRDT), bubbleData.getRowPtr((size_t)BP::DRDT_PRD),
               bubbleData.getRowPtr((size_t)BP::DXDT_OLD), bubbleData.getRowPtr((size_t)BP::ENERGY),
               bubbleData.getRowPtr((size_t)BP::DYDT_OLD), bubbleData.getRowPtr((size_t)BP::FREE_AREA),
               bubbleData.getRowPtr((size_t)BP::DZDT_OLD), bubbleData.getRowPtr((size_t)BP::ERROR),
               bubbleData.getRowPtr((size_t)BP::DRDT_OLD), bubbleData.getRowPtr((size_t)BP::VOLUME));
    CUDA_CALL(hipMemcpyAsync(static_cast<void *>(x),
                              static_cast<void *>(bubbleData.getRowPtr((size_t)BP::X_PRD)), sizeof(double) * (size_t)BP::X_PRD * bubbleData.getWidth(), hipMemcpyDeviceToDevice));

    int sharedMemSizeInBytes = 0;
    cubWrapper->reduce<int, int *, int *>(&hipcub::DeviceReduce::Max, sizes, static_cast<int *>(dnp), numCells);
    CUDA_CALL(hipMemcpy(static_cast<void *>(&sharedMemSizeInBytes), dnp, sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemset(dnp, 0, sizeof(int)));
    sharedMemSizeInBytes *= sharedMemSizeInBytes;
    sharedMemSizeInBytes *= 2;
    const int maxNumSharedVals = sharedMemSizeInBytes;
    sharedMemSizeInBytes *= sizeof(int);
    assertMemBelowLimit(sharedMemSizeInBytes);
    assert(sharedMemSizeInBytes > 0 && "Zero bytes of shared memory reserved!");

    gridSize.z *= CUBBLE_NUM_NEIGHBORS + 1;
    assertGridSizeBelowLimit(gridSize);

    ExecutionPolicy findPolicy(256, numBubbles);
    findPolicy.gridSize = gridSize;
    findPolicy.sharedMemBytes = sharedMemSizeInBytes;
    hipLaunchByPtr(findPolicy, findBubblePairs,
               x, y, z, r, offsets, sizes,
               neighborPairIndices.getRowPtr(0), neighborPairIndices.getRowPtr(1),
               numCells, numBubbles, env->getTfr() - env->getLbb(),
               maxNumSharedVals, (int)neighborPairIndices.getWidth());

    CUDA_CALL(hipMemcpy(&hostNumPairs, dnp, sizeof(int), hipMemcpyDeviceToHost));
}

void Simulator::updateData()
{
    const size_t numBytesToCopy = 4 * sizeof(double) * bubbleData.getWidth();

    double *x = bubbleData.getRowPtr((size_t)BP::X);
    double *xPrd = bubbleData.getRowPtr((size_t)BP::X_PRD);
    double *dxdt = bubbleData.getRowPtr((size_t)BP::DXDT);
    double *dxdtOld = bubbleData.getRowPtr((size_t)BP::DXDT_OLD);

    CUDA_CALL(hipMemcpyAsync(dxdtOld, dxdt, numBytesToCopy, hipMemcpyDeviceToDevice));
    CUDA_CALL(hipMemcpyAsync(x, xPrd, 2 * numBytesToCopy, hipMemcpyDeviceToDevice));
}

void Simulator::deleteSmallBubbles()
{
    NVTX_RANGE_PUSH_A("BubbleRemoval");
    ExecutionPolicy defaultPolicy(128, numBubbles);

    int *flag = aboveMinRadFlags.getRowPtr(0);
    double *r = bubbleData.getRowPtr((size_t)BP::R);
    double *volumes = bubbleData.getRowPtr((size_t)BP::VOLUME);
    hipMemset(dvm, 0, sizeof(double));

    hipLaunchByPtr(defaultPolicy, calculateRedistributedGasVolume,
               volumes, r, flag, env->getPi(), numBubbles);

    cubWrapper->reduce<double, double *, double *>(&hipcub::DeviceReduce::Sum, volumes, static_cast<double *>(dtv), numBubbles);

    int *newIdx = aboveMinRadFlags.getRowPtr(1);
    cubWrapper->scan<int *, int *>(&hipcub::DeviceScan::ExclusiveSum, flag, newIdx, numBubbles);

    hipLaunchByPtr(defaultPolicy, reorganizeKernel,
               numBubbles, ReorganizeType::CONDITIONAL_TO_INDEX, newIdx, flag,
               bubbleData.getRowPtr((size_t)BP::X), bubbleData.getRowPtr((size_t)BP::X_PRD),
               bubbleData.getRowPtr((size_t)BP::Y), bubbleData.getRowPtr((size_t)BP::Y_PRD),
               bubbleData.getRowPtr((size_t)BP::Z), bubbleData.getRowPtr((size_t)BP::Z_PRD),
               bubbleData.getRowPtr((size_t)BP::R), bubbleData.getRowPtr((size_t)BP::R_PRD),
               bubbleData.getRowPtr((size_t)BP::DXDT), bubbleData.getRowPtr((size_t)BP::DXDT_PRD),
               bubbleData.getRowPtr((size_t)BP::DYDT), bubbleData.getRowPtr((size_t)BP::DYDT_PRD),
               bubbleData.getRowPtr((size_t)BP::DZDT), bubbleData.getRowPtr((size_t)BP::DZDT_PRD),
               bubbleData.getRowPtr((size_t)BP::DRDT), bubbleData.getRowPtr((size_t)BP::DRDT_PRD),
               bubbleData.getRowPtr((size_t)BP::DXDT_OLD), bubbleData.getRowPtr((size_t)BP::ENERGY),
               bubbleData.getRowPtr((size_t)BP::DYDT_OLD), bubbleData.getRowPtr((size_t)BP::FREE_AREA),
               bubbleData.getRowPtr((size_t)BP::DZDT_OLD), bubbleData.getRowPtr((size_t)BP::ERROR),
               bubbleData.getRowPtr((size_t)BP::DRDT_OLD), bubbleData.getRowPtr((size_t)BP::VOLUME));
    CUDA_CALL(hipMemcpyAsync(static_cast<void *>(bubbleData.getRowPtr((size_t)BP::X)),
                              static_cast<void *>(bubbleData.getRowPtr((size_t)BP::X_PRD)), sizeof(double) * (size_t)BP::X_PRD * bubbleData.getWidth(), hipMemcpyDeviceToDevice));

    numBubbles = numBubblesAboveMinRad;
    hipLaunchByPtr(defaultPolicy, addVolume, r, numBubbles);

    NVTX_RANGE_POP();
}

dim3 Simulator::getGridSize()
{
    int numBubblesPerCell = env->getNumBubblesPerCell();
#if (NUM_DIM == 3)
    int numCellsPerDim = std::ceil(std::cbrt((float)numBubbles / numBubblesPerCell));
    dim3 gridSize(numCellsPerDim, numCellsPerDim, numCellsPerDim);
#else
    int numCellsPerDim = std::ceil(std::sqrt((float)numBubbles / numBubblesPerCell));
    dim3 gridSize(numCellsPerDim, numCellsPerDim, 1);
#endif

    return gridSize;
}

double Simulator::getVolumeOfBubbles()
{
    ExecutionPolicy defaultPolicy(128, numBubbles);
    double *r = bubbleData.getRowPtr((size_t)BP::R);
    double *volPtr = bubbleData.getRowPtr((size_t)BP::VOLUME);
    hipLaunchByPtr(defaultPolicy, calculateVolumes,
               r, volPtr, numBubbles, env->getPi());

    double volume = 0.0;
    cubWrapper->reduce<double, double *, double *>(&hipcub::DeviceReduce::Sum, volPtr, static_cast<double *>(dtv), numBubbles);
    CUDA_CALL(hipMemcpy(static_cast<void *>(&volume), dtv, sizeof(double), hipMemcpyDeviceToHost));

    return volume;
}

double Simulator::getAverageRadius()
{
    double *r = bubbleData.getRowPtr((size_t)BP::R);
    double avgRad = 0.0;
    cubWrapper->reduce<double, double *, double *>(&hipcub::DeviceReduce::Sum, r, static_cast<double *>(dtv), numBubbles);
    CUDA_CALL(hipMemcpy(static_cast<void *>(&avgRad), dtv, sizeof(double), hipMemcpyDeviceToHost));
    avgRad /= numBubbles;

    return avgRad;
}

void Simulator::getBubbles(std::vector<Bubble> &bubbles) const
{
    bubbles.clear();
    bubbles.resize(numBubbles);

    size_t memoryStride = bubbleData.getWidth();
    double *devX = bubbleData.getRowPtr((size_t)BP::X);
    std::vector<double> xyzr;
    xyzr.resize(memoryStride * 4);

    CUDA_CALL(hipMemcpy(xyzr.data(), devX, sizeof(double) * 4 * memoryStride, hipMemcpyDeviceToHost));

    for (size_t i = 0; i < numBubbles; ++i)
    {
        Bubble b;
        dvec pos(-1, -1, -1);
        pos.x = xyzr[i];
        pos.y = xyzr[i + memoryStride];
        pos.z = xyzr[i + 2 * memoryStride];
        b.setPos(pos);
        b.setRadius(xyzr[i + 3 * memoryStride]);
        bubbles[i] = b;
    }
}
} // namespace cubble