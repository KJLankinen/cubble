#include "hip/hip_runtime.h"
// -*- C++ -*-

#include "Simulator.h"
#include "Macros.h"
#include "CudaContainer.h"
#include "Cell.h"
#include "Vec.h"
#include "Util.h"

#include "cub/hipcub/hipcub.hpp"

#include <iostream>
#include <sstream>
#include <chrono>
#include <algorithm>
#include <vector>
#include <hiprand.h>

#include <hip/hip_runtime_api.h>
#include <nvToolsExt.h>

#include <hip/hip_runtime_api.h>
#include <nvToolsExt.h>


// ******************************
// Class functions run on CPU
// ******************************

cubble::Simulator::Simulator(std::shared_ptr<Env> e)
{
    env = e;
    
#if (NUM_DIM == 3)
    givenNumBubblesPerDim = std::ceil(std::cbrt((float)env->getNumBubbles()));
    numBubbles = givenNumBubblesPerDim * givenNumBubblesPerDim * givenNumBubblesPerDim;
#else
    givenNumBubblesPerDim = std::ceil(std::sqrt((float)env->getNumBubbles()));
    numBubbles = givenNumBubblesPerDim * givenNumBubblesPerDim;
#endif
    const dvec tfr = env->getLbb() + env->getAvgRad() * (double)givenNumBubblesPerDim * 2;
    env->setTfr(tfr);
    
    dmh = std::unique_ptr<cubble::DeviceMemoryHandler>(new DeviceMemoryHandler(numBubbles, neighborStride));
    dmh->reserveMemory();
    hostData.resize(dmh->getNumPermanentValuesInMemory(), 0);
    
    printRelevantInfoOfCurrentDevice();

    hipEventCreate(&start);
    hipEventCreate(&stop);
}

cubble::Simulator::~Simulator()
{
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

void cubble::Simulator::setupSimulation()
{
    NVTX_RANGE_PUSH_A(__FUNCTION__);

    generateBubbles();
    assignBubblesToCells(true);

    // Calculate some initial values which are needed
    // for the two-step Adams-Bashforth-Moulton perdictor-corrector method (ABMpc).

    double *x = dmh->getDataPtr(BubbleProperty::X);
    double *y = dmh->getDataPtr(BubbleProperty::Y);
    double *z = dmh->getDataPtr(BubbleProperty::Z);
    double *r = dmh->getDataPtr(BubbleProperty::R);
    
    double *dxdt = dmh->getDataPtr(BubbleProperty::DXDT);
    double *dydt = dmh->getDataPtr(BubbleProperty::DYDT);
    double *dzdt = dmh->getDataPtr(BubbleProperty::DZDT);
    double *drdt = dmh->getDataPtr(BubbleProperty::DRDT);
    
    double *dxdtOld = dmh->getDataPtr(BubbleProperty::DXDT_OLD);
    double *dydtOld = dmh->getDataPtr(BubbleProperty::DYDT_OLD);
    double *dzdtOld = dmh->getDataPtr(BubbleProperty::DZDT_OLD);
    double *drdtOld = dmh->getDataPtr(BubbleProperty::DRDT_OLD);
    
    double *energies = dmh->getDataPtr(BubbleProperty::ENERGY);
    double *freeArea = dmh->getDataPtr(BubbleProperty::FREE_AREA);

    double *ax = dmh->getDataPtr(BubblePairProperty::ACCELERATION_X);
    double *ay = dmh->getDataPtr(BubblePairProperty::ACCELERATION_Y);
    double *az = dmh->getDataPtr(BubblePairProperty::ACCELERATION_Z);
    double *ar = dmh->getDataPtr(BubblePairProperty::ACCELERATION_R);
    double *e = dmh->getDataPtr(BubblePairProperty::ENERGY);
    double *areaOverlap = dmh->getDataPtr(BubblePairProperty::OVERLAP_AREA);

    const dvec tfr = env->getTfr();
    const dvec lbb = env->getLbb();
    const double minRad = env->getMinRad();
    const size_t numThreads = 128;
    const size_t numBlocks = (size_t)std::ceil(numBubbles / (float)numThreads);
    const size_t numBlocksForAcc = (size_t)std::ceil(numBubbles * neighborStride / (float)numThreads);

    double timeStep = env->getTimeStep();

    createAccelerationArray<<<numBlocksForAcc, numThreads>>>(x, y, z, r,
							     ax, ay, az, ar, e, areaOverlap,
							     numberOfNeighbors.getDataPtr(),
							     neighborIndices.getDataPtr(),
							     tfr - lbb,
							     numBubbles,
							     neighborStride,
							     env->getPi(),
							     false,
							     false);
    
    calculateVelocityFromAccelerations<<<numBlocks, numThreads>>>(ax, ay, az, ar, e, areaOverlap,
								  dxdtOld, dydtOld, dzdtOld, drdtOld,
								  freeArea,
								  energies,
								  numBubbles,
								  neighborStride,
								  env->getFZeroPerMuZero(),
								  false,
								  false);
    
    eulerIntegration<<<numBlocks, numThreads>>>(x, y, z, r,
						dxdtOld, dydtOld, dzdtOld, drdtOld,
						tfr, lbb, timeStep, numBubbles);
    
    createAccelerationArray<<<numBlocksForAcc, numThreads>>>(x, y, z, r,
							     ax, ay, az, ar, e, areaOverlap,
							     numberOfNeighbors.getDataPtr(),
							     neighborIndices.getDataPtr(),
							     tfr - lbb,
							     numBubbles,
							     neighborStride,
							     env->getPi(),
							     false,
							     false);
    
    calculateVelocityFromAccelerations<<<numBlocks, numThreads>>>(ax, ay, az, ar, e, areaOverlap,
								  dxdtOld, dydtOld, dzdtOld, drdtOld,
								  freeArea,
								  energies,
								  numBubbles,
								  neighborStride,
								  env->getFZeroPerMuZero(),
								  false,
								  false);
    NVTX_RANGE_POP();
}

bool cubble::Simulator::integrate(bool useGasExchange, bool calculateEnergy)
{
    NVTX_RANGE_PUSH_A(__FUNCTION__);
    
    const dvec tfr = env->getTfr();
    const dvec lbb = env->getLbb();
    const double minRad = env->getMinRad();
    const size_t numThreads = 128;
    const size_t numBlocks = (size_t)std::ceil(numBubbles / (float)numThreads);
    const size_t numBlocksForAcc = (size_t)std::ceil(numBubbles * neighborStride / (float)numThreads);

    double timeStep = env->getTimeStep();
    double error = 0;

    size_t numIntegrationSteps = 0;

    double *x = dmh->getDataPtr(BubbleProperty::X);
    double *y = dmh->getDataPtr(BubbleProperty::Y);
    double *z = dmh->getDataPtr(BubbleProperty::Z);
    double *r = dmh->getDataPtr(BubbleProperty::R);
    
    double *xPrd = dmh->getDataPtr(BubbleProperty::X_PRD);
    double *yPrd = dmh->getDataPtr(BubbleProperty::Y_PRD);
    double *zPrd = dmh->getDataPtr(BubbleProperty::Z_PRD);
    double *rPrd = dmh->getDataPtr(BubbleProperty::R_PRD);
    
    double *dxdt = dmh->getDataPtr(BubbleProperty::DXDT);
    double *dydt = dmh->getDataPtr(BubbleProperty::DYDT);
    double *dzdt = dmh->getDataPtr(BubbleProperty::DZDT);
    double *drdt = dmh->getDataPtr(BubbleProperty::DRDT);
    
    double *dxdtPrd = dmh->getDataPtr(BubbleProperty::DXDT_PRD);
    double *dydtPrd = dmh->getDataPtr(BubbleProperty::DYDT_PRD);
    double *dzdtPrd = dmh->getDataPtr(BubbleProperty::DZDT_PRD);
    double *drdtPrd = dmh->getDataPtr(BubbleProperty::DRDT_PRD);
    
    double *dxdtOld = dmh->getDataPtr(BubbleProperty::DXDT_OLD);
    double *dydtOld = dmh->getDataPtr(BubbleProperty::DYDT_OLD);
    double *dzdtOld = dmh->getDataPtr(BubbleProperty::DZDT_OLD);
    double *drdtOld = dmh->getDataPtr(BubbleProperty::DRDT_OLD);

    double *energies = dmh->getDataPtr(BubbleProperty::ENERGY);
    double *errors = dmh->getDataPtr(BubbleProperty::ERROR);
    double *volumes = dmh->getDataPtr(BubbleProperty::VOLUME);
    double *freeArea = dmh->getDataPtr(BubbleProperty::FREE_AREA);

    double *ax = dmh->getDataPtr(BubblePairProperty::ACCELERATION_X);
    double *ay = dmh->getDataPtr(BubblePairProperty::ACCELERATION_Y);
    double *az = dmh->getDataPtr(BubblePairProperty::ACCELERATION_Z);
    double *ar = dmh->getDataPtr(BubblePairProperty::ACCELERATION_R);
    double *e = dmh->getDataPtr(BubblePairProperty::ENERGY);
    double *areaOverlap = dmh->getDataPtr(BubblePairProperty::OVERLAP_AREA);
    
    do
    {
	NVTX_RANGE_PUSH_A("Predict");
	predict<<<numBlocks, numThreads>>>(x, y, z, r,
					   xPrd, yPrd, zPrd, rPrd,
					   dxdt, dydt, dzdt, drdt,
					   dxdtOld, dydtOld, dzdtOld, drdtOld,
					   tfr, lbb, timeStep, numBubbles, useGasExchange);
	NVTX_RANGE_POP();
	NVTX_RANGE_PUSH_A("AccArr");

	createAccelerationArray<<<numBlocksForAcc, numThreads>>>(xPrd, yPrd, zPrd, rPrd,
								 ax, ay, az, ar, e, areaOverlap,
								 numberOfNeighbors.getDataPtr(),
								 neighborIndices.getDataPtr(),
								 tfr - lbb,
								 numBubbles,
								 neighborStride,
								 env->getPi(),
								 useGasExchange,
								 calculateEnergy);
	NVTX_RANGE_POP();
	NVTX_RANGE_PUSH_A("VelFromAcc");
        
	calculateVelocityFromAccelerations<<<numBlocks, numThreads>>>(ax, ay, az, ar, e, areaOverlap,
								      dxdtPrd, dydtPrd, dzdtPrd, drdtPrd,
								      freeArea,
								      energies,
								      numBubbles,
								      neighborStride,
								      env->getFZeroPerMuZero(),
								      calculateEnergy,
								      useGasExchange);
	
	NVTX_RANGE_POP();

	if (useGasExchange)
	{
	    calculateFreeAreaPerRadius<<<numBlocks, numThreads>>>(rPrd, freeArea, errors, numBubbles);
	    double invRho = cubReduction<double, double*, double*>(&hipcub::DeviceReduce::Sum, errors, numBubbles);
	    invRho /= cubReduction<double, double*, double*>(&hipcub::DeviceReduce::Sum, freeArea, numBubbles);
	    invRho = 1.0 / invRho;

	    calculateFinalRadiusChangeRate<<<numBlocks, numThreads>>>(drdtPrd,
								      rPrd,
								      freeArea,
								      numBubbles,
								      invRho,
								      env->getKappa(),
								      env->getKParameter());
	}
	
	NVTX_RANGE_PUSH_A("Correct");
        
	correct<<<numBlocks, numThreads>>>(x, y, z, r,
					   xPrd, yPrd, zPrd, rPrd,
					   dxdt, dydt, dzdt, drdt,
					   dxdtPrd, dydtPrd, dzdtPrd, drdtPrd,
					   errors,
					   tfr,
					   lbb,
					   timeStep,
					   numBubbles,
					   useGasExchange);
	NVTX_RANGE_POP();
	NVTX_RANGE_PUSH_A("CUB");
        
        error = cubReduction<double, double*, double*>(&hipcub::DeviceReduce::Max, errors, numBubbles);

	NVTX_RANGE_POP();

	if (error < env->getErrorTolerance() / 10 && timeStep < 0.1)
	    timeStep *= 1.9;
	else if (error > env->getErrorTolerance())
	    timeStep *= 0.5;

	++numIntegrationSteps;
    }
    while (error > env->getErrorTolerance());
	
    if (integrationStep == 15)
	hipProfilerStop();
    
    NVTX_RANGE_PUSH_A("UpdateData");
    // x, y, z, r are in memory continuously, so we can just make three copies with 4x the data of one component.
    size_t numBytesToCopy = 4 * sizeof(double) * dmh->getMemoryStride();
    hipMemcpyAsync(x, xPrd, numBytesToCopy, hipMemcpyDeviceToDevice);
    hipMemcpyAsync(dxdtOld, dxdt, numBytesToCopy, hipMemcpyDeviceToDevice);
    hipMemcpyAsync(dxdt, dxdtPrd, numBytesToCopy, hipMemcpyDeviceToDevice);
    NVTX_RANGE_POP();
    
    ++integrationStep;
    env->setTimeStep(timeStep);
    SimulationTime += timeStep;

    if (calculateEnergy)
	ElasticEnergy = cubReduction<double, double*, double*>(&hipcub::DeviceReduce::Sum, energies, numBubbles);

    double minRadius = cubReduction<double, double*, double*>(&hipcub::DeviceReduce::Min, r, numBubbles);
    if (minRadius < env->getMinRad())
    {
	NVTX_RANGE_PUSH_A("BubbleRemoval");
	
	hipMemcpyAsync(hostData.data(),
			dmh->getRawPtrToMemory(),
			dmh->getPermanentMemorySizeInBytes(),
			hipMemcpyDeviceToHost);
	
	minRadius = env->getMinRad();
	size_t memoryStride = dmh->getMemoryStride();
	size_t rIdx = (size_t)BubbleProperty::R;
	std::vector<int> idxVec;
	
	for (size_t i = 0; i < (size_t)BubbleProperty::NUM_VALUES; ++i)
	    idxVec.push_back(i);

	// Synchronize if memcpy not done yet.
	CUDA_CALL(hipDeviceSynchronize());

	double volumeMultiplier = 0;
	for (int i = (int)numBubbles - 1; i > -1; --i)
	{
	    double radius = hostData[rIdx * memoryStride + i];
	    assert(radius > 0 && "Radius is negative!");
	    if (radius < minRad)
	    {
		double volume = 0;
	        volume = radius * radius;
#if (NUM_DIM == 3)
		volume *= 1.333333333333333333333333 * radius;
#endif
		volume *= env->getPi();
	        volumeMultiplier += volume;
		
		for (size_t j = 0; j < idxVec.size(); ++j)
		    hostData[j * memoryStride + i] = hostData[j * memoryStride + (numBubbles - 1)];
		--numBubbles;
	    }
	}

	hipMemcpyAsync(dmh->getRawPtrToMemory(),
			hostData.data(),
			dmh->getPermanentMemorySizeInBytes(),
			hipMemcpyHostToDevice);

        volumeMultiplier /= getVolumeOfBubbles();
	volumeMultiplier += 1.0;

#if (NUM_DIM == 3)
	volumeMultiplier = std::cbrt(volumeMultiplier);
#else
	volumeMultiplier = std::sqrt(volumeMultiplier);
#endif

	addVolume<<<numBlocks, numThreads>>>(r, numBubbles, volumeMultiplier);
	
	NVTX_RANGE_POP();
	
	assignBubblesToCells(false);
    }
    else if (integrationStep % 100)
	assignBubblesToCells();

    NVTX_RANGE_POP();

    return numBubbles > env->getMinNumBubbles();
}

double cubble::Simulator::getVolumeOfBubbles() const
{
    NVTX_RANGE_PUSH_A(__FUNCTION__);
    const size_t numThreads = 128;
    const size_t numBlocks = (size_t)std::ceil(numBubbles / (float)numThreads);

    double *volPtr = dmh->getDataPtr(BubbleProperty::VOLUME);
    
    calculateVolumes<<<numBlocks, numThreads>>>(
	dmh->getDataPtr(BubbleProperty::R), volPtr, numBubbles, env->getPi());
    
    CUDA_CALL(hipDeviceSynchronize());
    CUDA_CALL(hipPeekAtLastError());

    double volume = cubReduction<double, double*, double*>(&hipcub::DeviceReduce::Sum, volPtr, numBubbles);
    
    NVTX_RANGE_POP();
    
    return volume;
}

double cubble::Simulator::getAverageRadius() const
{
    NVTX_RANGE_PUSH_A(__FUNCTION__);
    double *r = dmh->getDataPtr(BubbleProperty::R);
    double avgRad = cubReduction<double, double*, double*>(&hipcub::DeviceReduce::Sum, r, numBubbles);
    avgRad/= numBubbles;
    
    NVTX_RANGE_POP();
    
    return avgRad;
}

void cubble::Simulator::getBubbles(std::vector<Bubble> &bubbles) const
{
    NVTX_RANGE_PUSH_A(__FUNCTION__);
    bubbles.clear();
    bubbles.resize(numBubbles);

    size_t memoryStride = dmh->getMemoryStride();
    double *devX = dmh->getDataPtr(BubbleProperty::X);
    std::vector<double> xyzr;
    xyzr.resize(memoryStride * 4);

    hipMemcpy(xyzr.data(), devX, sizeof(double) * 4 * memoryStride, hipMemcpyDeviceToHost);
    
    for (size_t i = 0; i < numBubbles; ++i)
    {
	Bubble b;
	dvec pos(-1, -1, -1);
	pos.x = xyzr[i];
	pos.y = xyzr[i + memoryStride];
	pos.z = xyzr[i + 2 * memoryStride];
	b.setPos(pos);
	b.setRadius(xyzr[i + 3 * memoryStride]);
	bubbles[i] = b;
    }
    
    NVTX_RANGE_POP();
}

void cubble::Simulator::generateBubbles()
{
    NVTX_RANGE_PUSH_A(__FUNCTION__);
    
    std::cout << "Starting to generate data for bubbles." << std::endl;
    
    const int rngSeed = env->getRngSeed();
    const double avgRad = env->getAvgRad();
    const double stdDevRad = env->getStdDevRad();
    const dvec tfr = env->getTfr();
    const dvec lbb = env->getLbb();

    indices = CudaContainer<int>(numBubbles);
    numberOfNeighbors = CudaContainer<int>(numBubbles);
    neighborIndices = CudaContainer<int>(numBubbles * neighborStride);
    
    std::cout << "\tGenerating data..." << std::endl;

    double *x = dmh->getDataPtr(BubbleProperty::X);
    double *y = dmh->getDataPtr(BubbleProperty::Y);
    double *z = dmh->getDataPtr(BubbleProperty::Z);
    
    double *xPrd = dmh->getDataPtr(BubbleProperty::X_PRD);
    double *yPrd = dmh->getDataPtr(BubbleProperty::Y_PRD);
    double *zPrd = dmh->getDataPtr(BubbleProperty::Z_PRD);
    
    double *r = dmh->getDataPtr(BubbleProperty::R);
    double *w = dmh->getDataPtr(BubbleProperty::R_PRD);
    
    hiprandGenerator_t generator;
    CURAND_CALL(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_MTGP32));
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(generator, rngSeed));
    
    CURAND_CALL(hiprandGenerateUniformDouble(generator, x, numBubbles));
    CURAND_CALL(hiprandGenerateUniformDouble(generator, y, numBubbles));
    CURAND_CALL(hiprandGenerateUniformDouble(generator, z, numBubbles));
    CURAND_CALL(hiprandGenerateUniformDouble(generator, w, numBubbles));
    CURAND_CALL(hiprandGenerateNormalDouble(generator, r, numBubbles, avgRad, stdDevRad));

    CURAND_CALL(hiprandDestroyGenerator(generator));

    std::cout << "\tAssigning data to bubbles..." << std::endl;;

    const size_t numThreads = 128;
    const size_t numBlocks = (size_t)std::ceil((float)numBubbles / (float)numThreads);
    assignDataToBubbles<<<numBlocks, numThreads>>>(x, y, z,
						   xPrd, yPrd, zPrd,
						   r, w, givenNumBubblesPerDim, tfr, lbb, avgRad, numBubbles);
    NVTX_RANGE_POP();
}

void cubble::Simulator::assignBubblesToCells(bool useVerboseOutput)
{
    NVTX_RANGE_PUSH_A(__FUNCTION__);
    
    if (useVerboseOutput)
	std::cout << "Starting to assign bubbles to cells." << std::endl;
    
    dim3 gridSize = getGridSize();
    const int numCells = gridSize.x * gridSize.y * gridSize.z;
    const dvec domainDim(gridSize.x, gridSize.y, gridSize.z);
    const dvec cellSize = (env->getTfr() - env->getLbb()) / domainDim;
    const size_t numThreads = 128;
    const size_t numBlocks = (size_t)std::ceil(numBubbles / (float)numThreads);
    const int numDomains = (CUBBLE_NUM_NEIGHBORS + 1) * 4;

    double *x = dmh->getDataPtr(BubbleProperty::X);
    double *y = dmh->getDataPtr(BubbleProperty::Y);
    double *z = dmh->getDataPtr(BubbleProperty::Z);
    double *r = dmh->getDataPtr(BubbleProperty::R);

    const double minCellSize = cubReduction<double, double*, double*>(&hipcub::DeviceReduce::Max, r, numBubbles);
    
    if (useVerboseOutput)
	std::cout << "\tUsing grid size (" << gridSize.x
		  << ", " << gridSize.y
		  << ", " << gridSize.z
		  << ") with total of " << numCells << " cells." << std::endl;

#if NUM_DIM == 3    
    if (cellSize.x < minCellSize || cellSize.y < minCellSize || cellSize.z < minCellSize)
#else
    if (cellSize.x < minCellSize || cellSize.y < minCellSize)
#endif
    {
	std::stringstream ss;
	ss << "Size of cell (" << cellSize
	   << ") is smaller than the acceptable minimum cell size of "
	   << minCellSize
	   << " in at least one dimension."
	   << "\nEither decrease the number of bubbles or increase the size"
	   << " of the simulation box.";
	
	throw std::runtime_error(ss.str());
    }
    
    cells = CudaContainer<Cell>(numCells);

    hipMemset((void*)numberOfNeighbors.getDataPtr(), 0, sizeof(int) * numBubbles);
    hipMemset((void*)indices.getDataPtr(), 0, sizeof(int) * numBubbles);
    hipMemset((void*)neighborIndices.getDataPtr(), 0, sizeof(int) * numBubbles * neighborStride);
    
    if (useVerboseOutput)
	std::cout << "\tCalculating offsets..." << std::endl;
    
    calculateOffsets<<<numBlocks, numThreads>>>(x, y, z, cells.getDataPtr(), domainDim, numBubbles, numCells);

    CUDA_CALL(hipDeviceSynchronize());
    int cumulativeSum = 0;
    for (size_t i = 0; i < cells.getSize(); ++i)
    {
	const int numBubbles = cells[i].offset;
	cells[i].offset = cumulativeSum;
	cumulativeSum += numBubbles;
    }
    
    if (useVerboseOutput)
	std::cout << "\tAssigning bubbles to cells..." << std::endl;

    bubblesToCells<<<numBlocks, numThreads>>>(
	x, y, z, indices.getDataPtr(), cells.getDataPtr(), domainDim, numBubbles);

    gridSize.z *= numDomains;
    assertGridSizeBelowLimit(gridSize);

    if (useVerboseOutput)
	std::cout << "\tFinding neighbors for each bubble..." << std::endl;
    
    findNeighbors<<<gridSize, numThreads>>>(x, y, z, r,
					    indices.getDataPtr(),
					    cells.getDataPtr(),
					    numberOfNeighbors.getDataPtr(),
					    neighborIndices.getDataPtr(),
					    env->getTfr(),
					    env->getLbb(),
					    numBubbles,
					    numDomains,
					    cells.getSize(),
					    neighborStride);
    NVTX_RANGE_POP();
}

dim3 cubble::Simulator::getGridSize()
{
    NVTX_RANGE_PUSH_A(__FUNCTION__);
    int numBubblesPerCell = env->getNumBubblesPerCell();
#if NUM_DIM == 3
    int numCellsPerDim = std::ceil(std::cbrt((float)numBubbles / numBubblesPerCell));
    dim3 gridSize(numCellsPerDim, numCellsPerDim, numCellsPerDim);
#else
    int numCellsPerDim = std::ceil(std::sqrt((float)numBubbles / numBubblesPerCell));
    dim3 gridSize(numCellsPerDim, numCellsPerDim, 1);
#endif

    NVTX_RANGE_POP();
    return gridSize;
}


// ******************************
// Kernels
// ******************************

__global__
void cubble::calculateVolumes(double *r, double *volumes, int numBubbles, double pi)
{
    int tid = getGlobalTid();
    if (tid < numBubbles)
    {
	double radius = r[tid];
	double volume = radius * radius * pi;
#if (NUM_DIM == 3)
	volume *= radius * 1.33333333333333333333333333;
#endif
	
	volumes[tid] = volume;
    }   
}

__global__
void cubble::assignDataToBubbles(double *x,
				 double *y,
				 double *z,
				 double *xPrd,
				 double *yPrd,
				 double *zPrd,
				 double *r,
				 double *w,
				 int givenNumBubblesPerDim,
				 dvec tfr,
				 dvec lbb,
				 double avgRad,
				 int numBubbles)
{
    int tid = getGlobalTid();
    if (tid < numBubbles)
    {
	int xid = tid % givenNumBubblesPerDim;
	int yid = (tid / givenNumBubblesPerDim) % givenNumBubblesPerDim;
	
	dvec randomOffset(x[tid], y[tid], 0);
	dvec pos(0, 0, 0);
	pos.x = xid / (double)givenNumBubblesPerDim;
	pos.y = yid / (double)givenNumBubblesPerDim;
#if (NUM_DIM == 3)
	int zid = tid / (givenNumBubblesPerDim * givenNumBubblesPerDim);
	pos.z = zid / (double)givenNumBubblesPerDim;
	randomOffset.z = z[tid];
#endif

	randomOffset = dvec::normalize(randomOffset) * avgRad * w[tid];
	randomOffset = (randomOffset - lbb) / (tfr - lbb);
	pos = getWrappedPos(pos + randomOffset);

	x[tid] = pos.x;
	y[tid] = pos.y;
	z[tid] = pos.z;
	
	xPrd[tid] = pos.x;
	yPrd[tid] = pos.y;
	zPrd[tid] = pos.z;
	
	w[tid] = r[tid];
    }
}

__global__
void cubble::calculateOffsets(double *x,
			      double *y,
			      double *z,
			      Cell *cells,
			      dvec domainDim,
			      int numBubbles,
			      int numCells)
{   
    int tid = getGlobalTid();
    if (tid < numBubbles)
    {
	dvec pos = dvec(0, 0, 0);
	pos.x = x[tid];
	pos.y = y[tid];
	pos.z = z[tid];
	
        const ivec indexVec = (pos * domainDim).asType<int>();
	const int index = domainDim.x * domainDim.y * indexVec.z + domainDim.x * indexVec.y + indexVec.x;
	DEVICE_ASSERT(index < numCells);
	
	atomicAdd(&cells[index].offset, 1);
    }
}

__global__
void cubble::bubblesToCells(double *x,
			    double *y,
			    double *z,
			    int *indices,
			    Cell *cells,
			    dvec domainDim,
			    int numBubbles)
{
    int tid = getGlobalTid();
    if (tid < numBubbles)
    {
	dvec pos = dvec(0, 0, 0);
	pos.x = x[tid];
	pos.y = y[tid];
	pos.z = z[tid];
	
        const ivec indexVec = (pos * domainDim).asType<int>();
	const int index = domainDim.x * domainDim.y * indexVec.z + domainDim.x * indexVec.y + indexVec.x;
	const int offset = cells[index].offset + atomicAdd(&cells[index].size, 1);
        indices[offset] = tid;
    }
}

__global__
void cubble::findNeighbors(double *x,
			   double *y,
			   double *z,
			   double *r,
			   int *indices,
			   Cell *cells,
			   int *numberOfNeighbors,
			   int *neighborIndices,
			   dvec tfr,
			   dvec lbb,
			   int numBubbles,
			   int numDomains,
			   int numCells,
			   int neighborStride)
{
    DEVICE_ASSERT(numBubbles > 0);
    DEVICE_ASSERT(numDomains > 0);
    DEVICE_ASSERT(numCells > 0);
    DEVICE_ASSERT(!(numDomains & 1));
    
    ivec cellIdxVec(blockIdx.x, blockIdx.y, blockIdx.z / numDomains);
    ivec boxDim(gridDim.x, gridDim.y, gridDim.z / numDomains);
    
    int xBegin = -1;
    int xInterval = -1;
    int yBegin = -1;
    int yInterval = -1;
    bool isOwnCell = false;
    
    getDomainOffsetsAndIntervals(numBubbles,
				 numDomains,
				 numCells,
				 cellIdxVec,
				 boxDim,
				 cells,
				 xBegin,
				 xInterval,
				 yBegin,
				 yInterval,
				 isOwnCell);
    
    DEVICE_ASSERT(xBegin >= 0 && xInterval > 0 && yBegin >= 0 && yInterval > 0);
    
    int numPairs = xInterval * yInterval;
    int numRounds = 1 + (numPairs / blockDim.x);
    
    for (int round = 0; round < numRounds; ++round)
    {
        int pairIdx = round * blockDim.x + threadIdx.x;
	if (pairIdx < numPairs)
	{
	    int xid = pairIdx % xInterval;
	    int yid = pairIdx / xInterval;
	    DEVICE_ASSERT(yid < yInterval);
	    
	    int gid1 = indices[xBegin + xid];
	    int gid2 = indices[yBegin + yid];
	    
	    if (gid1 == gid2)
		continue;

	    dvec pos1, pos2;
	    pos1.x = x[gid1];
	    pos1.y = y[gid1];
	    pos1.z = z[gid1];
	    
	    pos2.x = x[gid2];
	    pos2.y = y[gid2];
	    pos2.z = z[gid2];
	    
	    dvec posVec = getShortestWrappedNormalizedVec(pos1, pos2);
	    const double length = (posVec * (tfr - lbb)).getSquaredLength();
	    
	    const double radii = r[gid1] + r[gid2];
	    
	    if (radii * radii > length)
	    {
		int index = atomicAdd(&numberOfNeighbors[gid1], 1);
		DEVICE_ASSERT(index < neighborStride);
		index = numBubbles * index + gid1;
		DEVICE_ASSERT(index < numBubbles * neighborStride);
		neighborIndices[index] = gid2;

		if (!isOwnCell)
		{
		    index = atomicAdd(&numberOfNeighbors[gid2], 1);
		    DEVICE_ASSERT(index < neighborStride);
		    index = numBubbles * index + gid2;
		    DEVICE_ASSERT(index < numBubbles * neighborStride);
		    neighborIndices[index] = gid1;
		}
	    }
	}
    }
}

__global__
void cubble::predict(double *x,
		     double *y,
		     double *z,
		     double *r,
		     
		     double *xPrd,
		     double *yPrd,
		     double *zPrd,
		     double *rPrd,
		     
		     double *dxdt,
		     double *dydt,
		     double *dzdt,
		     double *drdt,
		     
		     double *dxdtOld,
		     double *dydtOld,
		     double *dzdtOld,
		     double *drdtOld,
		     
		     dvec tfr,
		     dvec lbb,
		     double timeStep,
		     int numBubbles,
		     bool useGasExchange)
{
    const int tid = getGlobalTid();
    if (tid < numBubbles)
    {
	// Measure if it's faster to calculate these per component...
	const dvec interval = (tfr - lbb);
	dvec pos, vel, velOld;
	pos.x = x[tid];
	pos.y = y[tid];
	pos.z = z[tid];
	
	vel.x = dxdt[tid];
	vel.y = dydt[tid];
	vel.z = dzdt[tid];
	
	velOld.x = dxdtOld[tid];
	velOld.y = dydtOld[tid];
	velOld.z = dzdtOld[tid];

	pos = lbb + pos * interval;
	pos += 0.5 * timeStep * (3.0 * vel - velOld);
	pos = (pos - lbb) / interval;
	pos = getWrappedPos(pos);

	xPrd[tid] = pos.x;
	yPrd[tid] = pos.y;
	zPrd[tid] = pos.z;

	if (useGasExchange)
	    rPrd[tid] = r[tid] + 0.5 * timeStep * (3.0 * drdt[tid] - drdtOld[tid]);
    }
}

__global__
void cubble::createAccelerationArray(double *x,
				     double *y,
				     double *z,
				     double *r,

				     double *ax,
				     double *ay,
				     double *az,
				     double *ar,
				     double *e,
				     double *areaOverlap,
				     
				     int *numberOfNeighbors,
				     int *neighborIndices,
				     dvec interval,
				     int numBubbles,
				     int neighborStride,
				     double pi,
				     bool useGasExchange,
				     bool calculateEnergy)
{
    const int tid = getGlobalTid();
    if (tid < numBubbles * neighborStride)
    {
	const int idx1 = tid % numBubbles;
	const int neighborNum = tid / numBubbles;

	// Accelerations recide in temporary memory which is also used by reductions
	// and other temporary things, so they might contain garbage data.
	// Later these arrays are summed, so it's important to zero all the values.
	ax[tid] = 0;
	ay[tid] = 0;
	az[tid] = 0;
	ar[tid] = 0;
	areaOverlap[tid] = 0;
	e[tid] = 0;

	if (neighborNum < numberOfNeighbors[idx1])
	{
	    const int idx2 = neighborIndices[tid];
	    
	    double x1 = x[idx1];
	    double y1 = y[idx1];
	    double z1 = z[idx1];
	    double r1 = r[idx1];
	    
	    double x2 = x[idx2];
	    double y2 = y[idx2];
	    double z2 = z[idx2];
	    double r2 = r[idx2];
	    
	    double radii = r1 + r2;
	    
	    double magnitude = x1 - x2;
	    x2 = magnitude < -0.5 ? x2 - 1.0 : (magnitude > 0.5 ? x2 + 1.0 : x2);
	    x2 = x1 - x2;
	    x2 *= interval.x;
	    
	    magnitude = y1 - y2;
	    y2 = magnitude < -0.5 ? y2 - 1.0 : (magnitude > 0.5 ? y2 + 1.0 : y2);
	    y2 = y1 - y2;
	    y2 *= interval.y;
	    
	    magnitude = z1 - z2;
	    z2 = magnitude < -0.5 ? z2 - 1.0 : (magnitude > 0.5 ? z2 + 1.0 : z2);
	    z2 = z1 - z2;
	    z2 *= interval.z;
	    
	    magnitude = sqrt(x2 * x2 + y2 * y2 + z2 * z2);
	    DEVICE_ASSERT(magnitude > 0);
	    DEVICE_ASSERT(radii > 0);
	    
	    double tempVal = 0;
	    double invRadii = 1.0 / radii;
	    if (calculateEnergy)
	    {
	        tempVal = radii - magnitude;
		tempVal *= tempVal;
		tempVal *= invRadii;
		
		e[tid] = tempVal;
	    }
	    
	    tempVal = 1.0 / magnitude;
	    
	    x2 *= tempVal - invRadii;
	    y2 *= tempVal - invRadii;
	    z2 *= tempVal - invRadii;

	    ax[tid] = x2;
	    ay[tid] = y2;
	    az[tid] = z2;
	    
	    if (useGasExchange)
	    {
		if (magnitude > r1 && magnitude > r2)
		{
		    radii = r2 * r2;
		    tempVal = 0.5 * (radii - r1 * r1 + magnitude * magnitude) * tempVal;
		    tempVal *= tempVal;
		    tempVal = radii - tempVal;
		    DEVICE_ASSERT(tempVal > -0.001);
		    tempVal = tempVal < 0 ? -tempVal : tempVal;
		    DEVICE_ASSERT(tempVal >= 0);
		    
#if (NUM_DIM == 3)
		    tempVal = tempVal * 0.25;
		    tempVal /= r1 * r1;
#else
		    tempVal = sqrt(tempVal) / (pi * r1);
#endif
		    areaOverlap[tid] = tempVal;
		    
		    tempVal *= 1.0 / r2 - 1.0 / r1;
		}
		else
		    tempVal = 0.0;
		
		ar[tid] = tempVal;
	    }
	}
    }
}

__global__
void cubble::calculateVelocityFromAccelerations(double *ax,
						double *ay,
						double *az,
						double *ar,
						double *e,
						double *areaOverlap,
			
						double *dxdt,
						double *dydt,
						double *dzdt,
						double *drdt,

						double *freeArea,
						double *energies,

						int numBubbles,
						int neighborStride,
						double fZeroPerMuZero,
						bool calculateEnergy,
						bool useGasExchange)
{
    const int tid = getGlobalTid();
    if (tid < numBubbles)
    {
	double vx = 0.0;
	double vy = 0.0;
	double vz = 0.0;
	double vr = 0.0;
	double energy = 0;
	double a = 0;
	
	if (useGasExchange && calculateEnergy)
	{
	    for (int i = 0; i < neighborStride; ++i)
	    {
		vx += ax[tid + i * numBubbles];
		vy += ay[tid + i * numBubbles];
		vz += az[tid + i * numBubbles];
		vr += ar[tid + i * numBubbles];
		a += areaOverlap[tid + i * numBubbles];
		energy += e[tid + i * numBubbles];
	    }
	}
	else if (useGasExchange)
	{
	    for (int i = 0; i < neighborStride; ++i)
	    {
		vx += ax[tid + i * numBubbles];
		vy += ay[tid + i * numBubbles];
		vz += az[tid + i * numBubbles];
		vr += ar[tid + i * numBubbles];
		a += areaOverlap[tid + i * numBubbles];
	    }
	}
	else if (calculateEnergy)
	{
	    for (int i = 0; i < neighborStride; ++i)
	    {
		vx += ax[tid + i * numBubbles];
		vy += ay[tid + i * numBubbles];
		vz += az[tid + i * numBubbles];
		energy += e[tid + i * numBubbles];
	    }
	}
	else
	{
	    for (int i = 0; i < neighborStride; ++i)
	    {
		vx += ax[tid + i * numBubbles];
		vy += ay[tid + i * numBubbles];
		vz += az[tid + i * numBubbles];
	    }
	}
	    
	dxdt[tid] = vx * fZeroPerMuZero;
	dydt[tid] = vy * fZeroPerMuZero;
	dzdt[tid] = vz * fZeroPerMuZero;

	if (useGasExchange)
	{
	    drdt[tid] = vr;
	    if (!(a < 1.0))
	    {
		for (int i = 0; i < neighborStride; ++i)
		    printf("%d %d %d %f %f\n", i, tid, numBubbles, areaOverlap[tid + i * numBubbles], a);
	    }
	    
	    DEVICE_ASSERT(a < 1.0);
	    freeArea[tid] = 1.0 - a;
	}
	
	if (calculateEnergy)
	    energies[tid] = energy;
    }
}

__global__
void cubble::calculateFreeAreaPerRadius(double *r, double *freeArea, double *output, int numBubbles)
{
    const int tid = getGlobalTid();
    if (tid < numBubbles)
	output[tid] = freeArea[tid] * r[tid];
}

__global__
void cubble::calculateFinalRadiusChangeRate(double *drdt,
					    double *r,
					    double *freeArea,
					    int numBubbles,
					    double invRho,
					    double kappa,
					    double kParam)
{
    const int tid = getGlobalTid();
    if (tid < numBubbles)
    {
	double vr = drdt[tid];
	vr += kappa * freeArea[tid] * (invRho - 1.0 / r[tid]);
	drdt[tid] = kParam * vr;
    }
}

__global__
void cubble::correct(double *x,
		     double *y,
		     double *z,
		     double *r,
		     
		     double *xPrd,
		     double *yPrd,
		     double *zPrd,
		     double *rPrd,
		     
		     double *dxdt,
		     double *dydt,
		     double *dzdt,
		     double *drdt,
		     
		     double *dxdtPrd,
		     double *dydtPrd,
		     double *dzdtPrd,
		     double *drdtPrd,
		     
		     double *errors,
		     dvec tfr,
		     dvec lbb,
		     double timeStep,
		     int numBubbles,
		     bool useGasExchange)
{
    const int tid = getGlobalTid();
    if (tid < numBubbles)
    {   
	// Measure if it's faster to calculate these per component...
	const dvec interval = (tfr - lbb);
	dvec pos, posPrd, vel, velPrd;
	pos.x = x[tid];
	pos.y = y[tid];
	pos.z = z[tid];

	posPrd.x = xPrd[tid];
	posPrd.y = yPrd[tid];
	posPrd.z = zPrd[tid];
	
	vel.x = dxdt[tid];
	vel.y = dydt[tid];
	vel.z = dzdt[tid];
	
	velPrd.x = dxdtPrd[tid];
	velPrd.y = dydtPrd[tid];
	velPrd.z = dzdtPrd[tid];

	pos = lbb + pos * interval;
	pos += 0.5 * timeStep * (vel + velPrd);
	pos = (pos - lbb) / interval;
	pos = getWrappedPos(pos);

	double radError = 0;
	if (useGasExchange)
	{
	    const double radius = r[tid] + 0.5 * timeStep
		* (drdt[tid] + drdtPrd[tid]);
	    
	    radError = radius - rPrd[tid];
	    radError = radError < 0 ? -radError : radError;

	    rPrd[tid] = radius;
	}

	double error = (pos - posPrd).getAbsolute().getMaxComponent();
	error = error > radError ? error : radError;
	errors[tid] = error;

	xPrd[tid] = pos.x;
	yPrd[tid] = pos.y;
	zPrd[tid] = pos.z;
    }
}

__global__
void cubble::addVolume(double *r, int numBubbles, double volumeMultiplier)
{
    const int tid = getGlobalTid();
    if (tid < numBubbles)
	r[tid] = r[tid] * volumeMultiplier;
}

__global__
void cubble::eulerIntegration(double *x,
			      double *y,
			      double *z,
			      double *r,
			      
			      double *dxdt,
			      double *dydt,
			      double *dzdt,
			      double *drdt,
			      
			      dvec tfr,
			      dvec lbb,
			      double timeStep,
			      int numBubbles)
{
    const int tid = getGlobalTid();
    if (tid < numBubbles)
    {
	dvec interval = tfr - lbb;
	dvec pos(0, 0, 0);
	pos.x = x[tid];
	pos.y = y[tid];
	pos.z = z[tid];

	dvec vel(0, 0, 0);
	vel.x = dxdt[tid];
	vel.y = dydt[tid];
	vel.z = dzdt[tid];

	pos = lbb + pos * interval;
	pos += timeStep * vel;
	pos = (pos - lbb) / interval;
	pos = getWrappedPos(pos);
	
	x[tid] = pos.x;
	y[tid] = pos.y;
	z[tid] = pos.z;
	r[tid] = r[tid] + timeStep * drdt[tid];
    }
}


// ******************************
// Device functions
// ******************************

__forceinline__ __device__
int cubble::getNeighborCellIndex(ivec cellIdx, ivec dim, int neighborNum)
{
    // Switch statements and ifs that diverge inside one warp/block are
    // detrimental for performance. However, this should never diverge,
    // as all the threads of one block should always be in the same cell
    // going for the same neighbor.
    ivec idxVec = cellIdx;
    switch(neighborNum)
    {
    case 0:
	// self
	break;
    case 1:
	idxVec += ivec(-1, 1, 0);
	break;
    case 2:
	idxVec += ivec(-1, 0, 0);
	break;
    case 3:
	idxVec += ivec(-1, -1, 0);
	break;
    case 4:
	idxVec += ivec(0, -1, 0);
	break;
#if NUM_DIM == 3
    case 5:
	idxVec += ivec(-1, 1, -1);
	break;
    case 6:
	idxVec += ivec(-1, 0, -1);
	break;
    case 7:
	idxVec += ivec(-1, -1, -1);
	break;
    case 8:
	idxVec += ivec(0, 1, -1);
	break;
    case 9:
	idxVec += ivec(0, 0, -1);
	break;
    case 10:
	idxVec += ivec(0, -1, -1);
	break;
    case 11:
	idxVec += ivec(1, 1, -1);
	break;
    case 12:
	idxVec += ivec(1, 0, -1);
	break;
    case 13:
	idxVec += ivec(1, -1, -1);
	break;
#endif
    default:
	printf("Should never end up here!");
	break;
    }

    idxVec += dim;
    idxVec %= dim;

    return idxVec.z * dim.y * dim.x + idxVec.y * dim.x + idxVec.x;
}

__forceinline__ __device__
void cubble::getDomainOffsetsAndIntervals(int numBubbles,
					  int numDomains,
					  int numCells,
					  ivec cellIdxVec,
					  ivec boxDim,
					  Cell *cells,
					  int &outXBegin,
					  int &outXInterval,
					  int &outYBegin,
					  int &outYInterval,
					  bool &outIsOwnCell)
{
    int domain = blockIdx.z % numDomains;
    int di = (2 * domain) / numDomains;
    
    DEVICE_ASSERT((di == 0 && domain < (int)(0.5f * numDomains))
	   || (di == 1 && domain >= (int)(0.5f * numDomains)));
    
    int dj = domain % (int)(0.5f * numDomains);
    int djMod2 = dj % 2;

    // Find this cell
    int selfCellIndex = cellIdxVec.z * boxDim.x * boxDim.y
	+ cellIdxVec.y * boxDim.x
	+ cellIdxVec.x;
    DEVICE_ASSERT(selfCellIndex < numCells);
    Cell self = cells[selfCellIndex];

    // Find the neighbor of this cell
    int neighborCellIndex = getNeighborCellIndex(cellIdxVec, boxDim, dj / 2);
    DEVICE_ASSERT(neighborCellIndex < numCells);
    Cell neighbor = cells[neighborCellIndex];
    
    outIsOwnCell = selfCellIndex == neighborCellIndex;

    // Find the interval of values to use:
    // x-axis uses the right or the left half of the neighbor cell
    int halfSize = 0.5f * neighbor.size;
    outXBegin = neighbor.offset + djMod2 * halfSize;
    outXInterval = halfSize + djMod2 * (neighbor.size % 2);
    
    DEVICE_ASSERT(outXBegin + outXInterval <= numBubbles);
    DEVICE_ASSERT(outXBegin + outXInterval <= neighbor.size + neighbor.offset);
    DEVICE_ASSERT(outXInterval == halfSize || outXInterval == halfSize + 1);

    // y-axis uses the top or bottom half of this cell
    halfSize = 0.5f * self.size;
    outYBegin = self.offset + di * halfSize;
    outYInterval = halfSize + di * (self.size % 2);

    DEVICE_ASSERT(outYBegin + outYInterval <= numBubbles);
    DEVICE_ASSERT(outYInterval == halfSize || outYInterval == halfSize + 1);
    DEVICE_ASSERT(outYBegin + outYInterval <= self.size + self.offset);
}

__forceinline__ __device__
int cubble::getGlobalTid()
{
    // Simple helper function for calculating a 1D coordinate
    // from 1, 2 or 3 dimensional coordinates.
    int threadsPerBlock = blockDim.x * blockDim.y * blockDim.z;
    int blocksBefore = blockIdx.z * (gridDim.y * gridDim.x)
	+ blockIdx.y * gridDim.x
	+ blockIdx.x;
    int threadsBefore = blockDim.y * blockDim.x * threadIdx.z + blockDim.x * threadIdx.y;
    int tid = blocksBefore * threadsPerBlock + threadsBefore + threadIdx.x;

    return tid;
}

__forceinline__ __device__
cubble::dvec cubble::getShortestWrappedNormalizedVec(dvec pos1, dvec pos2)
{
    dvec temp = pos1 - pos2;
    pos2.x = temp.x < -0.5 ? pos2.x - 1.0 : (temp.x > 0.5 ? pos2.x + 1.0 : pos2.x);
    pos2.y = temp.y < -0.5 ? pos2.y - 1.0 : (temp.y > 0.5 ? pos2.y + 1.0 : pos2.y);
    pos2.z = temp.z < -0.5 ? pos2.z - 1.0 : (temp.z > 0.5 ? pos2.z + 1.0 : pos2.z);
    
    return pos1 - pos2;
}

__forceinline__ __device__
cubble::dvec cubble::getWrappedPos(dvec pos)
{
    // ASSUMPTION: Using normalized position
    // ASSUMPTION: Position never smaller/greater than -1/1
    pos.x = pos.x < 0 ? pos.x + 1.0 : (pos.x > 1 ? pos.x - 1.0 : pos.x);
    pos.y = pos.y < 0 ? pos.y + 1.0 : (pos.y > 1 ? pos.y - 1.0 : pos.y);
    pos.z = pos.z < 0 ? pos.z + 1.0 : (pos.z > 1 ? pos.z - 1.0 : pos.z);

    return pos;
}
