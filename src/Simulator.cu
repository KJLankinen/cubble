#include "hip/hip_runtime.h"
// -*- C++ -*-

#include "Simulator.h"
#include "Macros.h"
#include "Vec.h"
#include "Util.h"

#include "cub/hipcub/hipcub.hpp"

#include <iostream>
#include <sstream>
#include <chrono>
#include <algorithm>
#include <vector>
#include <hiprand.h>

#include <hip/hip_runtime_api.h>
#include <nvToolsExt.h>


// ******************************
// Class functions run on CPU
// ******************************

cubble::Simulator::Simulator(std::shared_ptr<Env> e)
{
    env = e;
    
#if (NUM_DIM == 3)
    givenNumBubblesPerDim = std::ceil(std::cbrt((float)env->getNumBubbles()));
    numBubbles = givenNumBubblesPerDim * givenNumBubblesPerDim * givenNumBubblesPerDim;
#else
    givenNumBubblesPerDim = std::ceil(std::sqrt((float)env->getNumBubbles()));
    numBubbles = givenNumBubblesPerDim * givenNumBubblesPerDim;
#endif
    const dvec tfr = env->getLbb() + env->getAvgRad() * (double)givenNumBubblesPerDim * 2;
    env->setTfr(tfr);

    bubbleData = FixedSizeDeviceArray<double>(numBubbles, (size_t)BubbleProperty::NUM_VALUES);
    aboveMinRadFlags = FixedSizeDeviceArray<int>(numBubbles, 2);
    indicesPerCell = FixedSizeDeviceArray<int>(numBubbles, 1);

    // TODO: Figure out a more sensible value for this.
    const int maxNumPairs = (CUBBLE_NUM_NEIGHBORS + 1) * env->getNumBubblesPerCell() * numBubbles;
    neighborPairIndices = FixedSizeDeviceArray<int>(maxNumPairs, 4);
    numPairs = FixedSizeDeviceArray<int>(1, 1);
    
    const dim3 gridSize = getGridSize();
    size_t numCells = gridSize.x * gridSize.y * gridSize.z;
    cellData = FixedSizeDeviceArray<int>(numCells, (size_t)CellProperty::NUM_VALUES);
    
    cubOutputData = FixedSizeDeviceArray<char>(sizeof(double), 1);
    cubTemporaryStorage = FixedSizeDeviceArray<char>(numBubbles * sizeof(double), 1);
    
    hostData.resize(bubbleData.getSize(), 0);
    
    printRelevantInfoOfCurrentDevice();
}

cubble::Simulator::~Simulator() {}

void cubble::Simulator::setupSimulation()
{
    NVTX_RANGE_PUSH_A(__FUNCTION__);
    
    generateBubbles();
    deleteSmallBubbles();
    updateCellsAndNeighbors();

    // Calculate some initial values which are needed
    // for the two-step Adams-Bashforth-Moulton perdictor-corrector method (ABMpc).

    double *x = bubbleData.getRowPtr((size_t)BubbleProperty::X);
    double *y = bubbleData.getRowPtr((size_t)BubbleProperty::Y);
    double *z = bubbleData.getRowPtr((size_t)BubbleProperty::Z);
    double *r = bubbleData.getRowPtr((size_t)BubbleProperty::R);
    
    double *dxdt = bubbleData.getRowPtr((size_t)BubbleProperty::DXDT);
    double *dydt = bubbleData.getRowPtr((size_t)BubbleProperty::DYDT);
    double *dzdt = bubbleData.getRowPtr((size_t)BubbleProperty::DZDT);
    double *drdt = bubbleData.getRowPtr((size_t)BubbleProperty::DRDT);
    
    double *dxdtOld = bubbleData.getRowPtr((size_t)BubbleProperty::DXDT_OLD);
    double *dydtOld = bubbleData.getRowPtr((size_t)BubbleProperty::DYDT_OLD);
    double *dzdtOld = bubbleData.getRowPtr((size_t)BubbleProperty::DZDT_OLD);
    double *drdtOld = bubbleData.getRowPtr((size_t)BubbleProperty::DRDT_OLD);
    
    double *energies = bubbleData.getRowPtr((size_t)BubbleProperty::ENERGY);
    double *freeArea = bubbleData.getRowPtr((size_t)BubbleProperty::FREE_AREA);

    int *firstIndices = neighborPairIndices.getRowPtr(0);
    int *secondIndices = neighborPairIndices.getRowPtr(1);

    const dvec tfr = env->getTfr();
    const dvec lbb = env->getLbb();
    const double minRad = env->getMinRad();
    const size_t numThreads = 128;
    const size_t numBlocks = (size_t)std::ceil(numBubbles / (float)numThreads);
    const size_t numBlocksAcc = (size_t)std::ceil(hostNumPairs / (float)numThreads);

    double timeStep = env->getTimeStep();
    
    size_t numBytesToReset = sizeof(double) * 6 * bubbleData.getWidth();
    CUDA_CALL(hipMemset(static_cast<void*>(energies), 0, numBytesToReset));

    std::cout << "Calculating some initial values as a part of setup." << std::endl;

    calculateVelocityAndGasExchange<<<numBlocksAcc, numThreads>>>(x, y, z, r,
								  dxdtOld, dydtOld, dzdtOld, drdtOld,
								  energies,
								  freeArea,
								  firstIndices,
								  secondIndices,
								  numBubbles,
								  hostNumPairs,
								  env->getFZeroPerMuZero(),
								  env->getPi(),
								  tfr - lbb,
								  false,
								  false);
    
    eulerIntegration<<<numBlocks, numThreads>>>(x, y, z, r,
						dxdtOld, dydtOld, dzdtOld, drdtOld,
						tfr, lbb, timeStep, numBubbles);

    if (deleteSmallBubbles())
	updateCellsAndNeighbors();

    CUDA_CALL(hipMemset(static_cast<void*>(energies), 0, numBytesToReset));
    
    calculateVelocityAndGasExchange<<<numBlocksAcc, numThreads>>>(x, y, z, r,
								  dxdtOld, dydtOld, dzdtOld, drdtOld,
								  energies,
								  freeArea,
								  firstIndices,
								  secondIndices,
								  numBubbles,
								  hostNumPairs,
								  env->getFZeroPerMuZero(),
								  env->getPi(),
								  env->getTfr() - env->getLbb(),
								  false,
								  false);
    
    NVTX_RANGE_POP();
}

bool cubble::Simulator::integrate(bool useGasExchange, bool calculateEnergy)
{
    NVTX_RANGE_PUSH_A(__FUNCTION__);
    
    const dvec tfr = env->getTfr();
    const dvec lbb = env->getLbb();
    const double minRad = env->getMinRad();
    const size_t numThreads = 128;
    const size_t numBlocks = (size_t)std::ceil(numBubbles / (float)numThreads);
    const size_t numBlocksAcc = (size_t)std::ceil(hostNumPairs / (float)numThreads);

    double timeStep = env->getTimeStep();
    double error = 0;

    double *x = bubbleData.getRowPtr((size_t)BubbleProperty::X);
    double *y = bubbleData.getRowPtr((size_t)BubbleProperty::Y);
    double *z = bubbleData.getRowPtr((size_t)BubbleProperty::Z);
    double *r = bubbleData.getRowPtr((size_t)BubbleProperty::R);
    
    double *xPrd = bubbleData.getRowPtr((size_t)BubbleProperty::X_PRD);
    double *yPrd = bubbleData.getRowPtr((size_t)BubbleProperty::Y_PRD);
    double *zPrd = bubbleData.getRowPtr((size_t)BubbleProperty::Z_PRD);
    double *rPrd = bubbleData.getRowPtr((size_t)BubbleProperty::R_PRD);
    
    double *dxdt = bubbleData.getRowPtr((size_t)BubbleProperty::DXDT);
    double *dydt = bubbleData.getRowPtr((size_t)BubbleProperty::DYDT);
    double *dzdt = bubbleData.getRowPtr((size_t)BubbleProperty::DZDT);
    double *drdt = bubbleData.getRowPtr((size_t)BubbleProperty::DRDT);
    
    double *dxdtPrd = bubbleData.getRowPtr((size_t)BubbleProperty::DXDT_PRD);
    double *dydtPrd = bubbleData.getRowPtr((size_t)BubbleProperty::DYDT_PRD);
    double *dzdtPrd = bubbleData.getRowPtr((size_t)BubbleProperty::DZDT_PRD);
    double *drdtPrd = bubbleData.getRowPtr((size_t)BubbleProperty::DRDT_PRD);
    
    double *dxdtOld = bubbleData.getRowPtr((size_t)BubbleProperty::DXDT_OLD);
    double *dydtOld = bubbleData.getRowPtr((size_t)BubbleProperty::DYDT_OLD);
    double *dzdtOld = bubbleData.getRowPtr((size_t)BubbleProperty::DZDT_OLD);
    double *drdtOld = bubbleData.getRowPtr((size_t)BubbleProperty::DRDT_OLD);

    double *energies = bubbleData.getRowPtr((size_t)BubbleProperty::ENERGY);
    double *errors = bubbleData.getRowPtr((size_t)BubbleProperty::ERROR);
    double *volumes = bubbleData.getRowPtr((size_t)BubbleProperty::VOLUME);
    double *freeArea = bubbleData.getRowPtr((size_t)BubbleProperty::FREE_AREA);

    int *firstIndices = neighborPairIndices.getRowPtr(0);
    int *secondIndices = neighborPairIndices.getRowPtr(1);

    do
    {
	predict<<<numBlocks, numThreads>>>(x, y, z, r,
					   xPrd, yPrd, zPrd, rPrd,
					   dxdt, dydt, dzdt, drdt,
					   dxdtOld, dydtOld, dzdtOld, drdtOld,
					   tfr, lbb, timeStep, numBubbles, useGasExchange);

	// Using atomicAdd, so these need to be reset to 0 every time before use.
	size_t numBytesToReset = sizeof(double) * 7 * bubbleData.getWidth();
	CUDA_CALL(hipMemset(static_cast<void*>(dxdtPrd), 0, numBytesToReset));

	calculateVelocityAndGasExchange<<<numBlocksAcc, numThreads>>>(xPrd, yPrd, zPrd, rPrd,
								      dxdtPrd, dydtPrd, dzdtPrd, drdtPrd,
								      energies,
								      freeArea,
								      firstIndices,
								      secondIndices,
								      numBubbles,
								      hostNumPairs,
								      env->getFZeroPerMuZero(),
								      env->getPi(),
								      env->getTfr() - env->getLbb(),
								      calculateEnergy,
								      useGasExchange);

	if (useGasExchange)
	{
	    calculateFreeAreaPerRadius<<<numBlocks, numThreads>>>(rPrd, freeArea, errors, env->getPi(), numBubbles);
	    double invRho = cubReduction<double, double*, double*>(&hipcub::DeviceReduce::Sum, errors, numBubbles);
	    invRho /= cubReduction<double, double*, double*>(&hipcub::DeviceReduce::Sum, freeArea, numBubbles);
	    calculateFinalRadiusChangeRate<<<numBlocks, numThreads>>>(drdtPrd,
								      rPrd,
								      freeArea,
								      numBubbles,
								      invRho,
								      1.0 / env->getPi(),
								      env->getKappa(),
								      env->getKParameter());
	}
        
	correct<<<numBlocks, numThreads>>>(x, y, z, r,
					   xPrd, yPrd, zPrd, rPrd,
					   dxdt, dydt, dzdt, drdt,
					   dxdtPrd, dydtPrd, dzdtPrd, drdtPrd,
					   errors,
					   aboveMinRadFlags.getRowPtr(0),
					   env->getMinRad(),
					   tfr,
					   lbb,
					   timeStep,
					   numBubbles,
					   useGasExchange);
        
        error = cubReduction<double, double*, double*>(&hipcub::DeviceReduce::Max, errors, numBubbles);

	if (error < env->getErrorTolerance() / 10 && timeStep < 0.1)
	    timeStep *= 1.9;
	else if (error > env->getErrorTolerance())
	    timeStep *= 0.5;
    }
    while (error > env->getErrorTolerance());

    updateData();
    
    ++integrationStep;
    env->setTimeStep(timeStep);
    SimulationTime += timeStep;

    if (calculateEnergy)
	ElasticEnergy = cubReduction<double, double*, double*>(&hipcub::DeviceReduce::Sum, energies, numBubbles);
    
    if (deleteSmallBubbles() || integrationStep % 50 == 0)
	updateCellsAndNeighbors();

    NVTX_RANGE_POP();

    return numBubbles > env->getMinNumBubbles();
}

void cubble::Simulator::generateBubbles()
{
    NVTX_RANGE_PUSH_A(__FUNCTION__);
    
    std::cout << "Starting to generate data for bubbles." << std::endl;
    
    const int rngSeed = env->getRngSeed();
    const double avgRad = env->getAvgRad();
    const double stdDevRad = env->getStdDevRad();
    const dvec tfr = env->getTfr();
    const dvec lbb = env->getLbb();

    double *x = bubbleData.getRowPtr((size_t)BubbleProperty::X);
    double *y = bubbleData.getRowPtr((size_t)BubbleProperty::Y);
    double *z = bubbleData.getRowPtr((size_t)BubbleProperty::Z);
    
    double *xPrd = bubbleData.getRowPtr((size_t)BubbleProperty::X_PRD);
    double *yPrd = bubbleData.getRowPtr((size_t)BubbleProperty::Y_PRD);
    double *zPrd = bubbleData.getRowPtr((size_t)BubbleProperty::Z_PRD);
    
    double *r = bubbleData.getRowPtr((size_t)BubbleProperty::R);
    double *w = bubbleData.getRowPtr((size_t)BubbleProperty::R_PRD);
    
    hiprandGenerator_t generator;
    CURAND_CALL(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_MTGP32));
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(generator, rngSeed));
    
    CURAND_CALL(hiprandGenerateUniformDouble(generator, x, numBubbles));
    CURAND_CALL(hiprandGenerateUniformDouble(generator, y, numBubbles));
    CURAND_CALL(hiprandGenerateUniformDouble(generator, z, numBubbles));
    CURAND_CALL(hiprandGenerateUniformDouble(generator, w, numBubbles));
    CURAND_CALL(hiprandGenerateNormalDouble(generator, r, numBubbles, avgRad, stdDevRad));

    CURAND_CALL(hiprandDestroyGenerator(generator));

    const size_t numThreads = 128;
    const size_t numBlocks = (size_t)std::ceil((float)numBubbles / (float)numThreads);
    assignDataToBubbles<<<numBlocks, numThreads>>>(x, y, z,
						   xPrd, yPrd, zPrd,
						   r, w,
						   aboveMinRadFlags.getRowPtr(0),
						   givenNumBubblesPerDim,
						   tfr,
						   lbb,
						   avgRad,
						   env->getMinRad(),
						   numBubbles);
    NVTX_RANGE_POP();
}

void cubble::Simulator::updateCellsAndNeighbors()
{
    NVTX_RANGE_PUSH_A(__FUNCTION__);

    dim3 gridSize = getGridSize();
    const int numCells = gridSize.x * gridSize.y * gridSize.z;
    const dvec domainDim(gridSize.x, gridSize.y, gridSize.z);
    const size_t numThreads = 256;
    size_t numBlocks = (size_t)std::ceil(numBubbles / (float)numThreads);

    NVTX_RANGE_PUSH_A("Memsets");
    cellData.setBytesToZero();
    indicesPerCell.setBytesToZero();
    neighborPairIndices.setBytesToZero();
    numPairs.setBytesToZero();
    NVTX_RANGE_POP();
    
    double *x = bubbleData.getRowPtr((size_t)BubbleProperty::X);
    double *y = bubbleData.getRowPtr((size_t)BubbleProperty::Y);
    double *z = bubbleData.getRowPtr((size_t)BubbleProperty::Z);
    double *r = bubbleData.getRowPtr((size_t)BubbleProperty::R);
    int *offsets = cellData.getRowPtr((size_t)CellProperty::OFFSET);
    int *sizes = cellData.getRowPtr((size_t)CellProperty::SIZE);
    
    NVTX_RANGE_PUSH_A("Offsets");
    calculateOffsets<<<numBlocks, numThreads>>>(x, y, z, sizes, domainDim, numBubbles, numCells);
    NVTX_RANGE_POP();

    NVTX_RANGE_PUSH_A("Exclusive sum");
    cubScan<int*, int*>(&hipcub::DeviceScan::ExclusiveSum, sizes, offsets, numCells);
    NVTX_RANGE_POP();

    NVTX_RANGE_PUSH_A("Memset sizes");
    CUDA_CALL(hipMemset(static_cast<void*>(sizes), 0, sizeof(int) * numCells));
    NVTX_RANGE_POP();

    NVTX_RANGE_PUSH_A("Bubbles2Cells");
    bubblesToCells<<<numBlocks, numThreads>>>(x, y, z,
					      indicesPerCell.getDataPtr(),
					      offsets,
					      sizes,
					      domainDim,
					      numBubbles);
    NVTX_RANGE_POP();

    gridSize.z *= CUBBLE_NUM_NEIGHBORS + 1;
    assertGridSizeBelowLimit(gridSize);

    NVTX_RANGE_PUSH_A("MaxNumCellRed");
    int sharedMemSizeInBytes = cubReduction<int, int*, int*>(&hipcub::DeviceReduce::Max, sizes, numCells);
    NVTX_RANGE_POP();

    sharedMemSizeInBytes *= sharedMemSizeInBytes;
    sharedMemSizeInBytes *= 2;
    sharedMemSizeInBytes *= sizeof(int);

    assertMemBelowLimit(sharedMemSizeInBytes);
    assert(sharedMemSizeInBytes > 0 && "Zero bytes of shared memory reserved!");

    NVTX_RANGE_PUSH_A("find");
    findBubblePairs<<<gridSize, numThreads, sharedMemSizeInBytes>>>(x, y, z, r,
								    indicesPerCell.getDataPtr(),
								    offsets,
								    sizes,
								    neighborPairIndices.getRowPtr(2),
								    neighborPairIndices.getRowPtr(3),
								    numPairs.getDataPtr(),
								    numCells,
								    numBubbles,
								    env->getTfr() - env->getLbb(),
								    sharedMemSizeInBytes / sizeof(int),
								    neighborPairIndices.getWidth());
    NVTX_RANGE_POP();

    NVTX_RANGE_PUSH_A("mecpy numpairs");
    CUDA_CALL(hipMemcpy(&hostNumPairs, static_cast<void*>(numPairs.getDataPtr()), sizeof(int), hipMemcpyDeviceToHost));
    NVTX_RANGE_POP();

    cubSortPairs<int, int>(&hipcub::DeviceRadixSort::SortPairs,
			   const_cast<const int*>(neighborPairIndices.getRowPtr(2)),
			   neighborPairIndices.getRowPtr(0),
			   const_cast<const int*>(neighborPairIndices.getRowPtr(3)),
			   neighborPairIndices.getRowPtr(1),
			   hostNumPairs);
    
    NVTX_RANGE_POP();
}

void cubble::Simulator::updateData()
{
    NVTX_RANGE_PUSH_A(__FUNCTION__);
    
    // x, y, z, r are in memory continuously, so we can just make three copies with 4x the data of one component.
    size_t numBytesToCopy = 4 * sizeof(double) * bubbleData.getWidth();

    double *x = bubbleData.getRowPtr((size_t)BubbleProperty::X);
    double *xPrd = bubbleData.getRowPtr((size_t)BubbleProperty::X_PRD);
    double *dxdt = bubbleData.getRowPtr((size_t)BubbleProperty::DXDT);
    double *dxdtPrd = bubbleData.getRowPtr((size_t)BubbleProperty::DXDT_PRD);
    double *dxdtOld = bubbleData.getRowPtr((size_t)BubbleProperty::DXDT_OLD);
    
    CUDA_CALL(hipMemcpyAsync(x, xPrd, numBytesToCopy, hipMemcpyDeviceToDevice));
    CUDA_CALL(hipMemcpyAsync(dxdtOld, dxdt, numBytesToCopy, hipMemcpyDeviceToDevice));
    CUDA_CALL(hipMemcpyAsync(dxdt, dxdtPrd, numBytesToCopy, hipMemcpyDeviceToDevice));

    NVTX_RANGE_POP();
}

bool cubble::Simulator::deleteSmallBubbles()
{
    NVTX_RANGE_PUSH_A(__FUNCTION__);

    int *flag = aboveMinRadFlags.getRowPtr(0);
    const int numBubblesAboveMinRad = cubReduction<int, int*, int*>(&hipcub::DeviceReduce::Sum,
								    flag,
								    numBubbles);

    bool atLeastOneBubbleDeleted = numBubblesAboveMinRad < numBubbles;
    if (atLeastOneBubbleDeleted)
    {
	NVTX_RANGE_PUSH_A("BubbleRemoval");
    
	const size_t numThreads = 128;
	const size_t numBlocks = (size_t)std::ceil(numBubbles / (float)numThreads);

	double *x = bubbleData.getRowPtr((size_t)BubbleProperty::X);
	double *y = bubbleData.getRowPtr((size_t)BubbleProperty::Y);
	double *z = bubbleData.getRowPtr((size_t)BubbleProperty::Z);
	double *r = bubbleData.getRowPtr((size_t)BubbleProperty::R);
	
	double *xPrd = bubbleData.getRowPtr((size_t)BubbleProperty::X_PRD);
	double *yPrd = bubbleData.getRowPtr((size_t)BubbleProperty::Y_PRD);
	double *zPrd = bubbleData.getRowPtr((size_t)BubbleProperty::Z_PRD);
	double *rPrd = bubbleData.getRowPtr((size_t)BubbleProperty::R_PRD);
	
	double *dxdt = bubbleData.getRowPtr((size_t)BubbleProperty::DXDT);
	double *dydt = bubbleData.getRowPtr((size_t)BubbleProperty::DYDT);
	double *dzdt = bubbleData.getRowPtr((size_t)BubbleProperty::DZDT);
	double *drdt = bubbleData.getRowPtr((size_t)BubbleProperty::DRDT);
	
	double *dxdtPrd = bubbleData.getRowPtr((size_t)BubbleProperty::DXDT_PRD);
	double *dydtPrd = bubbleData.getRowPtr((size_t)BubbleProperty::DYDT_PRD);
	double *dzdtPrd = bubbleData.getRowPtr((size_t)BubbleProperty::DZDT_PRD);
	double *drdtPrd = bubbleData.getRowPtr((size_t)BubbleProperty::DRDT_PRD);
	
	double *dxdtOld = bubbleData.getRowPtr((size_t)BubbleProperty::DXDT_OLD);
	double *dydtOld = bubbleData.getRowPtr((size_t)BubbleProperty::DYDT_OLD);
	double *dzdtOld = bubbleData.getRowPtr((size_t)BubbleProperty::DZDT_OLD);
	double *drdtOld = bubbleData.getRowPtr((size_t)BubbleProperty::DRDT_OLD);
	
	double *energies = bubbleData.getRowPtr((size_t)BubbleProperty::ENERGY);
	double *errors = bubbleData.getRowPtr((size_t)BubbleProperty::ERROR);
	double *volumes = bubbleData.getRowPtr((size_t)BubbleProperty::VOLUME);
	double *freeArea = bubbleData.getRowPtr((size_t)BubbleProperty::FREE_AREA);

	double *volumeMultiplier = errors + numBubblesAboveMinRad;
	hipMemset(static_cast<void*>(volumeMultiplier), 0, sizeof(double));
	
	calculateRedistributedGasVolume<<<numBlocks, numThreads>>>(volumes,
								   r,
								   flag,
								   volumeMultiplier,
								   env->getPi(),
								   numBubbles);

	int *newIdx = aboveMinRadFlags.getRowPtr(1);
	cubScan<int*, int*>(&hipcub::DeviceScan::ExclusiveSum, flag, newIdx, numBubbles);

	removeSmallBubbles<<<numBlocks, numThreads>>>(xPrd, yPrd, zPrd, rPrd,
						      x, y, z, r,
						      dxdtPrd, dydtPrd, dzdtPrd, drdtPrd,
						      dxdt, dydt, dzdt, drdt,
						      energies, freeArea, errors, volumes,
						      dxdtOld, dydtOld, dzdtOld, drdtOld,
						      newIdx,
						      flag,
						      numBubbles);
        
	const size_t numBytesToCopy = 2 * sizeof(double) * bubbleData.getWidth();
	CUDA_CALL(hipMemcpyAsync(x, xPrd, 2 * numBytesToCopy, hipMemcpyDeviceToDevice));
	CUDA_CALL(hipMemcpyAsync(dxdt, dxdtPrd, 2 * numBytesToCopy, hipMemcpyDeviceToDevice));
	CUDA_CALL(hipMemcpyAsync(dxdtOld, energies, numBytesToCopy, hipMemcpyDeviceToDevice));
	CUDA_CALL(hipMemcpyAsync(dzdtOld, errors, numBytesToCopy, hipMemcpyDeviceToDevice));
	
	numBubbles = numBubblesAboveMinRad;
	const double invTotalVolume = 1.0 / getVolumeOfBubbles();
	addVolume<<<numBlocks, numThreads>>>(r, volumeMultiplier, numBubbles, invTotalVolume);
	
	NVTX_RANGE_POP();
    }
    
    NVTX_RANGE_POP();

    return atLeastOneBubbleDeleted;
}

dim3 cubble::Simulator::getGridSize()
{
    int numBubblesPerCell = env->getNumBubblesPerCell();
#if (NUM_DIM == 3)
    int numCellsPerDim = std::ceil(std::cbrt((float)numBubbles / numBubblesPerCell));
    dim3 gridSize(numCellsPerDim, numCellsPerDim, numCellsPerDim);
#else
    int numCellsPerDim = std::ceil(std::sqrt((float)numBubbles / numBubblesPerCell));
    dim3 gridSize(numCellsPerDim, numCellsPerDim, 1);
#endif
    
    return gridSize;
}

double cubble::Simulator::getVolumeOfBubbles()
{
    NVTX_RANGE_PUSH_A(__FUNCTION__);

    const size_t numThreads = 128;
    const size_t numBlocks = (size_t)std::ceil(numBubbles / (float)numThreads);

    double *r = bubbleData.getRowPtr((size_t)BubbleProperty::R);
    double *volPtr = bubbleData.getRowPtr((size_t)BubbleProperty::VOLUME);
    calculateVolumes<<<numBlocks, numThreads>>>(r, volPtr, numBubbles, env->getPi());
    double volume = cubReduction<double, double*, double*>(&hipcub::DeviceReduce::Sum, volPtr, numBubbles);
    
    NVTX_RANGE_POP();
    
    return volume;
}

double cubble::Simulator::getAverageRadius()
{
    NVTX_RANGE_PUSH_A(__FUNCTION__);
    
    double *r = bubbleData.getRowPtr((size_t)BubbleProperty::R);
    double avgRad = cubReduction<double, double*, double*>(&hipcub::DeviceReduce::Sum, r, numBubbles);
    avgRad/= numBubbles;
    
    NVTX_RANGE_POP();
    
    return avgRad;
}

void cubble::Simulator::getBubbles(std::vector<Bubble> &bubbles) const
{
    NVTX_RANGE_PUSH_A(__FUNCTION__);
    
    bubbles.clear();
    bubbles.resize(numBubbles);

    size_t memoryStride = bubbleData.getWidth();
    double *devX = bubbleData.getRowPtr((size_t)BubbleProperty::X);
    std::vector<double> xyzr;
    xyzr.resize(memoryStride * 4);

    CUDA_CALL(hipMemcpy(xyzr.data(), devX, sizeof(double) * 4 * memoryStride, hipMemcpyDeviceToHost));
    
    for (size_t i = 0; i < numBubbles; ++i)
    {
	Bubble b;
	dvec pos(-1, -1, -1);
	pos.x = xyzr[i];
	pos.y = xyzr[i + memoryStride];
	pos.z = xyzr[i + 2 * memoryStride];
	b.setPos(pos);
	b.setRadius(xyzr[i + 3 * memoryStride]);
	bubbles[i] = b;
    }
    
    NVTX_RANGE_POP();
}


// ******************************
// Kernels
// ******************************

__global__
void cubble::calculateVolumes(double *r, double *volumes, int numBubbles, double pi)
{
    int tid = getGlobalTid();
    if (tid < numBubbles)
    {
	double radius = r[tid];
	double volume = radius * radius * pi;
#if (NUM_DIM == 3)
	volume *= radius * 1.33333333333333333333333333;
#endif
	
	volumes[tid] = volume;
    }   
}

__global__
void cubble::assignDataToBubbles(double *x,
				 double *y,
				 double *z,
				 double *xPrd,
				 double *yPrd,
				 double *zPrd,
				 double *r,
				 double *w,
				 int *aboveMinRadFlags,
				 int givenNumBubblesPerDim,
				 dvec tfr,
				 dvec lbb,
				 double avgRad,
				 double minRad,
				 int numBubbles)
{
    int tid = getGlobalTid();
    if (tid < numBubbles)
    {
	int xid = tid % givenNumBubblesPerDim;
	int yid = (tid / givenNumBubblesPerDim) % givenNumBubblesPerDim;
	
	dvec randomOffset(x[tid], y[tid], 0);
	dvec pos(0, 0, 0);
	pos.x = xid / (double)givenNumBubblesPerDim;
	pos.y = yid / (double)givenNumBubblesPerDim;
#if (NUM_DIM == 3)
	int zid = tid / (givenNumBubblesPerDim * givenNumBubblesPerDim);
	pos.z = zid / (double)givenNumBubblesPerDim;
	randomOffset.z = z[tid];
#endif

	randomOffset = dvec::normalize(randomOffset) * avgRad * w[tid];
	randomOffset = (randomOffset - lbb) / (tfr - lbb);
	pos = getWrappedPos(pos + randomOffset);

	x[tid] = pos.x;
	y[tid] = pos.y;
	z[tid] = pos.z;
	
	xPrd[tid] = pos.x;
	yPrd[tid] = pos.y;
	zPrd[tid] = pos.z;

	double radius = r[tid];
	r[tid] = radius > 0 ? radius : -radius;
	w[tid] = r[tid];
	aboveMinRadFlags[tid] = radius < minRad ? 0 : 1;
    }
}

__global__
void cubble::calculateOffsets(double *x,
			      double *y,
			      double *z,
			      int *sizes,
			      dvec domainDim,
			      int numBubbles,
			      int numCells)
{   
    int tid = getGlobalTid();
    if (tid < numBubbles)
    {
	dvec pos = dvec(0, 0, 0);
	pos.x = x[tid];
	pos.y = y[tid];
	pos.z = z[tid];
	
        const ivec indexVec = (pos * domainDim).asType<int>();
	const int index = domainDim.x * domainDim.y * indexVec.z + domainDim.x * indexVec.y + indexVec.x;
	DEVICE_ASSERT(index < numCells);
	
	atomicAdd(&sizes[index], 1);
    }
}

__global__
void cubble::bubblesToCells(double *x,
			    double *y,
			    double *z,
			    int *indices,
			    int *offsets,
			    int *sizes,
			    dvec domainDim,
			    int numBubbles)
{
    int tid = getGlobalTid();
    if (tid < numBubbles)
    {
	dvec pos = dvec(0, 0, 0);
	pos.x = x[tid];
	pos.y = y[tid];
	pos.z = z[tid];
	
        const ivec indexVec = (pos * domainDim).asType<int>();
	const int index = domainDim.x * domainDim.y * indexVec.z + domainDim.x * indexVec.y + indexVec.x;
	const int offset = offsets[index] + atomicAdd(&sizes[index], 1);
        indices[offset] = tid;
    }
}

__global__
void cubble::findBubblePairs(double *x,
			     double *y,
			     double *z,
			     double *r,
			     int *indices,
			     int *offsets,
			     int *sizes,
			     int *firstIndices,
			     int *secondIndices,
			     int *numPairs,
			     int numCells,
			     int numBubbles,
			     dvec interval,
			     int maxNumSharedVals,
			     int maxNumPairs)
{
    __shared__ int numLocalPairs[1];
    extern __shared__ int localPairs[];

    DEVICE_ASSERT(numCells > 0);
    DEVICE_ASSERT(numBubbles > 0);

    if (threadIdx.x == 0)
	numLocalPairs[0] = 0;
    
    __syncthreads();
    
#if (NUM_DIM == 3)
    const int numNeighborCells = 14;
#else
    const int numNeighborCells = 5;
#endif

    const int selfCellIndex = blockIdx.z / numNeighborCells * gridDim.y * gridDim.x
	+ blockIdx.y * gridDim.x
	+ blockIdx.x;
    const int neighborCellIndex = getNeighborCellIndex(ivec(blockIdx.x, blockIdx.y, blockIdx.z / numNeighborCells),
						       ivec(gridDim.x, gridDim.y, gridDim.z / numNeighborCells),
						       blockIdx.z % numNeighborCells);
    DEVICE_ASSERT(neighborCellIndex < numCells);
    DEVICE_ASSERT(selfCellIndex < numCells);

    const bool selfComparison = selfCellIndex == neighborCellIndex;
    const int selfSize = sizes[selfCellIndex];
    const int selfOffset = offsets[selfCellIndex];
    const int neighborSize = sizes[neighborCellIndex];
    const int neighborOffset = offsets[neighborCellIndex];
    int numComparisons = selfSize * neighborSize;

    DEVICE_ASSERT(selfOffset < numBubbles);
    DEVICE_ASSERT(neighborOffset < numBubbles);
    DEVICE_ASSERT(neighborSize < numBubbles);
    DEVICE_ASSERT(selfSize < numBubbles);

    int id = 0;
    for (int i = 0; i < (1 + numComparisons / blockDim.x); ++i)
    {
        id = i * blockDim.x + threadIdx.x;
	if (id < numComparisons)
	{
	    int idx1 = id / neighborSize;
	    int idx2 = id % neighborSize;

	    DEVICE_ASSERT(selfOffset + idx1 < numBubbles);
	    DEVICE_ASSERT(neighborOffset + idx2 < numBubbles);

	    idx1 = indices[selfOffset + idx1];
	    idx2 = indices[neighborOffset + idx2];

	    if (idx1 == idx2 || (selfComparison && idx2 < idx1))
		continue;
	    
	    DEVICE_ASSERT(idx1 < numBubbles);
	    DEVICE_ASSERT(idx2 < numBubbles);

	    double wrappedComponent = getWrappedCoordinate(x[idx1], x[idx2], interval.x);
	    double magnitude = wrappedComponent * wrappedComponent;
	    
	    wrappedComponent = getWrappedCoordinate(y[idx1], y[idx2], interval.y);
	    magnitude += wrappedComponent * wrappedComponent;
	    
	    wrappedComponent = getWrappedCoordinate(z[idx1], z[idx2], interval.z);
	    magnitude += wrappedComponent * wrappedComponent;
	    
	    wrappedComponent = r[idx1] + r[idx2];
	    wrappedComponent *= wrappedComponent;

	    if (magnitude < 1.1 * wrappedComponent)
	    {
		// Set the smaller index to idx1 and larger to idx2
		id = idx1;
		idx1 = idx1 > idx2 ? idx2 : idx1;
		idx2 = idx1 == idx2 ? id : idx2;
		    
		id = atomicAdd(numLocalPairs, 2);
		DEVICE_ASSERT(id < numComparisons * 2);
		DEVICE_ASSERT(id + 1 < maxNumSharedVals);
		localPairs[id] = idx1;
		localPairs[id + 1] = idx2;
	    }
	}
    }

    __syncthreads();

    numComparisons = numLocalPairs[0] / 2;

    __syncthreads();

    if (threadIdx.x == 0)
	numLocalPairs[0] = atomicAdd(numPairs, numComparisons);

    __syncthreads();
    
    for (int i = 0; i < (1 + numComparisons / blockDim.x); ++i)
    {
	id = i * blockDim.x + threadIdx.x;
	if (id < numComparisons)
	{
	    DEVICE_ASSERT(2 * id + 1 < maxNumSharedVals);
	    DEVICE_ASSERT(numLocalPairs[0] + id < maxNumPairs);
	    firstIndices[numLocalPairs[0] + id] = localPairs[2 * id];
	    secondIndices[numLocalPairs[0] + id] = localPairs[2 * id + 1];
	}
    }
}

__global__
void cubble::predict(double *x,
		     double *y,
		     double *z,
		     double *r,
		     
		     double *xPrd,
		     double *yPrd,
		     double *zPrd,
		     double *rPrd,
		     
		     double *dxdt,
		     double *dydt,
		     double *dzdt,
		     double *drdt,
		     
		     double *dxdtOld,
		     double *dydtOld,
		     double *dzdtOld,
		     double *drdtOld,
		     
		     dvec tfr,
		     dvec lbb,
		     double timeStep,
		     int numBubbles,
		     bool useGasExchange)
{
    const int tid = getGlobalTid();
    if (tid < numBubbles)
    {
	// Measure if it's faster to calculate these per component...
	const dvec interval = (tfr - lbb);
	dvec pos, vel, velOld;
	pos.x = x[tid];
	pos.y = y[tid];
	pos.z = z[tid];
	
	vel.x = dxdt[tid];
	vel.y = dydt[tid];
	vel.z = dzdt[tid];
	
	velOld.x = dxdtOld[tid];
	velOld.y = dydtOld[tid];
	velOld.z = dzdtOld[tid];

	pos = lbb + pos * interval;
	pos += 0.5 * timeStep * (3.0 * vel - velOld);
	pos = (pos - lbb) / interval;
	pos = getWrappedPos(pos);

	xPrd[tid] = pos.x;
	yPrd[tid] = pos.y;
	zPrd[tid] = pos.z;

	if (useGasExchange)
	    rPrd[tid] = r[tid] + 0.5 * timeStep * (3.0 * drdt[tid] - drdtOld[tid]);
    }
}

__global__
void cubble::calculateVelocityAndGasExchange(double *x,
					     double *y,
					     double *z,
					     double *r,
					     
					     double *dxdt,
					     double *dydt,
					     double *dzdt,
					     double *drdt,

					     double *energy,
					     double *freeArea,
					     
					     int *firstIndices,
					     int *secondIndices,
					     
					     int numBubbles,
					     int numPairs,
					     double fZeroPerMuZero,
					     double pi,
					     dvec interval,
					     bool calculateEnergy,
					     bool useGasExchange)
{
    // FYI: This kernel heavily reuses variables, since kernels can easily become register bound.
    // Pay attention to the last assignation of a variable.
    
    const int tid = getGlobalTid();
    if (tid < numPairs)
    {
	const int idx1 = firstIndices[tid];
	const int idx2 = secondIndices[tid];

	DEVICE_ASSERT(idx1 < numBubbles);
	DEVICE_ASSERT(idx2 < numBubbles);
	DEVICE_ASSERT(idx1 != idx2);

	double velX = getWrappedCoordinate(x[idx1], x[idx2], interval.x);
	double magnitude = velX * velX;
	
	double velY = getWrappedCoordinate(y[idx1], y[idx2], interval.y);
        magnitude += velY * velY;

	double velZ = 0;
#if (NUM_DIM == 3)
        velZ = getWrappedCoordinate(z[idx1], z[idx2], interval.z);
        magnitude += velZ * velZ;
#endif

	DEVICE_ASSERT(magnitude > 0);
	magnitude = sqrt(magnitude);

	const double radii = r[idx1] + r[idx2];
	if (magnitude <= radii)
	{
	    DEVICE_ASSERT(radii > 0);
	    const double invRadii = 1.0 / radii;

	    if (calculateEnergy)
	    {
	    	double potentialEnergy = radii - magnitude;
	    	potentialEnergy *= potentialEnergy;
	    	atomicAdd(&energy[idx1], potentialEnergy);
	    	atomicAdd(&energy[idx2], potentialEnergy);
	    }

            const double invMagnitude = 1.0 / magnitude;
            double generalVariable = fZeroPerMuZero * (radii - magnitude) * invRadii * invMagnitude;
	
            velX *= generalVariable;
	    velY *= generalVariable;
	    velZ *= generalVariable;

	    atomicAdd(&dxdt[idx1], velX);
	    atomicAdd(&dxdt[idx2], -velX);
	
	    atomicAdd(&dydt[idx1], velY);
	    atomicAdd(&dydt[idx2], -velY);
#if (NUM_DIM == 3)
	    atomicAdd(&dzdt[idx1], velZ);
	    atomicAdd(&dzdt[idx2], -velZ);
#endif

	    if (useGasExchange)
	    {
	    	velX = r[idx1];
	    	velY = r[idx2];
		if (magnitude < velX || magnitude < velY)
		{
		    velZ = velX < velY ? velX : velY;
		    velZ *= velZ;
		}
		else
		{
		    generalVariable = velY * velY;
		    velZ = 0.5 * (generalVariable - velX * velX + magnitude * magnitude) * invMagnitude;
		    velZ *= velZ;
		    velZ = generalVariable - velZ;
		    DEVICE_ASSERT(velZ > -0.0001);
		    velZ = velZ < 0 ? -velZ : velZ;
		    DEVICE_ASSERT(velZ >= 0);
		}
	    
#if (NUM_DIM == 3)
	    	velZ *= pi;
#else
	    	velZ = 2.0 * sqrt(velZ);
#endif
	    	atomicAdd(&freeArea[idx1], velZ);
	    	atomicAdd(&freeArea[idx2], velZ);
	    
	    	velZ *= 1.0 / velY - 1.0 / velX;
	    
	    	atomicAdd(&drdt[idx1], velZ);
	    	atomicAdd(&drdt[idx2], -velZ);
	    }
	}
    }
}

__global__
void cubble::calculateFreeAreaPerRadius(double *r, double *freeArea, double *output, double pi, int numBubbles)
{
    const int tid = getGlobalTid();
    if (tid < numBubbles)
    {
	double area = 2.0 * pi * r[tid];
#if (NUM_DIM == 3)
	area *= 2.0 * r[tid];
#endif
	area -= freeArea[tid];
	freeArea[tid] = area;
	output[tid] = freeArea[tid] / r[tid];
    }
}

__global__
void cubble::calculateFinalRadiusChangeRate(double *drdt,
					    double *r,
					    double *freeArea,
					    int numBubbles,
					    double invRho,
					    double invPi,
					    double kappa,
					    double kParam)
{
    const int tid = getGlobalTid();
    if (tid < numBubbles)
    {
	double invRadius = 1.0 / r[tid];
	double vr = kappa * freeArea[tid] * (invRho - invRadius);
	vr += drdt[tid];
	
        vr *= 0.5 * invPi * invRadius;
#if (NUM_DIM == 3)
	vr *= 0.5 * invRadius;
#endif
	
	drdt[tid] = kParam * vr;
    }
}

__global__
void cubble::correct(double *x,
		     double *y,
		     double *z,
		     double *r,
		     
		     double *xPrd,
		     double *yPrd,
		     double *zPrd,
		     double *rPrd,
		     
		     double *dxdt,
		     double *dydt,
		     double *dzdt,
		     double *drdt,
		     
		     double *dxdtPrd,
		     double *dydtPrd,
		     double *dzdtPrd,
		     double *drdtPrd,
		     
		     double *errors,
		     int *aboveMinRadFlags,
		     double minRad,
		     dvec tfr,
		     dvec lbb,
		     double timeStep,
		     int numBubbles,
		     bool useGasExchange)
{
    const int tid = getGlobalTid();
    if (tid < numBubbles)
    {   
	// Measure if it's faster to calculate these per component...
	const dvec interval = (tfr - lbb);
	dvec pos, posPrd, vel, velPrd;
	pos.x = x[tid];
	pos.y = y[tid];
	pos.z = z[tid];

	posPrd.x = xPrd[tid];
	posPrd.y = yPrd[tid];
	posPrd.z = zPrd[tid];
	
	vel.x = dxdt[tid];
	vel.y = dydt[tid];
	vel.z = dzdt[tid];
	
	velPrd.x = dxdtPrd[tid];
	velPrd.y = dydtPrd[tid];
	velPrd.z = dzdtPrd[tid];

	pos = lbb + pos * interval;
	pos += 0.5 * timeStep * (vel + velPrd);
	pos = (pos - lbb) / interval;
	pos = getWrappedPos(pos);

	double radError = 0;
	if (useGasExchange)
	{
	    const double radius = r[tid] + 0.5 * timeStep
		* (drdt[tid] + drdtPrd[tid]);
	    
	    radError = radius - rPrd[tid];
	    radError = radError < 0 ? -radError : radError;

	    rPrd[tid] = radius;
	    aboveMinRadFlags[tid] = radius < minRad ? 0 : 1;
	}
	else
	    aboveMinRadFlags[tid] = 1;

	double error = (pos - posPrd).getAbsolute().getMaxComponent();
	error = error > radError ? error : radError;
	errors[tid] = error;

	xPrd[tid] = pos.x;
	yPrd[tid] = pos.y;
	zPrd[tid] = pos.z;
    }
}

__global__
void cubble::addVolume(double *r, double *volumeMultiplier, int numBubbles, double invTotalVolume)
{
    const int tid = getGlobalTid();
    if (tid < numBubbles)
    {
        double multiplier = volumeMultiplier[0] * invTotalVolume;
	multiplier += 1.0;

#if (NUM_DIM == 3)
	multiplier = cbrt(multiplier);
#else
	multiplier = sqrt(multiplier);
#endif
	r[tid] = r[tid] * multiplier;
    }
}

__global__
void cubble::eulerIntegration(double *x,
			      double *y,
			      double *z,
			      double *r,
			      
			      double *dxdt,
			      double *dydt,
			      double *dzdt,
			      double *drdt,
			      
			      dvec tfr,
			      dvec lbb,
			      double timeStep,
			      int numBubbles)
{
    const int tid = getGlobalTid();
    if (tid < numBubbles)
    {
	dvec interval = tfr - lbb;
	dvec pos(0, 0, 0);
	pos.x = x[tid];
	pos.y = y[tid];
	pos.z = z[tid];

	dvec vel(0, 0, 0);
	vel.x = dxdt[tid];
	vel.y = dydt[tid];
	vel.z = dzdt[tid];

	pos = lbb + pos * interval;
	pos += timeStep * vel;
	pos = (pos - lbb) / interval;
	pos = getWrappedPos(pos);
	
	x[tid] = pos.x;
	y[tid] = pos.y;
	z[tid] = pos.z;
	r[tid] = r[tid] + timeStep * drdt[tid];
    }
}

 __global__
void cubble::calculateRedistributedGasVolume(double *volume,
	                                     double *r,
	                                     int *aboveMinRadFlags,
	                                     double *volumeMultiplier,
	                                     double pi,
                        	             int numBubbles)
{
    const int tid = getGlobalTid();
    if (tid < numBubbles)
    {
        const double radius = r[tid];
        double vol = pi * radius * radius;
#if (NUM_DIM == 3)
	vol *= 1.333333333333333333333333 * radius;
#endif
	volume[tid] = vol;

	if (aboveMinRadFlags[tid] == 0)
	    atomicAdd(volumeMultiplier, vol);
    }
}

__global__
void cubble::removeSmallBubbles(double *x,
				double *y,
				double *z,
				double *r,
			
				double *xTemp,
				double *yTemp,
				double *zTemp,
				double *rTemp,
			
				double *dxdt,
				double *dydt,
				double *dzdt,
				double *drdt,
			
				double *dxdtTemp,
				double *dydtTemp,
				double *dzdtTemp,
				double *drdtTemp,
			
				double *dxdtOld,
				double *dydtOld,
				double *dzdtOld,
				double *drdtOld,
			
				double *dxdtOldTemp,
				double *dydtOldTemp,
				double *dzdtOldTemp,
				double *drdtOldTemp,
			
				int *newIdx,
				int *flag,
				int numBubbles)
{
    const int tid = getGlobalTid();
    if (tid < numBubbles && flag[tid] == 1)
    {
	const int idx = newIdx[tid];
	x[idx] = xTemp[tid];
	y[idx] = yTemp[tid];
	z[idx] = zTemp[tid];
	r[idx] = rTemp[tid];
	
	dxdt[idx] = dxdtTemp[tid];
	dydt[idx] = dydtTemp[tid];
	dzdt[idx] = dzdtTemp[tid];
	drdt[idx] = drdtTemp[tid];
	
	dxdtOld[idx] = dxdtOldTemp[tid];
	dydtOld[idx] = dydtOldTemp[tid];
	dzdtOld[idx] = dzdtOldTemp[tid];
	drdtOld[idx] = drdtOldTemp[tid];
    }
}

// ******************************
// Device functions
// ******************************

__forceinline__ __device__
int cubble::getNeighborCellIndex(ivec cellIdx, ivec dim, int neighborNum)
{
    // Switch statements and ifs that diverge inside one warp/block are
    // detrimental for performance. However, this should never diverge,
    // as all the threads of one block should always be in the same cell
    // going for the same neighbor.
    ivec idxVec = cellIdx;
    switch(neighborNum)
    {
    case 0:
	// self
	break;
    case 1:
	idxVec += ivec(-1, 1, 0);
	break;
    case 2:
	idxVec += ivec(-1, 0, 0);
	break;
    case 3:
	idxVec += ivec(-1, -1, 0);
	break;
    case 4:
	idxVec += ivec(0, -1, 0);
	break;
#if NUM_DIM == 3
    case 5:
	idxVec += ivec(-1, 1, -1);
	break;
    case 6:
	idxVec += ivec(-1, 0, -1);
	break;
    case 7:
	idxVec += ivec(-1, -1, -1);
	break;
    case 8:
	idxVec += ivec(0, 1, -1);
	break;
    case 9:
	idxVec += ivec(0, 0, -1);
	break;
    case 10:
	idxVec += ivec(0, -1, -1);
	break;
    case 11:
	idxVec += ivec(1, 1, -1);
	break;
    case 12:
	idxVec += ivec(1, 0, -1);
	break;
    case 13:
	idxVec += ivec(1, -1, -1);
	break;
#endif
    default:
	printf("Should never end up here!");
	break;
    }

    idxVec += dim;
    idxVec %= dim;

    return idxVec.z * dim.y * dim.x + idxVec.y * dim.x + idxVec.x;
}

__forceinline__ __device__
int cubble::getGlobalTid()
{
    // Simple helper function for calculating a 1D coordinate
    // from 1, 2 or 3 dimensional coordinates.
    int threadsPerBlock = blockDim.x * blockDim.y * blockDim.z;
    int blocksBefore = blockIdx.z * (gridDim.y * gridDim.x)
	+ blockIdx.y * gridDim.x
	+ blockIdx.x;
    int threadsBefore = blockDim.y * blockDim.x * threadIdx.z + blockDim.x * threadIdx.y;
    int tid = blocksBefore * threadsPerBlock + threadsBefore + threadIdx.x;

    return tid;
}

__forceinline__ __device__
double cubble::getWrappedCoordinate(double val1, double val2, double multiplier)
{
    DEVICE_ASSERT(val1 <= 1.0 && val2 <= 1.0);
    DEVICE_ASSERT(val1 >= 0.0 && val2 >= 0.0);
    double difference = val1 - val2;
    val2 = difference < -0.5 ? val2 - 1.0 : (difference > 0.5 ? val2 + 1.0 : val2);
    val2 = val1 - val2;
    
    return val2 * multiplier;
}

__forceinline__ __device__
cubble::dvec cubble::getWrappedPos(dvec pos)
{
    // ASSUMPTION: Using normalized position
    // ASSUMPTION: Position never smaller/greater than -1/1
    pos.x = pos.x < 0 ? pos.x + 1.0 : (pos.x > 1 ? pos.x - 1.0 : pos.x);
    pos.y = pos.y < 0 ? pos.y + 1.0 : (pos.y > 1 ? pos.y - 1.0 : pos.y);
    pos.z = pos.z < 0 ? pos.z + 1.0 : (pos.z > 1 ? pos.z - 1.0 : pos.z);

    return pos;
}
