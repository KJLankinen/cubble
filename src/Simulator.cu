#include "hip/hip_runtime.h"
#include "CubWrapper.h"
#include "DataDefinitions.h"
#include "Kernels.cuh"
#include "Util.h"
#include "Vec.h"
#include "cub/hipcub/hipcub.hpp"
#include "nlohmann/json.hpp"
#include <array>
#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <fstream>
#include <iostream>
#include <nvToolsExt.h>
#include <sstream>
#include <string>
#include <vector>

namespace // anonymous
{
using namespace hipcubble;

#if (USE_PROFILING == 1)
void startProfiling(bool start) {
    if (start) {
        CUDA_CALL(hipProfilerStart());
    }
}

void stopProfiling(bool stop, bool &continueIntegration) {
    if (stop) {
        CUDA_CALL(hipDeviceSynchronize());
        CUDA_CALL(hipProfilerStop());
        continueIntegration = false;
    }
}
#endif

double calculateTotalEnergy(Params &params) {
    KERNEL_LAUNCH(resetKernel, params.defaultKernelSize, 0, 0, 0.0,
                  params.bubbles.count, params.bubbles.temp_doubles);
    KERNEL_LAUNCH(potentialEnergyKernel, params.pairKernelSize, 0, 0,
                  params.bubbles, params.pairs);
    return params.cw.reduce<double, double *, double *>(
        &hipcub::DeviceReduce::Sum, params.bubbles.temp_doubles,
        params.bubbles.count);
}

void transformPositions(Params &params, bool normalize) {
    KERNEL_LAUNCH(transformPositionsKernel, params.pairKernelSize, 0, 0,
                  normalize, params.bubbles);
}

double calculateVolumeOfBubbles(Params &params) {
    KERNEL_LAUNCH(calculateVolumes, params.pairKernelSize, 0, 0,
                  params.bubbles);
    return params.cw.reduce<double, double *, double *>(
        &hipcub::DeviceReduce::Sum, params.bubbles.temp_doubles,
        params.bubbles.count);
}

double getSimulationBoxVolume(Params &params) {
    dvec temp = params.hostConstants.interval;
    return (NUM_DIM == 3) ? temp.x * temp.y * temp.z : temp.x * temp.y;
}

void updateCellsAndNeighbors(Params &params) {
    NVTX_RANGE_PUSH_A("Neighbors");
    params.hostData.numNeighborsSearched++;

    KERNEL_LAUNCH(wrapKernel, params.pairKernelSize, 0, params.stream1,
                  params.bubbles);

    // Reset pairs arrays to zero
    uint64_t bytes = sizeof(int) * params.pairs.stride * 4;
    CUDA_CALL(hipMemsetAsync(static_cast<void *>(params.pairs.i), 0, bytes,
                              params.stream2));

    // Minimum size of cell is twice the sum of the skin and max bubble radius
    ivec cellDim = (params.hostConstants.interval /
                    (2 * (params.hostData.maxBubbleRadius +
                          params.hostConstants.skinRadius)))
                       .floor();
    cellDim.z = cellDim.z > 0 ? cellDim.z : 1;
    dim3 gridSize = dim3(cellDim.x, cellDim.y, cellDim.z);

    // Determine the maximum number of Morton numbers for the simulation box
    const int maxGridSize =
        gridSize.x > gridSize.y
            ? (gridSize.x > gridSize.z ? gridSize.x : gridSize.z)
            : (gridSize.y > gridSize.z ? gridSize.y : gridSize.z);
    int maxNumCells = 1;
    while (maxNumCells < maxGridSize)
        maxNumCells = maxNumCells << 1;

    if (NUM_DIM == 3)
        maxNumCells = maxNumCells * maxNumCells * maxNumCells;
    else
        maxNumCells = maxNumCells * maxNumCells;

#ifndef NDEBUG
    std::cout << "Max num cells: " << maxNumCells << ", grid size: ("
              << gridSize.x << ", " << gridSize.y << ", " << gridSize.z
              << "), avg num bubbles per cell: "
              << params.bubbles.count / (gridSize.x * gridSize.y * gridSize.z)
              << std::endl;
#endif
    assert(maxNumCells < params.pairs.stride);

    int *offsets = params.pairs.i;
    int *sizes = params.pairs.i + maxNumCells;
    int *cellIndices = params.pairs.i_copy;
    int *bubbleIndices = params.pairs.i_copy + 1 * params.bubbles.stride;
    int *sortedCellIndices = params.pairs.i_copy + 2 * params.bubbles.stride;
    int *sortedBubbleIndices = params.pairs.i_copy + 3 * params.bubbles.stride;

    // Assign each bubble to a particular cell, based on the bubbles
    // position. bubbleIndices and cellIndices will be filled by this
    // kernel. Wait until the event from stream1 is recorded.
    KERNEL_LAUNCH(assignBubblesToCells, params.pairKernelSize, 0, 0,
                  cellIndices, bubbleIndices, cellDim, params.bubbles);

    // Sort both the cell and bubble indices by the cellIndices array into
    // ascending order.
    params.cw.sortPairs<int, int>(
        &hipcub::DeviceRadixSort::SortPairs, const_cast<const int *>(cellIndices),
        sortedCellIndices, const_cast<const int *>(bubbleIndices),
        sortedBubbleIndices, params.bubbles.count);

    // Count the number of bubbles in each cell and store those values in sizes
    params.cw.histogram<int *, int, int, int>(
        &hipcub::DeviceHistogram::HistogramEven, cellIndices, sizes,
        maxNumCells + 1, 0, maxNumCells, params.bubbles.count);

    // Count the number of bubbles in the cells before this cell, for each cell
    params.cw.scan<int *, int *>(&hipcub::DeviceScan::ExclusiveSum, sizes, offsets,
                                 maxNumCells);

    // This kernel reorganizes the data by swapping values from one array to
    // another in a 'loopy' fashion. The pointers must be updated after the
    // kernel.
    KERNEL_LAUNCH(reorganizeByIndex, params.defaultKernelSize, 0, 0,
                  params.bubbles, const_cast<const int *>(sortedBubbleIndices));
    double *swapper = params.bubbles.xp;
    params.bubbles.xp = params.bubbles.x;
    params.bubbles.x = swapper;

    swapper = params.bubbles.yp;
    params.bubbles.yp = params.bubbles.y;
    params.bubbles.y = swapper;

    swapper = params.bubbles.zp;
    params.bubbles.zp = params.bubbles.z;
    params.bubbles.z = swapper;

    swapper = params.bubbles.rp;
    params.bubbles.rp = params.bubbles.r;
    params.bubbles.r = swapper;

    swapper = params.bubbles.dxdtp;
    params.bubbles.dxdtp = params.bubbles.dxdt;
    params.bubbles.dxdt = swapper;

    swapper = params.bubbles.dydtp;
    params.bubbles.dydtp = params.bubbles.dydt;
    params.bubbles.dydt = swapper;

    swapper = params.bubbles.dzdtp;
    params.bubbles.dzdtp = params.bubbles.dzdt;
    params.bubbles.dzdt = swapper;

    swapper = params.bubbles.drdtp;
    params.bubbles.drdtp = params.bubbles.drdt;
    params.bubbles.drdt = swapper;

    // Note that the order is reverse from the order in the kernel
    swapper = params.bubbles.error;
    params.bubbles.error = params.bubbles.path;
    params.bubbles.path = params.bubbles.drdto;
    params.bubbles.drdto = params.bubbles.dzdto;
    params.bubbles.dzdto = params.bubbles.dydto;
    params.bubbles.dydto = params.bubbles.dxdto;
    params.bubbles.dxdto = params.bubbles.flow_vx;
    params.bubbles.flow_vx = swapper;

    int *swapperI = params.bubbles.index;
    params.bubbles.index = params.bubbles.wrap_count_z;
    params.bubbles.wrap_count_z = params.bubbles.wrap_count_y;
    params.bubbles.wrap_count_y = params.bubbles.wrap_count_x;
    params.bubbles.wrap_count_x = params.bubbles.num_neighbors;
    params.bubbles.num_neighbors = swapperI;

    KernelSize kernelSizeNeighbor = KernelSize(gridSize, dim3(128, 1, 1));
    int zero = 0;
    CUDA_CALL(
        hipMemcpyToSymbol(HIP_SYMBOL(dNumPairs), static_cast<void *>(&zero), sizeof(int)));

    for (int i = 0; i < CUBBLE_NUM_NEIGHBORS + 1; ++i) {
        hipStream_t stream = (i % 2) ? params.stream2 : params.stream1;
        KERNEL_LAUNCH(neighborSearch, kernelSizeNeighbor, 0, stream, i,
                      maxNumCells, offsets, sizes, params.bubbles,
                      params.pairs);
    }

    CUDA_CALL(hipMemcpyFromSymbol(static_cast<void *>(&params.pairs.count),
                                   dNumPairs, sizeof(int)));

#ifndef NDEBUG
    std::cout << "Max num pairs: " << params.pairs.stride
              << ", actual num pairs: " << params.pairs.count << std::endl;
#endif

    params.cw.sortPairs<int, int>(&hipcub::DeviceRadixSort::SortPairs,
                                  const_cast<const int *>(params.pairs.i_copy),
                                  params.pairs.i,
                                  const_cast<const int *>(params.pairs.j_copy),
                                  params.pairs.j, params.pairs.count);
    NVTX_RANGE_POP();
}

void deleteSmallBubbles(Params &params, int numToBeDeleted) {
    NVTX_RANGE_PUSH_A("BubbleRemoval");

    KERNEL_LAUNCH(swapDataCountPairs, params.pairKernelSize, 0, 0,
                  params.bubbles, params.pairs);

    KERNEL_LAUNCH(addVolumeFixPairs, params.pairKernelSize, 0, 0,
                  params.bubbles, params.pairs);

    params.bubbles.count -= numToBeDeleted;
    params.defaultKernelSize = KernelSize(128, params.bubbles.count);
    const int numBlocks =
        std::min(1024, (int)std::ceil(params.bubbles.count / 128.0));
    params.pairKernelSize = KernelSize(dim3(numBlocks, 1, 1), dim3(128, 1, 1));

    NVTX_RANGE_POP();
}

void saveSnapshotToFile(Params &params) {
    // Should measure at some point how long it takes to save a snapshot
    // since there are many optimization possibilities here.
    calculateTotalEnergy(params);

    std::stringstream ss;
    ss << "snapshot.csv." << params.hostData.numSnapshots;
    std::ofstream file(ss.str().c_str(), std::ios::out);
    if (file.is_open()) {
        // Copy entire bubble struct to host memory
        uint64_t bytes = params.bubbles.getMemReq();
        std::vector<char> rawMem;
        rawMem.resize(bytes);
        void *memStart = static_cast<void *>(rawMem.data());
        // Async copy so host can sort pointers while copy is happening
        CUDA_CALL(hipMemcpyAsync(memStart, params.memory, bytes,
                                  hipMemcpyDeviceToHost, 0));

        // Get host pointer for each device pointer
        auto getHostPtr = [&params,
                           &memStart](auto devPtr) -> decltype(devPtr) {
            return static_cast<decltype(devPtr)>(memStart) +
                   (devPtr - static_cast<decltype(devPtr)>(params.memory));
        };
        double *x = getHostPtr(params.bubbles.x);
        double *y = getHostPtr(params.bubbles.y);
        double *z = getHostPtr(params.bubbles.z);
        double *r = getHostPtr(params.bubbles.r);
        double *vx = getHostPtr(params.bubbles.dxdt);
        double *vy = getHostPtr(params.bubbles.dydt);
        double *vz = getHostPtr(params.bubbles.dzdt);
        double *vr = getHostPtr(params.bubbles.drdt);
        double *path = getHostPtr(params.bubbles.path);
        double *error = getHostPtr(params.bubbles.error);
        double *energy = getHostPtr(params.bubbles.temp_doubles);
        int *index = getHostPtr(params.bubbles.index);

        // Starting to access the data, so need to sync to make sure all the
        // data is there
        CUDA_CALL(hipDeviceSynchronize());

        if (params.hostData.numSnapshots == 0) {
            // If this is the first snapshot, store current positions in the
            // previous
            for (uint64_t i = 0; i < (uint64_t)params.bubbles.count; ++i) {
                const int ind = index[i];
                params.previousX[ind] = x[i];
                params.previousY[ind] = y[i];
                params.previousZ[ind] = z[i];
            }
        }

        file << "x,y,z,r,vx,vy,vz,vtot,vr,path,energy,displacement,"
                "error,index\n ";
        for (uint64_t i = 0; i < (uint64_t)params.bubbles.count; ++i) {
            const int ind = index[i];
            const double xi = x[i];
            const double yi = y[i];
            const double zi = z[i];
            const double vxi = vx[i];
            const double vyi = vy[i];
            const double vzi = vz[i];
            const double px = params.previousX[ind];
            const double py = params.previousY[ind];
            const double pz = params.previousZ[ind];

            double displX = abs(xi - px);
            displX = displX > 0.5 * params.hostConstants.interval.x
                         ? displX - params.hostConstants.interval.x
                         : displX;
            double displY = abs(yi - py);
            displY = displY > 0.5 * params.hostConstants.interval.y
                         ? displY - params.hostConstants.interval.y
                         : displY;
            double displZ = abs(zi - pz);
            displZ = displZ > 0.5 * params.hostConstants.interval.z
                         ? displZ - params.hostConstants.interval.z
                         : displZ;

            file << xi;
            file << ",";
            file << yi;
            file << ",";
            file << zi;
            file << ",";
            file << r[i];
            file << ",";
            file << vxi;
            file << ",";
            file << vyi;
            file << ",";
            file << vzi;
            file << ",";
            file << sqrt(vxi * vxi + vyi * vyi + vzi * vzi);
            file << ",";
            file << vr[i];
            file << ",";
            file << path[i];
            file << ",";
            file << energy[i];
            file << ",";
            file << sqrt(displX * displX + displY * displY + displZ * displZ);
            file << ",";
            file << error[i];
            file << ",";
            file << ind;
            file << "\n";

            params.previousX[ind] = xi;
            params.previousY[ind] = yi;
            params.previousZ[ind] = zi;
        }

        ++params.hostData.numSnapshots;
    }
}

double stabilize(Params &params, int numStepsToRelax) {
    // This function integrates only the positions of the bubbles.
    // Gas exchange is not used. This is used for equilibrating the foam.
    double elapsedTime = 0.0;
    double error = 100000;
    params.hostData.energy1 = calculateTotalEnergy(params);

    for (int i = 0; i < numStepsToRelax; ++i) {
        do {
            KERNEL_LAUNCH(resetKernel, params.defaultKernelSize, 0, 0, 0.0,
                          params.bubbles.count, params.bubbles.dxdtp,
                          params.bubbles.dydtp, params.bubbles.dzdtp);

            KERNEL_LAUNCH(predictKernel, params.pairKernelSize, 0, 0,
                          params.hostData.timeStep, false, params.bubbles);

            KERNEL_LAUNCH(velocityPairKernel, params.pairKernelSize, 0, 0,
                          params.bubbles, params.pairs);

#if (PBC_X == 0 || PBC_Y == 0 || PBC_Z == 0)
            KERNEL_LAUNCH(velocityWallKernel, params.pairKernelSize, 0, 0,
                          params.bubbles);
#endif
            KERNEL_LAUNCH(correctKernel, params.pairKernelSize, 0, 0,
                          params.hostData.timeStep, false, params.bubbles);

            KERNEL_LAUNCH(endStepKernel, params.pairKernelSize, 0, 0,
                          (int)params.pairKernelSize.grid.x, params.bubbles);

            // endStepKernel reduced maximum error, copy it to host
            CUDA_CALL(hipMemcpyFromSymbol(static_cast<void *>(&error),
                                           dMaxError, sizeof(double)));

            if (error < params.hostData.errorTolerance &&
                params.hostData.timeStep < 0.1)
                params.hostData.timeStep *= 1.9;
            else if (error > params.hostData.errorTolerance)
                params.hostData.timeStep *= 0.5;

        } while (error > params.hostData.errorTolerance);

        // endStepKernel reduced maximum expansion, copy it to host
        double maxExpansion = 0.0;
        CUDA_CALL(hipMemcpyFromSymbol(static_cast<void *>(&maxExpansion),
                                       dMaxExpansion, sizeof(double)));

        // Update the current values with the calculated predictions
        double *swapper = params.bubbles.dxdto;
        params.bubbles.dxdto = params.bubbles.dxdt;
        params.bubbles.dxdt = params.bubbles.dxdtp;
        params.bubbles.dxdtp = swapper;

        swapper = params.bubbles.dydto;
        params.bubbles.dydto = params.bubbles.dydt;
        params.bubbles.dydt = params.bubbles.dydtp;
        params.bubbles.dydtp = swapper;

        swapper = params.bubbles.dzdto;
        params.bubbles.dzdto = params.bubbles.dzdt;
        params.bubbles.dzdt = params.bubbles.dzdtp;
        params.bubbles.dzdtp = swapper;

        swapper = params.bubbles.x;
        params.bubbles.x = params.bubbles.xp;
        params.bubbles.xp = swapper;

        swapper = params.bubbles.y;
        params.bubbles.y = params.bubbles.yp;
        params.bubbles.yp = swapper;

        swapper = params.bubbles.z;
        params.bubbles.z = params.bubbles.zp;
        params.bubbles.zp = swapper;

        elapsedTime += params.hostData.timeStep;

        if (maxExpansion >= 0.5 * params.hostConstants.skinRadius) {
            updateCellsAndNeighbors(params);
            // After updateCellsAndNeighbors r is correct,
            // but rp is trash. velocityPairKernel always uses
            // predicted values, so copy r to rp
            uint64_t bytes = params.bubbles.stride * sizeof(double);
            CUDA_CALL(hipMemcpyAsync(static_cast<void *>(params.bubbles.rp),
                                      static_cast<void *>(params.bubbles.r),
                                      bytes, hipMemcpyDeviceToDevice, 0));
        }
    }

    params.hostData.energy2 = calculateTotalEnergy(params);

    return elapsedTime;
}

void velocityCalculation(Params &params) {
    KERNEL_LAUNCH(velocityPairKernel, params.pairKernelSize, 0, params.stream2,
                  params.bubbles, params.pairs);

#if (USE_FLOW == 1)
    {
        KERNEL_LAUNCH(neighborVelocityKernel, params.pairKernelSize, 0,
                      params.stream2, params.bubbles, params.pairs);

        KERNEL_LAUNCH(flowVelocityKernel, params.pairKernelSize, 0,
                      params.stream2, params.bubbles);
    }
#endif

#if (PBC_X == 0 || PBC_Y == 0 || PBC_Z == 0)
    KERNEL_LAUNCH(velocityWallKernel, params.pairKernelSize, 0, params.stream2,
                  params.bubbles);
#endif
}

bool integrate(Params &params) {
    NVTX_RANGE_PUSH_A("Integration function");

    double error = 100000;
    uint32_t numLoopsDone = 0;
    double *hMaxRadius = static_cast<double *>(params.pinnedMemory);
    double *hMaxExpansion = hMaxRadius + 1;
    int *hNumToBeDeleted = reinterpret_cast<int *>(hMaxExpansion + 1);

    do {
        NVTX_RANGE_PUSH_A("Integration step");
        KERNEL_LAUNCH(resetKernel, params.defaultKernelSize, 0, params.stream2,
                      0.0, params.bubbles.count, params.bubbles.dxdtp,
                      params.bubbles.dydtp, params.bubbles.dzdtp,
                      params.bubbles.drdtp, params.bubbles.temp_doubles,
                      params.bubbles.temp_doubles2, params.bubbles.flow_vx,
                      params.bubbles.flow_vy, params.bubbles.flow_vz);

        KERNEL_LAUNCH(predictKernel, params.pairKernelSize, 0, params.stream1,
                      params.hostData.timeStep, true, params.bubbles);
        CUDA_CALL(hipEventRecord(params.event1, params.stream1));

        // Gas exchange can start immediately after predict, since they
        // are computed in the same stream
        KERNEL_LAUNCH(gasExchangeKernel, params.pairKernelSize, 0,
                      params.stream1, params.bubbles, params.pairs);
        KERNEL_LAUNCH(finalRadiusChangeRateKernel, params.pairKernelSize, 0,
                      params.stream1, params.bubbles);

        // Wait for the event recorded after predict kernel
        CUDA_CALL(hipStreamWaitEvent(params.stream2, params.event1, 0));
        velocityCalculation(params);

        KERNEL_LAUNCH(correctKernel, params.pairKernelSize, 0, 0,
                      params.hostData.timeStep, true, params.bubbles);

        CUDA_CALL(hipMemcpyFromSymbolAsync(
            static_cast<void *>(hNumToBeDeleted), dNumToBeDeleted, sizeof(int),
            0, hipMemcpyDeviceToHost, params.stream1));

        KERNEL_LAUNCH(endStepKernel, params.pairKernelSize, 0, params.stream2,
                      (int)params.pairKernelSize.grid.x, params.bubbles);

        // endStepKernel reduced maximum error, copy it to host
        CUDA_CALL(hipMemcpyFromSymbol(static_cast<void *>(&error), dMaxError,
                                       sizeof(double)));

        if (error < params.hostData.errorTolerance &&
            params.hostData.timeStep < 0.1)
            params.hostData.timeStep *= 1.9;
        else if (error > params.hostData.errorTolerance)
            params.hostData.timeStep *= 0.5;

        ++numLoopsDone;
        NVTX_RANGE_POP();
    } while (error > params.hostData.errorTolerance);

    // Increment the path of each bubble
    KERNEL_LAUNCH(incrementPath, params.pairKernelSize, 0, params.stream1,
                  params.bubbles);
    CUDA_CALL(hipEventRecord(params.event2, params.stream1));

    // endStepKernel reduced maximum radius and expansion, copy them to host
    // Should be pinned
    CUDA_CALL(hipMemcpyFromSymbolAsync(
        static_cast<void *>(hMaxExpansion), dMaxExpansion, sizeof(double), 0,
        hipMemcpyDeviceToHost, params.stream2));
    CUDA_CALL(hipMemcpyFromSymbolAsync(
        static_cast<void *>(hMaxRadius), dMaxRadius, sizeof(double), 0,
        hipMemcpyDeviceToHost, params.stream2));

    // Record event after both copies are done
    CUDA_CALL(hipEventRecord(params.event1, params.stream2));

    // Update values
    double *swapper = params.bubbles.dxdto;
    params.bubbles.dxdto = params.bubbles.dxdt;
    params.bubbles.dxdt = params.bubbles.dxdtp;
    params.bubbles.dxdtp = swapper;

    swapper = params.bubbles.dydto;
    params.bubbles.dydto = params.bubbles.dydt;
    params.bubbles.dydt = params.bubbles.dydtp;
    params.bubbles.dydtp = swapper;

    swapper = params.bubbles.dzdto;
    params.bubbles.dzdto = params.bubbles.dzdt;
    params.bubbles.dzdt = params.bubbles.dzdtp;
    params.bubbles.dzdtp = swapper;

    swapper = params.bubbles.drdto;
    params.bubbles.drdto = params.bubbles.drdt;
    params.bubbles.drdt = params.bubbles.drdtp;
    params.bubbles.drdtp = swapper;

    swapper = params.bubbles.x;
    params.bubbles.x = params.bubbles.xp;
    params.bubbles.xp = swapper;

    swapper = params.bubbles.y;
    params.bubbles.y = params.bubbles.yp;
    params.bubbles.yp = swapper;

    swapper = params.bubbles.z;
    params.bubbles.z = params.bubbles.zp;
    params.bubbles.zp = swapper;

    swapper = params.bubbles.r;
    params.bubbles.r = params.bubbles.rp;
    params.bubbles.rp = swapper;

    ++params.hostData.numIntegrationSteps;

    // As the total simulation time can reach very large numbers as the
    // simulation goes on it's better to keep track of the time as two separate
    // values. One large integer for the integer part and a double that is
    // <= 1.0 to which the potentially very small timeStep gets added. This
    // keeps the precision of the time relatively constant even when the
    // simulation has run a long time.
    params.hostData.timeFraction += params.hostData.timeStep;
    params.hostData.timeInteger += (uint64_t)params.hostData.timeFraction;
    params.hostData.timeFraction =
        params.hostData.timeFraction - (uint64_t)params.hostData.timeFraction;

    // Delete, if there are nonzero amount of bubbles with a radius
    // smaller than the minimum radius. See correctKernel for the
    // comparison & calculation.
    if (*hNumToBeDeleted > 0) {
        CUDA_CALL(hipEventSynchronize(params.event2));
        deleteSmallBubbles(params, *hNumToBeDeleted);
    }

    // If the boundary of the bubble with maximum sum of movement & expansion
    // has moved more than half of the "skin radius", reorder bubbles.
    // See correctKernel, comparePair for details.
    CUDA_CALL(hipEventSynchronize(params.event1));
    params.hostData.maxBubbleRadius = *hMaxRadius;
    if (*hMaxExpansion >= 0.5 * params.hostConstants.skinRadius) {
        updateCellsAndNeighbors(params);
    }

    bool continueSimulation =
        params.bubbles.count > params.hostData.minNumBubbles;
    continueSimulation &=
        (NUM_DIM == 3)
            ? params.hostData.maxBubbleRadius <
                  0.5 * params.hostConstants.interval.getMinComponent()
            : true;

    NVTX_RANGE_POP();
    return continueSimulation;
}

void deinit(Params &params) {
    CUDA_CALL(hipDeviceSynchronize());

    CUDA_CALL(hipFree(static_cast<void *>(params.deviceConstants)));
    CUDA_CALL(hipFree(params.memory));
    CUDA_CALL(hipHostFree(static_cast<void *>(params.pinnedMemory)));

    CUDA_CALL(hipEventDestroy(params.event1));
    CUDA_CALL(hipEventDestroy(params.event2));

    CUDA_CALL(hipStreamDestroy(params.stream2));
    CUDA_CALL(hipStreamDestroy(params.stream1));
}

void commonSetup(Params &params) {
    params.defaultKernelSize = KernelSize(128, params.bubbles.count);
    CUDA_ASSERT(hipStreamCreate(&params.stream2));
    CUDA_ASSERT(hipStreamCreate(&params.stream1));
    CUDA_CALL(hipEventCreate(&params.event1));
    CUDA_CALL(hipEventCreate(&params.event2));
    printRelevantInfoOfCurrentDevice();

    // Set device globals to zero
    double zero = 0.0;
    void *vz = reinterpret_cast<void *>(&zero);
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dTotalArea), vz, sizeof(double)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dTotalOverlapArea), vz, sizeof(double)));
    CUDA_CALL(
        hipMemcpyToSymbol(HIP_SYMBOL(dTotalOverlapAreaPerRadius), vz, sizeof(double)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dTotalAreaPerRadius), vz, sizeof(double)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dTotalVolumeNew), vz, sizeof(double)));

    bool falseB = false;
    vz = reinterpret_cast<void *>(&falseB);
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dErrorEncountered), vz, sizeof(bool)));

    int zeroI = 0;
    vz = reinterpret_cast<void *>(&zeroI);
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dNumPairs), vz, sizeof(int)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dNumPairsNew), vz, sizeof(int)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dNumToBeDeleted), vz, sizeof(int)));

    std::cout << "Reserving device memory to hold data." << std::endl;
    CUDA_CALL(
        hipHostMalloc(&params.pinnedMemory, sizeof(int) + 2 * sizeof(double)));

    // It seems to hold that in 3 dimensions the total number of
    // bubble pairs is 10x and in two dimensions 4x number of bubbles.
    // Note that these numbers depend on the "skin radius", i.e.
    // from how far are the neighbors looked for.
    const uint32_t avgNumNeighbors = (NUM_DIM == 3) ? 10 : 4;

    // Calculate the length of 'rows'. Will be divisible by 32, as that's the
    // warp size.
    params.bubbles.stride =
        params.bubbles.count +
        !!(params.bubbles.count % 32) * (32 - params.bubbles.count % 32);
    params.pairs.stride = avgNumNeighbors * params.bubbles.stride;

    uint64_t bytes = params.bubbles.getMemReq();
    bytes += params.pairs.getMemReq();

    CUDA_ASSERT(hipMalloc(reinterpret_cast<void **>(&params.memory), bytes));

    // Each named pointer is setup by these functions to point to
    // a different stride inside the continuous memory blob
    void *pairStart = params.bubbles.setupPointers(params.memory);
    pairStart = params.pairs.setupPointers(pairStart);

    params.previousX.resize(params.bubbles.stride);
    params.previousY.resize(params.bubbles.stride);
    params.previousZ.resize(params.bubbles.stride);

    const uint64_t megs = bytes / (1024 * 1024);
    const uint64_t kilos = (bytes - megs * 1024 * 1024) / 1024;
    bytes = (bytes - megs * 1024 * 1024 - kilos * 1024);
    std::cout << "Total device memory allocated: " << megs << "m " << kilos
              << "k " << bytes << "b" << std::endl;
}

void generateStartingData(Params &params, ivec bubblesPerDim, double stdDevRad,
                          int rngSeed) {
    std::cout << "Starting to generate data for bubbles." << std::endl;
    const double avgRad = params.hostData.avgRad;

    hiprandGenerator_t generator;
    CURAND_CALL(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_MTGP32));
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(generator, rngSeed));
    if (NUM_DIM == 3)
        CURAND_CALL(hiprandGenerateUniformDouble(generator, params.bubbles.z,
                                                params.bubbles.count));
    CURAND_CALL(hiprandGenerateUniformDouble(generator, params.bubbles.x,
                                            params.bubbles.count));
    CURAND_CALL(hiprandGenerateUniformDouble(generator, params.bubbles.y,
                                            params.bubbles.count));
    CURAND_CALL(hiprandGenerateUniformDouble(generator, params.bubbles.rp,
                                            params.bubbles.count));
    CURAND_CALL(hiprandGenerateNormalDouble(
        generator, params.bubbles.r, params.bubbles.count, avgRad, stdDevRad));
    CURAND_CALL(hiprandDestroyGenerator(generator));

    KERNEL_LAUNCH(assignDataToBubbles, params.pairKernelSize, 0, 0,
                  bubblesPerDim, avgRad, params.bubbles);

    params.hostConstants.averageSurfaceAreaIn =
        params.cw.reduce<double, double *, double *>(&hipcub::DeviceReduce::Sum,
                                                     params.bubbles.rp,
                                                     params.bubbles.count, 0);

    params.hostData.maxBubbleRadius =
        params.cw.reduce<double, double *, double *>(
            &hipcub::DeviceReduce::Max, params.bubbles.r, params.bubbles.count, 0);

    std::cout << "Updating neighbor lists." << std::endl;
    updateCellsAndNeighbors(params);

    // Calculate some initial values which are needed
    // for the two-step Adams-Bashforth-Moulton predictor-corrector method
    KERNEL_LAUNCH(
        resetKernel, params.defaultKernelSize, 0, 0, 0.0, params.bubbles.count,
        params.bubbles.dxdto, params.bubbles.dydto, params.bubbles.dzdto,
        params.bubbles.drdto, params.bubbles.dxdtp, params.bubbles.dydtp,
        params.bubbles.dzdtp, params.bubbles.drdtp, params.bubbles.path);

    std::cout << "Calculating some initial values as a part of setup."
              << std::endl;

    // After updateCellsAndNeighbors x, y, z, r are correct,
    // but all predicted are trash. velocityPairKernel always uses
    // predicted values, so copy currents to predicteds
    uint64_t bytes = params.bubbles.stride * sizeof(double);
    CUDA_CALL(hipMemcpyAsync(static_cast<void *>(params.bubbles.xp),
                              static_cast<void *>(params.bubbles.x), bytes,
                              hipMemcpyDeviceToDevice, 0));
    CUDA_CALL(hipMemcpyAsync(static_cast<void *>(params.bubbles.yp),
                              static_cast<void *>(params.bubbles.y), bytes,
                              hipMemcpyDeviceToDevice, 0));
    CUDA_CALL(hipMemcpyAsync(static_cast<void *>(params.bubbles.zp),
                              static_cast<void *>(params.bubbles.z), bytes,
                              hipMemcpyDeviceToDevice, 0));
    CUDA_CALL(hipMemcpyAsync(static_cast<void *>(params.bubbles.rp),
                              static_cast<void *>(params.bubbles.r), bytes,
                              hipMemcpyDeviceToDevice, 0));

    KERNEL_LAUNCH(velocityPairKernel, params.pairKernelSize, 0, 0,
                  params.bubbles, params.pairs);

    KERNEL_LAUNCH(eulerKernel, params.pairKernelSize, 0, 0,
                  params.hostData.timeStep, params.bubbles);

    // velocityPairKernel calculates to predicteds by accumulating values
    // using atomicAdd. They would have to be reset to zero after every
    // integration, but olds were set to zero above, so we can just swap.
    double *swapper = params.bubbles.dxdto;
    params.bubbles.dxdto = params.bubbles.dxdtp;
    params.bubbles.dxdtp = swapper;

    swapper = params.bubbles.dydto;
    params.bubbles.dydto = params.bubbles.dydtp;
    params.bubbles.dydtp = swapper;

    swapper = params.bubbles.dzdto;
    params.bubbles.dzdto = params.bubbles.dzdtp;
    params.bubbles.dzdtp = swapper;

    KERNEL_LAUNCH(velocityPairKernel, params.pairKernelSize, 0, 0,
                  params.bubbles, params.pairs);

    // The whole point of this part was to get integrated values into
    // dxdto & y & z, so swap again so that predicteds are in olds.
    swapper = params.bubbles.dxdto;
    params.bubbles.dxdto = params.bubbles.dxdtp;
    params.bubbles.dxdtp = swapper;

    swapper = params.bubbles.dydto;
    params.bubbles.dydto = params.bubbles.dydtp;
    params.bubbles.dydtp = swapper;

    swapper = params.bubbles.dzdto;
    params.bubbles.dzdto = params.bubbles.dzdtp;
    params.bubbles.dzdtp = swapper;
}

void initializeFromJson(const char *inputFileName, Params &params) {
    std::cout << "\n=====\nSetup\n====="
              << "Reading inputs from file \"" << inputFileName << "\""
              << std::endl;

    nlohmann::json inputJson;
    std::fstream file(inputFileName, std::ios::in);
    if (file.is_open()) {
        file >> inputJson;

        const double mu = inputJson["muZero"];
        assert(mu > 0);
        assert(inputJson["wallDragStrength"] >= 0.0 &&
               inputJson["wallDragStrength"] <= 1.0);

        params.hostData.avgRad = inputJson["avgRad"];
        params.hostConstants.minRad = 0.1 * params.hostData.avgRad;
        params.hostConstants.fZeroPerMuZero =
            (float)inputJson["sigmaZero"] * params.hostData.avgRad / mu;
        params.hostConstants.flowLbb = inputJson["flowLbb"];
        params.hostConstants.flowTfr = inputJson["flowTfr"];
        params.hostConstants.flowVel = inputJson["flowVel"];
        params.hostConstants.flowVel *= params.hostConstants.fZeroPerMuZero;
        params.hostConstants.kParameter = inputJson["kParameter"];
        params.hostConstants.kappa = inputJson["kappa"];
        params.hostConstants.skinRadius =
            (float)inputJson["skinRadius"] * params.hostData.avgRad;
        params.hostData.timeScalingFactor =
            params.hostConstants.kParameter /
            (params.hostData.avgRad * params.hostData.avgRad);
        params.hostData.errorTolerance = inputJson["errorTolerance"];
        params.hostConstants.wallDragStrength = inputJson["wallDragStrength"];
        params.hostData.snapshotFrequency = inputJson["snapshotFrequency"];
        params.hostData.minNumBubbles = inputJson["minNumBubbles"];
    } else
        throw std::runtime_error("Couldn't open input file!");

    // First calculate the size of the box and the starting number of bubbles
    dvec relDim = inputJson["boxRelDim"];
    assert(relDim.x > 0);
    assert(relDim.y > 0);
    assert(relDim.z > 0);

    relDim = relDim / relDim.x;
    const float d = 2 * params.hostData.avgRad;
    float x = (float)inputJson["numBubblesIn"] * d * d / relDim.y;
    ivec bubblesPerDim = ivec(0, 0, 0);

    if (NUM_DIM == 3) {
        x = x * d / relDim.z;
        x = std::cbrt(x);
        relDim = relDim * x;
        bubblesPerDim = ivec(std::ceil(relDim.x / d), std::ceil(relDim.y / d),
                             std::ceil(relDim.z / d));
        params.bubbles.count =
            bubblesPerDim.x * bubblesPerDim.y * bubblesPerDim.z;
    } else {
        x = std::sqrt(x);
        relDim = relDim * x;
        bubblesPerDim =
            ivec(std::ceil(relDim.x / d), std::ceil(relDim.y / d), 0);
        params.bubbles.count = bubblesPerDim.x * bubblesPerDim.y;
    }

    params.hostConstants.tfr =
        d * bubblesPerDim.asType<double>() + params.hostConstants.lbb;
    params.hostConstants.interval =
        params.hostConstants.tfr - params.hostConstants.lbb;
    params.hostData.timeStep = inputJson["timeStepIn"];

    // Allocate and copy constants to GPU
    CUDA_ASSERT(hipMalloc(reinterpret_cast<void **>(&params.deviceConstants),
                           sizeof(Constants)));
    CUDA_CALL(hipMemcpy(static_cast<void *>(params.deviceConstants),
                         static_cast<void *>(&params.hostConstants),
                         sizeof(Constants), hipMemcpyHostToDevice));
    // Copy to global pointer
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dConstants),
                                 static_cast<void *>(&params.deviceConstants),
                                 sizeof(Constants *)));

    // Reserve memory etc.
    commonSetup(params);
    generateStartingData(params, bubblesPerDim, inputJson["stdDevRad"],
                         inputJson["rngSeed"]);

    std::cout << "Letting bubbles settle after they've been created and before "
                 "scaling or stabilization."
              << std::endl;

    for (uint32_t i = 0; i < 3; ++i)
        stabilize(params, inputJson["numStepsToRelax"]);

    const double bubbleVolume = calculateVolumeOfBubbles(params);
    std::cout << "Volume ratios: current: "
              << bubbleVolume / getSimulationBoxVolume(params)
              << ", target: " << inputJson["phiTarget"]
              << "\nScaling the simulation box." << std::endl;

    transformPositions(params, true);

    relDim = inputJson["boxRelDim"];
    double t =
        bubbleVolume / ((float)inputJson["phiTarget"] * relDim.x * relDim.y);
    if (NUM_DIM == 3) {
        t /= relDim.z;
        t = std::cbrt(t);
    } else {
        t = std::sqrt(t);
        relDim.z = 0.0;
    }

    params.hostConstants.tfr = dvec(t, t, t) * relDim;
    params.hostConstants.interval =
        params.hostConstants.tfr - params.hostConstants.lbb;
    params.hostConstants.flowTfr =
        params.hostConstants.interval * params.hostConstants.flowTfr +
        params.hostConstants.lbb;
    params.hostConstants.flowLbb =
        params.hostConstants.interval * params.hostConstants.flowLbb +
        params.hostConstants.lbb;

    double mult = (double)inputJson["phiTarget"] *
                  getSimulationBoxVolume(params) / CUBBLE_PI;
#if (NUM_DIM == 3)
    mult = std::cbrt(0.75 * mult);
#else
    mult = std::sqrt(mult);
#endif
    params.hostConstants.bubbleVolumeMultiplier = mult;

    // Copy the updated constants to GPU
    CUDA_CALL(hipMemcpy(static_cast<void *>(params.deviceConstants),
                         static_cast<void *>(&params.hostConstants),
                         sizeof(Constants), hipMemcpyHostToDevice));

    transformPositions(params, false);
    updateCellsAndNeighbors(params);
    // After updateCellsAndNeighbors r is correct,
    // but rp is trash. velocityPairKernel always uses
    // predicted values, so copy r to rp
    uint64_t bytes = params.bubbles.stride * sizeof(double);
    CUDA_CALL(hipMemcpyAsync(static_cast<void *>(params.bubbles.rp),
                              static_cast<void *>(params.bubbles.r), bytes,
                              hipMemcpyDeviceToDevice, 0));

    for (uint32_t i = 0; i < 3; ++i)
        stabilize(params, inputJson["numStepsToRelax"]);

    std::cout << "Volume ratios: current: "
              << bubbleVolume / getSimulationBoxVolume(params)
              << ", target: " << inputJson["phiTarget"]
              << "\n\n=============\nStabilization\n=============" << std::endl;

    int numSteps = 0;
    const int failsafe = 500;

    std::cout << std::setw(10) << std::left << "#steps" << std::setw(12)
              << std::left << "dE" << std::setw(15) << std::left << "e1"
              << std::setw(15) << std::left << "e2" << std::setw(5) << std::left
              << "#searches" << std::endl;

    while (true) {
        double time = stabilize(params, inputJson["numStepsToRelax"]);
        double deltaEnergy =
            std::abs(1.0 - params.hostData.energy1 / params.hostData.energy2) /
            time;

        if (deltaEnergy < inputJson["maxDeltaEnergy"]) {
            std::cout << "Final delta energy " << deltaEnergy << " after "
                      << (numSteps + 1) * (int)inputJson["numStepsToRelax"]
                      << " steps."
                      << "\nEnergy before: " << params.hostData.energy1
                      << ", energy after: " << params.hostData.energy2
                      << ", time: " << time * params.hostData.timeScalingFactor
                      << std::endl;
            break;
        } else if (numSteps > failsafe) {
            std::cout << "Over " << failsafe * (int)inputJson["numStepsToRelax"]
                      << " steps taken and required delta energy not reached."
                      << " Check parameters." << std::endl;
            break;
        } else {
            std::cout << std::setw(10) << std::left
                      << (numSteps + 1) * (int)inputJson["numStepsToRelax"]
                      << std::setw(12) << std::left << std::setprecision(5)
                      << std::scientific << deltaEnergy << std::setw(15)
                      << std::left << std::setprecision(5) << std::fixed
                      << params.hostData.energy1 << std::setw(15) << std::left
                      << std::setprecision(5) << std::fixed
                      << params.hostData.energy2 << std::setw(5) << std::left
                      << params.hostData.numNeighborsSearched << std::endl;
            params.hostData.numNeighborsSearched = 0;
        }

        ++numSteps;
    }

    // TODO: Set starting positions
    // Avoiding batched memset, because the pointers might not be in order

    // Reset wrap counts to 0
    // Again avoiding batched memset, because the pointers might not be in order
    bytes = sizeof(int) * params.bubbles.stride;
    CUDA_CALL(hipMemset(params.bubbles.wrap_count_x, 0, bytes));
    CUDA_CALL(hipMemset(params.bubbles.wrap_count_y, 0, bytes));
    CUDA_CALL(hipMemset(params.bubbles.wrap_count_z, 0, bytes));

    // Reset errors since integration starts after this
    KERNEL_LAUNCH(resetKernel, params.defaultKernelSize, 0, 0, 0.0,
                  params.bubbles.count, params.bubbles.error);

    params.hostData.energy1 = calculateTotalEnergy(params);
    params.hostData.timeInteger = 0;
    params.hostData.timeFraction = 0.0;
    params.hostData.timesPrinted = 1;
    params.hostData.numIntegrationSteps = 0;
}

} // namespace

namespace cubble {
void run(std::string &&inputFileName) {
    Params params;
    initializeFromJson(inputFileName.c_str(), params);
    if (params.hostData.snapshotFrequency > 0.0)
        saveSnapshotToFile(params);

    std::cout << "\n==========\nIntegration\n==========" << std::endl;

    std::cout << std::setw(10) << std::left << "T" << std::setw(10) << std::left
              << "phi" << std::setw(10) << std::left << "R" << std::setw(10)
              << std::left << "#b" << std::setw(10) << std::left << "#pairs"
              << std::setw(10) << std::left << "#steps" << std::setw(10)
              << std::left << "#searches" << std::setw(10) << std::left
              << "min ts" << std::setw(10) << std::left << "max ts"
              << std::setw(10) << std::left << "avg ts" << std::endl;

    bool continueIntegration = true;
    double minTimestep = 9999999.9;
    double maxTimestep = -1.0;
    double avgTimestep = 0.0;
    bool resetErrors = false;

    // This is the simulation loop, which runs until (at least) one
    // end condition is met
    while (continueIntegration) {
        // Perform one integration step
        continueIntegration = integrate(params);

        // When profiling, we don't want to run the entire simulation until the
        // end, but rather just enough simulation steps to get a representative
        // view of the entire simulation
        CUDA_PROFILER_START(params.hostData.numIntegrationSteps == 2000);
        CUDA_PROFILER_STOP(params.hostData.numIntegrationSteps == 12000,
                           continueIntegration);

        // Track timestep
        minTimestep = params.hostData.timeStep < minTimestep
                          ? params.hostData.timeStep
                          : minTimestep;
        maxTimestep = params.hostData.timeStep > maxTimestep
                          ? params.hostData.timeStep
                          : maxTimestep;
        avgTimestep += params.hostData.timeStep;

        // Here we compare potentially very large integers (> 10e6) to each
        // other and small doubles (<= 1.0) to each other to preserve precision.
        const double nextPrintTime =
            params.hostData.timesPrinted / params.hostData.timeScalingFactor;
        const uint64_t nextPrintTimeInteger = (uint64_t)nextPrintTime;
        const double nextPrintTimeFraction =
            nextPrintTime - nextPrintTimeInteger;

        // Print stuff to stdout at the earliest possible moment
        // when simulation time is larger than scaled time
        if (params.hostData.timeInteger >= nextPrintTimeInteger &&
            params.hostData.timeFraction >= nextPrintTimeFraction) {
            // Define lambda for calculating averages of some values
            auto getAvg = [&params](double *p, Bubbles &bubbles) -> double {
                return params.cw.reduce<double, double *, double *>(
                           &hipcub::DeviceReduce::Sum, p, bubbles.count) /
                       bubbles.count;
            };

            params.hostData.energy2 = calculateTotalEnergy(params);
            const double dE =
                (params.hostData.energy2 - params.hostData.energy1) /
                params.hostData.energy2;
            const double relRad = getAvg(params.bubbles.r, params.bubbles) /
                                  params.hostData.avgRad;

            // Add values to data stream
            std::ofstream resultFile("results.dat", std::ios_base::app);
            if (resultFile.is_open()) {
                const double vx = getAvg(params.bubbles.dxdt, params.bubbles);
                const double vy = getAvg(params.bubbles.dydt, params.bubbles);
                const double vz = getAvg(params.bubbles.dzdt, params.bubbles);
                const double vr = getAvg(params.bubbles.drdt, params.bubbles);

                resultFile << params.hostData.timesPrinted << " " << relRad
                           << " " << params.bubbles.count << " "
                           << getAvg(params.bubbles.path, params.bubbles) << " "
                           << params.hostData.energy2 << " " << dE << " " << vx
                           << " " << vy << " " << vz << " "
                           << sqrt(vx * vx + vy * vy + vz * vz) << " " << vr
                           << "\n";
            } else {
                std::cout << "Couldn't open file stream to append results to!"
                          << std::endl;
            }

            const double phi = calculateVolumeOfBubbles(params) /
                               getSimulationBoxVolume(params);

            // Print some values
            std::cout << std::setw(10) << std::left
                      << params.hostData.timesPrinted << std::setw(10)
                      << std::left << std::setprecision(6) << std::fixed << phi
                      << std::setw(10) << std::left << std::setprecision(6)
                      << std::fixed << relRad << std::setw(10) << std::left
                      << params.bubbles.count << std::setw(10) << std::left
                      << params.pairs.count << std::setw(10) << std::left
                      << params.hostData.numStepsInTimeStep << std::setw(10)
                      << std::left << params.hostData.numNeighborsSearched
                      << std::setw(10) << std::left << minTimestep
                      << std::setw(10) << std::left << maxTimestep
                      << std::setw(10) << std::left
                      << avgTimestep / params.hostData.numStepsInTimeStep
                      << std::endl;

            ++params.hostData.timesPrinted;
            params.hostData.numStepsInTimeStep = 0;
            params.hostData.energy1 = params.hostData.energy2;
            params.hostData.numNeighborsSearched = 0;
            minTimestep = 9999999.9;
            maxTimestep = -1.0;
            avgTimestep = 0.0;
            resetErrors = true;
        }

        // Save snapshot
        if (params.hostData.snapshotFrequency > 0.0) {
            const double nextSnapshotTime = params.hostData.numSnapshots /
                                            params.hostData.snapshotFrequency /
                                            params.hostData.timeScalingFactor;
            const uint64_t nextSnapshotTimeInteger = (uint64_t)nextSnapshotTime;
            const double nextSnapshotTimeFraction =
                nextSnapshotTime - nextSnapshotTimeInteger;

            if (params.hostData.timeInteger >= nextSnapshotTimeInteger &&
                params.hostData.timeFraction >= nextSnapshotTimeFraction)
                saveSnapshotToFile(params);
        }

        if (resetErrors) {
            KERNEL_LAUNCH(resetKernel, params.defaultKernelSize, 0, 0, 0.0,
                          params.bubbles.count, params.bubbles.error);
            resetErrors = false;
        }

        ++params.hostData.numStepsInTimeStep;
    }

    if (params.hostData.snapshotFrequency > 0.0)
        saveSnapshotToFile(params);

    deinit(params);
}
} // namespace cubble
