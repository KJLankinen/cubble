#include "hip/hip_runtime.h"
#include "CubWrapper.h"
#include "Kernels.cuh"
#include "Macros.h"
#include "Vec.h"
#include "cub/hipcub/hipcub.hpp"
#include "nlohmann/json.hpp"
#include <array>
#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <fstream>
#include <iostream>
#include <nvToolsExt.h>
#include <sstream>
#include <string>
#include <vector>

namespace cubble
{
// Device double pointers
enum class DDP
{
  X,
  Y,
  Z,
  R,

  DXDT,
  DYDT,
  DZDT,
  DRDT,

  DXDTO,
  DYDTO,
  DZDTO,
  DRDTO,

  X0,
  Y0,
  Z0,

  PATH,
  DISTANCE,

  XP,
  YP,
  ZP,
  RP,

  DXDTP,
  DYDTP,
  DZDTP,
  DRDTP,

  ERROR,

  TEMP1,
  TEMP2,
  TEMP3,
  TEMP4,
  TEMP5,
  TEMP6,
  TEMP7,
  TEMP8,

  NUM_VALUES
};

// Device int pointers
enum class DIP
{
  FLAGS,

  WRAP_COUNT_X,
  WRAP_COUNT_Y,
  WRAP_COUNT_Z,

  WRAP_COUNT_XP,
  WRAP_COUNT_YP,
  WRAP_COUNT_ZP,

  PAIR1,
  PAIR2,

  TEMP1,
  TEMP2,

  NUM_VALUES
};

struct SimulationState
{
  double simulationTime        = 0.0;
  double energy1               = 0.0;
  double energy2               = 0.0;
  double maxBubbleRadius       = 0.0;
  double timeStep              = 0.0;
  uint64_t numIntegrationSteps = 0;
  uint64_t numStepsInTimeStep  = 0;
  uint32_t numSnapshots        = 0;
  uint32_t timesPrinted        = 0;
  int numBubbles               = 0;
  int maxNumCells              = 0;
  int numPairs                 = 0;
  dvec lbb                     = dvec(0.0, 0.0, 0.0);
  dvec tfr                     = dvec(0.0, 0.0, 0.0);
  dvec interval                = dvec(0.0, 0.0, 0.0);

  // Host pointers to device global variables
  int *mbpc      = nullptr;
  int *np        = nullptr;
  double *dtfapr = nullptr;
  double *dtfa   = nullptr;
  double *dvm    = nullptr;
  double *dtv    = nullptr;
  double *dir    = nullptr;
  double *dta    = nullptr;
  double *dasai  = nullptr;

  // Device data
  double *pinnedDoubles = nullptr;
  double *deviceDoubles = nullptr;
  int *deviceInts       = nullptr;
  int *pinnedInts       = nullptr;
  uint32_t dataStride   = 0;
  uint32_t pairStride   = 0;
  uint64_t memReqD      = 0;
  uint64_t memReqI      = 0;

  std::array<double *, (uint64_t)DDP::NUM_VALUES> ddps;
  std::array<int *, (uint64_t)DIP::NUM_VALUES> dips;
};

struct SimulationInputs
{
  int numBubblesPerCell = 0;
  int rngSeed           = 0;
  int numStepsToRelax   = 0;
  int numBubblesIn      = 0;
  int minNumBubbles     = 0;

  double avgRad            = 0.0;
  double stdDevRad         = 0.0;
  double minRad            = 0.0;
  double phiTarget         = 0.0;
  double muZero            = 0.0;
  double sigmaZero         = 0.0;
  double fZeroPerMuZero    = 0.0;
  double errorTolerance    = 0.0;
  double maxDeltaEnergy    = 0.0;
  double kParameter        = 0.0;
  double kappa             = 0.0;
  double timeScalingFactor = 0.0;
  double timeStepIn        = 0.0;

  std::string snapshotFilename = "";
  std::string dataFilename     = "";

  dvec boxRelDim = dvec(0.0, 0.0, 0.0);
  dvec flowLbb   = dvec(0.0, 0.0, 0.0);
  dvec flowTfr   = dvec(0.0, 0.0, 0.0);
  dvec flowVel   = dvec(0.0, 0.0, 0.0);
};

struct Params
{
  SimulationState state;
  SimulationInputs inputs;
  CubWrapper cw;

  hipStream_t velocityStream;
  hipStream_t gasStream;

  KernelSize pairKernelSize = KernelSize(dim3(256, 1, 1), dim3(128, 1, 1));
  KernelSize defaultKernelSize;
};

} // namespace cubble

namespace // anonymous
{
using namespace hipcubble;

void doBoundaryWrap(KernelSize ks, int sm, hipStream_t stream, bool wrapX, bool wrapY, bool wrapZ, int numValues,
                    double *x, double *y, double *z, dvec lbb, dvec tfr, int *mulX, int *mulY, int *mulZ, int *mulOldX,
                    int *mulOldY, int *mulOldZ)
{
  if (wrapX && wrapY && wrapZ)
    KERNEL_LAUNCH(boundaryWrapKernel, ks, sm, stream, numValues, x, lbb.x, tfr.x, mulX, mulOldX, y, lbb.y, tfr.y, mulY,
                  mulOldY, z, lbb.z, tfr.z, mulZ, mulOldZ);
  else if (wrapX && wrapY)
    KERNEL_LAUNCH(boundaryWrapKernel, ks, sm, stream, numValues, x, lbb.x, tfr.x, mulX, mulOldX, y, lbb.y, tfr.y, mulY,
                  mulOldY);
  else if (wrapX && wrapZ)
    KERNEL_LAUNCH(boundaryWrapKernel, ks, sm, stream, numValues, x, lbb.x, tfr.x, mulX, mulOldX, z, lbb.z, tfr.z, mulZ,
                  mulOldZ);
  else if (wrapY && wrapZ)
    KERNEL_LAUNCH(boundaryWrapKernel, ks, sm, stream, numValues, y, lbb.y, tfr.y, mulY, mulOldY, z, lbb.z, tfr.z, mulZ,
                  mulOldZ);
  else if (wrapX)
    KERNEL_LAUNCH(boundaryWrapKernel, ks, sm, stream, numValues, x, lbb.x, tfr.x, mulX, mulOldX);
  else if (wrapY)
    KERNEL_LAUNCH(boundaryWrapKernel, ks, sm, stream, numValues, y, lbb.y, tfr.y, mulY, mulOldY);
  else if (wrapZ)
    KERNEL_LAUNCH(boundaryWrapKernel, ks, sm, stream, numValues, z, lbb.z, tfr.z, mulZ, mulOldZ);
}

void doWallVelocity(KernelSize ks, int sm, hipStream_t stream, bool doX, bool doY, bool doZ, int numValues, int *first,
                    int *second, double *r, double *x, double *y, double *z, double *dxdt, double *dydt, double *dzdt,
                    dvec lbb, dvec tfr, Params &params)
{
  dvec interval = tfr - lbb;
  if (doX && doY && doZ)
  {
    KERNEL_LAUNCH(velocityWallKernel, ks, sm, stream, numValues, params.inputs.fZeroPerMuZero, first, second, r,
                  interval.x, lbb.x, !doX, x, dxdt, interval.y, lbb.y, !doY, y, dydt, interval.z, lbb.z, !doZ, z, dzdt);
  }
  else if (doX && doY)
  {
    KERNEL_LAUNCH(velocityWallKernel, ks, sm, stream, numValues, params.inputs.fZeroPerMuZero, first, second, r,
                  interval.x, lbb.x, !doX, x, dxdt, interval.y, lbb.y, !doY, y, dydt);
  }
  else if (doX && doZ)
  {
    KERNEL_LAUNCH(velocityWallKernel, ks, sm, stream, numValues, params.inputs.fZeroPerMuZero, first, second, r,
                  interval.x, lbb.x, !doX, x, dxdt, interval.z, lbb.z, !doZ, z, dzdt);
  }
  else if (doY && doZ)
  {
    KERNEL_LAUNCH(velocityWallKernel, ks, sm, stream, numValues, params.inputs.fZeroPerMuZero, first, second, r,
                  interval.y, lbb.y, !doY, y, dydt, interval.z, lbb.z, !doZ, z, dzdt);
  }
  else if (doX)
  {
    KERNEL_LAUNCH(velocityWallKernel, ks, sm, stream, numValues, params.inputs.fZeroPerMuZero, first, second, r,
                  interval.x, lbb.x, !doX, x, dxdt);
  }
  else if (doY)
  {
    KERNEL_LAUNCH(velocityWallKernel, ks, sm, stream, numValues, params.inputs.fZeroPerMuZero, first, second, r,
                  interval.y, lbb.y, !doY, y, dydt);
  }
  else if (doZ)
  {
    KERNEL_LAUNCH(velocityWallKernel, ks, sm, stream, numValues, params.inputs.fZeroPerMuZero, first, second, r,
                  interval.z, lbb.z, !doZ, z, dzdt);
  }
}

void startProfiling(bool start)
{
  if (start)
    hipProfilerStart();
}

void stopProfiling(bool stop, bool &continueIntegration)
{
  if (stop)
  {
    hipProfilerStop();
    continueIntegration = false;
  }
}

dim3 getGridSize(Params &params)
{
  const int totalNumCells = std::ceil((float)params.state.numBubbles / params.inputs.numBubblesPerCell);
  dvec relativeInterval   = params.state.interval / params.state.interval.x;
  float nx                = (float)totalNumCells / relativeInterval.y;
  if (NUM_DIM == 3)
    nx = std::cbrt(nx / relativeInterval.z);
  else
  {
    nx                 = std::sqrt(nx);
    relativeInterval.z = 0;
  }

  ivec grid = (nx * relativeInterval).floor() + 1;
  assert(grid.x > 0);
  assert(grid.y > 0);
  assert(grid.z > 0);

  return dim3(grid.x, grid.y, grid.z);
}

void updateCellsAndNeighbors(Params &params)
{
  dim3 gridSize = getGridSize(params);
  const ivec cellDim(gridSize.x, gridSize.y, gridSize.z);

  int *offsets             = params.state.dips[(uint32_t)DIP::PAIR1];
  int *sizes               = params.state.dips[(uint32_t)DIP::PAIR1] + params.state.maxNumCells;
  int *cellIndices         = params.state.dips[(uint32_t)DIP::TEMP1] + 0 * params.state.dataStride;
  int *bubbleIndices       = params.state.dips[(uint32_t)DIP::TEMP1] + 1 * params.state.dataStride;
  int *sortedCellIndices   = params.state.dips[(uint32_t)DIP::TEMP1] + 2 * params.state.dataStride;
  int *sortedBubbleIndices = params.state.dips[(uint32_t)DIP::TEMP1] + 3 * params.state.dataStride;

  const size_t resetBytes = sizeof(int) * params.state.pairStride * ((uint64_t)DIP::NUM_VALUES - (uint64_t)DIP::PAIR1);
  CUDA_CALL(hipMemset(params.state.dips[(uint32_t)DIP::PAIR1], 0, resetBytes));

  KERNEL_LAUNCH(assignBubblesToCells, params.pairKernelSize, 0, 0, params.state.ddps[(uint32_t)DDP::X],
                params.state.ddps[(uint32_t)DDP::Y], params.state.ddps[(uint32_t)DDP::Z], cellIndices, bubbleIndices,
                params.state.lbb, params.state.tfr, cellDim, params.state.numBubbles);

  params.cw.sortPairs<int, int>(&hipcub::DeviceRadixSort::SortPairs, const_cast<const int *>(cellIndices),
                                sortedCellIndices, const_cast<const int *>(bubbleIndices), sortedBubbleIndices,
                                params.state.numBubbles);

  params.cw.histogram<int *, int, int, int>(&hipcub::DeviceHistogram::HistogramEven, cellIndices, sizes,
                                            params.state.maxNumCells + 1, 0, params.state.maxNumCells,
                                            params.state.numBubbles);

  params.cw.scan<int *, int *>(&hipcub::DeviceScan::ExclusiveSum, sizes, offsets, params.state.maxNumCells);

  KERNEL_LAUNCH(
    reorganizeKernel, params.defaultKernelSize, 0, 0, params.state.numBubbles, ReorganizeType::COPY_FROM_INDEX,
    sortedBubbleIndices, sortedBubbleIndices, params.state.ddps[(uint32_t)DDP::X], params.state.ddps[(uint32_t)DDP::XP],
    params.state.ddps[(uint32_t)DDP::Y], params.state.ddps[(uint32_t)DDP::YP], params.state.ddps[(uint32_t)DDP::Z],
    params.state.ddps[(uint32_t)DDP::ZP], params.state.ddps[(uint32_t)DDP::R], params.state.ddps[(uint32_t)DDP::RP],
    params.state.ddps[(uint32_t)DDP::DXDT], params.state.ddps[(uint32_t)DDP::DXDTP],
    params.state.ddps[(uint32_t)DDP::DYDT], params.state.ddps[(uint32_t)DDP::DYDTP],
    params.state.ddps[(uint32_t)DDP::DZDT], params.state.ddps[(uint32_t)DDP::DZDTP],
    params.state.ddps[(uint32_t)DDP::DRDT], params.state.ddps[(uint32_t)DDP::DRDTP],
    params.state.ddps[(uint32_t)DDP::DXDTO], params.state.ddps[(uint32_t)DDP::ERROR],
    params.state.ddps[(uint32_t)DDP::DYDTO], params.state.ddps[(uint32_t)DDP::TEMP1],
    params.state.ddps[(uint32_t)DDP::DZDTO], params.state.ddps[(uint32_t)DDP::TEMP2],
    params.state.ddps[(uint32_t)DDP::DRDTO], params.state.ddps[(uint32_t)DDP::TEMP3],
    params.state.ddps[(uint32_t)DDP::X0], params.state.ddps[(uint32_t)DDP::TEMP4], params.state.ddps[(uint32_t)DDP::Y0],
    params.state.ddps[(uint32_t)DDP::TEMP5], params.state.ddps[(uint32_t)DDP::Z0],
    params.state.ddps[(uint32_t)DDP::TEMP6], params.state.ddps[(uint32_t)DDP::PATH],
    params.state.ddps[(uint32_t)DDP::TEMP7], params.state.ddps[(uint32_t)DDP::DISTANCE],
    params.state.ddps[(uint32_t)DDP::TEMP8], params.state.dips[(uint32_t)DIP::WRAP_COUNT_X],
    params.state.dips[(uint32_t)DIP::WRAP_COUNT_XP], params.state.dips[(uint32_t)DIP::WRAP_COUNT_Y],
    params.state.dips[(uint32_t)DIP::WRAP_COUNT_YP], params.state.dips[(uint32_t)DIP::WRAP_COUNT_Z],
    params.state.dips[(uint32_t)DIP::WRAP_COUNT_ZP]);

  CUDA_CALL(hipMemcpyAsync(static_cast<void *>(params.state.ddps[(uint32_t)DDP::X]),
                            static_cast<void *>(params.state.ddps[(uint32_t)DDP::XP]), params.state.memReqD / 2,
                            hipMemcpyDeviceToDevice));

  KernelSize kernelSizeNeighbor = KernelSize(gridSize, dim3(128, 1, 1));
  const double maxDistance      = 1.5 * params.state.maxBubbleRadius;

  CUDA_CALL(hipMemset(params.state.np, 0, sizeof(int)));

  for (int i = 0; i < CUBBLE_NUM_NEIGHBORS + 1; ++i)
  {
    hipStream_t stream = (i % 2) ? params.velocityStream : params.gasStream;
    if (NUM_DIM == 3)
      KERNEL_LAUNCH(neighborSearch, kernelSizeNeighbor, 0, stream, i, params.state.numBubbles, params.state.maxNumCells,
                    (int)params.state.pairStride, maxDistance, offsets, sizes, params.state.dips[(uint32_t)DIP::TEMP1],
                    params.state.dips[(uint32_t)DIP::TEMP2], params.state.ddps[(uint32_t)DDP::R],
                    params.state.interval.x, PBC_X == 1, params.state.ddps[(uint32_t)DDP::X], params.state.interval.y,
                    PBC_Y == 1, params.state.ddps[(uint32_t)DDP::Y], params.state.interval.z, PBC_Z == 1,
                    params.state.ddps[(uint32_t)DDP::Z]);
    else
      KERNEL_LAUNCH(neighborSearch, kernelSizeNeighbor, 0, stream, i, params.state.numBubbles, params.state.maxNumCells,
                    (int)params.state.pairStride, maxDistance, offsets, sizes, params.state.dips[(uint32_t)DIP::TEMP1],
                    params.state.dips[(uint32_t)DIP::TEMP2], params.state.ddps[(uint32_t)DDP::R],
                    params.state.interval.x, PBC_X == 1, params.state.ddps[(uint32_t)DDP::X], params.state.interval.y,
                    PBC_Y == 1, params.state.ddps[(uint32_t)DDP::Y]);
  }

  CUDA_CALL(
    hipMemcpy(static_cast<void *>(params.state.pinnedInts), params.state.np, sizeof(int), hipMemcpyDeviceToHost));
  params.state.numPairs = params.state.pinnedInts[0];
  params.cw.sortPairs<int, int>(
    &hipcub::DeviceRadixSort::SortPairs, const_cast<const int *>(params.state.dips[(uint32_t)DIP::TEMP1]),
    params.state.dips[(uint32_t)DIP::PAIR1], const_cast<const int *>(params.state.dips[(uint32_t)DIP::TEMP2]),
    params.state.dips[(uint32_t)DIP::PAIR2], params.state.numPairs);
}

void deleteSmallBubbles(Params &params, int numBubblesAboveMinRad)
{
  NVTX_RANGE_PUSH_A("BubbleRemoval");

  CUDA_CALL(hipMemset(static_cast<void *>(params.state.dvm), 0, sizeof(double)));
  KERNEL_LAUNCH(calculateRedistributedGasVolume, params.defaultKernelSize, 0, 0,
                params.state.ddps[(uint32_t)DDP::TEMP1], params.state.ddps[(uint32_t)DDP::R],
                params.state.dips[(uint32_t)DIP::FLAGS], params.state.numBubbles);

  params.cw.reduceNoCopy<double, double *, double *>(&hipcub::DeviceReduce::Sum, params.state.ddps[(uint32_t)DDP::TEMP1],
                                                     params.state.dtv, params.state.numBubbles);

  int *newIdx = params.state.dips[(uint32_t)DIP::TEMP1];
  params.cw.scan<int *, int *>(&hipcub::DeviceScan::ExclusiveSum, params.state.dips[(uint32_t)DIP::FLAGS], newIdx,
                               params.state.numBubbles);

  KERNEL_LAUNCH(reorganizeKernel, params.defaultKernelSize, 0, 0, params.state.numBubbles,
                ReorganizeType::CONDITIONAL_TO_INDEX, newIdx, params.state.dips[(uint32_t)DIP::FLAGS],
                params.state.ddps[(uint32_t)DDP::X], params.state.ddps[(uint32_t)DDP::XP],
                params.state.ddps[(uint32_t)DDP::Y], params.state.ddps[(uint32_t)DDP::YP],
                params.state.ddps[(uint32_t)DDP::Z], params.state.ddps[(uint32_t)DDP::ZP],
                params.state.ddps[(uint32_t)DDP::R], params.state.ddps[(uint32_t)DDP::RP],
                params.state.ddps[(uint32_t)DDP::DXDT], params.state.ddps[(uint32_t)DDP::DXDTP],
                params.state.ddps[(uint32_t)DDP::DYDT], params.state.ddps[(uint32_t)DDP::DYDTP],
                params.state.ddps[(uint32_t)DDP::DZDT], params.state.ddps[(uint32_t)DDP::DZDTP],
                params.state.ddps[(uint32_t)DDP::DRDT], params.state.ddps[(uint32_t)DDP::DRDTP],
                params.state.ddps[(uint32_t)DDP::DXDTO], params.state.ddps[(uint32_t)DDP::ERROR],
                params.state.ddps[(uint32_t)DDP::DYDTO], params.state.ddps[(uint32_t)DDP::TEMP1],
                params.state.ddps[(uint32_t)DDP::DZDTO], params.state.ddps[(uint32_t)DDP::TEMP2],
                params.state.ddps[(uint32_t)DDP::DRDTO], params.state.ddps[(uint32_t)DDP::TEMP3],
                params.state.ddps[(uint32_t)DDP::X0], params.state.ddps[(uint32_t)DDP::TEMP4],
                params.state.ddps[(uint32_t)DDP::Y0], params.state.ddps[(uint32_t)DDP::TEMP5],
                params.state.ddps[(uint32_t)DDP::Z0], params.state.ddps[(uint32_t)DDP::TEMP6],
                params.state.ddps[(uint32_t)DDP::PATH], params.state.ddps[(uint32_t)DDP::TEMP7],
                params.state.ddps[(uint32_t)DDP::DISTANCE], params.state.ddps[(uint32_t)DDP::TEMP8],
                params.state.dips[(uint32_t)DIP::WRAP_COUNT_X], params.state.dips[(uint32_t)DIP::WRAP_COUNT_XP],
                params.state.dips[(uint32_t)DIP::WRAP_COUNT_Y], params.state.dips[(uint32_t)DIP::WRAP_COUNT_YP],
                params.state.dips[(uint32_t)DIP::WRAP_COUNT_Z], params.state.dips[(uint32_t)DIP::WRAP_COUNT_ZP]);

  CUDA_CALL(hipMemcpyAsync(static_cast<void *>(params.state.ddps[(uint32_t)DDP::X]),
                            static_cast<void *>(params.state.ddps[(uint32_t)DDP::XP]), params.state.memReqD / 2,
                            hipMemcpyDeviceToDevice));

  params.state.numBubbles  = numBubblesAboveMinRad;
  params.defaultKernelSize = KernelSize(128, params.state.numBubbles);

  KERNEL_LAUNCH(addVolume, params.defaultKernelSize, 0, 0, params.state.ddps[(uint32_t)DDP::R],
                params.state.numBubbles);

  NVTX_RANGE_POP();
}

void saveSnapshotToFile(Params &params)
{
  std::stringstream ss;
  ss << params.inputs.snapshotFilename << ".csv." << params.state.numSnapshots;
  std::ofstream file(ss.str().c_str(), std::ios::out);
  if (file.is_open())
  {
    std::vector<double> hostData;
    const size_t numComp = 17;
    hostData.resize(params.state.dataStride * numComp);
    CUDA_CALL(hipMemcpy(hostData.data(), params.state.deviceDoubles,
                         sizeof(double) * numComp * params.state.dataStride, hipMemcpyDeviceToHost));

    file << "x,y,z,r,vx,vy,vz,path,dist\n";
    for (size_t i = 0; i < (size_t)params.state.numBubbles; ++i)
    {
      file << hostData[i + 0 * params.state.dataStride];
      file << ",";
      file << hostData[i + 1 * params.state.dataStride];
      file << ",";
      file << hostData[i + 2 * params.state.dataStride];
      file << ",";
      file << hostData[i + 3 * params.state.dataStride];
      file << ",";
      file << hostData[i + 4 * params.state.dataStride];
      file << ",";
      file << hostData[i + 5 * params.state.dataStride];
      file << ",";
      file << hostData[i + 6 * params.state.dataStride];
      file << ",";
      file << hostData[i + 15 * params.state.dataStride];
      file << ",";
      file << hostData[i + 16 * params.state.dataStride];
      file << "\n";
    }

    ++params.state.numSnapshots;
  }
}

double stabilize(Params &params)
{
  // This function integrates only the positions of the bubbles.
  // Gas exchange is not used. This is used for equilibrating the foam.

  double elapsedTime = 0.0;
  double error       = 100000;

  // Energy before stabilization
  KERNEL_LAUNCH(resetKernel, params.defaultKernelSize, 0, 0, 0.0, params.state.numBubbles,
                params.state.ddps[(uint32_t)DDP::TEMP4]);

  if (NUM_DIM == 3)
    KERNEL_LAUNCH(potentialEnergyKernel, params.pairKernelSize, 0, 0, params.state.numBubbles,
                  params.state.dips[(uint32_t)DIP::PAIR1], params.state.dips[(uint32_t)DIP::PAIR2],
                  params.state.ddps[(uint32_t)DDP::R], params.state.ddps[(uint32_t)DDP::TEMP4], params.state.interval.x,
                  PBC_X == 1, params.state.ddps[(uint32_t)DDP::X], params.state.interval.y, PBC_Y == 1,
                  params.state.ddps[(uint32_t)DDP::Y], params.state.interval.z, PBC_Z == 1,
                  params.state.ddps[(uint32_t)DDP::Z]);
  else
    KERNEL_LAUNCH(potentialEnergyKernel, params.pairKernelSize, 0, 0, params.state.numBubbles,
                  params.state.dips[(uint32_t)DIP::PAIR1], params.state.dips[(uint32_t)DIP::PAIR2],
                  params.state.ddps[(uint32_t)DDP::R], params.state.ddps[(uint32_t)DDP::TEMP4], params.state.interval.x,
                  PBC_X == 1, params.state.ddps[(uint32_t)DDP::X], params.state.interval.y, PBC_Y == 1,
                  params.state.ddps[(uint32_t)DDP::Y]);

  params.cw.reduceNoCopy<double, double *, double *>(&hipcub::DeviceReduce::Sum, params.state.ddps[(uint32_t)DDP::TEMP4],
                                                     params.state.dtfapr, params.state.numBubbles);
  CUDA_CALL(hipMemcpyAsync(static_cast<void *>(&params.state.pinnedDoubles[1]),
                            static_cast<void *>(params.state.dtfapr), sizeof(double), hipMemcpyDeviceToHost, 0));

  for (int i = 0; i < params.inputs.numStepsToRelax; ++i)
  {
    do
    {
      if (NUM_DIM == 3)
      {
        KERNEL_LAUNCH(resetKernel, params.defaultKernelSize, 0, 0, 0.0, params.state.numBubbles,
                      params.state.ddps[(uint32_t)DDP::DXDTP], params.state.ddps[(uint32_t)DDP::DYDTP],
                      params.state.ddps[(uint32_t)DDP::DZDTP], params.state.ddps[(uint32_t)DDP::ERROR],
                      params.state.ddps[(uint32_t)DDP::TEMP1], params.state.ddps[(uint32_t)DDP::TEMP2]);

        KERNEL_LAUNCH(predictKernel, params.defaultKernelSize, 0, 0, params.state.numBubbles, params.state.timeStep,
                      params.state.ddps[(uint32_t)DDP::XP], params.state.ddps[(uint32_t)DDP::X],
                      params.state.ddps[(uint32_t)DDP::DXDT], params.state.ddps[(uint32_t)DDP::DXDTO],
                      params.state.ddps[(uint32_t)DDP::YP], params.state.ddps[(uint32_t)DDP::Y],
                      params.state.ddps[(uint32_t)DDP::DYDT], params.state.ddps[(uint32_t)DDP::DYDTO],
                      params.state.ddps[(uint32_t)DDP::ZP], params.state.ddps[(uint32_t)DDP::Z],
                      params.state.ddps[(uint32_t)DDP::DZDT], params.state.ddps[(uint32_t)DDP::DZDTO]);

        KERNEL_LAUNCH(velocityPairKernel, params.pairKernelSize, 0, 0, params.inputs.fZeroPerMuZero,
                      params.state.dips[(uint32_t)DIP::PAIR1], params.state.dips[(uint32_t)DIP::PAIR2],
                      params.state.ddps[(uint32_t)DDP::RP], params.state.interval.x, params.state.lbb.x, PBC_X == 1,
                      params.state.ddps[(uint32_t)DDP::XP], params.state.ddps[(uint32_t)DDP::DXDTP],
                      params.state.interval.y, params.state.lbb.y, PBC_Y == 1, params.state.ddps[(uint32_t)DDP::YP],
                      params.state.ddps[(uint32_t)DDP::DYDTP], params.state.interval.z, params.state.lbb.z, PBC_Z == 1,
                      params.state.ddps[(uint32_t)DDP::ZP], params.state.ddps[(uint32_t)DDP::DZDTP]);

        doWallVelocity(params.pairKernelSize, 0, 0, PBC_X == 0, PBC_Y == 0, PBC_Z == 0, params.state.numBubbles,
                       params.state.dips[(uint32_t)DIP::PAIR1], params.state.dips[(uint32_t)DIP::PAIR2],
                       params.state.ddps[(uint32_t)DDP::RP], params.state.ddps[(uint32_t)DDP::XP],
                       params.state.ddps[(uint32_t)DDP::YP], params.state.ddps[(uint32_t)DDP::ZP],
                       params.state.ddps[(uint32_t)DDP::DXDTP], params.state.ddps[(uint32_t)DDP::DYDTP],
                       params.state.ddps[(uint32_t)DDP::DZDTP], params.state.lbb, params.state.tfr, params);

        KERNEL_LAUNCH(correctKernel, params.defaultKernelSize, 0, 0, params.state.numBubbles, params.state.timeStep,
                      params.state.ddps[(uint32_t)DDP::ERROR], params.state.ddps[(uint32_t)DDP::XP],
                      params.state.ddps[(uint32_t)DDP::X], params.state.ddps[(uint32_t)DDP::DXDT],
                      params.state.ddps[(uint32_t)DDP::DXDTP], params.state.ddps[(uint32_t)DDP::YP],
                      params.state.ddps[(uint32_t)DDP::Y], params.state.ddps[(uint32_t)DDP::DYDT],
                      params.state.ddps[(uint32_t)DDP::DYDTP], params.state.ddps[(uint32_t)DDP::ZP],
                      params.state.ddps[(uint32_t)DDP::Z], params.state.ddps[(uint32_t)DDP::DZDT],
                      params.state.ddps[(uint32_t)DDP::DZDTP]);
      }
      else // Two dimensional case
      {
        KERNEL_LAUNCH(resetKernel, params.defaultKernelSize, 0, 0, 0.0, params.state.numBubbles,
                      params.state.ddps[(uint32_t)DDP::DXDTP], params.state.ddps[(uint32_t)DDP::DYDTP],
                      params.state.ddps[(uint32_t)DDP::ERROR], params.state.ddps[(uint32_t)DDP::TEMP1],
                      params.state.ddps[(uint32_t)DDP::TEMP2]);

        KERNEL_LAUNCH(predictKernel, params.defaultKernelSize, 0, 0, params.state.numBubbles, params.state.timeStep,
                      params.state.ddps[(uint32_t)DDP::XP], params.state.ddps[(uint32_t)DDP::X],
                      params.state.ddps[(uint32_t)DDP::DXDT], params.state.ddps[(uint32_t)DDP::DXDTO],
                      params.state.ddps[(uint32_t)DDP::YP], params.state.ddps[(uint32_t)DDP::Y],
                      params.state.ddps[(uint32_t)DDP::DYDT], params.state.ddps[(uint32_t)DDP::DYDTO]);

        KERNEL_LAUNCH(velocityPairKernel, params.pairKernelSize, 0, 0, params.inputs.fZeroPerMuZero,
                      params.state.dips[(uint32_t)DIP::PAIR1], params.state.dips[(uint32_t)DIP::PAIR2],
                      params.state.ddps[(uint32_t)DDP::RP], params.state.interval.x, params.state.lbb.x, PBC_X == 1,
                      params.state.ddps[(uint32_t)DDP::XP], params.state.ddps[(uint32_t)DDP::DXDTP],
                      params.state.interval.y, params.state.lbb.y, PBC_Y == 1, params.state.ddps[(uint32_t)DDP::YP],
                      params.state.ddps[(uint32_t)DDP::DYDTP]);

        doWallVelocity(params.pairKernelSize, 0, 0, PBC_X == 0, PBC_Y == 0, PBC_Z == 0, params.state.numBubbles,
                       params.state.dips[(uint32_t)DIP::PAIR1], params.state.dips[(uint32_t)DIP::PAIR2],
                       params.state.ddps[(uint32_t)DDP::RP], params.state.ddps[(uint32_t)DDP::XP],
                       params.state.ddps[(uint32_t)DDP::YP], params.state.ddps[(uint32_t)DDP::ZP],
                       params.state.ddps[(uint32_t)DDP::DXDTP], params.state.ddps[(uint32_t)DDP::DYDTP],
                       params.state.ddps[(uint32_t)DDP::DZDTP], params.state.lbb, params.state.tfr, params);

        KERNEL_LAUNCH(correctKernel, params.defaultKernelSize, 0, 0, params.state.numBubbles, params.state.timeStep,
                      params.state.ddps[(uint32_t)DDP::ERROR], params.state.ddps[(uint32_t)DDP::XP],
                      params.state.ddps[(uint32_t)DDP::X], params.state.ddps[(uint32_t)DDP::DXDT],
                      params.state.ddps[(uint32_t)DDP::DXDTP], params.state.ddps[(uint32_t)DDP::YP],
                      params.state.ddps[(uint32_t)DDP::Y], params.state.ddps[(uint32_t)DDP::DYDT],
                      params.state.ddps[(uint32_t)DDP::DYDTP]);
      }

      doBoundaryWrap(params.defaultKernelSize, 0, 0, PBC_X == 1, PBC_Y == 1, PBC_Z == 1, params.state.numBubbles,
                     params.state.ddps[(uint32_t)DDP::XP], params.state.ddps[(uint32_t)DDP::YP],
                     params.state.ddps[(uint32_t)DDP::ZP], params.state.lbb, params.state.tfr,
                     params.state.dips[(uint32_t)DIP::WRAP_COUNT_X], params.state.dips[(uint32_t)DIP::WRAP_COUNT_Y],
                     params.state.dips[(uint32_t)DIP::WRAP_COUNT_Z], params.state.dips[(uint32_t)DIP::WRAP_COUNT_XP],
                     params.state.dips[(uint32_t)DIP::WRAP_COUNT_YP], params.state.dips[(uint32_t)DIP::WRAP_COUNT_ZP]);

      // Error
      error = params.cw.reduce<double, double *, double *>(
        &hipcub::DeviceReduce::Max, params.state.ddps[(uint32_t)DDP::ERROR], params.state.numBubbles);

      if (error < params.inputs.errorTolerance && params.state.timeStep < 0.1)
        params.state.timeStep *= 1.9;
      else if (error > params.inputs.errorTolerance)
        params.state.timeStep *= 0.5;

    } while (error > params.inputs.errorTolerance);

    // Update the current values with the calculated predictions
    const size_t numBytesToCopy = 3 * sizeof(double) * params.state.dataStride;
    CUDA_CALL(hipMemcpyAsync(params.state.ddps[(uint32_t)DDP::DXDTO], params.state.ddps[(uint32_t)DDP::DXDT],
                              numBytesToCopy, hipMemcpyDeviceToDevice, 0));
    CUDA_CALL(hipMemcpyAsync(params.state.ddps[(uint32_t)DDP::X], params.state.ddps[(uint32_t)DDP::XP], numBytesToCopy,
                              hipMemcpyDeviceToDevice, 0));
    CUDA_CALL(hipMemcpyAsync(params.state.ddps[(uint32_t)DDP::DXDT], params.state.ddps[(uint32_t)DDP::DXDTP],
                              numBytesToCopy, hipMemcpyDeviceToDevice, 0));

    elapsedTime += params.state.timeStep;

    if (i % 5000 == 0)
      updateCellsAndNeighbors(params);
  }

  // Energy after stabilization
  params.state.energy1 = params.state.pinnedDoubles[1];

  KERNEL_LAUNCH(resetKernel, params.defaultKernelSize, 0, 0, 0.0, params.state.numBubbles,
                params.state.ddps[(uint32_t)DDP::TEMP4]);

  if (NUM_DIM == 3)
    KERNEL_LAUNCH(potentialEnergyKernel, params.pairKernelSize, 0, 0, params.state.numBubbles,
                  params.state.dips[(uint32_t)DIP::PAIR1], params.state.dips[(uint32_t)DIP::PAIR2],
                  params.state.ddps[(uint32_t)DDP::R], params.state.ddps[(uint32_t)DDP::TEMP4], params.state.interval.x,
                  PBC_X == 1, params.state.ddps[(uint32_t)DDP::X], params.state.interval.y, PBC_Y == 1,
                  params.state.ddps[(uint32_t)DDP::Y], params.state.interval.z, PBC_Z == 1,
                  params.state.ddps[(uint32_t)DDP::Z]);
  else
    KERNEL_LAUNCH(potentialEnergyKernel, params.pairKernelSize, 0, 0, params.state.numBubbles,
                  params.state.dips[(uint32_t)DIP::PAIR1], params.state.dips[(uint32_t)DIP::PAIR2],
                  params.state.ddps[(uint32_t)DDP::R], params.state.ddps[(uint32_t)DDP::TEMP4], params.state.interval.x,
                  PBC_X == 1, params.state.ddps[(uint32_t)DDP::X], params.state.interval.y, PBC_Y == 1,
                  params.state.ddps[(uint32_t)DDP::Y]);

  params.state.energy2 = params.cw.reduce<double, double *, double *>(
    &hipcub::DeviceReduce::Sum, params.state.ddps[(uint32_t)DDP::TEMP4], params.state.numBubbles);

  return elapsedTime;
}

bool integrate(Params &params)
{
  NVTX_RANGE_PUSH_A("Integration function");

  double error          = 100000;
  uint32_t numLoopsDone = 0;

  do
  {
    NVTX_RANGE_PUSH_A("Integration step");

    if (NUM_DIM == 3)
    {
      // Reset
      KERNEL_LAUNCH(resetKernel, params.defaultKernelSize, 0, 0, 0.0, params.state.numBubbles,
                    params.state.ddps[(uint32_t)DDP::DXDTP], params.state.ddps[(uint32_t)DDP::DYDTP],
                    params.state.ddps[(uint32_t)DDP::DZDTP], params.state.ddps[(uint32_t)DDP::DRDTP],
                    params.state.ddps[(uint32_t)DDP::ERROR], params.state.ddps[(uint32_t)DDP::TEMP1],
                    params.state.ddps[(uint32_t)DDP::TEMP2]);

      // Predict
      KERNEL_LAUNCH(predictKernel, params.defaultKernelSize, 0, 0, params.state.numBubbles, params.state.timeStep,
                    params.state.ddps[(uint32_t)DDP::XP], params.state.ddps[(uint32_t)DDP::X],
                    params.state.ddps[(uint32_t)DDP::DXDT], params.state.ddps[(uint32_t)DDP::DXDTO],
                    params.state.ddps[(uint32_t)DDP::YP], params.state.ddps[(uint32_t)DDP::Y],
                    params.state.ddps[(uint32_t)DDP::DYDT], params.state.ddps[(uint32_t)DDP::DYDTO],
                    params.state.ddps[(uint32_t)DDP::ZP], params.state.ddps[(uint32_t)DDP::Z],
                    params.state.ddps[(uint32_t)DDP::DZDT], params.state.ddps[(uint32_t)DDP::DZDTO],
                    params.state.ddps[(uint32_t)DDP::RP], params.state.ddps[(uint32_t)DDP::R],
                    params.state.ddps[(uint32_t)DDP::DRDT], params.state.ddps[(uint32_t)DDP::DRDTO]);

      // Velocity
      KERNEL_LAUNCH(velocityPairKernel, params.pairKernelSize, 0, params.velocityStream, params.inputs.fZeroPerMuZero,
                    params.state.dips[(uint32_t)DIP::PAIR1], params.state.dips[(uint32_t)DIP::PAIR2],
                    params.state.ddps[(uint32_t)DDP::RP], params.state.interval.x, params.state.lbb.x, PBC_X == 1,
                    params.state.ddps[(uint32_t)DDP::XP], params.state.ddps[(uint32_t)DDP::DXDTP],
                    params.state.interval.y, params.state.lbb.y, PBC_Y == 1, params.state.ddps[(uint32_t)DDP::YP],
                    params.state.ddps[(uint32_t)DDP::DYDTP], params.state.interval.z, params.state.lbb.z, PBC_Z == 1,
                    params.state.ddps[(uint32_t)DDP::ZP], params.state.ddps[(uint32_t)DDP::DZDTP]);
      // Wall velocity
      doWallVelocity(params.pairKernelSize, 0, params.velocityStream, PBC_X == 0, PBC_Y == 0, PBC_Z == 0,
                     params.state.numBubbles, params.state.dips[(uint32_t)DIP::PAIR1],
                     params.state.dips[(uint32_t)DIP::PAIR2], params.state.ddps[(uint32_t)DDP::RP],
                     params.state.ddps[(uint32_t)DDP::XP], params.state.ddps[(uint32_t)DDP::YP],
                     params.state.ddps[(uint32_t)DDP::ZP], params.state.ddps[(uint32_t)DDP::DXDTP],
                     params.state.ddps[(uint32_t)DDP::DYDTP], params.state.ddps[(uint32_t)DDP::DZDTP], params.state.lbb,
                     params.state.tfr, params);

      // Flow velocity
      if (USE_FLOW == 1)
      {
        CUDA_CALL(hipMemset(params.state.dips[(uint32_t)DIP::TEMP1], 0, sizeof(int) * params.state.pairStride));
        int *numNeighbors = params.state.dips[(uint32_t)DIP::TEMP1];

        KERNEL_LAUNCH(neighborVelocityKernel, params.pairKernelSize, 0, params.velocityStream,
                      params.state.dips[(uint32_t)DIP::PAIR1], params.state.dips[(uint32_t)DIP::PAIR2], numNeighbors,
                      params.state.ddps[(uint32_t)DDP::TEMP1], params.state.ddps[(uint32_t)DDP::DXDTO],
                      params.state.ddps[(uint32_t)DDP::TEMP2], params.state.ddps[(uint32_t)DDP::DYDTO],
                      params.state.ddps[(uint32_t)DDP::TEMP3], params.state.ddps[(uint32_t)DDP::DZDTO]);

        KERNEL_LAUNCH(flowVelocityKernel, params.pairKernelSize, 0, params.velocityStream, params.state.numBubbles,
                      numNeighbors, params.state.ddps[(uint32_t)DDP::DXDTP], params.state.ddps[(uint32_t)DDP::DYDTP],
                      params.state.ddps[(uint32_t)DDP::DZDTP], params.state.ddps[(uint32_t)DDP::TEMP1],
                      params.state.ddps[(uint32_t)DDP::TEMP2], params.state.ddps[(uint32_t)DDP::TEMP3],
                      params.state.ddps[(uint32_t)DDP::XP], params.state.ddps[(uint32_t)DDP::YP],
                      params.state.ddps[(uint32_t)DDP::ZP], params.inputs.flowVel, params.inputs.flowTfr,
                      params.inputs.flowLbb);
      }

      // Correct
      KERNEL_LAUNCH(correctKernel, params.defaultKernelSize, 0, params.velocityStream, params.state.numBubbles,
                    params.state.timeStep, params.state.ddps[(uint32_t)DDP::ERROR],
                    params.state.ddps[(uint32_t)DDP::XP], params.state.ddps[(uint32_t)DDP::X],
                    params.state.ddps[(uint32_t)DDP::DXDT], params.state.ddps[(uint32_t)DDP::DXDTP],
                    params.state.ddps[(uint32_t)DDP::YP], params.state.ddps[(uint32_t)DDP::Y],
                    params.state.ddps[(uint32_t)DDP::DYDT], params.state.ddps[(uint32_t)DDP::DYDTP],
                    params.state.ddps[(uint32_t)DDP::ZP], params.state.ddps[(uint32_t)DDP::Z],
                    params.state.ddps[(uint32_t)DDP::DZDT], params.state.ddps[(uint32_t)DDP::DZDTP]);

      // Path lenghts & distances
      KERNEL_LAUNCH(pathLengthDistanceKernel, params.defaultKernelSize, 0, params.velocityStream,
                    params.state.numBubbles, params.state.ddps[(uint32_t)DDP::TEMP4],
                    params.state.ddps[(uint32_t)DDP::PATH], params.state.ddps[(uint32_t)DDP::DISTANCE],
                    params.state.ddps[(uint32_t)DDP::XP], params.state.ddps[(uint32_t)DDP::X],
                    params.state.ddps[(uint32_t)DDP::X0], params.state.dips[(uint32_t)DIP::WRAP_COUNT_XP],
                    params.state.interval.x, params.state.ddps[(uint32_t)DDP::YP], params.state.ddps[(uint32_t)DDP::Y],
                    params.state.ddps[(uint32_t)DDP::Y0], params.state.dips[(uint32_t)DIP::WRAP_COUNT_YP],
                    params.state.interval.y, params.state.ddps[(uint32_t)DDP::ZP], params.state.ddps[(uint32_t)DDP::Z],
                    params.state.ddps[(uint32_t)DDP::Z0], params.state.dips[(uint32_t)DIP::WRAP_COUNT_ZP],
                    params.state.interval.z);

      // Boundary wrap
      doBoundaryWrap(params.defaultKernelSize, 0, params.velocityStream, PBC_X == 1, PBC_Y == 1, PBC_Z == 1,
                     params.state.numBubbles, params.state.ddps[(uint32_t)DDP::XP],
                     params.state.ddps[(uint32_t)DDP::YP], params.state.ddps[(uint32_t)DDP::ZP], params.state.lbb,
                     params.state.tfr, params.state.dips[(uint32_t)DIP::WRAP_COUNT_X],
                     params.state.dips[(uint32_t)DIP::WRAP_COUNT_Y], params.state.dips[(uint32_t)DIP::WRAP_COUNT_Z],
                     params.state.dips[(uint32_t)DIP::WRAP_COUNT_XP], params.state.dips[(uint32_t)DIP::WRAP_COUNT_YP],
                     params.state.dips[(uint32_t)DIP::WRAP_COUNT_ZP]);

      // Gas exchange
      KERNEL_LAUNCH(gasExchangeKernel, params.pairKernelSize, 0, params.gasStream, params.state.numBubbles,
                    params.state.dips[(uint32_t)DIP::PAIR1], params.state.dips[(uint32_t)DIP::PAIR2],
                    params.state.ddps[(uint32_t)DDP::RP], params.state.ddps[(uint32_t)DDP::DRDTP],
                    params.state.ddps[(uint32_t)DDP::TEMP1], params.state.interval.x, PBC_X == 1,
                    params.state.ddps[(uint32_t)DDP::XP], params.state.interval.y, PBC_Y == 1,
                    params.state.ddps[(uint32_t)DDP::YP], params.state.interval.z, PBC_Z == 1,
                    params.state.ddps[(uint32_t)DDP::ZP]);
    }
    else // Two dimensions
    {
      // Reset
      KERNEL_LAUNCH(resetKernel, params.defaultKernelSize, 0, 0, 0.0, params.state.numBubbles,
                    params.state.ddps[(uint32_t)DDP::DXDTP], params.state.ddps[(uint32_t)DDP::DYDTP],
                    params.state.ddps[(uint32_t)DDP::DRDTP], params.state.ddps[(uint32_t)DDP::ERROR],
                    params.state.ddps[(uint32_t)DDP::TEMP1], params.state.ddps[(uint32_t)DDP::TEMP2]);

      // Predict
      KERNEL_LAUNCH(predictKernel, params.defaultKernelSize, 0, 0, params.state.numBubbles, params.state.timeStep,
                    params.state.ddps[(uint32_t)DDP::XP], params.state.ddps[(uint32_t)DDP::X],
                    params.state.ddps[(uint32_t)DDP::DXDT], params.state.ddps[(uint32_t)DDP::DXDTO],
                    params.state.ddps[(uint32_t)DDP::YP], params.state.ddps[(uint32_t)DDP::Y],
                    params.state.ddps[(uint32_t)DDP::DYDT], params.state.ddps[(uint32_t)DDP::DYDTO],
                    params.state.ddps[(uint32_t)DDP::RP], params.state.ddps[(uint32_t)DDP::R],
                    params.state.ddps[(uint32_t)DDP::DRDT], params.state.ddps[(uint32_t)DDP::DRDTO]);

      // Velocity
      KERNEL_LAUNCH(velocityPairKernel, params.pairKernelSize, 0, params.velocityStream, params.inputs.fZeroPerMuZero,
                    params.state.dips[(uint32_t)DIP::PAIR1], params.state.dips[(uint32_t)DIP::PAIR2],
                    params.state.ddps[(uint32_t)DDP::RP], params.state.interval.x, params.state.lbb.x, PBC_X == 1,
                    params.state.ddps[(uint32_t)DDP::XP], params.state.ddps[(uint32_t)DDP::DXDTP],
                    params.state.interval.y, params.state.lbb.y, PBC_Y == 1, params.state.ddps[(uint32_t)DDP::YP],
                    params.state.ddps[(uint32_t)DDP::DYDTP]);
      // Wall velocity
      doWallVelocity(params.pairKernelSize, 0, params.velocityStream, PBC_X == 0, PBC_Y == 0, false,
                     params.state.numBubbles, params.state.dips[(uint32_t)DIP::PAIR1],
                     params.state.dips[(uint32_t)DIP::PAIR2], params.state.ddps[(uint32_t)DDP::RP],
                     params.state.ddps[(uint32_t)DDP::XP], params.state.ddps[(uint32_t)DDP::YP],
                     params.state.ddps[(uint32_t)DDP::ZP], params.state.ddps[(uint32_t)DDP::DXDTP],
                     params.state.ddps[(uint32_t)DDP::DYDTP], params.state.ddps[(uint32_t)DDP::DZDTP], params.state.lbb,
                     params.state.tfr, params);

      // Flow velocity
      if (USE_FLOW == 1)
      {
        CUDA_CALL(hipMemset(params.state.dips[(uint32_t)DIP::TEMP1], 0, sizeof(int) * params.state.pairStride));
        int *numNeighbors = params.state.dips[(uint32_t)DIP::TEMP1];

        KERNEL_LAUNCH(neighborVelocityKernel, params.pairKernelSize, 0, params.velocityStream,
                      params.state.dips[(uint32_t)DIP::PAIR1], params.state.dips[(uint32_t)DIP::PAIR2], numNeighbors,
                      params.state.ddps[(uint32_t)DDP::TEMP1], params.state.ddps[(uint32_t)DDP::DXDTO],
                      params.state.ddps[(uint32_t)DDP::TEMP2], params.state.ddps[(uint32_t)DDP::DYDTO]);

        KERNEL_LAUNCH(flowVelocityKernel, params.pairKernelSize, 0, params.velocityStream, params.state.numBubbles,
                      numNeighbors, params.state.ddps[(uint32_t)DDP::DXDTP], params.state.ddps[(uint32_t)DDP::DYDTP],
                      params.state.ddps[(uint32_t)DDP::DZDTP], params.state.ddps[(uint32_t)DDP::TEMP1],
                      params.state.ddps[(uint32_t)DDP::TEMP2], params.state.ddps[(uint32_t)DDP::TEMP3],
                      params.state.ddps[(uint32_t)DDP::XP], params.state.ddps[(uint32_t)DDP::YP],
                      params.state.ddps[(uint32_t)DDP::ZP], params.inputs.flowVel, params.inputs.flowTfr,
                      params.inputs.flowLbb);
      }

      // Correct
      KERNEL_LAUNCH(correctKernel, params.defaultKernelSize, 0, params.velocityStream, params.state.numBubbles,
                    params.state.timeStep, params.state.ddps[(uint32_t)DDP::ERROR],
                    params.state.ddps[(uint32_t)DDP::XP], params.state.ddps[(uint32_t)DDP::X],
                    params.state.ddps[(uint32_t)DDP::DXDT], params.state.ddps[(uint32_t)DDP::DXDTP],
                    params.state.ddps[(uint32_t)DDP::YP], params.state.ddps[(uint32_t)DDP::Y],
                    params.state.ddps[(uint32_t)DDP::DYDT], params.state.ddps[(uint32_t)DDP::DYDTP]);

      // Path lenghts & distances
      KERNEL_LAUNCH(pathLengthDistanceKernel, params.defaultKernelSize, 0, params.velocityStream,
                    params.state.numBubbles, params.state.ddps[(uint32_t)DDP::TEMP4],
                    params.state.ddps[(uint32_t)DDP::PATH], params.state.ddps[(uint32_t)DDP::DISTANCE],
                    params.state.ddps[(uint32_t)DDP::XP], params.state.ddps[(uint32_t)DDP::X],
                    params.state.ddps[(uint32_t)DDP::X0], params.state.dips[(uint32_t)DIP::WRAP_COUNT_XP],
                    params.state.interval.x, params.state.ddps[(uint32_t)DDP::YP], params.state.ddps[(uint32_t)DDP::Y],
                    params.state.ddps[(uint32_t)DDP::Y0], params.state.dips[(uint32_t)DIP::WRAP_COUNT_YP],
                    params.state.interval.y);

      // Boundary wrap
      doBoundaryWrap(params.defaultKernelSize, 0, params.velocityStream, PBC_X == 1, PBC_Y == 1, false,
                     params.state.numBubbles, params.state.ddps[(uint32_t)DDP::XP],
                     params.state.ddps[(uint32_t)DDP::YP], params.state.ddps[(uint32_t)DDP::ZP], params.state.lbb,
                     params.state.tfr, params.state.dips[(uint32_t)DIP::WRAP_COUNT_X],
                     params.state.dips[(uint32_t)DIP::WRAP_COUNT_Y], params.state.dips[(uint32_t)DIP::WRAP_COUNT_Z],
                     params.state.dips[(uint32_t)DIP::WRAP_COUNT_XP], params.state.dips[(uint32_t)DIP::WRAP_COUNT_YP],
                     params.state.dips[(uint32_t)DIP::WRAP_COUNT_ZP]);

      // Gas exchange
      KERNEL_LAUNCH(gasExchangeKernel, params.pairKernelSize, 0, params.gasStream, params.state.numBubbles,
                    params.state.dips[(uint32_t)DIP::PAIR1], params.state.dips[(uint32_t)DIP::PAIR2],
                    params.state.ddps[(uint32_t)DDP::RP], params.state.ddps[(uint32_t)DDP::DRDTP],
                    params.state.ddps[(uint32_t)DDP::TEMP1], params.state.interval.x, PBC_X == 1,
                    params.state.ddps[(uint32_t)DDP::XP], params.state.interval.y, PBC_Y == 1,
                    params.state.ddps[(uint32_t)DDP::YP]);
    }

    // Free area
    KERNEL_LAUNCH(freeAreaKernel, params.defaultKernelSize, 0, params.gasStream, params.state.numBubbles,
                  params.state.ddps[(uint32_t)DDP::RP], params.state.ddps[(uint32_t)DDP::TEMP1],
                  params.state.ddps[(uint32_t)DDP::TEMP2], params.state.ddps[(uint32_t)DDP::TEMP3]);

    params.cw.reduceNoCopy<double, double *, double *>(&hipcub::DeviceReduce::Sum, params.state.ddps[(uint32_t)DDP::TEMP1],
                                                       params.state.dtfa, params.state.numBubbles, params.gasStream);
    params.cw.reduceNoCopy<double, double *, double *>(&hipcub::DeviceReduce::Sum, params.state.ddps[(uint32_t)DDP::TEMP2],
                                                       params.state.dtfapr, params.state.numBubbles, params.gasStream);
    params.cw.reduceNoCopy<double, double *, double *>(&hipcub::DeviceReduce::Sum, params.state.ddps[(uint32_t)DDP::TEMP3],
                                                       params.state.dta, params.state.numBubbles, params.gasStream);

    KERNEL_LAUNCH(finalRadiusChangeRateKernel, params.defaultKernelSize, 0, params.gasStream,
                  params.state.ddps[(uint32_t)DDP::DRDTP], params.state.ddps[(uint32_t)DDP::RP],
                  params.state.ddps[(uint32_t)DDP::TEMP1], params.state.numBubbles, params.inputs.kappa,
                  params.inputs.kParameter);

    // Radius correct
    KERNEL_LAUNCH(correctKernel, params.defaultKernelSize, 0, params.gasStream, params.state.numBubbles,
                  params.state.timeStep, params.state.ddps[(uint32_t)DDP::ERROR], params.state.ddps[(uint32_t)DDP::RP],
                  params.state.ddps[(uint32_t)DDP::R], params.state.ddps[(uint32_t)DDP::DRDT],
                  params.state.ddps[(uint32_t)DDP::DRDTP]);

    // Calculate how many bubbles are below the minimum size.
    // Also take note of maximum radius.
    KERNEL_LAUNCH(setFlagIfGreaterThanConstantKernel, params.defaultKernelSize, 0, params.gasStream,
                  params.state.numBubbles, params.state.dips[(uint32_t)DIP::FLAGS],
                  params.state.ddps[(uint32_t)DDP::RP], params.inputs.minRad);

    params.cw.reduceNoCopy<int, int *, int *>(&hipcub::DeviceReduce::Sum, params.state.dips[(uint32_t)DIP::FLAGS],
                                              params.state.mbpc, params.state.numBubbles, params.gasStream);
    params.cw.reduceNoCopy<double, double *, double *>(&hipcub::DeviceReduce::Max, params.state.ddps[(uint32_t)DDP::RP],
                                                       params.state.dtfa, params.state.numBubbles, params.gasStream);

    CUDA_CALL(hipMemcpyAsync(static_cast<void *>(params.state.pinnedInts), params.state.mbpc, sizeof(int),
                              hipMemcpyDeviceToHost, params.gasStream));
    CUDA_CALL(hipMemcpyAsync(static_cast<void *>(params.state.pinnedDoubles), params.state.dtfa, sizeof(double),
                              hipMemcpyDeviceToHost, params.gasStream));

    // Error
    error = params.cw.reduce<double, double *, double *>(
      &hipcub::DeviceReduce::Max, params.state.ddps[(uint32_t)DDP::ERROR], params.state.numBubbles);

    if (error < params.inputs.errorTolerance && params.state.timeStep < 0.1)
      params.state.timeStep *= 1.9;
    else if (error > params.inputs.errorTolerance)
      params.state.timeStep *= 0.5;

    ++numLoopsDone;

    NVTX_RANGE_POP();
  } while (error > params.inputs.errorTolerance);

  // Update values
  const size_t numBytesToCopy = 4 * sizeof(double) * params.state.dataStride;

  CUDA_CALL(hipMemcpyAsync(params.state.ddps[(uint32_t)DDP::DXDTO], params.state.ddps[(uint32_t)DDP::DXDT],
                            numBytesToCopy, hipMemcpyDeviceToDevice));
  CUDA_CALL(hipMemcpyAsync(params.state.ddps[(uint32_t)DDP::X], params.state.ddps[(uint32_t)DDP::XP],
                            2 * numBytesToCopy, hipMemcpyDeviceToDevice));
  CUDA_CALL(hipMemcpyAsync(params.state.ddps[(uint32_t)DDP::PATH], params.state.ddps[(uint32_t)DDP::TEMP4],
                            sizeof(double) * params.state.dataStride, hipMemcpyDeviceToDevice));
  CUDA_CALL(hipMemcpyAsync(params.state.dips[(uint32_t)DIP::WRAP_COUNT_XP],
                            params.state.dips[(uint32_t)DIP::WRAP_COUNT_X], params.state.dataStride * 3 * sizeof(int),
                            hipMemcpyDeviceToDevice));

  ++params.state.numIntegrationSteps;
  params.state.simulationTime += params.state.timeStep;
  params.state.maxBubbleRadius = params.state.pinnedDoubles[0];

  // Delete & reorder
  const int numBubblesAboveMinRad = params.state.pinnedInts[0];
  const bool shouldDeleteBubbles  = numBubblesAboveMinRad < params.state.numBubbles;

  if (shouldDeleteBubbles)
    deleteSmallBubbles(params, numBubblesAboveMinRad);

  if (shouldDeleteBubbles || params.state.numIntegrationSteps % 5000 == 0)
    updateCellsAndNeighbors(params);

  bool continueSimulation = params.state.numBubbles > params.inputs.minNumBubbles;
  continueSimulation &= (NUM_DIM == 3)
                          ? params.state.maxBubbleRadius < 0.5 * (params.state.tfr - params.state.lbb).getMinComponent()
                          : true;

  NVTX_RANGE_POP();

  return continueSimulation;
}

void transformPositions(Params &params, bool normalize)
{
  KERNEL_LAUNCH(transformPositionsKernel, params.pairKernelSize, 0, 0, normalize, params.state.numBubbles,
                params.state.lbb, params.state.tfr, params.state.ddps[(uint32_t)DDP::X],
                params.state.ddps[(uint32_t)DDP::Y], params.state.ddps[(uint32_t)DDP::Z]);
}

double calculateVolumeOfBubbles(Params &params)
{
  KERNEL_LAUNCH(calculateVolumes, params.defaultKernelSize, 0, 0, params.state.ddps[(uint32_t)DDP::R],
                params.state.ddps[(uint32_t)DDP::TEMP1], params.state.numBubbles);

  return params.cw.reduce<double, double *, double *>(&hipcub::DeviceReduce::Sum, params.state.ddps[(uint32_t)DDP::TEMP1],
                                                      params.state.numBubbles);
}

void deinit(Params &params)
{
  saveSnapshotToFile(params);

  CUDA_CALL(hipDeviceSynchronize());

  CUDA_CALL(hipFree(static_cast<void *>(params.state.deviceDoubles)));
  CUDA_CALL(hipFree(static_cast<void *>(params.state.deviceInts)));
  CUDA_CALL(hipHostFree(static_cast<void *>(params.state.pinnedInts)));
  CUDA_CALL(hipHostFree(static_cast<void *>(params.state.pinnedDoubles)));

  CUDA_CALL(hipStreamDestroy(params.velocityStream));
  CUDA_CALL(hipStreamDestroy(params.gasStream));
}

double getSimulationBoxVolume(Params &params)
{
  dvec temp = params.state.tfr - params.state.lbb;
  return (NUM_DIM == 3) ? temp.x * temp.y * temp.z : temp.x * temp.y;
}

#define JSON_READ(i, j, arg) \
  i.arg = j[#arg];           \
  std::cout << #arg << ": " << i.arg << std::endl

void readInputs(SimulationInputs &inputs, const char *inputFileName)
{
  std::cout << "Reading inputs from file \"" << inputFileName << "\"" << std::endl;
  nlohmann::json j;
  std::fstream file(inputFileName, std::ios::in);

  if (file.is_open())
  {
    file >> j;

    JSON_READ(inputs, j, phiTarget);
    JSON_READ(inputs, j, muZero);
    JSON_READ(inputs, j, sigmaZero);
    JSON_READ(inputs, j, avgRad);
    JSON_READ(inputs, j, stdDevRad);
    JSON_READ(inputs, j, errorTolerance);
    JSON_READ(inputs, j, timeStepIn);
    JSON_READ(inputs, j, rngSeed);
    JSON_READ(inputs, j, numBubblesPerCell);
    JSON_READ(inputs, j, snapshotFilename);
    JSON_READ(inputs, j, numStepsToRelax);
    JSON_READ(inputs, j, maxDeltaEnergy);
    JSON_READ(inputs, j, kParameter);
    JSON_READ(inputs, j, numBubblesIn);
    JSON_READ(inputs, j, kappa);
    JSON_READ(inputs, j, minNumBubbles);
    JSON_READ(inputs, j, dataFilename);
    JSON_READ(inputs, j, boxRelDim);
    JSON_READ(inputs, j, flowLbb);
    JSON_READ(inputs, j, flowTfr);
    JSON_READ(inputs, j, flowVel);

    assert(inputs.muZero > 0);
    assert(inputs.boxRelDim.x > 0);
    assert(inputs.boxRelDim.y > 0);
    assert(inputs.boxRelDim.z > 0);

    inputs.fZeroPerMuZero    = inputs.sigmaZero * inputs.avgRad / inputs.muZero;
    inputs.minRad            = 0.1 * inputs.avgRad;
    inputs.timeScalingFactor = inputs.kParameter / (inputs.avgRad * inputs.avgRad);
    inputs.flowVel *= inputs.fZeroPerMuZero;
  }
  else
    throw std::runtime_error("Couldn't open input file!");

  // First calculate the size of the box and the starting number of bubbles
  dvec relDim        = params.inputs.boxRelDim;
  relDim             = relDim / relDim.x;
  const float d      = 2 * params.inputs.avgRad;
  float x            = params.inputs.numBubblesIn * d * d / relDim.y;
  ivec bubblesPerDim = ivec(0, 0, 0);

  if (NUM_DIM == 3)
  {
    x                       = x * d / relDim.z;
    x                       = std::cbrt(x);
    relDim                  = relDim * x;
    bubblesPerDim           = ivec(std::ceil(relDim.x / d), std::ceil(relDim.y / d), std::ceil(relDim.z / d));
    params.state.numBubbles = bubblesPerDim.x * bubblesPerDim.y * bubblesPerDim.z;
  }
  else
  {
    x                       = std::sqrt(x);
    relDim                  = relDim * x;
    bubblesPerDim           = ivec(std::ceil(relDim.x / d), std::ceil(relDim.y / d), 0);
    params.state.numBubbles = bubblesPerDim.x * bubblesPerDim.y;
  }

  params.defaultKernelSize = KernelSize(128, params.state.numBubbles);
  params.state.tfr         = d * bubblesPerDim.asType<double>() + params.state.lbb;
  params.state.interval    = params.state.tfr - params.state.lbb;
  params.inputs.flowTfr    = params.state.interval * params.inputs.flowTfr + params.state.lbb;
  params.inputs.flowLbb    = params.state.interval * params.inputs.flowLbb + params.state.lbb;
  params.state.timeStep    = params.inputs.timeStepIn;

  // Determine the maximum number of Morton numbers for the simulation box
  dim3 gridDim         = getGridSize(params);
  const int maxGridDim = gridDim.x > gridDim.y ? (gridDim.x > gridDim.z ? gridDim.x : gridDim.z)
                                               : (gridDim.y > gridDim.z ? gridDim.y : gridDim.z);
  int maxNumCells = 1;
  while (maxNumCells < maxGridDim)
    maxNumCells = maxNumCells << 1;

  if (NUM_DIM == 3)
    maxNumCells = maxNumCells * maxNumCells * maxNumCells;
  else
    maxNumCells = maxNumCells * maxNumCells;

  params.state.maxNumCells = maxNumCells;

  std::cout << "Maximum (theoretical) number of cells: " << params.state.maxNumCells
            << ", actual grid dimensions: " << gridDim.x << ", " << gridDim.y << ", " << gridDim.z << std::endl;
}
#undef JSON_READ

void commonSetup(Params &params)
{
  // Get some device global symbol addresses to host pointers.
  CUDA_ASSERT(hipGetSymbolAddress(reinterpret_cast<void **>(&params.state.dtfa), dTotalFreeArea));
  CUDA_ASSERT(hipGetSymbolAddress(reinterpret_cast<void **>(&params.state.dtfapr), dTotalFreeAreaPerRadius));
  CUDA_ASSERT(hipGetSymbolAddress(reinterpret_cast<void **>(&params.state.mbpc), dMaxBubblesPerCell));
  CUDA_ASSERT(hipGetSymbolAddress(reinterpret_cast<void **>(&params.state.dvm), dVolumeMultiplier));
  CUDA_ASSERT(hipGetSymbolAddress(reinterpret_cast<void **>(&params.state.dtv), dTotalVolume));
  CUDA_ASSERT(hipGetSymbolAddress(reinterpret_cast<void **>(&params.state.np), dNumPairs));
  CUDA_ASSERT(hipGetSymbolAddress(reinterpret_cast<void **>(&params.state.dir), dInvRho));
  CUDA_ASSERT(hipGetSymbolAddress(reinterpret_cast<void **>(&params.state.dta), dTotalArea));
  CUDA_ASSERT(hipGetSymbolAddress(reinterpret_cast<void **>(&params.state.dasai), dAverageSurfaceAreaIn));

  // Streams
  CUDA_ASSERT(hipStreamCreate(&params.velocityStream));
  CUDA_ASSERT(hipStreamCreate(&params.gasStream));

  printRelevantInfoOfCurrentDevice();

  std::cout << "Reserving device memory to hold data." << std::endl;

  // Reserve pinned memory
  CUDA_ASSERT(hipHostMalloc(reinterpret_cast<void **>(&params.state.pinnedDoubles), 3 * sizeof(double)));
  CUDA_ASSERT(hipHostMalloc(reinterpret_cast<void **>(&params.state.pinnedInts), 1 * sizeof(int)));

  // Calculate the length of 'rows'. Will be divisible by 32, as that's the warp size.
  params.state.dataStride =
    params.state.numBubbles + !!(params.state.numBubbles % 32) * (32 - params.state.numBubbles % 32);

  // Doubles
  params.state.memReqD = sizeof(double) * (uint64_t)params.state.dataStride * (uint64_t)DDP::NUM_VALUES;
  CUDA_ASSERT(hipMalloc(reinterpret_cast<void **>(&params.state.deviceDoubles), params.state.memReqD));

  for (uint32_t i = 0; i < (uint32_t)DDP::NUM_VALUES; ++i)
    params.state.ddps[i] = params.state.deviceDoubles + i * params.state.dataStride;

  // Integers
  // 32 is just a guess, and roughly it seems to hold true with 3D sim.
  const uint32_t avgNumNeighbors = 32;
  params.state.pairStride        = avgNumNeighbors * params.state.dataStride;

  params.state.memReqI = sizeof(int) * (uint64_t)params.state.dataStride *
                         ((uint64_t)DIP::PAIR1 + avgNumNeighbors * ((uint64_t)DIP::NUM_VALUES - (uint64_t)DIP::PAIR1));
  CUDA_ASSERT(hipMalloc(reinterpret_cast<void **>(&params.state.deviceInts), params.state.memReqI));

  for (uint32_t i = 0; i < (uint32_t)DIP::PAIR2; ++i)
    params.state.dips[i] = params.state.deviceInts + i * params.state.dataStride;

  uint32_t j = 0;
  for (uint32_t i = (uint32_t)DIP::PAIR2; i < (uint32_t)DIP::NUM_VALUES; ++i)
    params.state.dips[i] = params.state.dips[(uint32_t)DIP::PAIR1] + avgNumNeighbors * ++j * params.state.dataStride;

  std::cout << "Memory requirement for data:\n\tdouble: " << params.state.memReqD
            << " bytes\n\tint: " << params.state.memReqI
            << " bytes\ntotal: " << params.state.memReqI + params.state.memReqD << " bytes" << std::endl;
}

void generateStartingData(Params &params)
{
  std::cout << "Starting to generate data for bubbles." << std::endl;
  const int rngSeed      = params.inputs.rngSeed;
  const double avgRad    = params.inputs.avgRad;
  const double stdDevRad = params.inputs.stdDevRad;

  hiprandGenerator_t generator;
  CURAND_CALL(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_MTGP32));
  CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(generator, rngSeed));
  if (NUM_DIM == 3)
    CURAND_CALL(hiprandGenerateUniformDouble(generator, params.state.ddps[(uint32_t)DDP::Z], params.state.numBubbles));
  CURAND_CALL(hiprandGenerateUniformDouble(generator, params.state.ddps[(uint32_t)DDP::X], params.state.numBubbles));
  CURAND_CALL(hiprandGenerateUniformDouble(generator, params.state.ddps[(uint32_t)DDP::Y], params.state.numBubbles));
  CURAND_CALL(hiprandGenerateUniformDouble(generator, params.state.ddps[(uint32_t)DDP::RP], params.state.numBubbles));
  CURAND_CALL(hiprandGenerateNormalDouble(generator, params.state.ddps[(uint32_t)DDP::R], params.state.numBubbles,
                                         avgRad, stdDevRad));
  CURAND_CALL(hiprandDestroyGenerator(generator));

  KERNEL_LAUNCH(assignDataToBubbles, params.defaultKernelSize, 0, 0, params.state.ddps[(uint32_t)DDP::X],
                params.state.ddps[(uint32_t)DDP::Y], params.state.ddps[(uint32_t)DDP::Z],
                params.state.ddps[(uint32_t)DDP::XP], params.state.ddps[(uint32_t)DDP::YP],
                params.state.ddps[(uint32_t)DDP::ZP], params.state.ddps[(uint32_t)DDP::R],
                params.state.ddps[(uint32_t)DDP::RP], params.state.dips[(uint32_t)DIP::FLAGS], bubblesPerDim,
                params.state.tfr, params.state.lbb, avgRad, params.inputs.minRad, params.state.numBubbles);

  params.cw.reduceNoCopy<double, double *, double *>(&hipcub::DeviceReduce::Sum, params.state.ddps[(uint32_t)DDP::RP],
                                                     params.state.dasai, params.state.numBubbles, 0);

  CUDA_CALL(hipMemcpyAsync(static_cast<void *>(params.state.ddps[(uint32_t)DDP::RP]),
                            static_cast<void *>(params.state.ddps[(uint32_t)DDP::R]),
                            sizeof(double) * params.state.dataStride, hipMemcpyDeviceToDevice, 0));

  std::cout << "Deleting small bubbles and updating neighbor lists." << std::endl;
  const int numBubblesAboveMinRad = params.cw.reduce<int, int *, int *>(
    &hipcub::DeviceReduce::Sum, params.state.dips[(uint32_t)DIP::FLAGS], params.state.numBubbles);
  if (numBubblesAboveMinRad < params.state.numBubbles)
    deleteSmallBubbles(params, numBubblesAboveMinRad);

  params.state.maxBubbleRadius = params.cw.reduce<double, double *, double *>(
    &hipcub::DeviceReduce::Max, params.state.ddps[(uint32_t)DDP::R], params.state.numBubbles);

  updateCellsAndNeighbors(params);

  // Calculate some initial values which are needed
  // for the two-step Adams-Bashforth-Moulton prEdictor-corrector method
  KERNEL_LAUNCH(resetKernel, params.defaultKernelSize, 0, 0, 0.0, params.state.numBubbles,
                params.state.ddps[(uint32_t)DDP::DXDTO], params.state.ddps[(uint32_t)DDP::DYDTO],
                params.state.ddps[(uint32_t)DDP::DZDTO], params.state.ddps[(uint32_t)DDP::DRDTO],
                params.state.ddps[(uint32_t)DDP::DISTANCE], params.state.ddps[(uint32_t)DDP::PATH]);

  std::cout << "Calculating some initial values as a part of setup." << std::endl;
  if (NUM_DIM == 3)
  {
    KERNEL_LAUNCH(velocityPairKernel, params.pairKernelSize, 0, 0, params.inputs.fZeroPerMuZero,
                  params.state.dips[(uint32_t)DIP::PAIR1], params.state.dips[(uint32_t)DIP::PAIR2],
                  params.state.ddps[(uint32_t)DDP::R], params.state.interval.x, params.state.lbb.x, PBC_X == 1,
                  params.state.ddps[(uint32_t)DDP::X], params.state.ddps[(uint32_t)DDP::DXDTO], params.state.interval.y,
                  params.state.lbb.y, PBC_Y == 1, params.state.ddps[(uint32_t)DDP::Y],
                  params.state.ddps[(uint32_t)DDP::DYDTO], params.state.interval.z, params.state.lbb.z, PBC_Z == 1,
                  params.state.ddps[(uint32_t)DDP::Z], params.state.ddps[(uint32_t)DDP::DZDTO]);

    KERNEL_LAUNCH(eulerKernel, params.defaultKernelSize, 0, 0, params.state.numBubbles, params.state.timeStep,
                  params.state.ddps[(uint32_t)DDP::X], params.state.ddps[(uint32_t)DDP::DXDTO],
                  params.state.ddps[(uint32_t)DDP::Y], params.state.ddps[(uint32_t)DDP::DYDTO],
                  params.state.ddps[(uint32_t)DDP::Z], params.state.ddps[(uint32_t)DDP::DZDTO]);

    doBoundaryWrap(params.defaultKernelSize, 0, 0, PBC_X == 1, PBC_Y == 1, PBC_Z == 1, params.state.numBubbles,
                   params.state.ddps[(uint32_t)DDP::XP], params.state.ddps[(uint32_t)DDP::YP],
                   params.state.ddps[(uint32_t)DDP::ZP], params.state.lbb, params.state.tfr,
                   params.state.dips[(uint32_t)DIP::WRAP_COUNT_X], params.state.dips[(uint32_t)DIP::WRAP_COUNT_Y],
                   params.state.dips[(uint32_t)DIP::WRAP_COUNT_Z], params.state.dips[(uint32_t)DIP::WRAP_COUNT_XP],
                   params.state.dips[(uint32_t)DIP::WRAP_COUNT_YP], params.state.dips[(uint32_t)DIP::WRAP_COUNT_ZP]);

    KERNEL_LAUNCH(resetKernel, params.defaultKernelSize, 0, 0, 0.0, params.state.numBubbles,
                  params.state.ddps[(uint32_t)DDP::DXDTO], params.state.ddps[(uint32_t)DDP::DYDTO],
                  params.state.ddps[(uint32_t)DDP::DZDTO], params.state.ddps[(uint32_t)DDP::DRDTO]);

    KERNEL_LAUNCH(velocityPairKernel, params.pairKernelSize, 0, 0, params.inputs.fZeroPerMuZero,
                  params.state.dips[(uint32_t)DIP::PAIR1], params.state.dips[(uint32_t)DIP::PAIR2],
                  params.state.ddps[(uint32_t)DDP::R], params.state.interval.x, params.state.lbb.x, PBC_X == 1,
                  params.state.ddps[(uint32_t)DDP::X], params.state.ddps[(uint32_t)DDP::DXDTO], params.state.interval.y,
                  params.state.lbb.y, PBC_Y == 1, params.state.ddps[(uint32_t)DDP::Y],
                  params.state.ddps[(uint32_t)DDP::DYDTO], params.state.interval.z, params.state.lbb.z, PBC_Z == 1,
                  params.state.ddps[(uint32_t)DDP::Z], params.state.ddps[(uint32_t)DDP::DZDTO]);
  }
  else
  {
    KERNEL_LAUNCH(velocityPairKernel, params.pairKernelSize, 0, 0, params.inputs.fZeroPerMuZero,
                  params.state.dips[(uint32_t)DIP::PAIR1], params.state.dips[(uint32_t)DIP::PAIR2],
                  params.state.ddps[(uint32_t)DDP::R], params.state.interval.x, params.state.lbb.x, PBC_X == 1,
                  params.state.ddps[(uint32_t)DDP::X], params.state.ddps[(uint32_t)DDP::DXDTO], params.state.interval.y,
                  params.state.lbb.y, PBC_Y == 1, params.state.ddps[(uint32_t)DDP::Y],
                  params.state.ddps[(uint32_t)DDP::DYDTO]);

    KERNEL_LAUNCH(eulerKernel, params.defaultKernelSize, 0, 0, params.state.numBubbles, params.state.timeStep,
                  params.state.ddps[(uint32_t)DDP::X], params.state.ddps[(uint32_t)DDP::DXDTO],
                  params.state.ddps[(uint32_t)DDP::Y], params.state.ddps[(uint32_t)DDP::DYDTO]);

    doBoundaryWrap(params.defaultKernelSize, 0, 0, PBC_X == 1, PBC_Y == 1, false, params.state.numBubbles,
                   params.state.ddps[(uint32_t)DDP::XP], params.state.ddps[(uint32_t)DDP::YP],
                   params.state.ddps[(uint32_t)DDP::ZP], params.state.lbb, params.state.tfr,
                   params.state.dips[(uint32_t)DIP::WRAP_COUNT_X], params.state.dips[(uint32_t)DIP::WRAP_COUNT_Y],
                   params.state.dips[(uint32_t)DIP::WRAP_COUNT_Z], params.state.dips[(uint32_t)DIP::WRAP_COUNT_XP],
                   params.state.dips[(uint32_t)DIP::WRAP_COUNT_YP], params.state.dips[(uint32_t)DIP::WRAP_COUNT_ZP]);

    KERNEL_LAUNCH(resetKernel, params.defaultKernelSize, 0, 0, 0.0, params.state.numBubbles,
                  params.state.ddps[(uint32_t)DDP::DXDTO], params.state.ddps[(uint32_t)DDP::DYDTO],
                  params.state.ddps[(uint32_t)DDP::DRDTO]);

    KERNEL_LAUNCH(velocityPairKernel, params.pairKernelSize, 0, 0, params.inputs.fZeroPerMuZero,
                  params.state.dips[(uint32_t)DIP::PAIR1], params.state.dips[(uint32_t)DIP::PAIR2],
                  params.state.ddps[(uint32_t)DDP::R], params.state.interval.x, params.state.lbb.x, PBC_X == 1,
                  params.state.ddps[(uint32_t)DDP::X], params.state.ddps[(uint32_t)DDP::DXDTO], params.state.interval.y,
                  params.state.lbb.y, PBC_Y == 1, params.state.ddps[(uint32_t)DDP::Y],
                  params.state.ddps[(uint32_t)DDP::DYDTO]);
  }
}

void deserializeState(Params &params, const char *inputFileName)
{
  // Deserialize the saved state of the simulation.
}

void deserializeData(Params &params, const char *inputFileName)
{
  // Deserialize the data from the binary and copy it to the device.
  // When serializing data, already 'remove' the useless bits at the end of each 'row'.
}

void initializeFromJson(const char *inputFileName, Params &params, std::stringstream &dataStream)
{
  // Initialize everything, starting with an input .json file.
  // The end state of this function is 'prepared state' that can then be used immediately to run the integration loop.

  std::cout << "\n=====\nSetup\n=====" << std::endl;
  readInputs(params.inputs, inputFileName);
  commonSetup(params);
  generateStartingData(params);
  saveSnapshotToFile(params); // 0

  std::cout << "Letting bubbles settle after they've been created and before "
               "scaling or stabilization."
            << std::endl;

  stabilize(params);
  saveSnapshotToFile(params); // 1

  const double bubbleVolume = calculateVolumeOfBubbles(params);

  std::cout << "Volume ratios: current: " << bubbleVolume / getSimulationBoxVolume(params)
            << ", target: " << params.inputs.phiTarget << "\nScaling the simulation box." << std::endl;

  transformPositions(params, true);

  dvec relativeSize     = params.inputs.boxRelDim;
  relativeSize.z        = (NUM_DIM == 2) ? 1 : relativeSize.z;
  double t              = bubbleVolume / (params.inputs.phiTarget * relativeSize.x * relativeSize.y * relativeSize.z);
  t                     = (NUM_DIM == 3) ? std::cbrt(t) : std::sqrt(t);
  params.state.tfr      = dvec(t, t, t) * relativeSize;
  params.state.interval = params.state.tfr - params.state.lbb;

  transformPositions(params, false);
  saveSnapshotToFile(params); // 2

  std::cout << "Volume ratios: current: " << bubbleVolume / getSimulationBoxVolume(params)
            << ", target: " << params.inputs.phiTarget
            << "\n\n=============\nStabilization\n=============" << std::endl;

  int numSteps       = 0;
  const int failsafe = 500;
  std::cout << "#steps\tdE/t\te1\te2" << std::endl;
  while (true)
  {
    double time        = stabilize(params);
    double deltaEnergy = std::abs(params.state.energy2 - params.state.energy1) / time;
    deltaEnergy *= 0.5 * params.inputs.sigmaZero;

    if (deltaEnergy < params.inputs.maxDeltaEnergy)
    {
      std::cout << "Final delta energy " << deltaEnergy << " after " << (numSteps + 1) * params.inputs.numStepsToRelax
                << " steps."
                << " Energy before: " << params.state.energy1 << ", energy after: " << params.state.energy2
                << ", time: " << time * params.inputs.timeScalingFactor << std::endl;
      break;
    }
    else if (numSteps > failsafe)
    {
      std::cout << "Over " << failsafe * params.inputs.numStepsToRelax
                << " steps taken and required delta energy not reached."
                << " Check parameters." << std::endl;
      break;
    }
    else
    {
      std::cout << (numSteps + 1) * params.inputs.numStepsToRelax << "\t" << deltaEnergy << "\t" << params.state.energy1
                << "\t" << params.state.energy2 << std::endl;
    }

    ++numSteps;
  }

  saveSnapshotToFile(params); // 3

  // Set starting positions and reset wrap counts to 0
  const size_t numBytesToCopy = 3 * sizeof(double) * params.state.dataStride;
  CUDA_CALL(hipMemcpy(params.state.ddps[(uint32_t)DDP::X0], params.state.ddps[(uint32_t)DDP::X], numBytesToCopy,
                       hipMemcpyDeviceToDevice));
  CUDA_CALL(hipMemset(params.state.dips[(uint32_t)DIP::WRAP_COUNT_X], 0, 6 * params.state.dataStride * sizeof(int)));

  // Calculate the energy at starting positions
  KERNEL_LAUNCH(resetKernel, params.defaultKernelSize, 0, 0, 0.0, params.state.numBubbles,
                params.state.ddps[(uint32_t)DDP::TEMP4]);

  if (NUM_DIM == 3)
  {
    KERNEL_LAUNCH(potentialEnergyKernel, params.pairKernelSize, 0, 0, params.state.numBubbles,
                  params.state.dips[(uint32_t)DIP::PAIR1], params.state.dips[(uint32_t)DIP::PAIR2],
                  params.state.ddps[(uint32_t)DDP::R], params.state.ddps[(uint32_t)DDP::TEMP4], params.state.interval.x,
                  PBC_X == 1, params.state.ddps[(uint32_t)DDP::X], params.state.interval.y, PBC_Y == 1,
                  params.state.ddps[(uint32_t)DDP::Y], params.state.interval.z, PBC_Z == 1,
                  params.state.ddps[(uint32_t)DDP::Z]);
  }
  else
  {
    KERNEL_LAUNCH(potentialEnergyKernel, params.pairKernelSize, 0, 0, params.state.numBubbles,
                  params.state.dips[(uint32_t)DIP::PAIR1], params.state.dips[(uint32_t)DIP::PAIR2],
                  params.state.ddps[(uint32_t)DDP::R], params.state.ddps[(uint32_t)DDP::TEMP4], params.state.interval.x,
                  PBC_X == 1, params.state.ddps[(uint32_t)DDP::X], params.state.interval.y, PBC_Y == 1,
                  params.state.ddps[(uint32_t)DDP::Y]);
  }

  params.state.energy1 = params.cw.reduce<double, double *, double *>(
    &hipcub::DeviceReduce::Sum, params.state.ddps[(uint32_t)DDP::TEMP4], params.state.numBubbles);
  params.state.simulationTime      = 0.0;
  params.state.timesPrinted        = 1;
  params.state.numIntegrationSteps = 0;
}

void initializeFromBinary(const char *inputFileName, Params &params, std::stringstream &dataStream)
{
  // This function initializes the simulation state from a binary dump.
  // The end state of this function is 'prepared state' that can then be used immediately to run the integration loop.
  deserializeState(params, inputFileName);
  commonSetup(params);
  deserializeData(params, oldDataStride, oldPairStride);
}

} // namespace

namespace cubble
{
void run(const char *inputFileName)
{
  Params params;
  std::stringstream dataStream;

  initializeFromJson(inputFileName, params, dataStream);
  // initializeFromBinary(inputFileName, params, dataStream);

  std::cout << "\n==========\nIntegration\n==========" << std::endl;
  bool continueIntegration = true;
  std::cout << "T\tphi\tR\t#b\tdE\t\t#steps\t#pairs" << std::endl;
  while (continueIntegration)
  {
    continueIntegration = integrate(params);
    CUDA_PROFILER_START(params.state.numIntegrationSteps == 2000);
    CUDA_PROFILER_STOP(params.state.numIntegrationSteps == 2200, continueIntegration);

    const double scaledTime = params.state.simulationTime * params.inputs.timeScalingFactor;
    if ((int)scaledTime >= params.state.timesPrinted)
    {
      // Calculate total energy
      KERNEL_LAUNCH(resetKernel, params.defaultKernelSize, 0, 0, 0.0, params.state.numBubbles,
                    params.state.ddps[(uint32_t)DDP::TEMP4]);

      if (NUM_DIM == 3)
        KERNEL_LAUNCH(potentialEnergyKernel, params.pairKernelSize, 0, 0, params.state.numBubbles,
                      params.state.dips[(uint32_t)DIP::PAIR1], params.state.dips[(uint32_t)DIP::PAIR2],
                      params.state.ddps[(uint32_t)DDP::R], params.state.ddps[(uint32_t)DDP::TEMP4],
                      params.state.interval.x, PBC_X == 1, params.state.ddps[(uint32_t)DDP::X], params.state.interval.y,
                      PBC_Y == 1, params.state.ddps[(uint32_t)DDP::Y], params.state.interval.z, PBC_Z == 1,
                      params.state.ddps[(uint32_t)DDP::Z]);
      else
        KERNEL_LAUNCH(potentialEnergyKernel, params.pairKernelSize, 0, 0, params.state.numBubbles,
                      params.state.dips[(uint32_t)DIP::PAIR1], params.state.dips[(uint32_t)DIP::PAIR2],
                      params.state.ddps[(uint32_t)DDP::R], params.state.ddps[(uint32_t)DDP::TEMP4],
                      params.state.interval.x, PBC_X == 1, params.state.ddps[(uint32_t)DDP::X], params.state.interval.y,
                      PBC_Y == 1, params.state.ddps[(uint32_t)DDP::Y]);

      auto getSum = [](double *p, Params &params) -> double {
        return params.cw.reduce<double, double *, double *>(&hipcub::DeviceReduce::Sum, p, params.state.numBubbles);
      };

      auto getAvg = [getSum](double *p, Params &params) -> double {
        return getSum(p, params) / params.state.numBubbles;
      };

      params.state.energy2        = getSum(params.state.ddps[(uint32_t)DDP::TEMP4], params);
      const double dE             = (params.state.energy2 - params.state.energy1) / params.state.energy2;
      const double relativeRadius = getAvg(params.state.ddps[(uint32_t)DDP::R], params) / params.inputs.avgRad;

      // Add values to data stream
      dataStream << (int)scaledTime << " " << relativeRadius << " "
                 << params.state.maxBubbleRadius / params.inputs.avgRad << " " << params.state.numBubbles << " "
                 << getAvg(params.state.ddps[(uint32_t)DDP::PATH], params) << " "
                 << getAvg(params.state.ddps[(uint32_t)DDP::DISTANCE], params) << " " << dE << "\n";

      // Print some values
      std::cout << (int)scaledTime << "\t" << calculateVolumeOfBubbles(params) / getSimulationBoxVolume(params) << "\t"
                << relativeRadius << "\t" << params.state.numBubbles << "\t" << dE << "\t"
                << params.state.numStepsInTimeStep << "\t" << params.state.numPairs << std::endl;

      // Only write snapshots when t* is a power of 2.
      if ((params.state.timesPrinted & (params.state.timesPrinted - 1)) == 0)
        saveSnapshotToFile(params);

      ++params.state.timesPrinted;
      params.state.numStepsInTimeStep = 0;
      params.state.energy1            = params.state.energy2;
    }

    ++params.state.numStepsInTimeStep;
  }

  // Append when continued
  std::ofstream file(params.inputs.dataFilename);
  file << dataStream.str() << std::endl;

  deinit(params);
}
} // namespace cubble
