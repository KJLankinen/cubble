#include "hip/hip_runtime.h"
// -*- C++ -*-

#include "Simulator.cuh"
#include "Macros.h"
#include "Vec.h"
#include "BubbleKernels.cuh"
#include "UtilityKernels.cuh"
#include "IntegrationKernels.cuh"

#include "cub.cuh"
#include <iostream>
#include <sstream>
#include <chrono>
#include <algorithm>
#include <vector>
#include <hiprand.h>
#include <hip/hip_runtime_api.h>
#include <nvToolsExt.h>

namespace cubble
{
typedef BubbleProperty BP;
Simulator::Simulator(std::shared_ptr<Env> e)
{
    env = e;
    fpvec relDim = env->getBoxRelativeDimensions();
    relDim /= relDim.x;
    const CubbleFloatType d = 2 * env->getAvgRad();
#if (NUM_DIM == 3)
    const CubbleFloatType x = std::cbrt(env->getNumBubbles() * d * d * d / (relDim.y * relDim.z));
    fpvec tfr = relDim * x;
    const ivec bubblesPerDim(std::ceil(tfr.x / d), std::ceil(tfr.y / d), std::ceil(tfr.z / d));
    numBubbles = bubblesPerDim.x * bubblesPerDim.y * bubblesPerDim.z;
#else
    const CubbleFloatType x = std::sqrt(env->getNumBubbles() * d * d / relDim.y);
    fpvec tfr = relDim * x;
    tfr.z = 0;
    const ivec bubblesPerDim(std::ceil(tfr.x / d), std::ceil(tfr.y / d), 0);
    numBubbles = bubblesPerDim.x * bubblesPerDim.y;
#endif
    bubblesPerDimAtStart = bubblesPerDim;
    tfr = d * bubblesPerDim.asType<CubbleFloatType>();
    env->setTfr(tfr + env->getLbb());

    cubWrapper = std::make_shared<CubWrapper>(env, numBubbles);
    bubbleData = DeviceArray<CubbleFloatType>(numBubbles, (size_t)BP::NUM_VALUES);
    aboveMinRadFlags = DeviceArray<int>(numBubbles, 2);
    bubbleCellIndices = DeviceArray<int>(numBubbles, 4);
    pairs = DeviceArray<int>(8 * numBubbles, 4);

    const dim3 gridSize = getGridSize();
    size_t numCells = gridSize.x * gridSize.y * gridSize.z;
    cellData = DeviceArray<int>(numCells, (size_t)CellProperty::NUM_VALUES);

    hostData.resize(bubbleData.getSize(), 0);

    CUDA_CALL(hipGetSymbolAddress((void **)&dtfa, dTotalFreeArea));
    assert(dtfa != nullptr);
    CUDA_CALL(hipGetSymbolAddress((void **)&dtfapr, dTotalFreeAreaPerRadius));
    assert(dtfapr != nullptr);
    CUDA_CALL(hipGetSymbolAddress((void **)&mbpc, dMaxBubblesPerCell));
    assert(mbpc != nullptr);
    CUDA_CALL(hipGetSymbolAddress((void **)&dvm, dVolumeMultiplier));
    assert(dvm != nullptr);
    CUDA_CALL(hipGetSymbolAddress((void **)&dtv, dTotalVolume));
    assert(dtv != nullptr);
    CUDA_CALL(hipGetSymbolAddress((void **)&np, dNumPairs));
    assert(np != nullptr);
    CUDA_CALL(hipGetSymbolAddress((void **)&dir, dInvRho));
    assert(dir != nullptr);
    CUDA_CALL(hipGetSymbolAddress((void **)&dta, dTotalArea));
    assert(dta != nullptr);
    CUDA_CALL(hipGetSymbolAddress((void **)&dasai, dAverageSurfaceAreaIn));
    assert(dasai != nullptr);

    CUDA_CALL(hipStreamCreateWithFlags(&nonBlockingStream1, hipStreamNonBlocking));
    CUDA_CALL(hipStreamCreateWithFlags(&nonBlockingStream2, hipStreamNonBlocking));
    CUDA_CALL(hipEventCreateWithFlags(&blockingEvent1, hipEventBlockingSync));
    CUDA_CALL(hipEventCreateWithFlags(&blockingEvent2, hipEventBlockingSync));

    for (size_t i = 0; i < CUBBLE_NUM_NEIGHBORS + 1; ++i)
    {
        neighborStreamVec.emplace_back();
        neighborEventVec.emplace_back();
        CUDA_CALL(hipStreamCreateWithFlags(&neighborStreamVec[i], hipStreamNonBlocking));
        CUDA_CALL(hipEventCreate(&neighborEventVec[i]));
    }

    pinnedInt = PinnedHostArray<int>(1);
    pinnedDouble = PinnedHostArray<CubbleFloatType>(1);

    printRelevantInfoOfCurrentDevice();
}

Simulator::~Simulator()
{
    CUDA_CALL(hipStreamDestroy(nonBlockingStream1));
    CUDA_CALL(hipStreamDestroy(nonBlockingStream2));
    CUDA_CALL(hipEventDestroy(blockingEvent1));
    CUDA_CALL(hipEventDestroy(blockingEvent2));

    for (size_t i = 0; i < CUBBLE_NUM_NEIGHBORS + 1; ++i)
    {
        CUDA_CALL(hipStreamDestroy(neighborStreamVec[i]));
        CUDA_CALL(hipEventDestroy(neighborEventVec[i]));
    }
}

void Simulator::setupSimulation()
{
    generateBubbles();

    const int numBubblesAboveMinRad = cubWrapper->reduce<int, int *, int *>(&hipcub::DeviceReduce::Sum, aboveMinRadFlags.getRowPtr(0), numBubbles);
    if (numBubblesAboveMinRad < numBubbles)
        deleteSmallBubbles(numBubblesAboveMinRad);

    updateCellsAndNeighbors();

    // Calculate some initial values which are needed
    // for the two-step Adams-Bashforth-Moulton perdictor-corrector method (ABMpc).

    CubbleFloatType *x = bubbleData.getRowPtr((size_t)BP::X);
    CubbleFloatType *y = bubbleData.getRowPtr((size_t)BP::Y);
    CubbleFloatType *z = bubbleData.getRowPtr((size_t)BP::Z);
    CubbleFloatType *r = bubbleData.getRowPtr((size_t)BP::R);

    CubbleFloatType *dxdt = bubbleData.getRowPtr((size_t)BP::DXDT);
    CubbleFloatType *dydt = bubbleData.getRowPtr((size_t)BP::DYDT);
    CubbleFloatType *dzdt = bubbleData.getRowPtr((size_t)BP::DZDT);
    CubbleFloatType *drdt = bubbleData.getRowPtr((size_t)BP::DRDT);

    CubbleFloatType *dxdtOld = bubbleData.getRowPtr((size_t)BP::DXDT_OLD);
    CubbleFloatType *dydtOld = bubbleData.getRowPtr((size_t)BP::DYDT_OLD);
    CubbleFloatType *dzdtOld = bubbleData.getRowPtr((size_t)BP::DZDT_OLD);
    CubbleFloatType *drdtOld = bubbleData.getRowPtr((size_t)BP::DRDT_OLD);

    CubbleFloatType *energies = bubbleData.getRowPtr((size_t)BP::ENERGY);
    CubbleFloatType *freeArea = bubbleData.getRowPtr((size_t)BP::FREE_AREA);

    const fpvec tfr = env->getTfr();
    const fpvec lbb = env->getLbb();
    const fpvec interval = tfr - lbb;
    ExecutionPolicy defaultPolicy(128, numBubbles);
    ExecutionPolicy pairPolicy;
    pairPolicy.blockSize = dim3(128, 1, 1);
    pairPolicy.stream = 0;
    pairPolicy.gridSize = dim3(256, 1, 1);
    pairPolicy.sharedMemBytes = 0;

    CubbleFloatType timeStep = env->getTimeStep();

    CUDA_LAUNCH(resetKernel, defaultPolicy,
                (CubbleFloatType)0.0, numBubbles,
                dxdtOld, dydtOld, dzdtOld, drdtOld);

    std::cout << "Calculating some initial values as a part of setup." << std::endl;

    CUDA_LAUNCH(velocityPairKernel, pairPolicy,
                env->getFZeroPerMuZero(), pairs.getRowPtr(0), pairs.getRowPtr(1), r,
                interval.x, lbb.x, PBC_X == 1, x, dxdtOld,
                interval.y, lbb.y, PBC_Y == 1, y, dydtOld
#if (NUM_DIM == 3)
                ,
                interval.z, lbb.z, PBC_Z == 1, z, dzdtOld
#endif
    );

    CUDA_LAUNCH(eulerKernel, defaultPolicy,
                numBubbles, timeStep,
                x, dxdtOld,
                y, dydtOld
#if (NUM_DIM == 3)
                ,
                z, dzdtOld
#endif
    );

#if (PBC_X == 1 || PBC_Y == 1 || PBC_Z == 1)
    CUDA_LAUNCH(boundaryWrapKernel, defaultPolicy,
                numBubbles
#if (PBC_X == 1)
                ,
                x, lbb.x, tfr.x
#endif
#if (PBC_Y == 1)
                ,
                y, lbb.y, tfr.y
#endif
#if (PBC_Z == 1 && NUM_DIM == 3)
                ,
                z, lbb.z, tfr.z
#endif
    );
#endif

    CUDA_LAUNCH(resetKernel, defaultPolicy,
                (CubbleFloatType)0.0, numBubbles,
                dxdtOld, dydtOld, dzdtOld, drdtOld);

    CUDA_LAUNCH(velocityPairKernel, pairPolicy,
                env->getFZeroPerMuZero(), pairs.getRowPtr(0), pairs.getRowPtr(1), r,
                interval.x, lbb.x, PBC_X == 1, x, dxdtOld,
                interval.y, lbb.y, PBC_Y == 1, y, dydtOld
#if (NUM_DIM == 3)
                ,
                interval.z, lbb.z, PBC_Z == 1, z, dzdtOld
#endif
    );
}

bool Simulator::integrate(bool useGasExchange)
{
    ExecutionPolicy defaultPolicy(128, numBubbles);

    ExecutionPolicy pairPolicy;
    pairPolicy.blockSize = dim3(128, 1, 1);
    pairPolicy.stream = 0;
    pairPolicy.gridSize = dim3(256, 1, 1);
    pairPolicy.sharedMemBytes = 0;

    CubbleFloatType timeStep = env->getTimeStep();
    CubbleFloatType error = 100000;
    size_t numLoopsDone = 0;

    do
    {
        NVTX_RANGE_PUSH_A("Integration step");

        doReset(defaultPolicy);
        doPrediction(defaultPolicy, timeStep, useGasExchange, blockingEvent2);
        doVelocity(pairPolicy);
        if (useGasExchange)
            doGasExchange(pairPolicy, blockingEvent2, pairPolicy.stream);
        doCorrection(defaultPolicy, timeStep, useGasExchange, nonBlockingStream2);

        error = doError();
        if (error < env->getErrorTolerance() && timeStep < 0.1)
            timeStep *= 1.9;
        else if (error > env->getErrorTolerance())
            timeStep *= 0.5;

        ++numLoopsDone;

        NVTX_RANGE_POP();
    } while (error > env->getErrorTolerance());

    doBoundaryWrap(defaultPolicy);
    doBubbleSizeChecks(defaultPolicy, nonBlockingStream1, blockingEvent1);
    updateData();

    ++integrationStep;
    env->setTimeStep(timeStep);
    SimulationTime += timeStep;

    CUDA_CALL(hipEventSynchronize(blockingEvent1));

    const int numBubblesAboveMinRad = pinnedInt.get()[0];
    const bool shouldDeleteBubbles = numBubblesAboveMinRad < numBubbles;

    if (shouldDeleteBubbles)
        deleteSmallBubbles(numBubblesAboveMinRad);

    if (shouldDeleteBubbles || integrationStep % 50 == 0)
        updateCellsAndNeighbors();

    bool continueSimulation = numBubbles > env->getMinNumBubbles();

    maxBubbleRadius = pinnedDouble.get()[0];
#if (NUM_DIM == 3)
    continueSimulation &= maxBubbleRadius < 0.5 * (env->getTfr() - env->getLbb()).getMinComponent();
#endif

    return continueSimulation;
}

void Simulator::doPrediction(const ExecutionPolicy &policy, CubbleFloatType timeStep, bool useGasExchange, hipEvent_t &eventToMark)
{
    CubbleFloatType *x = bubbleData.getRowPtr((size_t)BP::X);
    CubbleFloatType *y = bubbleData.getRowPtr((size_t)BP::Y);
    CubbleFloatType *z = bubbleData.getRowPtr((size_t)BP::Z);
    CubbleFloatType *r = bubbleData.getRowPtr((size_t)BP::R);

    CubbleFloatType *xPrd = bubbleData.getRowPtr((size_t)BP::X_PRD);
    CubbleFloatType *yPrd = bubbleData.getRowPtr((size_t)BP::Y_PRD);
    CubbleFloatType *zPrd = bubbleData.getRowPtr((size_t)BP::Z_PRD);
    CubbleFloatType *rPrd = bubbleData.getRowPtr((size_t)BP::R_PRD);

    CubbleFloatType *dxdt = bubbleData.getRowPtr((size_t)BP::DXDT);
    CubbleFloatType *dydt = bubbleData.getRowPtr((size_t)BP::DYDT);
    CubbleFloatType *dzdt = bubbleData.getRowPtr((size_t)BP::DZDT);
    CubbleFloatType *drdt = bubbleData.getRowPtr((size_t)BP::DRDT);

    CubbleFloatType *dxdtOld = bubbleData.getRowPtr((size_t)BP::DXDT_OLD);
    CubbleFloatType *dydtOld = bubbleData.getRowPtr((size_t)BP::DYDT_OLD);
    CubbleFloatType *dzdtOld = bubbleData.getRowPtr((size_t)BP::DZDT_OLD);
    CubbleFloatType *drdtOld = bubbleData.getRowPtr((size_t)BP::DRDT_OLD);

    if (useGasExchange)
    {
        CUDA_LAUNCH(predictKernel, policy,
                    numBubbles, timeStep,
                    xPrd, x, dxdt, dxdtOld,
                    yPrd, y, dydt, dydtOld,
#if (NUM_DIM == 3)
                    zPrd, z, dzdt, dzdtOld,
#endif
                    rPrd, r, drdt, drdtOld);
    }
    else
    {
        CUDA_LAUNCH(predictKernel, policy,
                    numBubbles, timeStep,
                    xPrd, x, dxdt, dxdtOld,
                    yPrd, y, dydt, dydtOld
#if (NUM_DIM == 3)
                    ,
                    zPrd, z, dzdt, dzdtOld
#endif
        );
    }

    CUDA_CALL(hipEventRecord(eventToMark, policy.stream));
}

void Simulator::doCorrection(const ExecutionPolicy &policy, CubbleFloatType timeStep, bool useGasExchange, hipStream_t &streamThatShouldWait)
{
    CubbleFloatType *x = bubbleData.getRowPtr((size_t)BP::X);
    CubbleFloatType *y = bubbleData.getRowPtr((size_t)BP::Y);
    CubbleFloatType *z = bubbleData.getRowPtr((size_t)BP::Z);
    CubbleFloatType *r = bubbleData.getRowPtr((size_t)BP::R);

    CubbleFloatType *xPrd = bubbleData.getRowPtr((size_t)BP::X_PRD);
    CubbleFloatType *yPrd = bubbleData.getRowPtr((size_t)BP::Y_PRD);
    CubbleFloatType *zPrd = bubbleData.getRowPtr((size_t)BP::Z_PRD);
    CubbleFloatType *rPrd = bubbleData.getRowPtr((size_t)BP::R_PRD);

    CubbleFloatType *dxdt = bubbleData.getRowPtr((size_t)BP::DXDT);
    CubbleFloatType *dydt = bubbleData.getRowPtr((size_t)BP::DYDT);
    CubbleFloatType *dzdt = bubbleData.getRowPtr((size_t)BP::DZDT);
    CubbleFloatType *drdt = bubbleData.getRowPtr((size_t)BP::DRDT);

    CubbleFloatType *dxdtPrd = bubbleData.getRowPtr((size_t)BP::DXDT_PRD);
    CubbleFloatType *dydtPrd = bubbleData.getRowPtr((size_t)BP::DYDT_PRD);
    CubbleFloatType *dzdtPrd = bubbleData.getRowPtr((size_t)BP::DZDT_PRD);
    CubbleFloatType *drdtPrd = bubbleData.getRowPtr((size_t)BP::DRDT_PRD);

    CubbleFloatType *errors = bubbleData.getRowPtr((size_t)BP::ERROR);

    if (useGasExchange)
    {
        CUDA_LAUNCH(correctKernel, policy,
                    numBubbles, timeStep, errors,
                    xPrd, x, dxdt, dxdtPrd,
                    yPrd, y, dydt, dydtPrd,
#if (NUM_DIM == 3)
                    zPrd, z, dzdt, dzdtPrd,
#endif
                    rPrd, r, drdt, drdtPrd);
    }
    else
    {
        CUDA_LAUNCH(correctKernel, policy,
                    numBubbles, timeStep, errors,
                    xPrd, x, dxdt, dxdtPrd,
                    yPrd, y, dydt, dydtPrd
#if (NUM_DIM == 3)
                    ,
                    zPrd, z, dzdt, dzdtPrd
#endif
        );
    }

    CUDA_CALL(hipEventRecord(blockingEvent2, policy.stream));
    CUDA_CALL(hipStreamWaitEvent(streamThatShouldWait, blockingEvent2, 0));
}

void Simulator::doGasExchange(ExecutionPolicy policy, const hipEvent_t &eventToWaitOn, hipStream_t &streamThatShouldWait)
{
    ExecutionPolicy gasExchangePolicy(128, numBubbles);
    gasExchangePolicy.stream = nonBlockingStream2;
    policy.stream = gasExchangePolicy.stream;

    CUDA_CALL(hipStreamWaitEvent(gasExchangePolicy.stream, eventToWaitOn, 0));

    CubbleFloatType *xPrd = bubbleData.getRowPtr((size_t)BP::X_PRD);
    CubbleFloatType *yPrd = bubbleData.getRowPtr((size_t)BP::Y_PRD);
    CubbleFloatType *zPrd = bubbleData.getRowPtr((size_t)BP::Z_PRD);
    CubbleFloatType *rPrd = bubbleData.getRowPtr((size_t)BP::R_PRD);
    CubbleFloatType *drdtPrd = bubbleData.getRowPtr((size_t)BP::DRDT_PRD);
    CubbleFloatType *errors = bubbleData.getRowPtr((size_t)BP::ERROR);
    CubbleFloatType *freeArea = bubbleData.getRowPtr((size_t)BP::FREE_AREA);
    CubbleFloatType *volume = bubbleData.getRowPtr((size_t)BP::VOLUME);

    const fpvec tfr = env->getTfr();
    const fpvec lbb = env->getLbb();
    const fpvec interval = tfr - lbb;

    CUDA_LAUNCH(gasExchangeKernel, policy,
                numBubbles,
                env->getPi(),
                pairs.getRowPtr(0),
                pairs.getRowPtr(1),
                rPrd,
                drdtPrd,
                freeArea,
                interval.x, PBC_X == 1, xPrd,
                interval.y, PBC_Y == 1, yPrd
#if (NUM_DIM == 3)
                ,
                interval.z, PBC_Z == 1, zPrd
#endif
    );

    CUDA_LAUNCH(freeAreaKernel, gasExchangePolicy,
                numBubbles, env->getPi(), rPrd, freeArea, errors, volume);

    cubWrapper->reduceNoCopy<CubbleFloatType, CubbleFloatType *, CubbleFloatType *>(&hipcub::DeviceReduce::Sum, errors, dtfapr, numBubbles, gasExchangePolicy.stream);
    cubWrapper->reduceNoCopy<CubbleFloatType, CubbleFloatType *, CubbleFloatType *>(&hipcub::DeviceReduce::Sum, freeArea, dtfa, numBubbles, gasExchangePolicy.stream);
    cubWrapper->reduceNoCopy<CubbleFloatType, CubbleFloatType *, CubbleFloatType *>(&hipcub::DeviceReduce::Sum, volume, dta, numBubbles, gasExchangePolicy.stream);

    CUDA_LAUNCH(finalRadiusChangeRateKernel, gasExchangePolicy,
                drdtPrd, rPrd, freeArea, numBubbles, (CubbleFloatType)1.0 / env->getPi(), env->getKappa(), env->getKParameter());

    CUDA_CALL(hipEventRecord(blockingEvent2, gasExchangePolicy.stream));
    CUDA_CALL(hipStreamWaitEvent(streamThatShouldWait, blockingEvent2, 0));
}

void Simulator::doVelocity(const ExecutionPolicy &policy)
{
    const fpvec tfr = env->getTfr();
    const fpvec lbb = env->getLbb();
    const fpvec interval = tfr - lbb;

    CubbleFloatType *xPrd = bubbleData.getRowPtr((size_t)BP::X_PRD);
    CubbleFloatType *yPrd = bubbleData.getRowPtr((size_t)BP::Y_PRD);
    CubbleFloatType *zPrd = bubbleData.getRowPtr((size_t)BP::Z_PRD);
    CubbleFloatType *rPrd = bubbleData.getRowPtr((size_t)BP::R_PRD);
    CubbleFloatType *dxdtPrd = bubbleData.getRowPtr((size_t)BP::DXDT_PRD);
    CubbleFloatType *dydtPrd = bubbleData.getRowPtr((size_t)BP::DYDT_PRD);
    CubbleFloatType *dzdtPrd = bubbleData.getRowPtr((size_t)BP::DZDT_PRD);

    CUDA_LAUNCH(velocityPairKernel, policy,
                env->getFZeroPerMuZero(), pairs.getRowPtr(0), pairs.getRowPtr(1), rPrd,
                interval.x, lbb.x, PBC_X == 1, xPrd, dxdtPrd,
                interval.y, lbb.y, PBC_Y == 1, yPrd, dydtPrd
#if (NUM_DIM == 3)
                ,
                interval.z, lbb.z, PBC_Z == 1, zPrd, dzdtPrd
#endif
    );

#if (PBC_X == 0 || PBC_Y == 0 || PBC_Z == 0)
    CUDA_LAUNCH(velocityWallKernel, policy,
                numBubbles, env->getFZeroPerMuZero(), pairs.getRowPtr(0), pairs.getRowPtr(1), rPrd
#if (PBC_X == 0)
                ,
                interval.x, lbb.x, PBC_X == 1, xPrd, dxdtPrd
#endif
#if (PBC_Y == 0)
                ,
                interval.y, lbb.y, PBC_Y == 1, yPrd, dydtPrd
#endif
#if (NUM_DIM == 3 && PBC_Z == 0)
                ,
                interval.z, lbb.z, PBC_Z == 1, zPrd, dzdtPrd
#endif
    );
#endif
}

void Simulator::doReset(const ExecutionPolicy &policy)
{
    CUDA_LAUNCH(resetKernel, policy,
                (CubbleFloatType)0.0, numBubbles,
                bubbleData.getRowPtr((size_t)BP::DXDT_PRD),
                bubbleData.getRowPtr((size_t)BP::DYDT_PRD),
                bubbleData.getRowPtr((size_t)BP::DZDT_PRD),
                bubbleData.getRowPtr((size_t)BP::DRDT_PRD),
                bubbleData.getRowPtr((size_t)BP::FREE_AREA),
                bubbleData.getRowPtr((size_t)BP::ENERGY));
}

CubbleFloatType Simulator::doError()
{
    cubWrapper->reduceNoCopy<CubbleFloatType, CubbleFloatType *, CubbleFloatType *>(&hipcub::DeviceReduce::Max, bubbleData.getRowPtr((size_t)BP::ERROR), dtfa, numBubbles, nonBlockingStream2);
    CUDA_CALL(hipMemcpyAsync(static_cast<void *>(pinnedDouble.get()), static_cast<void *>(dtfa), sizeof(CubbleFloatType), hipMemcpyDeviceToHost, nonBlockingStream2));
    CUDA_CALL(hipEventRecord(blockingEvent2, nonBlockingStream2));
    CUDA_CALL(hipEventSynchronize(blockingEvent2));

    return pinnedDouble.get()[0];
}

void Simulator::doBoundaryWrap(const ExecutionPolicy &policy)
{
    const fpvec tfr = env->getTfr();
    const fpvec lbb = env->getLbb();

    CubbleFloatType *xPrd = bubbleData.getRowPtr((size_t)BP::X_PRD);
    CubbleFloatType *yPrd = bubbleData.getRowPtr((size_t)BP::Y_PRD);
    CubbleFloatType *zPrd = bubbleData.getRowPtr((size_t)BP::Z_PRD);

#if (PBC_X == 1 || PBC_Y == 1 || PBC_Z == 1)
    CUDA_LAUNCH(boundaryWrapKernel, policy,
                numBubbles
#if (PBC_X == 1)
                ,
                xPrd, lbb.x, tfr.x
#endif
#if (PBC_Y == 1)
                ,
                yPrd, lbb.y, tfr.y
#endif
#if (PBC_Z == 1 && NUM_DIM == 3)
                ,
                zPrd, lbb.z, tfr.z
#endif
    );
#endif
}

void Simulator::doBubbleSizeChecks(ExecutionPolicy policy, hipStream_t &streamToUse, hipEvent_t &eventToMark)
{
    CubbleFloatType *rPrd = bubbleData.getRowPtr((size_t)BP::R_PRD);
    policy.stream = streamToUse;

    CUDA_LAUNCH(setFlagIfGreaterThanConstantKernel, policy,
                numBubbles,
                aboveMinRadFlags.getRowPtr(0),
                rPrd,
                env->getMinRad());

    cubWrapper->reduceNoCopy<int, int *, int *>(&hipcub::DeviceReduce::Sum, aboveMinRadFlags.getRowPtr(0), static_cast<int *>(mbpc), numBubbles, streamToUse);
    CUDA_CALL(hipMemcpyAsync(static_cast<void *>(pinnedInt.get()), mbpc, sizeof(int), hipMemcpyDeviceToHost, streamToUse));

    cubWrapper->reduceNoCopy<CubbleFloatType, CubbleFloatType *, CubbleFloatType *>(&hipcub::DeviceReduce::Max, rPrd, static_cast<CubbleFloatType *>(dtfa), numBubbles, streamToUse);
    CUDA_CALL(hipMemcpyAsync(static_cast<void *>(pinnedDouble.get()), dtfa, sizeof(CubbleFloatType), hipMemcpyDeviceToHost, streamToUse));

    CUDA_CALL(hipEventRecord(eventToMark, streamToUse));
}

void Simulator::generateBubbles()
{
    std::cout << "Starting to generate data for bubbles." << std::endl;

    const int rngSeed = env->getRngSeed();
    const CubbleFloatType avgRad = env->getAvgRad();
    const CubbleFloatType stdDevRad = env->getStdDevRad();
    const fpvec tfr = env->getTfr();
    const fpvec lbb = env->getLbb();

    CubbleFloatType *x = bubbleData.getRowPtr((size_t)BP::X);
    CubbleFloatType *y = bubbleData.getRowPtr((size_t)BP::Y);
    CubbleFloatType *z = bubbleData.getRowPtr((size_t)BP::Z);

    CubbleFloatType *xPrd = bubbleData.getRowPtr((size_t)BP::X_PRD);
    CubbleFloatType *yPrd = bubbleData.getRowPtr((size_t)BP::Y_PRD);
    CubbleFloatType *zPrd = bubbleData.getRowPtr((size_t)BP::Z_PRD);

    CubbleFloatType *r = bubbleData.getRowPtr((size_t)BP::R);
    CubbleFloatType *w = bubbleData.getRowPtr((size_t)BP::R_PRD);

    hiprandGenerator_t generator;
    CURAND_CALL(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_MTGP32));
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(generator, rngSeed));

#if (CUBBLE_FLOAT_TYPE == CUBBLE_FLOAT)
    CURAND_CALL(hiprandGenerateUniform(generator, x, numBubbles));
    CURAND_CALL(hiprandGenerateUniform(generator, y, numBubbles));
#if (NUM_DIM == 3)
    CURAND_CALL(hiprandGenerateUniform(generator, z, numBubbles));
#endif
    CURAND_CALL(hiprandGenerateUniform(generator, w, numBubbles));
    CURAND_CALL(hiprandGenerateNormal(generator, r, numBubbles, avgRad, stdDevRad));
#else
    CURAND_CALL(hiprandGenerateUniformDouble(generator, x, numBubbles));
    CURAND_CALL(hiprandGenerateUniformDouble(generator, y, numBubbles));
#if (NUM_DIM == 3)
    CURAND_CALL(hiprandGenerateUniformDouble(generator, z, numBubbles));
#endif
    CURAND_CALL(hiprandGenerateUniformDouble(generator, w, numBubbles));
    CURAND_CALL(hiprandGenerateNormalDouble(generator, r, numBubbles, avgRad, stdDevRad));
#endif

    CURAND_CALL(hiprandDestroyGenerator(generator));

    ExecutionPolicy defaultPolicy(128, numBubbles);
    assert(bubblesPerDimAtStart.x > 0);
    assert(bubblesPerDimAtStart.y > 0);
#if (NUM_DIM == 3)
    assert(bubblesPerDimAtStart.z > 0);
#endif
    CUDA_LAUNCH(assignDataToBubbles, defaultPolicy,
                x, y, z, xPrd, yPrd, zPrd, r, w,
                aboveMinRadFlags.getRowPtr(0), bubblesPerDimAtStart,
                tfr, lbb, avgRad, env->getMinRad(), env->getPi(), numBubbles);

    cubWrapper->reduceNoCopy<CubbleFloatType, CubbleFloatType *, CubbleFloatType *>(&hipcub::DeviceReduce::Sum, w, dasai, numBubbles, defaultPolicy.stream);
    CUDA_CALL(hipMemcpyAsync(static_cast<void *>(w), static_cast<void *>(r),
                              sizeof(CubbleFloatType) * bubbleData.getWidth(),
                              hipMemcpyDeviceToDevice,
                              defaultPolicy.stream));
}

void Simulator::updateCellsAndNeighbors()
{
    dim3 gridSize = getGridSize();
    const int numCells = gridSize.x * gridSize.y * gridSize.z;
    const ivec cellDim(gridSize.x, gridSize.y, gridSize.z);

    CubbleFloatType *x = bubbleData.getRowPtr((size_t)BP::X);
    CubbleFloatType *y = bubbleData.getRowPtr((size_t)BP::Y);
    CubbleFloatType *z = bubbleData.getRowPtr((size_t)BP::Z);
    CubbleFloatType *r = bubbleData.getRowPtr((size_t)BP::R);
    int *offsets = cellData.getRowPtr((size_t)CellProperty::OFFSET);
    int *sizes = cellData.getRowPtr((size_t)CellProperty::SIZE);

    cellData.setBytesToZero();
    bubbleCellIndices.setBytesToZero();

    ExecutionPolicy defaultPolicy = {};
    defaultPolicy.blockSize = dim3(128, 1, 1);
    defaultPolicy.gridSize = dim3(256, 1, 1);
    ExecutionPolicy asyncCopyDDPolicy(128, numBubbles, 0, nonBlockingStream1);
    CUDA_LAUNCH(assignBubblesToCells, defaultPolicy,
                x, y, z, bubbleCellIndices.getRowPtr(2), bubbleCellIndices.getRowPtr(3), env->getLbb(), env->getTfr(), cellDim, numBubbles);

    int *cellIndices = bubbleCellIndices.getRowPtr(0);
    int *bubbleIndices = bubbleCellIndices.getRowPtr(1);

    cubWrapper->sortPairs<int, int>(&hipcub::DeviceRadixSort::SortPairs,
                                    const_cast<const int *>(bubbleCellIndices.getRowPtr(2)),
                                    cellIndices,
                                    const_cast<const int *>(bubbleCellIndices.getRowPtr(3)),
                                    bubbleIndices,
                                    numBubbles);

    CUDA_CALL(hipEventRecord(blockingEvent1));
    CUDA_CALL(hipStreamWaitEvent(nonBlockingStream1, blockingEvent1, 0));

    cubWrapper->histogram<int *, int, int, int>(&hipcub::DeviceHistogram::HistogramEven,
                                                bubbleCellIndices.getRowPtr(2),
                                                sizes,
                                                numCells + 1,
                                                0,
                                                numCells,
                                                numBubbles);

    cubWrapper->scan<int *, int *>(&hipcub::DeviceScan::ExclusiveSum, sizes, offsets, numCells);
    CUDA_CALL(hipEventRecord(blockingEvent2));

    CUDA_LAUNCH(reorganizeKernel, asyncCopyDDPolicy,
                numBubbles, ReorganizeType::COPY_FROM_INDEX, bubbleIndices, bubbleIndices,
                bubbleData.getRowPtr((size_t)BP::X), bubbleData.getRowPtr((size_t)BP::X_PRD),
                bubbleData.getRowPtr((size_t)BP::Y), bubbleData.getRowPtr((size_t)BP::Y_PRD),
                bubbleData.getRowPtr((size_t)BP::Z), bubbleData.getRowPtr((size_t)BP::Z_PRD),
                bubbleData.getRowPtr((size_t)BP::R), bubbleData.getRowPtr((size_t)BP::R_PRD),
                bubbleData.getRowPtr((size_t)BP::DXDT), bubbleData.getRowPtr((size_t)BP::DXDT_PRD),
                bubbleData.getRowPtr((size_t)BP::DYDT), bubbleData.getRowPtr((size_t)BP::DYDT_PRD),
                bubbleData.getRowPtr((size_t)BP::DZDT), bubbleData.getRowPtr((size_t)BP::DZDT_PRD),
                bubbleData.getRowPtr((size_t)BP::DRDT), bubbleData.getRowPtr((size_t)BP::DRDT_PRD),
                bubbleData.getRowPtr((size_t)BP::DXDT_OLD), bubbleData.getRowPtr((size_t)BP::ENERGY),
                bubbleData.getRowPtr((size_t)BP::DYDT_OLD), bubbleData.getRowPtr((size_t)BP::FREE_AREA),
                bubbleData.getRowPtr((size_t)BP::DZDT_OLD), bubbleData.getRowPtr((size_t)BP::ERROR),
                bubbleData.getRowPtr((size_t)BP::DRDT_OLD), bubbleData.getRowPtr((size_t)BP::VOLUME));
    CUDA_CALL(hipMemcpyAsync(static_cast<void *>(x),
                              static_cast<void *>(bubbleData.getRowPtr((size_t)BP::X_PRD)),
                              sizeof(CubbleFloatType) * (size_t)BP::X_PRD * bubbleData.getWidth(),
                              hipMemcpyDeviceToDevice,
                              nonBlockingStream1));

    CUDA_CALL(hipEventRecord(blockingEvent1, nonBlockingStream1));

    fpvec interval = env->getTfr() - env->getLbb();

    ExecutionPolicy findPolicy;
    findPolicy.blockSize = dim3(128, 1, 1);
    findPolicy.gridSize = gridSize;
    findPolicy.sharedMemBytes = 0;

    CUDA_CALL(hipMemset(np, 0, sizeof(int)));

    for (int i = 0; i < CUBBLE_NUM_NEIGHBORS + 1; ++i)
    {
        findPolicy.stream = neighborStreamVec[i];
        CUDA_CALL(hipStreamWaitEvent(neighborStreamVec[i], blockingEvent1, 0));
        CUDA_CALL(hipStreamWaitEvent(neighborStreamVec[i], blockingEvent2, 0));
        CUDA_LAUNCH(neighborSearch, findPolicy,
                    i, numBubbles, numCells, static_cast<int>(pairs.getWidth()),
                    offsets, sizes, pairs.getRowPtr(2), pairs.getRowPtr(3), r,
                    interval.x, PBC_X == 1, x,
                    interval.y, PBC_Y == 1, y
#if (NUM_DIM == 3)
                    ,
                    interval.z, PBC_Z == 1, z
#endif
        );

        CUDA_CALL(hipEventRecord(neighborEventVec[i], neighborStreamVec[i]));
        CUDA_CALL(hipStreamWaitEvent(0, neighborEventVec[i], 0));
    }

    CUDA_CALL(hipMemcpy(static_cast<void *>(pinnedInt.get()), np, sizeof(int), hipMemcpyDeviceToHost));
    int numPairs = pinnedInt.get()[0];
    cubWrapper->sortPairs<int, int>(&hipcub::DeviceRadixSort::SortPairs,
                                    const_cast<const int *>(pairs.getRowPtr(2)),
                                    pairs.getRowPtr(0),
                                    const_cast<const int *>(pairs.getRowPtr(3)),
                                    pairs.getRowPtr(1),
                                    numPairs);
}

void Simulator::updateData()
{
    CUDA_CALL(hipStreamWaitEvent(0, blockingEvent1, 0));
    const size_t numBytesToCopy = 4 * sizeof(CubbleFloatType) * bubbleData.getWidth();

    CubbleFloatType *x = bubbleData.getRowPtr((size_t)BP::X);
    CubbleFloatType *xPrd = bubbleData.getRowPtr((size_t)BP::X_PRD);
    CubbleFloatType *dxdt = bubbleData.getRowPtr((size_t)BP::DXDT);
    CubbleFloatType *dxdtOld = bubbleData.getRowPtr((size_t)BP::DXDT_OLD);

    CUDA_CALL(hipMemcpyAsync(dxdtOld, dxdt, numBytesToCopy, hipMemcpyDeviceToDevice));
    CUDA_CALL(hipMemcpyAsync(x, xPrd, 2 * numBytesToCopy, hipMemcpyDeviceToDevice));
}

void Simulator::deleteSmallBubbles(int numBubblesAboveMinRad)
{
    NVTX_RANGE_PUSH_A("BubbleRemoval");
    ExecutionPolicy defaultPolicy(128, numBubbles);

    int *flag = aboveMinRadFlags.getRowPtr(0);
    CubbleFloatType *r = bubbleData.getRowPtr((size_t)BP::R);
    CubbleFloatType *volumes = bubbleData.getRowPtr((size_t)BP::VOLUME);

    CUDA_CALL(hipMemset(static_cast<void *>(dvm), 0, sizeof(CubbleFloatType)));
    CUDA_LAUNCH(calculateRedistributedGasVolume, defaultPolicy,
                volumes, r, flag, env->getPi(), numBubbles);

    cubWrapper->reduceNoCopy<CubbleFloatType, CubbleFloatType *, CubbleFloatType *>(&hipcub::DeviceReduce::Sum, volumes, dtv, numBubbles);

    int *newIdx = aboveMinRadFlags.getRowPtr(1);
    cubWrapper->scan<int *, int *>(&hipcub::DeviceScan::ExclusiveSum, flag, newIdx, numBubbles);

    CUDA_LAUNCH(reorganizeKernel, defaultPolicy,
                numBubbles, ReorganizeType::CONDITIONAL_TO_INDEX, newIdx, flag,
                bubbleData.getRowPtr((size_t)BP::X), bubbleData.getRowPtr((size_t)BP::X_PRD),
                bubbleData.getRowPtr((size_t)BP::Y), bubbleData.getRowPtr((size_t)BP::Y_PRD),
                bubbleData.getRowPtr((size_t)BP::Z), bubbleData.getRowPtr((size_t)BP::Z_PRD),
                bubbleData.getRowPtr((size_t)BP::R), bubbleData.getRowPtr((size_t)BP::R_PRD),
                bubbleData.getRowPtr((size_t)BP::DXDT), bubbleData.getRowPtr((size_t)BP::DXDT_PRD),
                bubbleData.getRowPtr((size_t)BP::DYDT), bubbleData.getRowPtr((size_t)BP::DYDT_PRD),
                bubbleData.getRowPtr((size_t)BP::DZDT), bubbleData.getRowPtr((size_t)BP::DZDT_PRD),
                bubbleData.getRowPtr((size_t)BP::DRDT), bubbleData.getRowPtr((size_t)BP::DRDT_PRD),
                bubbleData.getRowPtr((size_t)BP::DXDT_OLD), bubbleData.getRowPtr((size_t)BP::ENERGY),
                bubbleData.getRowPtr((size_t)BP::DYDT_OLD), bubbleData.getRowPtr((size_t)BP::FREE_AREA),
                bubbleData.getRowPtr((size_t)BP::DZDT_OLD), bubbleData.getRowPtr((size_t)BP::ERROR),
                bubbleData.getRowPtr((size_t)BP::DRDT_OLD), bubbleData.getRowPtr((size_t)BP::VOLUME));
    CUDA_CALL(hipMemcpyAsync(static_cast<void *>(bubbleData.getRowPtr((size_t)BP::X)),
                              static_cast<void *>(bubbleData.getRowPtr((size_t)BP::X_PRD)),
                              sizeof(CubbleFloatType) * (size_t)BP::X_PRD * bubbleData.getWidth(),
                              hipMemcpyDeviceToDevice));

    numBubbles = numBubblesAboveMinRad;
    CUDA_LAUNCH(addVolume, defaultPolicy, r, numBubbles);

    NVTX_RANGE_POP();
}

dim3 Simulator::getGridSize()
{
    const int totalNumCells = std::ceil((CubbleFloatType)numBubbles / env->getNumBubblesPerCell());
    fpvec interval = env->getTfr() - env->getLbb();
    interval /= interval.x;
#if (NUM_DIM == 3)
    CubbleFloatType nx = std::cbrt((CubbleFloatType)totalNumCells / (interval.y * interval.z));
#else
    CubbleFloatType nx = std::sqrt((CubbleFloatType)totalNumCells / interval.y);
    interval.z = 0;
#endif
    ivec grid = (nx * interval).floor() + 1;
    assert(grid.x > 0);
    assert(grid.y > 0);
    assert(grid.z > 0);

    return dim3(grid.x, grid.y, grid.z);
}

void Simulator::calculateEnergy()
{
    ExecutionPolicy pairPolicy;
    pairPolicy.blockSize = dim3(128, 1, 1);
    pairPolicy.stream = 0;
    pairPolicy.gridSize = dim3(256, 1, 1);
    pairPolicy.sharedMemBytes = 0;

    const fpvec tfr = env->getTfr();
    const fpvec lbb = env->getLbb();
    const fpvec interval = tfr - lbb;

    CUDA_LAUNCH(potentialEnergyKernel, pairPolicy,
                numBubbles,
                pairs.getRowPtr(0),
                pairs.getRowPtr(1),
                bubbleData.getRowPtr((size_t)BP::R),
                bubbleData.getRowPtr((size_t)BP::ENERGY),
                interval.x, PBC_X == 1, bubbleData.getRowPtr((size_t)BP::X),
                interval.y, PBC_Y == 1, bubbleData.getRowPtr((size_t)BP::Y)
#if (NUM_DIM == 3)
                                            ,
                interval.z, PBC_Z == 1, bubbleData.getRowPtr((size_t)BP::Z)
#endif
    );

    ElasticEnergy = cubWrapper->reduce<CubbleFloatType, CubbleFloatType *, CubbleFloatType *>(&hipcub::DeviceReduce::Sum,
                                                                                              bubbleData.getRowPtr((size_t)BP::ENERGY),
                                                                                              numBubbles);
}

CubbleFloatType Simulator::getVolumeOfBubbles()
{
    ExecutionPolicy defaultPolicy(128, numBubbles);
    CubbleFloatType *r = bubbleData.getRowPtr((size_t)BP::R);
    CubbleFloatType *volPtr = bubbleData.getRowPtr((size_t)BP::VOLUME);
    CUDA_LAUNCH(calculateVolumes, defaultPolicy,
                r, volPtr, numBubbles, env->getPi());
    CubbleFloatType volume = cubWrapper->reduce<CubbleFloatType, CubbleFloatType *, CubbleFloatType *>(&hipcub::DeviceReduce::Sum, volPtr, numBubbles);

    return volume;
}

CubbleFloatType Simulator::getAverageRadius()
{
    CubbleFloatType *r = bubbleData.getRowPtr((size_t)BP::R);
    CubbleFloatType avgRad = cubWrapper->reduce<CubbleFloatType, CubbleFloatType *, CubbleFloatType *>(&hipcub::DeviceReduce::Sum, r, numBubbles);
    avgRad /= numBubbles;

    return avgRad;
}

void Simulator::getBubbles(std::vector<Bubble> &bubbles) const
{
    bubbles.clear();
    bubbles.resize(numBubbles);

    size_t memoryStride = bubbleData.getWidth();
    CubbleFloatType *devX = bubbleData.getRowPtr((size_t)BP::X);
    std::vector<CubbleFloatType> xyzr;
    xyzr.resize(memoryStride * 4);

    CUDA_CALL(hipMemcpy(xyzr.data(), devX, sizeof(CubbleFloatType) * 4 * memoryStride, hipMemcpyDeviceToHost));

    for (size_t i = 0; i < numBubbles; ++i)
    {
        Bubble b;
        fpvec pos(-1, -1, -1);
        pos.x = xyzr[i];
        pos.y = xyzr[i + memoryStride];
        pos.z = xyzr[i + 2 * memoryStride];
        b.setPos(pos);
        b.setRadius(xyzr[i + 3 * memoryStride]);
        bubbles[i] = b;
    }
}

CubbleFloatType Simulator::getInvRho()
{
    CubbleFloatType invRho = 0;
    CUDA_CALL(hipMemcpy(static_cast<void *>(&invRho), static_cast<void *>(dir), sizeof(CubbleFloatType), hipMemcpyDeviceToHost));

    return invRho;
}

void Simulator::transformPositions(bool normalize)
{
    ExecutionPolicy policy;
    policy.gridSize = dim3(256, 1, 1);
    policy.blockSize = dim3(128, 1, 1);
    policy.stream = 0;
    policy.sharedMemBytes = 0;

    CUDA_LAUNCH(transformPositionsKernel, policy,
                normalize, numBubbles, env->getLbb(), env->getTfr(),
                bubbleData.getRowPtr((size_t)BP::X),
                bubbleData.getRowPtr((size_t)BP::Y),
                bubbleData.getRowPtr((size_t)BP::Z));
}
} // namespace cubble
